#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

/*----------------------------------------------------------------------------80
The neuroevolution potential (NEP)
Ref: Zheyong Fan et al., Neuroevolution machine learning potentials:
Combining high accuracy and low cost in atomistic simulations and application to
heat transport, Phys. Rev. B. 104, 104309 (2021).
------------------------------------------------------------------------------*/

#include "nep.cuh"
#include "utilities/error.cuh"
#include "utilities/nep_utilities.cuh"
#include <vector>

NEP2::NEP2(FILE* fid, char* input_dir, int num_types, const Neighbor& neighbor)
{
  if (num_types == 1) {
    printf("Use the NEP potential with %d atom type.\n", num_types);
  } else {
    printf("Use the NEP potential with %d atom types.\n", num_types);
  }

  char name[20];

  // need to read the num_types atom symbols, although they are not used here
  for (int n = 0; n < num_types; ++n) {
    int count = fscanf(fid, "%s", name);
    PRINT_SCANF_ERROR(count, 1, "reading error for NEP potential.");
  }

  paramb.num_types = num_types;

  int count = fscanf(fid, "%s%f%f", name, &paramb.rc_radial, &paramb.rc_angular);
  PRINT_SCANF_ERROR(count, 3, "reading error for NEP potential.");
  printf("    radial cutoff = %g A.\n", paramb.rc_radial);
  printf("    angular cutoff = %g A.\n", paramb.rc_angular);

  count = fscanf(fid, "%s%d%d", name, &paramb.n_max_radial, &paramb.n_max_angular);
  PRINT_SCANF_ERROR(count, 3, "reading error for NEP potential.");
  printf("    n_max_radial = %d.\n", paramb.n_max_radial);
  printf("    n_max_angular = %d.\n", paramb.n_max_angular);

  count = fscanf(fid, "%s%d", name, &paramb.L_max);
  PRINT_SCANF_ERROR(count, 2, "reading error for NEP potential.");
  printf("    l_max = %d.\n", paramb.L_max);

  int num_neurons2;
  count = fscanf(fid, "%s%d%d", name, &annmb.num_neurons1, &num_neurons2);
  PRINT_SCANF_ERROR(count, 3, "reading error for NEP potential.");

  rc = paramb.rc_radial; // largest cutoff

  paramb.rcinv_radial = 1.0f / paramb.rc_radial;
  paramb.rcinv_angular = 1.0f / paramb.rc_angular;
  annmb.dim = (paramb.n_max_radial + 1) + (paramb.n_max_angular + 1) * paramb.L_max;

  printf("    ANN = %d-%d-1.\n", annmb.dim, annmb.num_neurons1);

  annmb.num_para = (annmb.dim + 2) * annmb.num_neurons1 + 1;
  printf("    number of neural network parameters = %d.\n", annmb.num_para);
  int num_para_descriptor =
    (num_types == 1) ? 0 : num_types * num_types * (paramb.n_max_radial + paramb.n_max_angular + 2);
  printf("    number of descriptor parameters = %d.\n", num_para_descriptor);
  annmb.num_para += num_para_descriptor;
  printf("    total number of parameters = %d\n", annmb.num_para);

  nep_data.f12x.resize(neighbor.NN.size() * neighbor.MN);
  nep_data.f12y.resize(neighbor.NN.size() * neighbor.MN);
  nep_data.f12z.resize(neighbor.NN.size() * neighbor.MN);
  nep_data.NN.resize(neighbor.NN.size());
  nep_data.NL.resize(neighbor.NN.size() * neighbor.MN);
  nep_data.Fp.resize(neighbor.NN.size() * annmb.dim);
  nep_data.sum_fxyz.resize(neighbor.NN.size() * (paramb.n_max_angular + 1) * NUM_OF_ABC);

  update_potential(fid);
}

NEP2::~NEP2(void)
{
  // nothing
}

void NEP2::update_potential(const float* parameters, ANN& ann)
{
  ann.w0 = parameters;
  ann.b0 = ann.w0 + ann.num_neurons1 * ann.dim;
  ann.w1 = ann.b0 + ann.num_neurons1;
  ann.b1 = ann.w1 + ann.num_neurons1;
  if (paramb.num_types > 1) {
    ann.c = ann.b1 + 1;
  }
}

void NEP2::update_potential(FILE* fid)
{
  std::vector<float> parameters(annmb.num_para);
  for (int n = 0; n < annmb.num_para; ++n) {
    int count = fscanf(fid, "%f", &parameters[n]);
    PRINT_SCANF_ERROR(count, 1, "reading error for NEP potential.");
  }
  CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_parameters), parameters.data(), sizeof(float) * annmb.num_para));
  float* address_c_parameters;
  CHECK(hipGetSymbolAddress((void**)&address_c_parameters, c_parameters));
  update_potential(address_c_parameters, annmb);

  for (int d = 0; d < annmb.dim; ++d) {
    int count = fscanf(fid, "%f", &paramb.q_scaler[d]);
    PRINT_SCANF_ERROR(count, 1, "reading error for NEP potential.");
  }
}

static __device__ void
apply_ann_one_layer(const NEP2::ANN& ann, float* q, float& energy, float* energy_derivative)
{
  for (int n = 0; n < ann.num_neurons1; ++n) {
    float w0_times_q = 0.0f;
    for (int d = 0; d < ann.dim; ++d) {
      w0_times_q += ann.w0[n * ann.dim + d] * q[d];
    }
    float x1 = tanh(w0_times_q - ann.b0[n]);
    energy += ann.w1[n] * x1;
    for (int d = 0; d < ann.dim; ++d) {
      float y1 = (1.0f - x1 * x1) * ann.w0[n * ann.dim + d];
      energy_derivative[d] += ann.w1[n] * y1;
    }
  }
  energy -= ann.b1[0];
}

static __global__ void find_neighbor_angular(
  NEP2::ParaMB paramb,
  const int N,
  const int N1,
  const int N2,
  const Box box,
  const int* g_NN,
  const int* g_NL,
  const double* __restrict__ g_x,
  const double* __restrict__ g_y,
  const double* __restrict__ g_z,
  int* g_NN_angular,
  int* g_NL_angular)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1;
  if (n1 < N2) {
    double x1 = g_x[n1];
    double y1 = g_y[n1];
    double z1 = g_z[n1];
    int count = 0;
    for (int i1 = 0; i1 < g_NN[n1]; ++i1) {
      int n2 = g_NL[n1 + N * i1];
      double x12double = g_x[n2] - x1;
      double y12double = g_y[n2] - y1;
      double z12double = g_z[n2] - z1;
      apply_mic(box, x12double, y12double, z12double);
      float x12 = float(x12double), y12 = float(y12double), z12 = float(z12double);
      float d12sq = x12 * x12 + y12 * y12 + z12 * z12;
      if (d12sq < paramb.rc_angular * paramb.rc_angular) {
        g_NL_angular[count++ * N + n1] = n2;
      }
    }
    g_NN_angular[n1] = count;
  }
}

static __global__ void find_descriptor(
  NEP2::ParaMB paramb,
  NEP2::ANN annmb,
  const int N,
  const int N1,
  const int N2,
  const Box box,
  const int* g_NN,
  const int* g_NL,
  const int* g_NN_angular,
  const int* g_NL_angular,
  const int* __restrict__ g_type,
  const double* __restrict__ g_x,
  const double* __restrict__ g_y,
  const double* __restrict__ g_z,
  double* g_pe,
  float* g_Fp,
  float* g_sum_fxyz)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1;
  if (n1 < N2) {
    int t1 = g_type[n1];
    double x1 = g_x[n1];
    double y1 = g_y[n1];
    double z1 = g_z[n1];
    float q[MAX_DIM] = {0.0f};

    // get radial descriptors
    for (int i1 = 0; i1 < g_NN[n1]; ++i1) {
      int n2 = g_NL[n1 + N * i1];
      double x12double = g_x[n2] - x1;
      double y12double = g_y[n2] - y1;
      double z12double = g_z[n2] - z1;
      apply_mic(box, x12double, y12double, z12double);
      float x12 = float(x12double), y12 = float(y12double), z12 = float(z12double);
      float d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12);
      float fc12;
      find_fc(paramb.rc_radial, paramb.rcinv_radial, d12, fc12);
      int t2 = g_type[n2];
      float fn12[MAX_NUM_N];
      find_fn(paramb.n_max_radial, paramb.rcinv_radial, d12, fc12, fn12);
      for (int n = 0; n <= paramb.n_max_radial; ++n) {
        float c = (paramb.num_types == 1)
                    ? 1.0f
                    : annmb.c[(n * paramb.num_types + t1) * paramb.num_types + t2];
        q[n] += fn12[n] * c;
      }
    }

    // get angular descriptors
    for (int n = 0; n <= paramb.n_max_angular; ++n) {
      float s[NUM_OF_ABC] = {0.0f};
      for (int i1 = 0; i1 < g_NN_angular[n1]; ++i1) {
        int n2 = g_NL_angular[n1 + N * i1];
        double x12double = g_x[n2] - x1;
        double y12double = g_y[n2] - y1;
        double z12double = g_z[n2] - z1;
        apply_mic(box, x12double, y12double, z12double);
        float x12 = float(x12double), y12 = float(y12double), z12 = float(z12double);
        float d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12);
        float fc12;
        find_fc(paramb.rc_angular, paramb.rcinv_angular, d12, fc12);
        int t2 = g_type[n2];
        float fn;
        find_fn(n, paramb.rcinv_angular, d12, fc12, fn);
        fn *=
          (paramb.num_types == 1)
            ? 1.0f
            : annmb
                .c[((paramb.n_max_radial + 1 + n) * paramb.num_types + t1) * paramb.num_types + t2];
        accumulate_s(d12, x12, y12, z12, fn, s);
      }
      find_q(paramb.n_max_angular + 1, n, s, q + (paramb.n_max_radial + 1));
      for (int abc = 0; abc < NUM_OF_ABC; ++abc) {
        g_sum_fxyz[(n * NUM_OF_ABC + abc) * N + n1] = s[abc] * YLM[abc];
      }
    }

    // nomalize descriptor
    for (int d = 0; d < annmb.dim; ++d) {
      q[d] = q[d] * paramb.q_scaler[d];
    }

    // get energy and energy gradient
    float F = 0.0f, Fp[MAX_DIM] = {0.0f};
    apply_ann_one_layer(annmb, q, F, Fp);
    g_pe[n1] += F;

    for (int d = 0; d < annmb.dim; ++d) {
      g_Fp[d * N + n1] = Fp[d] * paramb.q_scaler[d];
    }
  }
}

static __global__ void find_force_radial(
  NEP2::ParaMB paramb,
  NEP2::ANN annmb,
  const int N,
  const int N1,
  const int N2,
  const Box box,
  const int* g_NN,
  const int* g_NL,
  const int* __restrict__ g_type,
  const double* __restrict__ g_x,
  const double* __restrict__ g_y,
  const double* __restrict__ g_z,
  const float* __restrict__ g_Fp,
  double* g_fx,
  double* g_fy,
  double* g_fz,
  double* g_virial)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1;
  if (n1 < N2) {
    int t1 = g_type[n1];
    float s_fx = 0.0f;
    float s_fy = 0.0f;
    float s_fz = 0.0f;
    float s_sxx = 0.0f;
    float s_sxy = 0.0f;
    float s_sxz = 0.0f;
    float s_syx = 0.0f;
    float s_syy = 0.0f;
    float s_syz = 0.0f;
    float s_szx = 0.0f;
    float s_szy = 0.0f;
    float s_szz = 0.0f;
    double x1 = g_x[n1];
    double y1 = g_y[n1];
    double z1 = g_z[n1];
    for (int i1 = 0; i1 < g_NN[n1]; ++i1) {
      int n2 = g_NL[n1 + N * i1];
      int t2 = g_type[n2];
      double x12double = g_x[n2] - x1;
      double y12double = g_y[n2] - y1;
      double z12double = g_z[n2] - z1;
      apply_mic(box, x12double, y12double, z12double);
      float r12[3] = {float(x12double), float(y12double), float(z12double)};
      float d12 = sqrt(r12[0] * r12[0] + r12[1] * r12[1] + r12[2] * r12[2]);
      float d12inv = 1.0f / d12;
      float fc12, fcp12;
      find_fc_and_fcp(paramb.rc_radial, paramb.rcinv_radial, d12, fc12, fcp12);
      float fn12[MAX_NUM_N];
      float fnp12[MAX_NUM_N];
      find_fn_and_fnp(paramb.n_max_radial, paramb.rcinv_radial, d12, fc12, fcp12, fn12, fnp12);
      float f12[3] = {0.0f};
      float f21[3] = {0.0f};
      for (int n = 0; n <= paramb.n_max_radial; ++n) {
        float tmp12 = g_Fp[n1 + n * N] * fnp12[n] * d12inv;
        float tmp21 = g_Fp[n2 + n * N] * fnp12[n] * d12inv;
        tmp12 *= (paramb.num_types == 1)
                   ? 1.0f
                   : annmb.c[(n * paramb.num_types + t1) * paramb.num_types + t2];
        tmp21 *= (paramb.num_types == 1)
                   ? 1.0f
                   : annmb.c[(n * paramb.num_types + t2) * paramb.num_types + t1];
        for (int d = 0; d < 3; ++d) {
          f12[d] += tmp12 * r12[d];
          f21[d] -= tmp21 * r12[d];
        }
      }
      s_fx += f12[0] - f21[0];
      s_fy += f12[1] - f21[1];
      s_fz += f12[2] - f21[2];
      s_sxx += r12[0] * f21[0];
      s_sxy += r12[0] * f21[1];
      s_sxz += r12[0] * f21[2];
      s_syx += r12[1] * f21[0];
      s_syy += r12[1] * f21[1];
      s_syz += r12[1] * f21[2];
      s_szx += r12[2] * f21[0];
      s_szy += r12[2] * f21[1];
      s_szz += r12[2] * f21[2];
    }
    g_fx[n1] += s_fx;
    g_fy[n1] += s_fy;
    g_fz[n1] += s_fz;
    // save virial
    // xx xy xz    0 3 4
    // yx yy yz    6 1 5
    // zx zy zz    7 8 2
    g_virial[n1 + 0 * N] += s_sxx;
    g_virial[n1 + 1 * N] += s_syy;
    g_virial[n1 + 2 * N] += s_szz;
    g_virial[n1 + 3 * N] += s_sxy;
    g_virial[n1 + 4 * N] += s_sxz;
    g_virial[n1 + 5 * N] += s_syz;
    g_virial[n1 + 6 * N] += s_syx;
    g_virial[n1 + 7 * N] += s_szx;
    g_virial[n1 + 8 * N] += s_szy;
  }
}

static __global__ void find_partial_force_angular(
  NEP2::ParaMB paramb,
  NEP2::ANN annmb,
  const int N,
  const int N1,
  const int N2,
  const Box box,
  const int* g_NN_angular,
  const int* g_NL_angular,
  const int* __restrict__ g_type,
  const double* __restrict__ g_x,
  const double* __restrict__ g_y,
  const double* __restrict__ g_z,
  const float* __restrict__ g_Fp,
  const float* __restrict__ g_sum_fxyz,
  double* g_f12x,
  double* g_f12y,
  double* g_f12z)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1;
  if (n1 < N2) {

    float Fp[MAX_DIM_ANGULAR] = {0.0f};
    float sum_fxyz[NUM_OF_ABC * MAX_NUM_N];
    for (int d = 0; d < (paramb.n_max_angular + 1) * paramb.L_max; ++d) {
      Fp[d] = g_Fp[(paramb.n_max_radial + 1 + d) * N + n1];
    }
    for (int d = 0; d < (paramb.n_max_angular + 1) * NUM_OF_ABC; ++d) {
      sum_fxyz[d] = g_sum_fxyz[d * N + n1];
    }

    int t1 = g_type[n1];
    double x1 = g_x[n1];
    double y1 = g_y[n1];
    double z1 = g_z[n1];
    for (int i1 = 0; i1 < g_NN_angular[n1]; ++i1) {
      int index = i1 * N + n1;
      int n2 = g_NL_angular[n1 + N * i1];
      double x12double = g_x[n2] - x1;
      double y12double = g_y[n2] - y1;
      double z12double = g_z[n2] - z1;
      apply_mic(box, x12double, y12double, z12double);
      float r12[3] = {float(x12double), float(y12double), float(z12double)};
      float d12 = sqrt(r12[0] * r12[0] + r12[1] * r12[1] + r12[2] * r12[2]);
      float fc12, fcp12;
      find_fc_and_fcp(paramb.rc_angular, paramb.rcinv_angular, d12, fc12, fcp12);
      int t2 = g_type[n2];
      float f12[3] = {0.0f};
      for (int n = 0; n <= paramb.n_max_angular; ++n) {
        float fn;
        float fnp;
        find_fn_and_fnp(n, paramb.rcinv_angular, d12, fc12, fcp12, fn, fnp);
        const float c =
          (paramb.num_types == 1)
            ? 1.0f
            : annmb
                .c[((paramb.n_max_radial + 1 + n) * paramb.num_types + t1) * paramb.num_types + t2];
        fn *= c;
        fnp *= c;
        accumulate_f12(
          n, n1, paramb.n_max_radial + 1, paramb.n_max_angular + 1, d12, r12, fn, fnp, Fp, sum_fxyz,
          f12);
      }
      g_f12x[index] = f12[0] * 2.0f;
      g_f12y[index] = f12[1] * 2.0f;
      g_f12z[index] = f12[2] * 2.0f;
    }
  }
}

#ifdef USE_ZBL
static __device__ void find_phi_and_phip(float a, float b, float x, float& phi, float& phip)
{
  phi = a * exp(-b * x);
  phip = -b * phi;
}

static __device__ void find_f_and_fp(float d12, float& f, float& fp)
{
  float d12inv = 1 / d12;
  float d12inv_p = -1 / (d12 * d12);
  float Zbl_para[8] = {0.18175, 3.1998, 0.50986, 0.94229, 0.28022, 0.4029, 0.02817, 0.20162};
  float Z = 26;
  float e = 1.6023 * 1e-19;
  float ep = 8.8542 * 1e-12;
  double a = 0.46848 / (2 * powf(Z, 0.23f));
  float x = d12 / a;
  float A = (1 / (4 * 3.1415927f * ep) * Z * Z * e * e) * 1e10;
  float phi[4], phip[4];
  find_phi_and_phip(Zbl_para[0], Zbl_para[1], x, phi[0], phip[0]);
  find_phi_and_phip(Zbl_para[2], Zbl_para[3], x, phi[1], phip[1]);
  find_phi_and_phip(Zbl_para[4], Zbl_para[5], x, phi[2], phip[2]);
  find_phi_and_phip(Zbl_para[6], Zbl_para[7], x, phi[3], phip[3]);
  float PHI = phi[0] + phi[1] + phi[2] + phi[3];
  float PHIP = (phip[0] + phip[1] + phip[2] + phip[3]) / a;
  float fc, fcp;
  float r1 = 1.0;
  float r2 = 2.2;
  find_fc_and_fcp_zbl(r1, r2, d12, fc, fcp);
  f = fc * A * PHI * d12inv;
  fp = A * (fcp * PHI * d12inv + fc * PHIP * d12inv + fc * PHI * d12inv_p);
}

static __global__ void find_force_ZBL(
  const int N,
  const int N1,
  const int N2,
  const Box box,
  const int* g_NN,
  const int* g_NL,
  const double* __restrict__ g_x,
  const double* __restrict__ g_y,
  const double* __restrict__ g_z,
  double* g_fx,
  double* g_fy,
  double* g_fz,
  double* g_virial,
  double* g_pe)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1;
  if (n1 < N2) {
    float s_pe = 0.0f;
    float s_fx = 0.0f;
    float s_fy = 0.0f;
    float s_fz = 0.0f;
    float s_sxx = 0.0f;
    float s_sxy = 0.0f;
    float s_sxz = 0.0f;
    float s_syx = 0.0f;
    float s_syy = 0.0f;
    float s_syz = 0.0f;
    float s_szx = 0.0f;
    float s_szy = 0.0f;
    float s_szz = 0.0f;
    double x1 = g_x[n1];
    double y1 = g_y[n1];
    double z1 = g_z[n1];
    for (int i1 = 0; i1 < g_NN[n1]; ++i1) {
      int n2 = g_NL[n1 + N * i1];
      double x12double = g_x[n2] - x1;
      double y12double = g_y[n2] - y1;
      double z12double = g_z[n2] - z1;
      apply_mic(box, x12double, y12double, z12double);
      float r12[3] = {float(x12double), float(y12double), float(z12double)};
      float d12 = sqrt(r12[0] * r12[0] + r12[1] * r12[1] + r12[2] * r12[2]);
      float d12inv = 1.0f / d12;
      float f, fp;
      find_f_and_fp(d12, f, fp);
      float f2 = fp * d12inv * 0.5f;
      float f12[3] = {r12[0] * f2, r12[1] * f2, r12[2] * f2};
      float f21[3] = {-r12[0] * f2, -r12[1] * f2, -r12[2] * f2};
      s_fx += f12[0] - f21[0];
      s_fy += f12[1] - f21[1];
      s_fz += f12[2] - f21[2];
      s_sxx -= r12[0] * f12[0];
      s_sxy -= r12[0] * f12[1];
      s_sxz -= r12[0] * f12[2];
      s_syx -= r12[1] * f12[0];
      s_syy -= r12[1] * f12[1];
      s_syz -= r12[1] * f12[2];
      s_szx -= r12[2] * f12[0];
      s_szy -= r12[2] * f12[1];
      s_szz -= r12[2] * f12[2];
      s_pe += f * 0.5f;
    }
    g_fx[n1] += s_fx;
    g_fy[n1] += s_fy;
    g_fz[n1] += s_fz;
    g_virial[n1 + 0 * N] += s_sxx;
    g_virial[n1 + 1 * N] += s_syy;
    g_virial[n1 + 2 * N] += s_szz;
    g_virial[n1 + 3 * N] += s_sxy;
    g_virial[n1 + 4 * N] += s_sxz;
    g_virial[n1 + 5 * N] += s_syz;
    g_virial[n1 + 6 * N] += s_syx;
    g_virial[n1 + 7 * N] += s_szx;
    g_virial[n1 + 8 * N] += s_szy;
    g_pe[n1] += s_pe;
  }
}

#endif

void NEP2::compute(
  const int type_shift,
  const Box& box,
  const Neighbor& neighbor,
  const GPU_Vector<int>& type,
  const GPU_Vector<double>& position_per_atom,
  GPU_Vector<double>& potential_per_atom,
  GPU_Vector<double>& force_per_atom,
  GPU_Vector<double>& virial_per_atom)
{
  const int BLOCK_SIZE = 64;
  const int N = type.size();
  const int grid_size = (N2 - N1 - 1) / BLOCK_SIZE + 1;

  find_neighbor_angular<<<grid_size, BLOCK_SIZE>>>(
    paramb, N, N1, N2, box, neighbor.NN_local.data(), neighbor.NL_local.data(),
    position_per_atom.data(), position_per_atom.data() + N, position_per_atom.data() + N * 2,
    nep_data.NN.data(), nep_data.NL.data());
  CUDA_CHECK_KERNEL

  find_descriptor<<<grid_size, BLOCK_SIZE>>>(
    paramb, annmb, N, N1, N2, box, neighbor.NN_local.data(), neighbor.NL_local.data(),
    nep_data.NN.data(), nep_data.NL.data(), type.data(), position_per_atom.data(),
    position_per_atom.data() + N, position_per_atom.data() + N * 2, potential_per_atom.data(),
    nep_data.Fp.data(), nep_data.sum_fxyz.data());
  CUDA_CHECK_KERNEL

  find_force_radial<<<grid_size, BLOCK_SIZE>>>(
    paramb, annmb, N, N1, N2, box, neighbor.NN_local.data(), neighbor.NL_local.data(), type.data(),
    position_per_atom.data(), position_per_atom.data() + N, position_per_atom.data() + N * 2,
    nep_data.Fp.data(), force_per_atom.data(), force_per_atom.data() + N,
    force_per_atom.data() + N * 2, virial_per_atom.data());
  CUDA_CHECK_KERNEL

  find_partial_force_angular<<<grid_size, BLOCK_SIZE>>>(
    paramb, annmb, N, N1, N2, box, nep_data.NN.data(), nep_data.NL.data(), type.data(),
    position_per_atom.data(), position_per_atom.data() + N, position_per_atom.data() + N * 2,
    nep_data.Fp.data(), nep_data.sum_fxyz.data(), nep_data.f12x.data(), nep_data.f12y.data(),
    nep_data.f12z.data());
  CUDA_CHECK_KERNEL
  find_properties_many_body(
    box, nep_data.NN.data(), nep_data.NL.data(), nep_data.f12x.data(), nep_data.f12y.data(),
    nep_data.f12z.data(), position_per_atom, force_per_atom, virial_per_atom);
  CUDA_CHECK_KERNEL

#ifdef USE_ZBL
  find_force_ZBL<<<grid_size, BLOCK_SIZE>>>(
    N, N1, N2, box, nep_data.NN.data(), nep_data.NL.data(), position_per_atom.data(),
    position_per_atom.data() + N, position_per_atom.data() + N * 2, force_per_atom.data(),
    force_per_atom.data() + N, force_per_atom.data() + N * 2, virial_per_atom.data(),
    potential_per_atom.data());
  CUDA_CHECK_KERNEL
#endif
}
