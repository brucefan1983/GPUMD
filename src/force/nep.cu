#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

/*----------------------------------------------------------------------------80
The neuroevolution potential (NEP)
Ref: Zheyong Fan et al., Neuroevolution machine learning potentials:
Combining high accuracy and low cost in atomistic simulations and application to
heat transport, Phys. Rev. B. 104, 104309 (2021).
------------------------------------------------------------------------------*/

#include "nep.cuh"
#include "nep_small_box.cuh"
#include "utilities/common.cuh"
#include "utilities/error.cuh"
#include "utilities/nep_utilities.cuh"
#include <string>
#include <vector>

const int NUM_ELEMENTS = 103;
const std::string ELEMENTS[NUM_ELEMENTS] = {
  "H",  "He", "Li", "Be", "B",  "C",  "N",  "O",  "F",  "Ne", "Na", "Mg", "Al", "Si", "P",
  "S",  "Cl", "Ar", "K",  "Ca", "Sc", "Ti", "V",  "Cr", "Mn", "Fe", "Co", "Ni", "Cu", "Zn",
  "Ga", "Ge", "As", "Se", "Br", "Kr", "Rb", "Sr", "Y",  "Zr", "Nb", "Mo", "Tc", "Ru", "Rh",
  "Pd", "Ag", "Cd", "In", "Sn", "Sb", "Te", "I",  "Xe", "Cs", "Ba", "La", "Ce", "Pr", "Nd",
  "Pm", "Sm", "Eu", "Gd", "Tb", "Dy", "Ho", "Er", "Tm", "Yb", "Lu", "Hf", "Ta", "W",  "Re",
  "Os", "Ir", "Pt", "Au", "Hg", "Tl", "Pb", "Bi", "Po", "At", "Rn", "Fr", "Ra", "Ac", "Th",
  "Pa", "U",  "Np", "Pu", "Am", "Cm", "Bk", "Cf", "Es", "Fm", "Md", "No", "Lr"};

NEP2::NEP2(FILE* fid, char* input_dir, int num_types, bool enable_zbl, const Neighbor& neighbor)
{
  if (num_types == 1) {
    printf("Use the NEP potential with %d atom type.\n", num_types);
  } else {
    printf("Use the NEP potential with %d atom types.\n", num_types);
  }

  char name[20];

  for (int n = 0; n < num_types; ++n) {
    int count = fscanf(fid, "%s", name);
    PRINT_SCANF_ERROR(count, 1, "reading error for NEP potential.");
    std::string element(name);
    int atomic_number = 0;
    for (int m = 0; m < NUM_ELEMENTS; ++m) {
      if (element == ELEMENTS[m]) {
        atomic_number = m + 1;
        break;
      }
    }
    zbl.atomic_numbers[n] = atomic_number;
    printf("    type %d (%s with Z = %g).\n", n, name, zbl.atomic_numbers[n]);
  }

  paramb.num_types = num_types;

  if (enable_zbl) {
    int count = fscanf(fid, "%s%f%f", name, &zbl.rc_inner, &zbl.rc_outer);
    PRINT_SCANF_ERROR(count, 3, "reading error for NEP potential.");
    zbl.enabled = true;
    printf(
      "    has ZBL with inner cutoff %g A and outer cutoff %g A.\n", zbl.rc_inner, zbl.rc_outer);
  }

  int count = fscanf(fid, "%s%f%f", name, &paramb.rc_radial, &paramb.rc_angular);
  PRINT_SCANF_ERROR(count, 3, "reading error for NEP potential.");
  printf("    radial cutoff = %g A.\n", paramb.rc_radial);
  printf("    angular cutoff = %g A.\n", paramb.rc_angular);

  count = fscanf(fid, "%s%d%d", name, &paramb.n_max_radial, &paramb.n_max_angular);
  PRINT_SCANF_ERROR(count, 3, "reading error for NEP potential.");
  printf("    n_max_radial = %d.\n", paramb.n_max_radial);
  printf("    n_max_angular = %d.\n", paramb.n_max_angular);

  count = fscanf(fid, "%s%d", name, &paramb.L_max);
  PRINT_SCANF_ERROR(count, 2, "reading error for NEP potential.");
  printf("    l_max = %d.\n", paramb.L_max);

  paramb.num_Alm_products = (paramb.L_max + 1) * (paramb.L_max + 1) - 1; // 0,3,8,15,24

  int num_neurons2;
  count = fscanf(fid, "%s%d%d", name, &annmb.num_neurons1, &num_neurons2);
  PRINT_SCANF_ERROR(count, 3, "reading error for NEP potential.");

  rc = paramb.rc_radial; // largest cutoff

  paramb.rcinv_radial = 1.0f / paramb.rc_radial;
  paramb.rcinv_angular = 1.0f / paramb.rc_angular;
  annmb.dim = (paramb.n_max_radial + 1) + (paramb.n_max_angular + 1) * paramb.L_max;

  printf("    ANN = %d-%d-1.\n", annmb.dim, annmb.num_neurons1);

  annmb.num_para = (annmb.dim + 2) * annmb.num_neurons1 + 1;
  printf("    number of neural network parameters = %d.\n", annmb.num_para);
  int num_para_descriptor =
    (num_types == 1) ? 0 : num_types * num_types * (paramb.n_max_radial + paramb.n_max_angular + 2);
  printf("    number of descriptor parameters = %d.\n", num_para_descriptor);
  annmb.num_para += num_para_descriptor;
  printf("    total number of parameters = %d\n", annmb.num_para);

  float rc_factor = paramb.rc_angular / paramb.rc_radial;
  int angular_neighbor_size = int(ceil(neighbor.MN * rc_factor * rc_factor));
  nep_data.f12x.resize(neighbor.NN.size() * angular_neighbor_size);
  nep_data.f12y.resize(neighbor.NN.size() * angular_neighbor_size);
  nep_data.f12z.resize(neighbor.NN.size() * angular_neighbor_size);
  nep_data.NN.resize(neighbor.NN.size());
  nep_data.NL.resize(neighbor.NN.size() * angular_neighbor_size);
  nep_data.Fp.resize(neighbor.NN.size() * annmb.dim);
  nep_data.sum_fxyz.resize(
    neighbor.NN.size() * (paramb.n_max_angular + 1) * paramb.num_Alm_products);
  nep_data.parameters.resize(annmb.num_para);

  update_potential(fid);
}

NEP2::~NEP2(void)
{
  // nothing
}

void NEP2::update_potential(const float* parameters, ANN& ann)
{
  ann.w0 = parameters;
  ann.b0 = ann.w0 + ann.num_neurons1 * ann.dim;
  ann.w1 = ann.b0 + ann.num_neurons1;
  ann.b1 = ann.w1 + ann.num_neurons1;
  if (paramb.num_types > 1) {
    ann.c = ann.b1 + 1;
  }
}

void NEP2::update_potential(FILE* fid)
{
  std::vector<float> parameters(annmb.num_para);
  for (int n = 0; n < annmb.num_para; ++n) {
    int count = fscanf(fid, "%f", &parameters[n]);
    PRINT_SCANF_ERROR(count, 1, "reading error for NEP potential.");
  }
  nep_data.parameters.copy_from_host(parameters.data());
  update_potential(nep_data.parameters.data(), annmb);

  for (int d = 0; d < annmb.dim; ++d) {
    int count = fscanf(fid, "%f", &paramb.q_scaler[d]);
    PRINT_SCANF_ERROR(count, 1, "reading error for NEP potential.");
  }
}

static __global__ void find_neighbor_angular(
  NEP2::ParaMB paramb,
  const int N,
  const int N1,
  const int N2,
  const Box box,
  const int* g_NN,
  const int* g_NL,
  const double* __restrict__ g_x,
  const double* __restrict__ g_y,
  const double* __restrict__ g_z,
  int* g_NN_angular,
  int* g_NL_angular)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1;
  if (n1 < N2) {
    double x1 = g_x[n1];
    double y1 = g_y[n1];
    double z1 = g_z[n1];
    int count = 0;
    for (int i1 = 0; i1 < g_NN[n1]; ++i1) {
      int n2 = g_NL[n1 + N * i1];
      double x12double = g_x[n2] - x1;
      double y12double = g_y[n2] - y1;
      double z12double = g_z[n2] - z1;
      apply_mic(box, x12double, y12double, z12double);
      float x12 = float(x12double), y12 = float(y12double), z12 = float(z12double);
      float d12sq = x12 * x12 + y12 * y12 + z12 * z12;
      if (d12sq < paramb.rc_angular * paramb.rc_angular) {
        g_NL_angular[count++ * N + n1] = n2;
      }
    }
    g_NN_angular[n1] = count;
  }
}

static __global__ void find_descriptor(
  NEP2::ParaMB paramb,
  NEP2::ANN annmb,
  const int N,
  const int N1,
  const int N2,
  const Box box,
  const int* g_NN,
  const int* g_NL,
  const int* g_NN_angular,
  const int* g_NL_angular,
  const int* __restrict__ g_type,
  const double* __restrict__ g_x,
  const double* __restrict__ g_y,
  const double* __restrict__ g_z,
  double* g_pe,
  float* g_Fp,
  float* g_sum_fxyz)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1;
  if (n1 < N2) {
    int t1 = g_type[n1];
    double x1 = g_x[n1];
    double y1 = g_y[n1];
    double z1 = g_z[n1];
    float q[MAX_DIM] = {0.0f};

    // get radial descriptors
    for (int i1 = 0; i1 < g_NN[n1]; ++i1) {
      int n2 = g_NL[n1 + N * i1];
      double x12double = g_x[n2] - x1;
      double y12double = g_y[n2] - y1;
      double z12double = g_z[n2] - z1;
      apply_mic(box, x12double, y12double, z12double);
      float x12 = float(x12double), y12 = float(y12double), z12 = float(z12double);
      float d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12);
      float fc12;
      find_fc(paramb.rc_radial, paramb.rcinv_radial, d12, fc12);
      int t2 = g_type[n2];
      float fn12[MAX_NUM_N];
      find_fn(paramb.n_max_radial, paramb.rcinv_radial, d12, fc12, fn12);
      for (int n = 0; n <= paramb.n_max_radial; ++n) {
        float c = (paramb.num_types == 1)
                    ? 1.0f
                    : annmb.c[(n * paramb.num_types + t1) * paramb.num_types + t2];
        q[n] += fn12[n] * c;
      }
    }

    // get angular descriptors
    for (int n = 0; n <= paramb.n_max_angular; ++n) {
      float s[NUM_OF_ABC] = {0.0f};
      for (int i1 = 0; i1 < g_NN_angular[n1]; ++i1) {
        int n2 = g_NL_angular[n1 + N * i1];
        double x12double = g_x[n2] - x1;
        double y12double = g_y[n2] - y1;
        double z12double = g_z[n2] - z1;
        apply_mic(box, x12double, y12double, z12double);
        float x12 = float(x12double), y12 = float(y12double), z12 = float(z12double);
        float d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12);
        float fc12;
        find_fc(paramb.rc_angular, paramb.rcinv_angular, d12, fc12);
        int t2 = g_type[n2];
        float fn;
        find_fn(n, paramb.rcinv_angular, d12, fc12, fn);
        fn *=
          (paramb.num_types == 1)
            ? 1.0f
            : annmb
                .c[((paramb.n_max_radial + 1 + n) * paramb.num_types + t1) * paramb.num_types + t2];
        accumulate_s(paramb.L_max, d12, x12, y12, z12, fn, s);
      }
      find_q(paramb.L_max, paramb.n_max_angular + 1, n, s, q + (paramb.n_max_radial + 1));
      for (int abc = 0; abc < paramb.num_Alm_products; ++abc) {
        g_sum_fxyz[(n * paramb.num_Alm_products + abc) * N + n1] = s[abc] * YLM[abc];
      }
    }

    // nomalize descriptor
    for (int d = 0; d < annmb.dim; ++d) {
      q[d] = q[d] * paramb.q_scaler[d];
    }

    // get energy and energy gradient
    float F = 0.0f, Fp[MAX_DIM] = {0.0f};
    apply_ann_one_layer(
      annmb.dim, annmb.num_neurons1, annmb.w0, annmb.b0, annmb.w1, annmb.b1, q, F, Fp);
    g_pe[n1] += F;

    for (int d = 0; d < annmb.dim; ++d) {
      g_Fp[d * N + n1] = Fp[d] * paramb.q_scaler[d];
    }
  }
}

static __global__ void find_force_radial(
  NEP2::ParaMB paramb,
  NEP2::ANN annmb,
  const int N,
  const int N1,
  const int N2,
  const Box box,
  const int* g_NN,
  const int* g_NL,
  const int* __restrict__ g_type,
  const double* __restrict__ g_x,
  const double* __restrict__ g_y,
  const double* __restrict__ g_z,
  const float* __restrict__ g_Fp,
  double* g_fx,
  double* g_fy,
  double* g_fz,
  double* g_virial)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1;
  if (n1 < N2) {
    int t1 = g_type[n1];
    float s_fx = 0.0f;
    float s_fy = 0.0f;
    float s_fz = 0.0f;
    float s_sxx = 0.0f;
    float s_sxy = 0.0f;
    float s_sxz = 0.0f;
    float s_syx = 0.0f;
    float s_syy = 0.0f;
    float s_syz = 0.0f;
    float s_szx = 0.0f;
    float s_szy = 0.0f;
    float s_szz = 0.0f;
    double x1 = g_x[n1];
    double y1 = g_y[n1];
    double z1 = g_z[n1];
    for (int i1 = 0; i1 < g_NN[n1]; ++i1) {
      int n2 = g_NL[n1 + N * i1];
      int t2 = g_type[n2];
      double x12double = g_x[n2] - x1;
      double y12double = g_y[n2] - y1;
      double z12double = g_z[n2] - z1;
      apply_mic(box, x12double, y12double, z12double);
      float r12[3] = {float(x12double), float(y12double), float(z12double)};
      float d12 = sqrt(r12[0] * r12[0] + r12[1] * r12[1] + r12[2] * r12[2]);
      float d12inv = 1.0f / d12;
      float fc12, fcp12;
      find_fc_and_fcp(paramb.rc_radial, paramb.rcinv_radial, d12, fc12, fcp12);
      float fn12[MAX_NUM_N];
      float fnp12[MAX_NUM_N];
      find_fn_and_fnp(paramb.n_max_radial, paramb.rcinv_radial, d12, fc12, fcp12, fn12, fnp12);
      float f12[3] = {0.0f};
      float f21[3] = {0.0f};
      for (int n = 0; n <= paramb.n_max_radial; ++n) {
        float tmp12 = g_Fp[n1 + n * N] * fnp12[n] * d12inv;
        float tmp21 = g_Fp[n2 + n * N] * fnp12[n] * d12inv;
        tmp12 *= (paramb.num_types == 1)
                   ? 1.0f
                   : annmb.c[(n * paramb.num_types + t1) * paramb.num_types + t2];
        tmp21 *= (paramb.num_types == 1)
                   ? 1.0f
                   : annmb.c[(n * paramb.num_types + t2) * paramb.num_types + t1];
        for (int d = 0; d < 3; ++d) {
          f12[d] += tmp12 * r12[d];
          f21[d] -= tmp21 * r12[d];
        }
      }
      s_fx += f12[0] - f21[0];
      s_fy += f12[1] - f21[1];
      s_fz += f12[2] - f21[2];
      s_sxx += r12[0] * f21[0];
      s_sxy += r12[0] * f21[1];
      s_sxz += r12[0] * f21[2];
      s_syx += r12[1] * f21[0];
      s_syy += r12[1] * f21[1];
      s_syz += r12[1] * f21[2];
      s_szx += r12[2] * f21[0];
      s_szy += r12[2] * f21[1];
      s_szz += r12[2] * f21[2];
    }
    g_fx[n1] += s_fx;
    g_fy[n1] += s_fy;
    g_fz[n1] += s_fz;
    // save virial
    // xx xy xz    0 3 4
    // yx yy yz    6 1 5
    // zx zy zz    7 8 2
    g_virial[n1 + 0 * N] += s_sxx;
    g_virial[n1 + 1 * N] += s_syy;
    g_virial[n1 + 2 * N] += s_szz;
    g_virial[n1 + 3 * N] += s_sxy;
    g_virial[n1 + 4 * N] += s_sxz;
    g_virial[n1 + 5 * N] += s_syz;
    g_virial[n1 + 6 * N] += s_syx;
    g_virial[n1 + 7 * N] += s_szx;
    g_virial[n1 + 8 * N] += s_szy;
  }
}

static __global__ void find_partial_force_angular(
  NEP2::ParaMB paramb,
  NEP2::ANN annmb,
  const int N,
  const int N1,
  const int N2,
  const Box box,
  const int* g_NN_angular,
  const int* g_NL_angular,
  const int* __restrict__ g_type,
  const double* __restrict__ g_x,
  const double* __restrict__ g_y,
  const double* __restrict__ g_z,
  const float* __restrict__ g_Fp,
  const float* __restrict__ g_sum_fxyz,
  double* g_f12x,
  double* g_f12y,
  double* g_f12z)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1;
  if (n1 < N2) {

    float Fp[MAX_DIM_ANGULAR] = {0.0f};
    float sum_fxyz[NUM_OF_ABC * MAX_NUM_N];
    for (int d = 0; d < (paramb.n_max_angular + 1) * paramb.L_max; ++d) {
      Fp[d] = g_Fp[(paramb.n_max_radial + 1 + d) * N + n1];
    }
    for (int d = 0; d < (paramb.n_max_angular + 1) * paramb.num_Alm_products; ++d) {
      sum_fxyz[d] = g_sum_fxyz[d * N + n1];
    }

    int t1 = g_type[n1];
    double x1 = g_x[n1];
    double y1 = g_y[n1];
    double z1 = g_z[n1];
    for (int i1 = 0; i1 < g_NN_angular[n1]; ++i1) {
      int index = i1 * N + n1;
      int n2 = g_NL_angular[n1 + N * i1];
      double x12double = g_x[n2] - x1;
      double y12double = g_y[n2] - y1;
      double z12double = g_z[n2] - z1;
      apply_mic(box, x12double, y12double, z12double);
      float r12[3] = {float(x12double), float(y12double), float(z12double)};
      float d12 = sqrt(r12[0] * r12[0] + r12[1] * r12[1] + r12[2] * r12[2]);
      float fc12, fcp12;
      find_fc_and_fcp(paramb.rc_angular, paramb.rcinv_angular, d12, fc12, fcp12);
      int t2 = g_type[n2];
      float f12[3] = {0.0f};
      for (int n = 0; n <= paramb.n_max_angular; ++n) {
        float fn;
        float fnp;
        find_fn_and_fnp(n, paramb.rcinv_angular, d12, fc12, fcp12, fn, fnp);
        const float c =
          (paramb.num_types == 1)
            ? 1.0f
            : annmb
                .c[((paramb.n_max_radial + 1 + n) * paramb.num_types + t1) * paramb.num_types + t2];
        fn *= c;
        fnp *= c;
        accumulate_f12(
          n, paramb.L_max, paramb.n_max_angular + 1, d12, r12, fn, fnp, Fp, sum_fxyz, f12);
      }
      g_f12x[index] = f12[0] * 2.0f;
      g_f12y[index] = f12[1] * 2.0f;
      g_f12z[index] = f12[2] * 2.0f;
    }
  }
}

static __global__ void find_force_ZBL(
  const int N,
  const NEP2::ZBL zbl,
  const int N1,
  const int N2,
  const Box box,
  const int* g_NN,
  const int* g_NL,
  const int* __restrict__ g_type,
  const double* __restrict__ g_x,
  const double* __restrict__ g_y,
  const double* __restrict__ g_z,
  double* g_fx,
  double* g_fy,
  double* g_fz,
  double* g_virial,
  double* g_pe)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1;
  if (n1 < N2) {
    float s_pe = 0.0f;
    float s_fx = 0.0f;
    float s_fy = 0.0f;
    float s_fz = 0.0f;
    float s_sxx = 0.0f;
    float s_sxy = 0.0f;
    float s_sxz = 0.0f;
    float s_syx = 0.0f;
    float s_syy = 0.0f;
    float s_syz = 0.0f;
    float s_szx = 0.0f;
    float s_szy = 0.0f;
    float s_szz = 0.0f;
    double x1 = g_x[n1];
    double y1 = g_y[n1];
    double z1 = g_z[n1];
    float zi = zbl.atomic_numbers[g_type[n1]];
    float pow_zi = pow(zi, 0.23f);
    for (int i1 = 0; i1 < g_NN[n1]; ++i1) {
      int n2 = g_NL[n1 + N * i1];
      double x12double = g_x[n2] - x1;
      double y12double = g_y[n2] - y1;
      double z12double = g_z[n2] - z1;
      apply_mic(box, x12double, y12double, z12double);
      float r12[3] = {float(x12double), float(y12double), float(z12double)};
      float d12 = sqrt(r12[0] * r12[0] + r12[1] * r12[1] + r12[2] * r12[2]);
      float d12inv = 1.0f / d12;
      float f, fp;
      float zj = zbl.atomic_numbers[g_type[n2]];
      float a_inv = (pow_zi + pow(zj, 0.23f)) * 2.134563f;
      float zizj = K_C_SP * zi * zj;
      find_f_and_fp_zbl(zizj, a_inv, zbl.rc_inner, zbl.rc_outer, d12, d12inv, f, fp);
      float f2 = fp * d12inv * 0.5f;
      float f12[3] = {r12[0] * f2, r12[1] * f2, r12[2] * f2};
      float f21[3] = {-r12[0] * f2, -r12[1] * f2, -r12[2] * f2};
      s_fx += f12[0] - f21[0];
      s_fy += f12[1] - f21[1];
      s_fz += f12[2] - f21[2];
      s_sxx -= r12[0] * f12[0];
      s_sxy -= r12[0] * f12[1];
      s_sxz -= r12[0] * f12[2];
      s_syx -= r12[1] * f12[0];
      s_syy -= r12[1] * f12[1];
      s_syz -= r12[1] * f12[2];
      s_szx -= r12[2] * f12[0];
      s_szy -= r12[2] * f12[1];
      s_szz -= r12[2] * f12[2];
      s_pe += f * 0.5f;
    }
    g_fx[n1] += s_fx;
    g_fy[n1] += s_fy;
    g_fz[n1] += s_fz;
    g_virial[n1 + 0 * N] += s_sxx;
    g_virial[n1 + 1 * N] += s_syy;
    g_virial[n1 + 2 * N] += s_szz;
    g_virial[n1 + 3 * N] += s_sxy;
    g_virial[n1 + 4 * N] += s_sxz;
    g_virial[n1 + 5 * N] += s_syz;
    g_virial[n1 + 6 * N] += s_syx;
    g_virial[n1 + 7 * N] += s_szx;
    g_virial[n1 + 8 * N] += s_szy;
    g_pe[n1] += s_pe;
  }
}

// large box fo MD applications
void NEP2::compute_large_box(
  const int type_shift,
  const Box& box,
  const Neighbor& neighbor,
  const GPU_Vector<int>& type,
  const GPU_Vector<double>& position_per_atom,
  GPU_Vector<double>& potential_per_atom,
  GPU_Vector<double>& force_per_atom,
  GPU_Vector<double>& virial_per_atom)
{
  const int BLOCK_SIZE = 64;
  const int N = type.size();
  const int grid_size = (N2 - N1 - 1) / BLOCK_SIZE + 1;

  find_neighbor_angular<<<grid_size, BLOCK_SIZE>>>(
    paramb, N, N1, N2, box, neighbor.NN_local.data(), neighbor.NL_local.data(),
    position_per_atom.data(), position_per_atom.data() + N, position_per_atom.data() + N * 2,
    nep_data.NN.data(), nep_data.NL.data());
  CUDA_CHECK_KERNEL

  find_descriptor<<<grid_size, BLOCK_SIZE>>>(
    paramb, annmb, N, N1, N2, box, neighbor.NN_local.data(), neighbor.NL_local.data(),
    nep_data.NN.data(), nep_data.NL.data(), type.data(), position_per_atom.data(),
    position_per_atom.data() + N, position_per_atom.data() + N * 2, potential_per_atom.data(),
    nep_data.Fp.data(), nep_data.sum_fxyz.data());
  CUDA_CHECK_KERNEL

  find_force_radial<<<grid_size, BLOCK_SIZE>>>(
    paramb, annmb, N, N1, N2, box, neighbor.NN_local.data(), neighbor.NL_local.data(), type.data(),
    position_per_atom.data(), position_per_atom.data() + N, position_per_atom.data() + N * 2,
    nep_data.Fp.data(), force_per_atom.data(), force_per_atom.data() + N,
    force_per_atom.data() + N * 2, virial_per_atom.data());
  CUDA_CHECK_KERNEL

  find_partial_force_angular<<<grid_size, BLOCK_SIZE>>>(
    paramb, annmb, N, N1, N2, box, nep_data.NN.data(), nep_data.NL.data(), type.data(),
    position_per_atom.data(), position_per_atom.data() + N, position_per_atom.data() + N * 2,
    nep_data.Fp.data(), nep_data.sum_fxyz.data(), nep_data.f12x.data(), nep_data.f12y.data(),
    nep_data.f12z.data());
  CUDA_CHECK_KERNEL
  find_properties_many_body(
    box, nep_data.NN.data(), nep_data.NL.data(), nep_data.f12x.data(), nep_data.f12y.data(),
    nep_data.f12z.data(), position_per_atom, force_per_atom, virial_per_atom);
  CUDA_CHECK_KERNEL

  if (zbl.enabled) {
    find_force_ZBL<<<grid_size, BLOCK_SIZE>>>(
      N, zbl, N1, N2, box, nep_data.NN.data(), nep_data.NL.data(), type.data(),
      position_per_atom.data(), position_per_atom.data() + N, position_per_atom.data() + N * 2,
      force_per_atom.data(), force_per_atom.data() + N, force_per_atom.data() + N * 2,
      virial_per_atom.data(), potential_per_atom.data());
    CUDA_CHECK_KERNEL
  }
}

// small box possibly used for active learning:
void NEP2::compute_small_box(
  const int type_shift,
  const Box& box,
  const Neighbor& neighbor,
  const GPU_Vector<int>& type,
  const GPU_Vector<double>& position_per_atom,
  GPU_Vector<double>& potential_per_atom,
  GPU_Vector<double>& force_per_atom,
  GPU_Vector<double>& virial_per_atom)
{
  const int BLOCK_SIZE = 64;
  const int N = type.size();
  const int grid_size = (N2 - N1 - 1) / BLOCK_SIZE + 1;

  const int size_x12 = neighbor.NL_local.size();
  GPU_Vector<int> NN_radial(neighbor.NN_local.size());
  GPU_Vector<int> NL_radial(size_x12);
  GPU_Vector<float> r12(size_x12 * 6);

  find_neighbor_list_small_box<<<grid_size, BLOCK_SIZE>>>(
    paramb, N, N1, N2, box, ebox, position_per_atom.data(), position_per_atom.data() + N,
    position_per_atom.data() + N * 2, NN_radial.data(), NL_radial.data(), nep_data.NN.data(),
    nep_data.NL.data(), r12.data(), r12.data() + size_x12, r12.data() + size_x12 * 2,
    r12.data() + size_x12 * 3, r12.data() + size_x12 * 4, r12.data() + size_x12 * 5);
  CUDA_CHECK_KERNEL

  find_descriptor_small_box<<<grid_size, BLOCK_SIZE>>>(
    paramb, annmb, N, N1, N2, NN_radial.data(), NL_radial.data(), nep_data.NN.data(),
    nep_data.NL.data(), type.data(), r12.data(), r12.data() + size_x12, r12.data() + size_x12 * 2,
    r12.data() + size_x12 * 3, r12.data() + size_x12 * 4, r12.data() + size_x12 * 5,
    potential_per_atom.data(), nep_data.Fp.data(), nep_data.sum_fxyz.data());
  CUDA_CHECK_KERNEL

  find_force_radial_small_box<<<grid_size, BLOCK_SIZE>>>(
    paramb, annmb, N, N1, N2, NN_radial.data(), NL_radial.data(), type.data(), r12.data(),
    r12.data() + size_x12, r12.data() + size_x12 * 2, nep_data.Fp.data(), force_per_atom.data(),
    force_per_atom.data() + N, force_per_atom.data() + N * 2, virial_per_atom.data());
  CUDA_CHECK_KERNEL

  find_force_angular_small_box<<<grid_size, BLOCK_SIZE>>>(
    paramb, annmb, N, N1, N2, nep_data.NN.data(), nep_data.NL.data(), type.data(),
    r12.data() + size_x12 * 3, r12.data() + size_x12 * 4, r12.data() + size_x12 * 5,
    nep_data.Fp.data(), nep_data.sum_fxyz.data(), force_per_atom.data(), force_per_atom.data() + N,
    force_per_atom.data() + N * 2, virial_per_atom.data());
  CUDA_CHECK_KERNEL

  if (zbl.enabled) {
    find_force_ZBL_small_box<<<grid_size, BLOCK_SIZE>>>(
      N, zbl, N1, N2, nep_data.NN.data(), nep_data.NL.data(), type.data(), r12.data(),
      r12.data() + size_x12, r12.data() + size_x12 * 2, force_per_atom.data(),
      force_per_atom.data() + N, force_per_atom.data() + N * 2, virial_per_atom.data(),
      potential_per_atom.data());
    CUDA_CHECK_KERNEL
  }
}

static void get_expanded_box(const double rc, const Box& box, NEP2::ExpandedBox& ebox)
{
  double volume = box.get_volume();
  double thickness_x = volume / box.get_area(0);
  double thickness_y = volume / box.get_area(1);
  double thickness_z = volume / box.get_area(2);
  ebox.num_cells[0] = int(ceil(2.0 * rc / thickness_x));
  ebox.num_cells[1] = int(ceil(2.0 * rc / thickness_y));
  ebox.num_cells[2] = int(ceil(2.0 * rc / thickness_z));
  if (ebox.num_cells[0] * ebox.num_cells[1] * ebox.num_cells[2] > 1) {
    if (box.triclinic) {
      ebox.h[0] = box.cpu_h[0] * ebox.num_cells[0];
      ebox.h[3] = box.cpu_h[3] * ebox.num_cells[0];
      ebox.h[6] = box.cpu_h[6] * ebox.num_cells[0];
      ebox.h[1] = box.cpu_h[1] * ebox.num_cells[1];
      ebox.h[4] = box.cpu_h[4] * ebox.num_cells[1];
      ebox.h[7] = box.cpu_h[7] * ebox.num_cells[1];
      ebox.h[2] = box.cpu_h[2] * ebox.num_cells[2];
      ebox.h[5] = box.cpu_h[5] * ebox.num_cells[2];
      ebox.h[8] = box.cpu_h[8] * ebox.num_cells[2];

      ebox.h[9] = ebox.h[4] * ebox.h[8] - ebox.h[5] * ebox.h[7];
      ebox.h[10] = ebox.h[2] * ebox.h[7] - ebox.h[1] * ebox.h[8];
      ebox.h[11] = ebox.h[1] * ebox.h[5] - ebox.h[2] * ebox.h[4];
      ebox.h[12] = ebox.h[5] * ebox.h[6] - ebox.h[3] * ebox.h[8];
      ebox.h[13] = ebox.h[0] * ebox.h[8] - ebox.h[2] * ebox.h[6];
      ebox.h[14] = ebox.h[2] * ebox.h[3] - ebox.h[0] * ebox.h[5];
      ebox.h[15] = ebox.h[3] * ebox.h[7] - ebox.h[4] * ebox.h[6];
      ebox.h[16] = ebox.h[1] * ebox.h[6] - ebox.h[0] * ebox.h[7];
      ebox.h[17] = ebox.h[0] * ebox.h[4] - ebox.h[1] * ebox.h[3];
      double det = ebox.h[0] * (ebox.h[4] * ebox.h[8] - ebox.h[5] * ebox.h[7]) +
                   ebox.h[1] * (ebox.h[5] * ebox.h[6] - ebox.h[3] * ebox.h[8]) +
                   ebox.h[2] * (ebox.h[3] * ebox.h[7] - ebox.h[4] * ebox.h[6]);
      for (int n = 9; n < 18; n++) {
        ebox.h[n] /= det;
      }
    } else {
      ebox.h[0] = box.cpu_h[0] * ebox.num_cells[0];
      ebox.h[1] = box.cpu_h[1] * ebox.num_cells[1];
      ebox.h[2] = box.cpu_h[2] * ebox.num_cells[2];
      ebox.h[3] = ebox.h[0] * 0.5;
      ebox.h[4] = ebox.h[1] * 0.5;
      ebox.h[5] = ebox.h[2] * 0.5;
    }
  }
}

void NEP2::compute(
  const int type_shift,
  const Box& box,
  const Neighbor& neighbor,
  const GPU_Vector<int>& type,
  const GPU_Vector<double>& position_per_atom,
  GPU_Vector<double>& potential_per_atom,
  GPU_Vector<double>& force_per_atom,
  GPU_Vector<double>& virial_per_atom)
{
  get_expanded_box(paramb.rc_radial, box, ebox);

  if (ebox.num_cells[0] * ebox.num_cells[1] * ebox.num_cells[2] > 1) {
    compute_small_box(
      type_shift, box, neighbor, type, position_per_atom, potential_per_atom, force_per_atom,
      virial_per_atom);
  } else {
    compute_large_box(
      type_shift, box, neighbor, type, position_per_atom, potential_per_atom, force_per_atom,
      virial_per_atom);
  }
}
