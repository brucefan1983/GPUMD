#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan and GPUMD development team
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

/*----------------------------------------------------------------------------80
The minimal Tersoff potential, as developed in the following paper:

Z. Fan, Y. Wang, X. Gu, P. Qian, Y. Su, and T. Ala-Nissila,
A minimal Tersoff potential for diamond silicon with improved
descriptions of elastic and phonon transport properties,
J. Phys.: Condens. Matter 32, 135901 (2020).
------------------------------------------------------------------------------*/

#include "neighbor.cuh"
#include "tersoff_mini.cuh"
#include "utilities/common.cuh"
#include "utilities/error.cuh"
#include "utilities/gpu_macro.cuh"
#include <cstring>

#define BLOCK_SIZE_FORCE 64

Tersoff_mini::Tersoff_mini(FILE* fid, int num_of_types, const int num_atoms)
{
  num_types = num_of_types;
  printf("Use Tersoff-mini (%d-element) potential with element(s):\n", num_types);
  for (int n = 0; n < num_types; ++n) {
    char atom_symbol[10];
    int count = fscanf(fid, "%s", atom_symbol);
    PRINT_SCANF_ERROR(count, 1, "Reading error for Tersoff-mini potential.");
    printf(" %s", atom_symbol);
  }
  printf("\n");

  int n_entries = 2 * num_types - 1; // 1 or 3 entries

  const char err[] = "Reading error for Tersoff-mini potential.\n";
  rc = 0.0;
  int count;
  double d0, a, r0, s, beta, n, h, r1, r2;
  for (int i = 0; i < n_entries; i++) {
    count = fscanf(fid, "%lf%lf%lf%lf%lf%lf%lf%lf%lf", &d0, &a, &r0, &s, &beta, &n, &h, &r1, &r2);
    PRINT_SCANF_ERROR(count, 9, err);
    if (d0 <= 0.0)
      PRINT_INPUT_ERROR(err);
    if (a <= 0.0)
      PRINT_INPUT_ERROR(err);
    if (r0 <= 0.0)
      PRINT_INPUT_ERROR(err);
    if (beta < 0.0)
      PRINT_INPUT_ERROR(err);
    if (n < 0.0)
      PRINT_INPUT_ERROR(err);
    if (h < -1.0 || h > 1.0)
      PRINT_INPUT_ERROR(err);
    if (r1 < 0.0)
      PRINT_INPUT_ERROR(err);
    if (r2 <= 0.0)
      PRINT_INPUT_ERROR(err);
    if (r2 <= r1)
      PRINT_INPUT_ERROR(err);

    para.a[i] = d0 / (s - 1.0) * exp(sqrt(2.0 * s) * a * r0);
    para.b[i] = s * d0 / (s - 1.0) * exp(sqrt(2.0 / s) * a * r0);
    para.lambda[i] = sqrt(2.0 * s) * a;
    para.mu[i] = sqrt(2.0 / s) * a;
    para.beta[i] = beta;
    para.n[i] = n;
    para.h[i] = h;
    para.r1[i] = r1;
    para.r2[i] = r2;
    para.pi_factor[i] = PI / (r2 - r1);
    para.minus_half_over_n[i] = -0.5 / n;
    rc = r2 > rc ? r2 : rc;
  }

  const int num_of_neighbors = 50 * num_atoms;
  tersoff_mini_data.b.resize(num_of_neighbors);
  tersoff_mini_data.bp.resize(num_of_neighbors);
  tersoff_mini_data.f12x.resize(num_of_neighbors);
  tersoff_mini_data.f12y.resize(num_of_neighbors);
  tersoff_mini_data.f12z.resize(num_of_neighbors);
  tersoff_mini_data.NN.resize(num_atoms);
  tersoff_mini_data.NL.resize(num_of_neighbors);
  tersoff_mini_data.cell_count.resize(num_atoms);
  tersoff_mini_data.cell_count_sum.resize(num_atoms);
  tersoff_mini_data.cell_contents.resize(num_atoms);
}

Tersoff_mini::~Tersoff_mini(void)
{
  // nothing
}

static __device__ void find_fr_and_frp(double a, double lambda, double d12, double& fr, double& frp)
{
  fr = a * exp(-lambda * d12);
  frp = -lambda * fr;
}

static __device__ void find_fa_and_fap(double b, double mu, double d12, double& fa, double& fap)
{
  fa = b * exp(-mu * d12);
  fap = -mu * fa;
}

static __device__ void find_fa(double b, double mu, double d12, double& fa)
{
  fa = b * exp(-mu * d12);
}

static __device__ void
find_fc_and_fcp(double r1, double r2, double pi_factor, double d12, double& fc, double& fcp)
{
  if (d12 < r1) {
    fc = 1.0;
    fcp = 0.0;
  } else if (d12 < r2) {
    fc = 0.5 * cos(pi_factor * (d12 - r1)) + 0.5;
    fcp = -sin(pi_factor * (d12 - r1)) * pi_factor * 0.5;
  } else {
    fc = 0.0;
    fcp = 0.0;
  }
}

static __device__ void find_fc(double r1, double r2, double pi_factor, double d12, double& fc)
{
  if (d12 < r1) {
    fc = 1.0;
  } else if (d12 < r2) {
    fc = 0.5 * cos(pi_factor * (d12 - r1)) + 0.5;
  } else {
    fc = 0.0;
  }
}

static __device__ void find_g_and_gp(double h, double cos, double& g, double& gp)
{
  double tmp = cos - h;
  g = tmp * tmp;
  gp = 2.0 * tmp;
}

static __device__ void find_g(double h, double cos, double& g)
{
  double tmp = cos - h;
  g = tmp * tmp;
}

// step 1: pre-compute all the bond-order functions and their derivatives
static __global__ void find_force_step1(
  const int number_of_particles,
  const int N1,
  const int N2,
  const Box box,
  const int num_types,
  const int* g_neighbor_number,
  const int* g_neighbor_list,
  const int* g_type,
  const Tersoff_mini_Para para,
  const double* __restrict__ g_x,
  const double* __restrict__ g_y,
  const double* __restrict__ g_z,
  double* g_b,
  double* g_bp)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1;
  if (n1 < N2) {
    int neighbor_number = g_neighbor_number[n1];
    int type1 = g_type[n1];
    double x1 = g_x[n1];
    double y1 = g_y[n1];
    double z1 = g_z[n1];
    for (int i1 = 0; i1 < neighbor_number; ++i1) {
      int n2 = g_neighbor_list[n1 + number_of_particles * i1];
      int type12 = type1 + g_type[n2];
      double x12 = g_x[n2] - x1;
      double y12 = g_y[n2] - y1;
      double z12 = g_z[n2] - z1;
      apply_mic(box, x12, y12, z12);
      double d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12);
      double zeta = 0.0;
      for (int i2 = 0; i2 < neighbor_number; ++i2) {
        int n3 = g_neighbor_list[n1 + number_of_particles * i2];
        int type13 = type1 + g_type[n3];
        if (n3 == n2) {
          continue;
        } // ensure that n3 != n2
        double x13 = g_x[n3] - x1;
        double y13 = g_y[n3] - y1;
        double z13 = g_z[n3] - z1;
        apply_mic(box, x13, y13, z13);
        double d13 = sqrt(x13 * x13 + y13 * y13 + z13 * z13);
        double cos123 = (x12 * x13 + y12 * y13 + z12 * z13) / (d12 * d13);
        double fc13, g123;
        find_fc(para.r1[type13], para.r2[type13], para.pi_factor[type13], d13, fc13);
        find_g(para.h[type12], cos123, g123);
        zeta += fc13 * g123;
      }

      double bzn, b12;
      bzn = pow(para.beta[type12] * zeta, para.n[type12]);
      b12 = pow(1.0 + bzn, para.minus_half_over_n[type12]);
      if (zeta < 1.0e-16) // avoid division by 0
      {
        g_b[i1 * number_of_particles + n1] = 1.0;
        g_bp[i1 * number_of_particles + n1] = 0.0;
      } else {
        g_b[i1 * number_of_particles + n1] = b12;
        g_bp[i1 * number_of_particles + n1] = -b12 * bzn * 0.5 / ((1.0 + bzn) * zeta);
      }
    }
  }
}

// step 2: calculate all the partial forces dU_i/dr_ij
static __global__ void find_force_step2(
  const int number_of_particles,
  const int N1,
  const int N2,
  const Box box,
  const int num_types,
  const int* g_neighbor_number,
  const int* g_neighbor_list,
  const int* g_type,
  const Tersoff_mini_Para para,
  const double* __restrict__ g_b,
  const double* __restrict__ g_bp,
  const double* __restrict__ g_x,
  const double* __restrict__ g_y,
  const double* __restrict__ g_z,
  double* g_potential,
  double* g_f12x,
  double* g_f12y,
  double* g_f12z)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1;
  if (n1 < N2) {
    int neighbor_number = g_neighbor_number[n1];
    int type1 = g_type[n1];
    double x1 = g_x[n1];
    double y1 = g_y[n1];
    double z1 = g_z[n1];
    double pot_energy = 0.0;
    for (int i1 = 0; i1 < neighbor_number; ++i1) {
      int index = i1 * number_of_particles + n1;
      int n2 = g_neighbor_list[index];
      int type12 = type1 + g_type[n2];

      double x12 = g_x[n2] - x1;
      double y12 = g_y[n2] - y1;
      double z12 = g_z[n2] - z1;
      apply_mic(box, x12, y12, z12);
      double d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12);
      double d12inv = 1.0 / d12;
      double fc12, fcp12, fa12, fap12, fr12, frp12;
      find_fc_and_fcp(para.r1[type12], para.r2[type12], para.pi_factor[type12], d12, fc12, fcp12);
      find_fa_and_fap(para.b[type12], para.mu[type12], d12, fa12, fap12);
      find_fr_and_frp(para.a[type12], para.lambda[type12], d12, fr12, frp12);

      // (i,j) part
      double b12 = g_b[index];
      double factor3 = (fcp12 * (fr12 - b12 * fa12) + fc12 * (frp12 - b12 * fap12)) * d12inv;
      double f12x = x12 * factor3 * 0.5;
      double f12y = y12 * factor3 * 0.5;
      double f12z = z12 * factor3 * 0.5;

      // accumulate potential energy
      pot_energy += fc12 * (fr12 - b12 * fa12) * 0.5;

      // (i,j,k) part
      double bp12 = g_bp[index];
      for (int i2 = 0; i2 < neighbor_number; ++i2) {
        int index_2 = n1 + number_of_particles * i2;
        int n3 = g_neighbor_list[index_2];
        if (n3 == n2) {
          continue;
        }
        int type13 = type1 + g_type[n3];
        double x13 = g_x[n3] - x1;
        double y13 = g_y[n3] - y1;
        double z13 = g_z[n3] - z1;
        apply_mic(box, x13, y13, z13);
        double d13 = sqrt(x13 * x13 + y13 * y13 + z13 * z13);
        double fc13, fa13;
        find_fc(para.r1[type13], para.r2[type13], para.pi_factor[type13], d13, fc13);
        find_fa(para.b[type13], para.mu[type13], d13, fa13);
        double bp13 = g_bp[index_2];
        double one_over_d12d13 = 1.0 / (d12 * d13);
        double cos123 = (x12 * x13 + y12 * y13 + z12 * z13) * one_over_d12d13;
        double cos123_over_d12d12 = cos123 * d12inv * d12inv;
        double g123, gp123;
        find_g_and_gp(para.h[type12], cos123, g123, gp123);
        // derivatives with cosine
        double dc = -fc12 * bp12 * fa12 * fc13 * gp123 - fc12 * bp13 * fa13 * fc13 * gp123;
        // derivatives with rij
        double dr = -fcp12 * bp13 * fa13 * g123 * fc13 * d12inv;
        double cos_d = x13 * one_over_d12d13 - x12 * cos123_over_d12d12;
        f12x += (x12 * dr + dc * cos_d) * 0.5;
        cos_d = y13 * one_over_d12d13 - y12 * cos123_over_d12d12;
        f12y += (y12 * dr + dc * cos_d) * 0.5;
        cos_d = z13 * one_over_d12d13 - z12 * cos123_over_d12d12;
        f12z += (z12 * dr + dc * cos_d) * 0.5;
      }
      g_f12x[index] = f12x;
      g_f12y[index] = f12y;
      g_f12z[index] = f12z;
    }
    // save potential
    g_potential[n1] += pot_energy;
  }
}

// Wrapper of force evaluation for the SBOP potential
void Tersoff_mini::compute(
  Box& box,
  const GPU_Vector<int>& type,
  const GPU_Vector<double>& position_per_atom,
  GPU_Vector<double>& potential_per_atom,
  GPU_Vector<double>& force_per_atom,
  GPU_Vector<double>& virial_per_atom)
{
  const int number_of_atoms = type.size();
  const int grid_size = (N2 - N1 - 1) / BLOCK_SIZE_FORCE + 1;

#ifdef USE_FIXED_NEIGHBOR
  static int num_calls = 0;
#endif
#ifdef USE_FIXED_NEIGHBOR
  if (num_calls++ == 0) {
#endif
    find_neighbor(
      N1,
      N2,
      rc,
      box,
      type,
      position_per_atom,
      tersoff_mini_data.cell_count,
      tersoff_mini_data.cell_count_sum,
      tersoff_mini_data.cell_contents,
      tersoff_mini_data.NN,
      tersoff_mini_data.NL);
#ifdef USE_FIXED_NEIGHBOR
  }
#endif

  // pre-compute the bond order functions and their derivatives
  find_force_step1<<<grid_size, BLOCK_SIZE_FORCE>>>(
    number_of_atoms,
    N1,
    N2,
    box,
    num_types,
    tersoff_mini_data.NN.data(),
    tersoff_mini_data.NL.data(),
    type.data(),
    para,
    position_per_atom.data(),
    position_per_atom.data() + number_of_atoms,
    position_per_atom.data() + number_of_atoms * 2,
    tersoff_mini_data.b.data(),
    tersoff_mini_data.bp.data());
  GPU_CHECK_KERNEL

  // pre-compute the partial forces
  find_force_step2<<<grid_size, BLOCK_SIZE_FORCE>>>(
    number_of_atoms,
    N1,
    N2,
    box,
    num_types,
    tersoff_mini_data.NN.data(),
    tersoff_mini_data.NL.data(),
    type.data(),
    para,
    tersoff_mini_data.b.data(),
    tersoff_mini_data.bp.data(),
    position_per_atom.data(),
    position_per_atom.data() + number_of_atoms,
    position_per_atom.data() + number_of_atoms * 2,
    potential_per_atom.data(),
    tersoff_mini_data.f12x.data(),
    tersoff_mini_data.f12y.data(),
    tersoff_mini_data.f12z.data());
  GPU_CHECK_KERNEL

  // the final step: calculate force and related quantities
  find_properties_many_body(
    box,
    tersoff_mini_data.NN.data(),
    tersoff_mini_data.NL.data(),
    tersoff_mini_data.f12x.data(),
    tersoff_mini_data.f12y.data(),
    tersoff_mini_data.f12z.data(),
    position_per_atom,
    force_per_atom,
    virial_per_atom);
}
