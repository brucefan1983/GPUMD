#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

/*----------------------------------------------------------------------------80
The EAM potential. Currently two analytical versions:
[1] X. W. Zhou et al. Phys. Rev. B 69, 144113 (2004).
[2] X. D. Dai et al. JPCM 18, 4527 (2006).
------------------------------------------------------------------------------*/

#include "eam.cuh"
#include "utilities/error.cuh"
#define BLOCK_SIZE_FORCE 64

EAM::EAM(FILE* fid, char* name, int num_types, const int number_of_atoms)
{
  if (strcmp(name, "eam_zhou_2004") == 0) {
    initialize_eam2004zhou(fid, num_types);
  } else if (strcmp(name, "eam_dai_2006") == 0) {
    initialize_eam2006dai(fid);
    if (num_types > 1) {
      PRINT_INPUT_ERROR(
        "eam_dai_2006 has not been implemented for systems with two or more atom types.");
    }
  }

  eam_data.Fp.resize(number_of_atoms);
}

void EAM::initialize_eam2004zhou(FILE* fid, int num_types)
{
  printf("Use the EAM-type potential in the following reference:\n");
  printf("    X. W. Zhou et al., PRB 69, 144113 (2004).\n");
  potential_model = 0;

  rc = 0.0;
  for (int type = 0; type < num_types; ++type) {
    double x[21];
    for (int n = 0; n < 21; n++) {
      int count = fscanf(fid, "%lf", &x[n]);
      PRINT_SCANF_ERROR(count, 1, "Reading error for EAM potential.");
    }
    eam2004zhou.re_inv[type] = 1.0 / x[0];
    eam2004zhou.fe[type] = x[1];
    eam2004zhou.rho_e_inv[type] = 1.0 / x[2];
    eam2004zhou.rho_s_inv[type] = 1.0 / x[3];
    eam2004zhou.alpha[type] = x[4];
    eam2004zhou.beta[type] = x[5];
    eam2004zhou.A[type] = x[6];
    eam2004zhou.B[type] = x[7];
    eam2004zhou.kappa[type] = x[8];
    eam2004zhou.lambda[type] = x[9];
    eam2004zhou.Fn0[type] = x[10];
    eam2004zhou.Fn1[type] = x[11];
    eam2004zhou.Fn2[type] = x[12];
    eam2004zhou.Fn3[type] = x[13];
    eam2004zhou.F0[type] = x[14];
    eam2004zhou.F1[type] = x[15];
    eam2004zhou.F2[type] = x[16];
    eam2004zhou.F3[type] = x[17];
    eam2004zhou.eta[type] = x[18];
    eam2004zhou.Fe[type] = x[19];
    eam2004zhou.rc[type] = x[20];
    eam2004zhou.rho_n[type] = x[2] * 0.85;
    eam2004zhou.rho_0[type] = x[2] * 1.15;
    eam2004zhou.rho_n_inv[type] = 1.0 / eam2004zhou.rho_n[type];
    if (rc < eam2004zhou.rc[type]) {
      rc = eam2004zhou.rc[type];
    }
  }
}

void EAM::initialize_eam2006dai(FILE* fid)
{
  printf("Use the EAM-type potential in the following reference:\n");
  printf("    X. D. Dai et al., JPCM 18, 4527 (2006).\n");
  potential_model = 1;

  double x[9];
  for (int n = 0; n < 9; n++) {
    int count = fscanf(fid, "%lf", &x[n]);
    PRINT_SCANF_ERROR(count, 1, "Reading error for EAM potential.");
  }
  eam2006dai.A = x[0];
  eam2006dai.d = x[1];
  eam2006dai.c = x[2];
  eam2006dai.c0 = x[3];
  eam2006dai.c1 = x[4];
  eam2006dai.c2 = x[5];
  eam2006dai.c3 = x[6];
  eam2006dai.c4 = x[7];
  eam2006dai.B = x[8];
  eam2006dai.rc = (eam2006dai.c > eam2006dai.d) ? eam2006dai.c : eam2006dai.d;
  rc = eam2006dai.rc;
}

EAM::~EAM(void)
{
  // nothing
}

// pair function (phi and phip have been intentionally halved here)
static __device__ void
find_phi(const EAM2004Zhou& eam, const int type, const double d12, double& phi, double& phip)
{
  double r_ratio = d12 * eam.re_inv[type];
  double tmp1 = (r_ratio - eam.kappa[type]) * (r_ratio - eam.kappa[type]);   // 2
  tmp1 *= tmp1;                                                              // 4
  tmp1 *= tmp1 * tmp1 * tmp1 * tmp1;                                         // 20
  double tmp2 = (r_ratio - eam.lambda[type]) * (r_ratio - eam.lambda[type]); // 2
  tmp2 *= tmp2;                                                              // 4
  tmp2 *= tmp2 * tmp2 * tmp2 * tmp2;                                         // 20
  double phi1 = 0.5 * eam.A[type] * exp(-eam.alpha[type] * (r_ratio - 1.0)) / (1.0 + tmp1);
  double phi2 = 0.5 * eam.B[type] * exp(-eam.beta[type] * (r_ratio - 1.0)) / (1.0 + tmp2);
  phi = phi1 - phi2;
  phip = (phi2 * eam.re_inv[type]) *
           (eam.beta[type] + 20.0 * tmp2 / (r_ratio - eam.lambda[type]) / (1.0 + tmp2)) -
         (phi1 * eam.re_inv[type]) *
           (eam.alpha[type] + 20.0 * tmp1 / (r_ratio - eam.kappa[type]) / (1.0 + tmp1));
}

// density function f(r)
static __device__ void find_f(const EAM2004Zhou& eam, const int type, const double d12, double& f)
{
  double r_ratio = d12 * eam.re_inv[type];
  double tmp = (r_ratio - eam.lambda[type]) * (r_ratio - eam.lambda[type]); // 2
  tmp *= tmp;                                                               // 4
  tmp *= tmp * tmp * tmp * tmp;                                             // 20
  f = eam.fe[type] * exp(-eam.beta[type] * (r_ratio - 1.0)) / (1.0 + tmp);
}

// derivative of the density function f'(r)
static __device__ void find_fp(const EAM2004Zhou& eam, const int type, const double d12, double& fp)
{
  double r_ratio = d12 * eam.re_inv[type];
  double tmp = (r_ratio - eam.lambda[type]) * (r_ratio - eam.lambda[type]); // 2
  tmp *= tmp;                                                               // 4
  tmp *= tmp * tmp * tmp * tmp;                                             // 20
  double f = eam.fe[type] * exp(-eam.beta[type] * (r_ratio - 1.0)) / (1.0 + tmp);
  fp = -(f * eam.re_inv[type]) *
       (eam.beta[type] + 20.0 * tmp / (r_ratio - eam.lambda[type]) / (1.0 + tmp));
}

static __device__ void
find_f_and_fp(const EAM2004Zhou& eam, const int type, const double d12, double& f, double& fp)
{
  double r_ratio = d12 * eam.re_inv[type];
  double tmp = (r_ratio - eam.lambda[type]) * (r_ratio - eam.lambda[type]); // 2
  tmp *= tmp;                                                               // 4
  tmp *= tmp * tmp * tmp * tmp;                                             // 20
  f = eam.fe[type] * exp(-eam.beta[type] * (r_ratio - 1.0)) / (1.0 + tmp);
  fp = -(f * eam.re_inv[type]) *
       (eam.beta[type] + 20.0 * tmp / (r_ratio - eam.lambda[type]) / (1.0 + tmp));
}

// pair function for EAM2004Zhou
static __device__ void find_phi(
  const EAM2004Zhou& eam,
  const int type1,
  const int type2,
  const double d12,
  double& phi,
  double& phip)
{
  if (type1 == type2) {
    find_phi(eam, type1, d12, phi, phip);
  } else {
    double phi1, phip1;
    find_phi(eam, type1, d12, phi1, phip1);
    double phi2, phip2;
    find_phi(eam, type2, d12, phi2, phip2);
    double f1, fp1;
    find_f_and_fp(eam, type1, d12, f1, fp1);
    double f2, fp2;
    find_f_and_fp(eam, type2, d12, f2, fp2);
    double f1inv = 1.0 / f1;
    double f2inv = 1.0 / f2;
    phi = 0.5 * (phi1 * f2 * f1inv + phi2 * f1 * f2inv);
    phip = (phip1 * f2 + phi1 * (fp2 - f2 * fp1 * f1inv)) * f1inv;
    phip += (phip2 * f1 + phi2 * (fp1 - f1 * fp2 * f2inv)) * f2inv;
    phip *= 0.5;
  }
}

// embedding function
static __device__ void
find_F(const EAM2004Zhou& eam, const int type, const double rho, double& F, double& Fp)
{
  if (rho < eam.rho_n[type]) {
    double x = rho * eam.rho_n_inv[type] - 1.0;
    F = ((eam.Fn3[type] * x + eam.Fn2[type]) * x + eam.Fn1[type]) * x + eam.Fn0[type];
    Fp = ((3.0 * eam.Fn3[type] * x + 2.0 * eam.Fn2[type]) * x + eam.Fn1[type]) / eam.rho_n[type];
  } else if (rho < eam.rho_0[type]) {
    double x = rho * eam.rho_e_inv[type] - 1.0;
    F = ((eam.F3[type] * x + eam.F2[type]) * x + eam.F1[type]) * x + eam.F0[type];
    Fp = ((3.0 * eam.F3[type] * x + 2.0 * eam.F2[type]) * x + eam.F1[type]) * eam.rho_e_inv[type];
  } else {
    double x = rho * eam.rho_s_inv[type];
    double x_eta = pow(x, eam.eta[type]);
    F = eam.Fe[type] * (1.0 - eam.eta[type] * log(x)) * x_eta;
    Fp = (eam.eta[type] / rho) * (F - eam.Fe[type] * x_eta);
  }
}

// pair function (phi and phip have been intentionally halved here)
static __device__ void find_phi(const EAM2006Dai& fs, const double d12, double& phi, double& phip)
{
  if (d12 > fs.c) {
    phi = 0.0;
    phip = 0.0;
  } else {
    double tmp = ((((fs.c4 * d12 + fs.c3) * d12 + fs.c2) * d12 + fs.c1) * d12 + fs.c0);

    phi = 0.5 * (d12 - fs.c) * (d12 - fs.c) * tmp;

    phip = 2.0 * (d12 - fs.c) * tmp;
    phip += (((4.0 * fs.c4 * d12 + 3.0 * fs.c3) * d12 + 2.0 * fs.c2) * d12 + fs.c1) * (d12 - fs.c) *
            (d12 - fs.c);
    phip *= 0.5;
  }
}

// density function f(r)
static __device__ void find_f(const EAM2006Dai& fs, const double d12, double& f)
{
  if (d12 > fs.d) {
    f = 0.0;
  } else {
    double tmp = (d12 - fs.d) * (d12 - fs.d);
    f = tmp + fs.B * fs.B * tmp * tmp;
  }
}

// derivative of the density function f'(r)
static __device__ void find_fp(const EAM2006Dai& fs, const double d12, double& fp)
{
  if (d12 > fs.d) {
    fp = 0.0;
  } else {
    double tmp = 2.0 * (d12 - fs.d);
    fp = tmp * (1.0 + fs.B * fs.B * tmp * (d12 - fs.d));
  }
}

// embedding function
static __device__ void find_F(const EAM2006Dai& fs, const double rho, double& F, double& Fp)
{
  double sqrt_rho = sqrt(rho);
  F = -fs.A * sqrt_rho;
  Fp = -fs.A * 0.5 / sqrt_rho;
}

// Calculate the embedding energy and its derivative
template <int potential_model>
static __global__ void find_force_eam_step1(
  const EAM2004Zhou eam2004zhou,
  const EAM2006Dai eam2006dai,
  const int N,
  const int N1,
  const int N2,
  const Box box,
  const int* g_NN,
  const int* g_NL,
  const int* g_type,
  const double* __restrict__ g_x,
  const double* __restrict__ g_y,
  const double* __restrict__ g_z,
  double* g_Fp,
  double* g_pe)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1; // particle index

  if (n1 < N2) {
    int NN = g_NN[n1];

    double x1 = g_x[n1];
    double y1 = g_y[n1];
    double z1 = g_z[n1];

    // Calculate the density
    double rho = 0.0;
    for (int i1 = 0; i1 < NN; ++i1) {
      int n2 = g_NL[n1 + N * i1];
      double x12 = g_x[n2] - x1;
      double y12 = g_y[n2] - y1;
      double z12 = g_z[n2] - z1;
      apply_mic(box, x12, y12, z12);
      double d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12);
      double rho12 = 0.0;
      if (potential_model == 0) {
        find_f(eam2004zhou, g_type[n2], d12, rho12); // density is contributed by n2
      }
      if (potential_model == 1) {
        find_f(eam2006dai, d12, rho12);
      }
      rho += rho12;
    }

    // Calculate the embedding energy F and its derivative Fp
    double F, Fp;
    if (potential_model == 0)
      find_F(eam2004zhou, g_type[n1], rho, F, Fp); // embedding energy is for n1
    if (potential_model == 1)
      find_F(eam2006dai, rho, F, Fp);

    g_pe[n1] += F; // many-body potential energy
    g_Fp[n1] = Fp;
  }
}

// Force evaluation kernel
template <int potential_model>
static __global__ void find_force_eam_step2(
  const EAM2004Zhou eam2004zhou,
  const EAM2006Dai eam2006dai,
  const int N,
  const int N1,
  const int N2,
  const Box box,
  const int* g_NN,
  const int* g_NL,
  const int* g_type,
  const double* __restrict__ g_Fp,
  const double* __restrict__ g_x,
  const double* __restrict__ g_y,
  const double* __restrict__ g_z,
  double* g_fx,
  double* g_fy,
  double* g_fz,
  double* g_virial,
  double* g_pe)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1;
  double s_fx = 0.0;  // force_x
  double s_fy = 0.0;  // force_y
  double s_fz = 0.0;  // force_z
  double s_pe = 0.0;  // potential energy
  double s_sxx = 0.0; // virial_stress_xx
  double s_sxy = 0.0; // virial_stress_xy
  double s_sxz = 0.0; // virial_stress_xz
  double s_syx = 0.0; // virial_stress_yx
  double s_syy = 0.0; // virial_stress_yy
  double s_syz = 0.0; // virial_stress_yz
  double s_szx = 0.0; // virial_stress_zx
  double s_szy = 0.0; // virial_stress_zy
  double s_szz = 0.0; // virial_stress_zz

  if (n1 < N2) {
    int type1 = g_type[n1];
    int NN = g_NN[n1];
    double x1 = g_x[n1];
    double y1 = g_y[n1];
    double z1 = g_z[n1];
    double Fp1 = g_Fp[n1];

    for (int i1 = 0; i1 < NN; ++i1) {
      int n2 = g_NL[n1 + N * i1];
      int type2 = g_type[n2];
      double Fp2 = g_Fp[n2];
      double x12 = g_x[n2] - x1;
      double y12 = g_y[n2] - y1;
      double z12 = g_z[n2] - z1;
      apply_mic(box, x12, y12, z12);
      double d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12);

      double phi, phip, fp1, fp2;
      if (potential_model == 0) {
        find_phi(eam2004zhou, type1, type2, d12, phi, phip);
        if (type1 == type2) {
          find_fp(eam2004zhou, type1, d12, fp1);
          fp2 = fp1;
        } else {
          find_fp(eam2004zhou, type1, d12, fp1);
          find_fp(eam2004zhou, type2, d12, fp2);
        }
      }
      if (potential_model == 1) {
        find_phi(eam2006dai, d12, phi, phip);
        find_fp(eam2006dai, d12, fp1);
        fp2 = fp1;
      }

      double d12inv = 1.0 / d12;
      phip *= d12inv;
      fp1 *= d12inv;
      fp2 *= d12inv;
      double f12x = x12 * (phip + Fp1 * fp2);
      double f12y = y12 * (phip + Fp1 * fp2);
      double f12z = z12 * (phip + Fp1 * fp2);
      double f21x = -x12 * (phip + Fp2 * fp1);
      double f21y = -y12 * (phip + Fp2 * fp1);
      double f21z = -z12 * (phip + Fp2 * fp1);

      // two-body potential energy
      s_pe += phi;

      // per atom force
      s_fx += f12x - f21x;
      s_fy += f12y - f21y;
      s_fz += f12z - f21z;

      // per-atom virial
      s_sxx += x12 * f21x;
      s_sxy += x12 * f21y;
      s_sxz += x12 * f21z;
      s_syx += y12 * f21x;
      s_syy += y12 * f21y;
      s_syz += y12 * f21z;
      s_szx += z12 * f21x;
      s_szy += z12 * f21y;
      s_szz += z12 * f21z;
    }

    // save force
    g_fx[n1] += s_fx;
    g_fy[n1] += s_fy;
    g_fz[n1] += s_fz;

    // save virial
    // xx xy xz    0 3 4
    // yx yy yz    6 1 5
    // zx zy zz    7 8 2
    g_virial[n1 + 0 * N] += s_sxx;
    g_virial[n1 + 1 * N] += s_syy;
    g_virial[n1 + 2 * N] += s_szz;
    g_virial[n1 + 3 * N] += s_sxy;
    g_virial[n1 + 4 * N] += s_sxz;
    g_virial[n1 + 5 * N] += s_syz;
    g_virial[n1 + 6 * N] += s_syx;
    g_virial[n1 + 7 * N] += s_szx;
    g_virial[n1 + 8 * N] += s_szy;

    // save potential energy
    g_pe[n1] += s_pe;
  }
}

// Force evaluation wrapper
void EAM::compute(
  const int type_shift,
  const Box& box,
  const Neighbor& neighbor,
  const GPU_Vector<int>& type,
  const GPU_Vector<double>& position_per_atom,
  GPU_Vector<double>& potential_per_atom,
  GPU_Vector<double>& force_per_atom,
  GPU_Vector<double>& virial_per_atom)
{
  const int number_of_atoms = type.size();
  int grid_size = (N2 - N1 - 1) / BLOCK_SIZE_FORCE + 1;

  if (potential_model == 0) {
    find_force_eam_step1<0><<<grid_size, BLOCK_SIZE_FORCE>>>(
      eam2004zhou, eam2006dai, number_of_atoms, N1, N2, box, neighbor.NN_local.data(),
      neighbor.NL_local.data(), type.data(), position_per_atom.data(),
      position_per_atom.data() + number_of_atoms, position_per_atom.data() + number_of_atoms * 2,
      eam_data.Fp.data(), potential_per_atom.data());
    CUDA_CHECK_KERNEL

    find_force_eam_step2<0><<<grid_size, BLOCK_SIZE_FORCE>>>(
      eam2004zhou, eam2006dai, number_of_atoms, N1, N2, box, neighbor.NN_local.data(),
      neighbor.NL_local.data(), type.data(), eam_data.Fp.data(), position_per_atom.data(),
      position_per_atom.data() + number_of_atoms, position_per_atom.data() + number_of_atoms * 2,
      force_per_atom.data(), force_per_atom.data() + number_of_atoms,
      force_per_atom.data() + 2 * number_of_atoms, virial_per_atom.data(),
      potential_per_atom.data());
    CUDA_CHECK_KERNEL
  }

  if (potential_model == 1) {
    find_force_eam_step1<1><<<grid_size, BLOCK_SIZE_FORCE>>>(
      eam2004zhou, eam2006dai, number_of_atoms, N1, N2, box, neighbor.NN_local.data(),
      neighbor.NL_local.data(), type.data(), position_per_atom.data(),
      position_per_atom.data() + number_of_atoms, position_per_atom.data() + number_of_atoms * 2,
      eam_data.Fp.data(), potential_per_atom.data());
    CUDA_CHECK_KERNEL

    find_force_eam_step2<1><<<grid_size, BLOCK_SIZE_FORCE>>>(
      eam2004zhou, eam2006dai, number_of_atoms, N1, N2, box, neighbor.NN_local.data(),
      neighbor.NL_local.data(), type.data(), eam_data.Fp.data(), position_per_atom.data(),
      position_per_atom.data() + number_of_atoms, position_per_atom.data() + number_of_atoms * 2,
      force_per_atom.data(), force_per_atom.data() + number_of_atoms,
      force_per_atom.data() + 2 * number_of_atoms, virial_per_atom.data(),
      potential_per_atom.data());
    CUDA_CHECK_KERNEL
  }
}
