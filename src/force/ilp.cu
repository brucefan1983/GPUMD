#include "hip/hip_runtime.h"

/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

/*----------------------------------------------------------------------------80
The class dealing with the Lennard-Jones (LJ) pairwise potentials.
------------------------------------------------------------------------------*/

#include "ilp.cuh"
#include "neighbor.cuh"
#include "utilities/error.cuh"

// TODO: best size here: 128
#define BLOCK_SIZE_FORCE 128

ILP::ILP(FILE* fid, int num_types, int num_atoms)
{
  printf("Use %d-element ILP potential with elements:\n", num_types);
  if (!(num_types >= 1 && num_types <= MAX_TYPE_ILP)) {
    PRINT_INPUT_ERROR("Incorrect number of ILP parameters.\n");
  }
  for (int n = 0; n < num_types; ++n) {
    char atom_symbol[10];
    int count = fscanf(fid, "%s", atom_symbol);
    PRINT_SCANF_ERROR(count, 1, "Reading error for ILP potential.");
    printf(" %s", atom_symbol);
  }
  printf("\n");

  // read parameters
  double beta, alpha, delta, epsilon, C, d, sR, reff, C6, S, rcut;
  rc = 0.0;
  for (int n = 0; n < num_types; ++n) {
    for (int m = 0; m < num_types; ++m) {
      int count = fscanf(fid, "%lf%lf%lf%lf%lf%lf%lf%lf%lf%lf", \
      &beta, &alpha, &delta, &epsilon, &C, &d, &sR, &reff, &C6, &S, &rcut);
      PRINT_SCANF_ERROR(count, 10, "Reading error for ILP potential.");

      ilp_para.C[n][m] = C;
      ilp_para.C_6[n][m] = C6;
      ilp_para.d[n][m] = d;
      ilp_para.d_Seff[n][m] = d / sR / reff;
      ilp_para.epsilon[n][m] = epsilon;
      ilp_para.z0[n][m] = beta;
      ilp_para.lambda[n][m] = alpha / beta;
      ilp_para.delta2inv[n][m] = 1.0 / (delta * delta); //TODO: how faster?
      ilp_para.S[n][m] = S;
      ilp_para.r_cut[n][m] = rcut;

      // TODO: ILP has taper function, check if necessary
      if (rc < rcut)
        rc = rcut;
    }
  }

  ilp_data.NN.resize(num_atoms);
  ilp_data.NL.resize(num_atoms * CUDA_MAX_NL);
  ilp_data.cell_count.resize(num_atoms);
  ilp_data.cell_count_sum.resize(num_atoms);
  ilp_data.cell_contents.resize(num_atoms);
}

ILP::~ILP(void)
{
  // TODO
}

// calculate the normals and its derivatives
static __device__ void calc_normal(void)
{
  // TODO
}

// calculate the van der Waals force and energy
static __device__ void calc_vdW(void)
{
  // TODO
}

// calculate the repulsive force and energy
static __device__ void calc_rep(void)
{
  // TODO
}

// force evaluation kernel
static __global__ void gpu_find_force(
  ILP_Para ilp)
{
  // TODO
}

// find force and related quantities
void ILP::compute(
  Box &box,
  const GPU_Vector<int> &type,
  const GPU_Vector<double> &position_per_atom,
  GPU_Vector<double> &potential_per_atom,
  GPU_Vector<double> &force_per_atom,
  GPU_Vector<double> &virial_per_atom)
{
  const int number_of_atoms = type.size();
  int grid_size = (N2 - N1 - 1) / BLOCK_SIZE_FORCE + 1;

// what's this??
#ifdef USE_FIXED_NEIGHBOR
  static int num_calls = 0;
#endif
#ifdef USE_FIXED_NEIGHBOR
  if (num_calls++ == 0) {
#endif
    find_neighbor(
      N1,
      N2,
      rc,
      box,
      type,
      position_per_atom,
      ilp_data.cell_count,
      ilp_data.cell_count_sum,
      ilp_data.cell_contents,
      ilp_data.NN,
      ilp_data.NL);
#ifdef USE_FIXED_NEIGHBOR
  }
#endif

  gpu_find_force<<<grid_size, BLOCK_SIZE_FORCE>>>(ilp_para);
  // TODO
  CUDA_CHECK_KERNEL
}
