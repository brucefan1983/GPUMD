#include "hip/hip_runtime.h"

/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

/*----------------------------------------------------------------------------80
The class dealing with the Lennard-Jones (LJ) pairwise potentials.
------------------------------------------------------------------------------*/

#include "ilp.cuh"
#include "neighbor.cuh"
#include "utilities/error.cuh"

// TODO: best size here: 128
#define BLOCK_SIZE_FORCE 128

ILP::ILP(FILE* fid, int num_types, int num_atoms)
{
  printf("Use %d-element ILP potential with elements:\n", num_types);
  if (!(num_types >= 1 && num_types <= MAX_TYPE_ILP)) {
    PRINT_INPUT_ERROR("Incorrect number of ILP parameters.\n");
  }
  for (int n = 0; n < num_types; ++n) {
    char atom_symbol[10];
    int count = fscanf(fid, "%s", atom_symbol);
    PRINT_SCANF_ERROR(count, 1, "Reading error for ILP potential.");
    printf(" %s", atom_symbol);
  }
  printf("\n");

  // read parameters
  double beta, alpha, delta, epsilon, C, d, sR, reff, C6, S, rcut;
  rc = 0.0;
  for (int n = 0; n < num_types; ++n) {
    for (int m = 0; m < num_types; ++m) {
      int count = fscanf(fid, "%lf%lf%lf%lf%lf%lf%lf%lf%lf%lf%lf", \
      &beta, &alpha, &delta, &epsilon, &C, &d, &sR, &reff, &C6, &S, &rcut);
      PRINT_SCANF_ERROR(count, 11, "Reading error for ILP potential.");

      ilp_para.C[n][m] = C;
      ilp_para.C_6[n][m] = C6;
      ilp_para.d[n][m] = d;
      ilp_para.d_Seff[n][m] = d / sR / reff;
      ilp_para.epsilon[n][m] = epsilon;
      ilp_para.z0[n][m] = beta;
      ilp_para.lambda[n][m] = alpha / beta;
      ilp_para.delta2inv[n][m] = 1.0 / (delta * delta); //TODO: how faster?
      ilp_para.S[n][m] = S;
      ilp_para.r_cut[n][m] = rcut;
      // TODO: meV???
      double meV = 1e-3 * S;
      ilp_para.C[n][m] *= meV;
      ilp_para.C_6[n][m] *= meV;
      ilp_para.epsilon[n][m] *= meV;

      // TODO: ILP has taper function, check if necessary
      if (rc < rcut)
        rc = rcut;
    }
  }

  ilp_data.NN.resize(num_atoms);
  ilp_data.NL.resize(num_atoms * CUDA_MAX_NL);
  ilp_data.cell_count.resize(num_atoms);
  ilp_data.cell_count_sum.resize(num_atoms);
  ilp_data.cell_contents.resize(num_atoms);

  // init ilp neighbor list
  ilp_data.ilp_NN.resize(num_atoms);
  ilp_data.ilp_NL.resize(num_atoms * CUDA_MAX_NL);

  ilp_data.f12x.resize(num_atoms);
  ilp_data.f12y.resize(num_atoms);
  ilp_data.f12z.resize(num_atoms);

  // init constant cutoff coeff
  double h_tap_coeff[8] = \
    {1.0, 0.0, 0.0, 0.0, -35.0, 84.0, 70.0, 20.0};
  hipMemcpyToSymbol(HIP_SYMBOL(Tap_coeff), h_tap_coeff, 8 * sizeof(double));
  CUDA_CHECK_KERNEL
}

ILP::~ILP(void)
{
  // TODO
}

// TODO: set inline???
// calculate the long-range cutoff term
static __device__ double calc_Tap(const double r_ij, const double Rcut)
{
  double Tap, r;

  r = r_ij / Rcut;
  if (r >= 1.0) {
    Tap = 0.0;
  } else {
    Tap = Tap_coeff[7];
    for (int i = 6; i >= 0; --i) {
      Tap = Tap * r + Tap_coeff[i];
    }
  }

  return Tap;
}

// TODO: set inline???
// calculate the derivatives of long-range cutoff term
static __device__ double calc_dTap(const double r_ij, const double Rcut)
{
  double dTap, r;
  
  r = r_ij / Rcut;
  if (r >= Rcut) {
    dTap = 0.0;
  } else {
    dTap = 7.0 * Tap_coeff[7];
    for (int i = 6; i > 0; --i) {
      dTap = dTap * r + i * Tap_coeff[i];
    }
    dTap /= Rcut;
  }

  return dTap;
}

// create ILP neighbor list from main neighbor list to calculate normals
static __global__ void ILP_neighbor(
  const int number_of_particles,
  const int N1,
  const int N2,
  const Box box,
  const int *g_neighbor_number,
  const int *g_neighbor_list,
  const int *g_type,
  ILP_Para ilp_para,
  const double* __restrict__ g_x,
  const double* __restrict__ g_y,
  const double* __restrict__ g_z,
  int *ilp_neighbor_number,
  int *ilp_neighbor_list,
  const int *group_label)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1; // particle index

  if (n1 < N2) {
    printf("***** ilp neighbor GPU *****\n");
    int count = 0;
    int neighbor_number = g_neighbor_number[n1];
    int type1 = g_type[n1];
    double x1 = g_x[n1];
    double y1 = g_y[n1];
    double z1 = g_z[n1];

    printf("***** n1: %d, neigh: %d *****\n", n1, neighbor_number);
    for (int i1 = 0; i1 < neighbor_number; ++i1) {
      int n2 = g_neighbor_list[n1 + number_of_particles * i1];
      int type2 = g_type[n2];
      printf("***** n1: %d, n2: %d *****\n", n1, n2);
      printf("***** group lable[0]: %d, [1]: %d *****\n", group_label[0], group_label[1]);
      printf("***** %f *****\n", ilp_para.r_cut[0][0]);

      double x12 = g_x[n2] - x1;
      double y12 = g_y[n2] - y1;
      double z12 = g_z[n2] - z1;
      apply_mic(box, x12, y12, z12);
      double d12sq = x12 * x12 + y12 * y12 + z12 * z12;
      double r_cut = ilp_para.r_cut[type1][type2];

      printf("***** type1: %d, type2: %d *****\n", type1, type2);
      printf("***** size of r_cut: %d*****\n", sizeof(r_cut));
//      printf("***** size of r_cut[0]: %d*****\n", sizeof(*r_cut[0]));
//      printf("***** r_cut: %p *****\n", r_cut);
//      printf("***** r_cut[1]: %p *****\n", r_cut[1]);
//      printf("***** *r_cut: %p *****\n", *r_cut);
//      printf("***** **r_cut: %p *****\n", **r_cut);
//
      // TODO: store cutILPsq to calc fast
      double cutILPsq = r_cut * r_cut;

      if (group_label[n1] == group_label[n2] && d12sq < cutILPsq && d12sq != 0) {
        printf("----- count: %d -----\n", count);
        ilp_neighbor_list[count++ * number_of_particles + n1] = n2;
      }
    }
    printf("===== get here =====\n");
    ilp_neighbor_number[n1] = count;

    if (count > 3) {
      // TODO: error, there are too many neighbors for some atoms, 
      // please check your configuration
    }
  }
  // TODO: check group id before calc potential(calc in defferent layers)
}

// calculate the normals and its derivatives
static __device__ void calc_normal(
  double (&vet)[3][3],
  int cont,
  double (&normal)[3],
  double (&dnormdri)[3][3],
  double (&dnormal)[3][3][3])
{
  int id, ip, m;
  double nn2, nn;
  double pv12[3], pv31[3], pv23[3], n1[3], dni[3];
  double dnn[3][3], dpvdri[3][3];
  double dn1[3][3][3], dpv12[3][3][3], dpv23[3][3][3], dpv31[3][3][3];

  // initialize the arrays
  for (id = 0; id < 3; id++) {
    pv12[id] = 0.0;
    pv31[id] = 0.0;
    pv23[id] = 0.0;
    n1[id] = 0.0;
    dni[id] = 0.0;
    for (ip = 0; ip < 3; ip++) {
      dnn[ip][id] = 0.0;
      dpvdri[ip][id] = 0.0;
      for (m = 0; m < 3; m++) {
        dpv12[ip][id][m] = 0.0;
        dpv31[ip][id][m] = 0.0;
        dpv23[ip][id][m] = 0.0;
        dn1[ip][id][m] = 0.0;
      }
    }
  }

  if (cont <= 1) {
    for (id = 0; id < 3; ++id) {
      normal[id] = 0.0;
      for (ip = 0; ip < 3; ++ip) {
        dnormdri[id][ip] = 0.0;
        for (m = 0; m < 3; ++m) {
          dnormal[id][ip][m] = 0.0;
        }
      }
    }
  } else if (cont == 2) {
    pv12[0] = vet[0][1] * vet[1][2] - vet[1][1] * vet[0][2];
    pv12[1] = vet[0][2] * vet[1][0] - vet[1][2] * vet[0][0];
    pv12[2] = vet[0][0] * vet[1][1] - vet[1][0] * vet[0][1];
    // derivatives of pv12[0] to ri
    dpvdri[0][0] = 0.0;
    dpvdri[0][1] = vet[0][2] - vet[1][2];
    dpvdri[0][2] = vet[1][1] - vet[0][1];
    // derivatives of pv12[1] to ri
    dpvdri[1][0] = vet[1][2] - vet[0][2];
    dpvdri[1][1] = 0.0;
    dpvdri[1][2] = vet[0][0] - vet[1][0];
    // derivatives of pv12[2] to ri
    dpvdri[2][0] = vet[0][1] - vet[1][1];
    dpvdri[2][1] = vet[1][0] - vet[0][0];
    dpvdri[2][2] = 0.0;

    dpv12[0][0][0] = 0.0;
    dpv12[0][1][0] = vet[1][2];
    dpv12[0][2][0] = -vet[1][1];
    dpv12[1][0][0] = -vet[1][2];
    dpv12[1][1][0] = 0.0;
    dpv12[1][2][0] = vet[1][0];
    dpv12[2][0][0] = vet[1][1];
    dpv12[2][1][0] = -vet[1][0];
    dpv12[2][2][0] = 0.0;

    // derivatives respect to the second neighbor, atom l
    dpv12[0][0][1] = 0.0;
    dpv12[0][1][1] = -vet[0][2];
    dpv12[0][2][1] = vet[0][1];
    dpv12[1][0][1] = vet[0][2];
    dpv12[1][1][1] = 0.0;
    dpv12[1][2][1] = -vet[0][0];
    dpv12[2][0][1] = -vet[0][1];
    dpv12[2][1][1] = vet[0][0];
    dpv12[2][2][1] = 0.0;

    // derivatives respect to the third neighbor, atom n
    // derivatives of pv12 to rn is zero
    for (id = 0; id < 3; id++) {
      for (ip = 0; ip < 3; ip++) { dpv12[id][ip][2] = 0.0; }
    }

    n1[0] = pv12[0];
    n1[1] = pv12[1];
    n1[2] = pv12[2];
    // the magnitude of the normal vector
    nn2 = n1[0] * n1[0] + n1[1] * n1[1] + n1[2] * n1[2];
    nn = sqrt(nn2);
    
    // TODO
    // if (nn == 0) error->one(FLERR, "The magnitude of the normal vector is zero");
    // the unit normal vector
    normal[0] = n1[0] / nn;
    normal[1] = n1[1] / nn;
    normal[2] = n1[2] / nn;
    // derivatives of nn, dnn:3x1 vector
    dni[0] = (n1[0] * dpvdri[0][0] + n1[1] * dpvdri[1][0] + n1[2] * dpvdri[2][0]) / nn;
    dni[1] = (n1[0] * dpvdri[0][1] + n1[1] * dpvdri[1][1] + n1[2] * dpvdri[2][1]) / nn;
    dni[2] = (n1[0] * dpvdri[0][2] + n1[1] * dpvdri[1][2] + n1[2] * dpvdri[2][2]) / nn;
    // derivatives of unit vector ni respect to ri, the result is 3x3 matrix
    for (id = 0; id < 3; id++) {
      for (ip = 0; ip < 3; ip++) {
        dnormdri[id][ip] = dpvdri[id][ip] / nn - n1[id] * dni[ip] / nn2;
      }
    }
    // derivatives of non-normalized normal vector, dn1:3x3x3 array
    for (id = 0; id < 3; id++) {
      for (ip = 0; ip < 3; ip++) {
        for (m = 0; m < 3; m++) { dn1[id][ip][m] = dpv12[id][ip][m]; }
      }
    }
    // derivatives of nn, dnn:3x3 vector
    // dnn[id][m]: the derivative of nn respect to r[id][m], id,m=0,1,2
    // r[id][m]: the id's component of atom m
    for (m = 0; m < 3; m++) {
      for (id = 0; id < 3; id++) {
        dnn[id][m] = (n1[0] * dn1[0][id][m] + n1[1] * dn1[1][id][m] + n1[2] * dn1[2][id][m]) / nn;
      }
    }
    // dnormal[id][ip][m][i]: the derivative of normal[id] respect to r[ip][m], id,ip=0,1,2
    // for atom m, which is a neighbor atom of atom i, m=0,jnum-1
    for (m = 0; m < 3; m++) {
      for (id = 0; id < 3; id++) {
        for (ip = 0; ip < 3; ip++) {
          dnormal[id][ip][m] = dn1[id][ip][m] / nn - n1[id] * dnn[ip][m] / nn2;
        }
      }
    }
    // TODO
  } else if (cont == 3) {
    pv12[0] = vet[0][1] * vet[1][2] - vet[1][1] * vet[0][2];
    pv12[1] = vet[0][2] * vet[1][0] - vet[1][2] * vet[0][0];
    pv12[2] = vet[0][0] * vet[1][1] - vet[1][0] * vet[0][1];
    // derivatives respect to the first neighbor, atom k
    dpv12[0][0][0] = 0.0;
    dpv12[0][1][0] = vet[1][2];
    dpv12[0][2][0] = -vet[1][1];
    dpv12[1][0][0] = -vet[1][2];
    dpv12[1][1][0] = 0.0;
    dpv12[1][2][0] = vet[1][0];
    dpv12[2][0][0] = vet[1][1];
    dpv12[2][1][0] = -vet[1][0];
    dpv12[2][2][0] = 0.0;
    // derivatives respect to the second neighbor, atom l
    dpv12[0][0][1] = 0.0;
    dpv12[0][1][1] = -vet[0][2];
    dpv12[0][2][1] = vet[0][1];
    dpv12[1][0][1] = vet[0][2];
    dpv12[1][1][1] = 0.0;
    dpv12[1][2][1] = -vet[0][0];
    dpv12[2][0][1] = -vet[0][1];
    dpv12[2][1][1] = vet[0][0];
    dpv12[2][2][1] = 0.0;

    // derivatives respect to the third neighbor, atom n
    for (id = 0; id < 3; id++) {
      for (ip = 0; ip < 3; ip++) { dpv12[id][ip][2] = 0.0; }
    }

    pv31[0] = vet[2][1] * vet[0][2] - vet[0][1] * vet[2][2];
    pv31[1] = vet[2][2] * vet[0][0] - vet[0][2] * vet[2][0];
    pv31[2] = vet[2][0] * vet[0][1] - vet[0][0] * vet[2][1];
    // derivatives respect to the first neighbor, atom k
    dpv31[0][0][0] = 0.0;
    dpv31[0][1][0] = -vet[2][2];
    dpv31[0][2][0] = vet[2][1];
    dpv31[1][0][0] = vet[2][2];
    dpv31[1][1][0] = 0.0;
    dpv31[1][2][0] = -vet[2][0];
    dpv31[2][0][0] = -vet[2][1];
    dpv31[2][1][0] = vet[2][0];
    dpv31[2][2][0] = 0.0;
    // derivatives respect to the third neighbor, atom n
    dpv31[0][0][2] = 0.0;
    dpv31[0][1][2] = vet[0][2];
    dpv31[0][2][2] = -vet[0][1];
    dpv31[1][0][2] = -vet[0][2];
    dpv31[1][1][2] = 0.0;
    dpv31[1][2][2] = vet[0][0];
    dpv31[2][0][2] = vet[0][1];
    dpv31[2][1][2] = -vet[0][0];
    dpv31[2][2][2] = 0.0;
    // derivatives respect to the second neighbor, atom l
    for (id = 0; id < 3; id++) {
      for (ip = 0; ip < 3; ip++) { dpv31[id][ip][1] = 0.0; }
    }

    pv23[0] = vet[1][1] * vet[2][2] - vet[2][1] * vet[1][2];
    pv23[1] = vet[1][2] * vet[2][0] - vet[2][2] * vet[1][0];
    pv23[2] = vet[1][0] * vet[2][1] - vet[2][0] * vet[1][1];
    // derivatives respect to the second neighbor, atom k
    for (id = 0; id < 3; id++) {
      for (ip = 0; ip < 3; ip++) { dpv23[id][ip][0] = 0.0; }
    }
    // derivatives respect to the second neighbor, atom l
    dpv23[0][0][1] = 0.0;
    dpv23[0][1][1] = vet[2][2];
    dpv23[0][2][1] = -vet[2][1];
    dpv23[1][0][1] = -vet[2][2];
    dpv23[1][1][1] = 0.0;
    dpv23[1][2][1] = vet[2][0];
    dpv23[2][0][1] = vet[2][1];
    dpv23[2][1][1] = -vet[2][0];
    dpv23[2][2][1] = 0.0;
    // derivatives respect to the third neighbor, atom n
    dpv23[0][0][2] = 0.0;
    dpv23[0][1][2] = -vet[1][2];
    dpv23[0][2][2] = vet[1][1];
    dpv23[1][0][2] = vet[1][2];
    dpv23[1][1][2] = 0.0;
    dpv23[1][2][2] = -vet[1][0];
    dpv23[2][0][2] = -vet[1][1];
    dpv23[2][1][2] = vet[1][0];
    dpv23[2][2][2] = 0.0;

    //############################################################################################
    // average the normal vectors by using the 3 neighboring planes
    n1[0] = (pv12[0] + pv31[0] + pv23[0]) / cont;
    n1[1] = (pv12[1] + pv31[1] + pv23[1]) / cont;
    n1[2] = (pv12[2] + pv31[2] + pv23[2]) / cont;
    // the magnitude of the normal vector
    nn2 = n1[0] * n1[0] + n1[1] * n1[1] + n1[2] * n1[2];
    nn = sqrt(nn2);
    // TODO
    // if (nn == 0) error->one(FLERR, "The magnitude of the normal vector is zero");
    // the unit normal vector
    normal[0] = n1[0] / nn;
    normal[1] = n1[1] / nn;
    normal[2] = n1[2] / nn;

    // for the central atoms, dnormdri is always zero
    for (id = 0; id < 3; id++) {
      for (ip = 0; ip < 3; ip++) { dnormdri[id][ip] = 0.0; }
    }

    // derivatives of non-normalized normal vector, dn1:3x3x3 array
    for (id = 0; id < 3; id++) {
      for (ip = 0; ip < 3; ip++) {
        for (m = 0; m < 3; m++) {
          dn1[id][ip][m] = (dpv12[id][ip][m] + dpv23[id][ip][m] + dpv31[id][ip][m]) / cont;
        }
      }
    }
    // derivatives of nn, dnn:3x3 vector
    // dnn[id][m]: the derivative of nn respect to r[id][m], id,m=0,1,2
    // r[id][m]: the id's component of atom m
    for (m = 0; m < 3; m++) {
      for (id = 0; id < 3; id++) {
        dnn[id][m] = (n1[0] * dn1[0][id][m] + n1[1] * dn1[1][id][m] + n1[2] * dn1[2][id][m]) / nn;
      }
    }
    // dnormal[id][ip][m][i]: the derivative of normal[id] respect to r[ip][m], id,ip=0,1,2
    // for atom m, which is a neighbor atom of atom i, m=0,jnum-1
    for (m = 0; m < 3; m++) {
      for (id = 0; id < 3; id++) {
        for (ip = 0; ip < 3; ip++) {
          dnormal[id][ip][m] = dn1[id][ip][m] / nn - n1[id] * dnn[ip][m] / nn2;
        }
      }
    }// TODO
  } else {
    // TODO: too many neighbors for calculating normals
  }
  // TODO
}

// calculate the van der Waals force and energy
static __device__ void calc_vdW(
  double r,
  double d,
  double d_Seff,
  double C_6,
  double Tap,
  double dTap,
  double &p2_vdW,
  double &f2_vdW)
{
  double rsq, r2inv, r6inv, r8inv;
  double TSvdw, TSvdw2inv, Vilp;
  double fpair, fsum;

  rsq = r * r;
  r2inv = 1.0 / rsq;
  r6inv = r2inv * r2inv * r2inv;
  r8inv = r2inv * r6inv;

  TSvdw = 1.0 + exp(-d_Seff * r + d);
  TSvdw2inv = pow(TSvdw, -2.0);
  Vilp = -C_6 * r6inv / TSvdw;

  // derivatives
  fpair = -6.0 * C_6 * r8inv / TSvdw + \
    C_6 * d_Seff * (TSvdw - 1) * TSvdw2inv * r8inv * r;
  fsum = fpair * Tap - Vilp * dTap / r;

  p2_vdW = Tap * Vilp;
  f2_vdW = fsum;
  
}

// calculate the repulsive force and energy
static __device__ void calc_rep(
  double (&delxyz)[3],
  double r,
  double C,
  double lambda_,
  double delta2inv,
  double epsilon,
  double z0,
  double (&normal)[3],
  double (&dnormdri)[3][3],
  double (&dnormal)[3][3][3])
{
  double prodnorm1, rsq, rhosq1, rdsq1, exp0, exp1, frho1, Erep, Vilp;
  double fpair, fpair1, fsum, delx, dely, delz, fkcx, fkcy, fkcz;
  double dprodnorm1[3] = {0.0, 0.0, 0.0};
  double fp1[3] = {0.0, 0.0, 0.0};
  double fprod1[3] = {0.0, 0.0, 0.0};
  double delki[3] = {0.0, 0.0, 0.0};
  double fk[3] = {0.0, 0.0, 0.0};

  delx = delxyz[0];
  dely = delxyz[1];
  delz = delxyz[2];

  rsq = r * r;
  // calculate the transverse distance
  prodnorm1 = normal[0] * delx + normal[1] * dely + normal[2] * delz;
  rhosq1 = rsq - prodnorm1 * prodnorm1;
  rdsq1 = rhosq1 * delta2inv;

  // store exponents
  exp0 = exp(-lambda_ * (r - z0));
  exp1 = exp(-rdsq1);

  frho1 = exp1 * C;
  Erep = 0.5 * epsilon + frho1;
  Vilp = exp0 * Erep;
  // TODO

  // derivatives
  fpair = lambda_ * exp0 / r * Erep;
  fpair1 = 2.0 * exp0 * frho1 * delta2inv;
  fsum = fpair + fpair1;

  // derivatives of the product of rij and ni, the resutl is a vector
  dprodnorm1[0] = 
    dnormdri[0][0] * delx + dnormdri[1][0] * dely + dnormdri[2][0] * delz;
  dprodnorm1[1] = 
    dnormdri[0][1] * delx + dnormdri[1][1] * dely + dnormdri[2][1] * delz;
  dprodnorm1[2] = 
    dnormdri[0][2] * delx + dnormdri[1][2] * dely + dnormdri[2][2] * delz;
  fp1[0] = prodnorm1 * normal[0] * fpair1;
  fp1[1] = prodnorm1 * normal[1] * fpair1;
  fp1[2] = prodnorm1 * normal[2] * fpair1;
  fprod1[0] = prodnorm1 * dprodnorm1[0] * fpair1;
  fprod1[1] = prodnorm1 * dprodnorm1[1] * fpair1;
  fprod1[2] = prodnorm1 * dprodnorm1[2] * fpair1;

  // fkcx = (delx * fsum - fp1[0]) * Tap - Vilp * dTap * delx / r;
  // fkcy = (dely * fsum - fp1[1]) * Tap - Vilp * dTap * dely / r;
  // fkcz = (delz * fsum - fp1[2]) * Tap - Vilp * dTap * delz / r;
  // TODO
  
}

// force evaluation kernel
static __global__ void gpu_find_force(
  ILP_Para ilp_para,
  const int number_of_particles,
  const int N1,
  const int N2,
  const Box box,
  const int *g_neighbor_number,
  const int *g_neighbor_list,
  int *g_ilp_neighbor_number,
  int *g_ilp_neighbor_list,
  const int *group_label,
  const int *g_type,
  const double *__restrict__ g_x,
  const double *__restrict__ g_y,
  const double *__restrict__ g_z,
  double *g_fx,
  double *g_fy,
  double *g_fz,
  double *g_virial,
  double *g_potential)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1; // particle index
  double s_fx = 0.0;                                   // force_x
  double s_fy = 0.0;                                   // force_y
  double s_fz = 0.0;                                   // force_z
  double s_pe = 0.0;                                   // potential energy
  double s_sxx = 0.0;                                  // virial_stress_xx
  double s_sxy = 0.0;                                  // virial_stress_xy
  double s_sxz = 0.0;                                  // virial_stress_xz
  double s_syx = 0.0;                                  // virial_stress_yx
  double s_syy = 0.0;                                  // virial_stress_yy
  double s_syz = 0.0;                                  // virial_stress_yz
  double s_szx = 0.0;                                  // virial_stress_zx
  double s_szy = 0.0;                                  // virial_stress_zy
  double s_szz = 0.0;                                  // virial_stress_zz

  double r = 0.0;
  double rsq = 0.0;
  double Rcut = 0.0;
  // double r2inv, r6inv, r8inv;

  if (n1 < N2) {
    printf("********* ILP find force GPU **********\n");
    double x12, y12, z12;
    int neighor_number = g_neighbor_number[n1];
    int type1 = g_type[n1];
    double x1 = g_x[n1];
    double y1 = g_y[n1];
    double z1 = g_z[n1];

    // calculate the normal
    // TODO: loop the ILP_neigh to create the vet and cont
    int cont = 0;
    // TODO: how to initialize normals
    double normal[3];
    double dnormdri[3][3];
    double dnormal[3][3][3];

    double vet[3][3];
    int id, ip, m;
    for (id = 0; id < 3; ++id) {
      normal[id] = 0.0;
      for (ip = 0; ip < 3; ++ip) {
        vet[id][ip] = 0.0;
        dnormdri[id][ip] = 0.0;
        for (m = 0; m < 3; ++m) {
          dnormal[id][ip][m] = 0.0;
        }
      }
    }

    int ilp_neighbor_number = g_ilp_neighbor_number[n1];
    for (int i1 = 0; i1 < ilp_neighbor_number; ++i1) {
      int n2_ilp = g_ilp_neighbor_list[n1 + number_of_particles * i1];
      x12 = g_x[n2_ilp] - x1;
      y12 = g_y[n2_ilp] - y1;
      z12 = g_z[n2_ilp] - z1;
      vet[cont][0] = x12;
      vet[cont][1] = y12;
      vet[cont][2] = z12;
      ++cont;
    }

    printf("********* ILP calc normal **********\n");
    calc_normal(vet, cont, normal, dnormdri, dnormal);

    // calculate energy and force
    for (int i1 = 0; i1 < neighor_number; ++i1) {
      int n2 = g_neighbor_list[n1 + number_of_particles * i1];
      int type2 = g_type[n2];

      x12 = g_x[n2] - x1;
      y12 = g_y[n2] - y1;
      z12 = g_z[n2] - z1;
      apply_mic(box, x12, y12, z12);

      // calculate distance between atoms
      rsq = x12 * x12 + y12 * y12 + z12 * z12;
      r = sqrt(rsq);
      Rcut = ilp_para.r_cut[type1][type2];
      // TODO: not in the same layer
      if (r >= Rcut || group_label[n1] == group_label[n2]) {
        continue;
      }

      printf("********* ILP calc Tap **********\n");
      double Tap, dTap;
      Tap = calc_Tap(r, Rcut);
      dTap = calc_dTap(r, Rcut);

      printf("********* ILP calc vdW **********\n");
      double p2_vdW, f2_vdW;
      calc_vdW(
        r,
        ilp_para.d[type1][type2],
        ilp_para.d_Seff[type1][type2],
        ilp_para.C_6[type1][type2],
        Tap,
        dTap,
        p2_vdW,
        f2_vdW);
      
      double f12x = f2_vdW * x12 * 0.5;
      double f12y = f2_vdW * y12 * 0.5;
      double f12z = f2_vdW * z12 * 0.5;
      double f21x = -f12x;
      double f21y = -f12y;
      double f21z = -f12z;

      s_fx += f12x - f21x;
      s_fy += f12y - f21y;
      s_fz += f12z - f21z;

      s_pe += p2_vdW * 0.5;
      s_sxx += x12 * f21x;
      s_sxy += x12 * f21y;
      s_sxz += x12 * f21z;
      s_syx += y12 * f21x;
      s_syy += y12 * f21y;
      s_syz += y12 * f21z;
      s_szx += z12 * f21x;
      s_szy += z12 * f21y;
      s_szz += z12 * f21z;

      
      printf("********* ILP calc rep **********\n");
      double delxyz[3] = {x12, y12, z12};
      // calc_rep(
      //   delxyz,
      //   r,
      //   ilp_para->C[type1][type2],
      //   ilp_para->lambda[type1][type2],
      //   ilp_para->delta2inv[type1][type2],
      //   ilp_para->epsilon[type1][type2],
      //   ilp_para->z0[type1][type2],
      //   normal,
      //   dnormdri,
      //   dnormal);

      double C = ilp_para.C[type1][type2];
      double lambda_ = ilp_para.lambda[type1][type2];
      double delta2inv = ilp_para.delta2inv[type1][type2];
      double epsilon = ilp_para.epsilon[type1][type2];
      double z0 = ilp_para.z0[type1][type2];
      // calc_rep
      double prodnorm1, rhosq1, rdsq1, exp0, exp1, frho1, Erep, Vilp;
      double fpair, fpair1, fsum, delx, dely, delz, fkcx, fkcy, fkcz;
      double dprodnorm1[3] = {0.0, 0.0, 0.0};
      double fp1[3] = {0.0, 0.0, 0.0};
      double fprod1[3] = {0.0, 0.0, 0.0};
      double delki[3] = {0.0, 0.0, 0.0};
      double fk[3] = {0.0, 0.0, 0.0};

      delx = delxyz[0];
      dely = delxyz[1];
      delz = delxyz[2];

      // rsq = r * r;
      // calculate the transverse distance
      prodnorm1 = normal[0] * delx + normal[1] * dely + normal[2] * delz;
      rhosq1 = rsq - prodnorm1 * prodnorm1;
      rdsq1 = rhosq1 * delta2inv;

      // store exponents
      exp0 = exp(-lambda_ * (r - z0));
      exp1 = exp(-rdsq1);

      frho1 = exp1 * C;
      Erep = 0.5 * epsilon + frho1;
      Vilp = exp0 * Erep;
      // TODO

      // derivatives
      fpair = lambda_ * exp0 / r * Erep;
      fpair1 = 2.0 * exp0 * frho1 * delta2inv;
      fsum = fpair + fpair1;

      // derivatives of the product of rij and ni, the resutl is a vector
      dprodnorm1[0] = 
        dnormdri[0][0] * delx + dnormdri[1][0] * dely + dnormdri[2][0] * delz;
      dprodnorm1[1] = 
        dnormdri[0][1] * delx + dnormdri[1][1] * dely + dnormdri[2][1] * delz;
      dprodnorm1[2] = 
        dnormdri[0][2] * delx + dnormdri[1][2] * dely + dnormdri[2][2] * delz;
      fp1[0] = prodnorm1 * normal[0] * fpair1;
      fp1[1] = prodnorm1 * normal[1] * fpair1;
      fp1[2] = prodnorm1 * normal[2] * fpair1;
      fprod1[0] = prodnorm1 * dprodnorm1[0] * fpair1;
      fprod1[1] = prodnorm1 * dprodnorm1[1] * fpair1;
      fprod1[2] = prodnorm1 * dprodnorm1[2] * fpair1;

      fkcx = (delx * fsum - fp1[0]) * Tap - Vilp * dTap * delx / r;
      fkcy = (dely * fsum - fp1[1]) * Tap - Vilp * dTap * dely / r;
      fkcz = (delz * fsum - fp1[2]) * Tap - Vilp * dTap * delz / r;

      s_fx += fkcx - fprod1[0] * Tap;
      s_fy += fkcy - fprod1[1] * Tap;
      s_fz += fkcz - fprod1[2] * Tap;

      // TODO: write data of other atoms, need atomic operation???
      g_fx[n2] -= fkcx;
      g_fy[n2] -= fkcy;
      g_fz[n2] -= fkcz;

      for (int kk = 0; kk < ilp_neighbor_number; ++kk) {
        int n2_ilp = g_ilp_neighbor_list[n1 + number_of_particles * kk];
        if (n2_ilp == n1) continue;
        // derivatives of the product of rij and ni respect to rk, k=0,1,2, where atom k is the neighbors of atom i
        dprodnorm1[0] = dnormal[0][0][kk] * delx + dnormal[1][0][kk] * dely +
            dnormal[2][0][kk] * delz;
        dprodnorm1[1] = dnormal[0][1][kk] * delx + dnormal[1][1][kk] * dely +
            dnormal[2][1][kk] * delz;
        dprodnorm1[2] = dnormal[0][2][kk] * delx + dnormal[1][2][kk] * dely +
            dnormal[2][2][kk] * delz;
        fk[0] = (-prodnorm1 * dprodnorm1[0] * fpair1) * Tap;
        fk[1] = (-prodnorm1 * dprodnorm1[1] * fpair1) * Tap;
        fk[2] = (-prodnorm1 * dprodnorm1[2] * fpair1) * Tap;

        // TODO: write data of other atoms, need atomic operation???
        g_fx[n2_ilp] += fk[0];
        g_fy[n2_ilp] += fk[1];
        g_fz[n2_ilp] += fk[2];

        delki[0] = g_x[n2_ilp] - x1;
        delki[1] = g_y[n2_ilp] - y1;
        delki[2] = g_z[n2_ilp] - z1;

      }
      s_pe += Tap * Vilp;

    }

    // TODO

    // save force
    g_fx[n1] += s_fx;
    g_fy[n1] += s_fy;
    g_fz[n1] += s_fz;

    // save virial
    // xx xy xz    0 3 4
    // yx yy yz    6 1 5
    // zx zy zz    7 8 2
    g_virial[n1 + 0 * number_of_particles] += s_sxx;
    g_virial[n1 + 1 * number_of_particles] += s_syy;
    g_virial[n1 + 2 * number_of_particles] += s_szz;
    g_virial[n1 + 3 * number_of_particles] += s_sxy;
    g_virial[n1 + 4 * number_of_particles] += s_sxz;
    g_virial[n1 + 5 * number_of_particles] += s_syz;
    g_virial[n1 + 6 * number_of_particles] += s_syx;
    g_virial[n1 + 7 * number_of_particles] += s_szx;
    g_virial[n1 + 8 * number_of_particles] += s_szy;

    // save potential
    g_potential[n1] += s_pe;

  }
}

void ILP::compute(
  Box &box,
  const GPU_Vector<int> &type,
  const GPU_Vector<double> &position_per_atom,
  GPU_Vector<double> &potential_per_atom,
  GPU_Vector<double> &force_per_atom,
  GPU_Vector<double> &virial_per_atom)
{
  // TODO
}
// find force and related quantities
void ILP::compute(
  Box &box,
  const GPU_Vector<int> &type,
  const GPU_Vector<double> &position_per_atom,
  GPU_Vector<double> &potential_per_atom,
  GPU_Vector<double> &force_per_atom,
  GPU_Vector<double> &virial_per_atom,
  std::vector<Group> &group)
{
  printf("********* ILP compute **********\n");
  const int number_of_atoms = type.size();
  int grid_size = (N2 - N1 - 1) / BLOCK_SIZE_FORCE + 1;

// what's this??
#ifdef USE_FIXED_NEIGHBOR
  static int num_calls = 0;
#endif
#ifdef USE_FIXED_NEIGHBOR
  if (num_calls++ == 0) {
#endif
    find_neighbor(
      N1,
      N2,
      rc,
      box,
      type,
      position_per_atom,
      ilp_data.cell_count,
      ilp_data.cell_count_sum,
      ilp_data.cell_contents,
      ilp_data.NN,
      ilp_data.NL);
#ifdef USE_FIXED_NEIGHBOR
  }
#endif

  const double* x = position_per_atom.data();
  const double* y = position_per_atom.data() + number_of_atoms;
  const double* z = position_per_atom.data() + number_of_atoms * 2;
  const int *NN = ilp_data.NN.data();
  const int *NL = ilp_data.NL.data();
  int *ilp_NL = ilp_data.ilp_NL.data();
  int *ilp_NN = ilp_data.ilp_NN.data();

  // find ILP neighbor list
  // TODO: __global__ ???
  // TODO: assume the first group column is for ILP
  printf("********* ILP neigh **********\n");
  const int *group_label = group[0].label.data();
  ILP_neighbor<<<grid_size, BLOCK_SIZE_FORCE>>>(
    number_of_atoms, N1, N2, box, NN, NL, \
    type.data(), ilp_para, x, y, z, ilp_NN, \
    ilp_NL, group_label);
  CUDA_CHECK_KERNEL

  printf("********* ILP find force **********\n");
  gpu_find_force<<<grid_size, BLOCK_SIZE_FORCE>>>(
    ilp_para,
    number_of_atoms,
    N1,
    N2,
    box,
    NN,
    NL,
    ilp_NN,
    ilp_NL,
    group_label,
    type.data(),
    x,
    y,
    z,
    force_per_atom.data(),
    force_per_atom.data() + number_of_atoms,
    force_per_atom.data() + number_of_atoms * 2,
    virial_per_atom.data(),
    potential_per_atom.data());
  // TODO
  CUDA_CHECK_KERNEL
}
