#include "hip/hip_runtime.h"

/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

/*----------------------------------------------------------------------------80
The class dealing with the Lennard-Jones (LJ) pairwise potentials.
------------------------------------------------------------------------------*/

#include "ilp.cuh"
#include "neighbor.cuh"
#include "utilities/error.cuh"

#define BLOCK_SIZE_FORCE 128

ILP::ILP(FILE* fid, int num_types, int num_atoms)
{
  printf("Use %d-element ILP potential with elements:\n", num_types);
  if (!(num_types >= 1 && num_types <= MAX_TYPE_ILP)) {
    PRINT_INPUT_ERROR("Incorrect number of ILP parameters.\n");
  }
  for (int n = 0; n < num_types; ++n) {
    char atom_symbol[10];
    int count = fscanf(fid, "%s", atom_symbol);
    PRINT_SCANF_ERROR(count, 1, "Reading error for ILP potential.");
    printf(" %s", atom_symbol);
  }
  printf("\n");

  // read parameters
  float beta, alpha, delta, epsilon, C, d, sR;
  float reff, C6, S, rcut_ilp, rcut_global;
  rc = 0.0;
  for (int n = 0; n < num_types; ++n) {
    for (int m = 0; m < num_types; ++m) {
      int count = fscanf(fid, "%f%f%f%f%f%f%f%f%f%f%f%f", \
      &beta, &alpha, &delta, &epsilon, &C, &d, &sR, &reff, &C6, &S, \
      &rcut_ilp, &rcut_global);
      PRINT_SCANF_ERROR(count, 12, "Reading error for ILP potential.");

      ilp_para.C[n][m] = C;
      ilp_para.C_6[n][m] = C6;
      ilp_para.d[n][m] = d;
      ilp_para.d_Seff[n][m] = d / sR / reff;
      ilp_para.epsilon[n][m] = epsilon;
      ilp_para.z0[n][m] = beta;
      ilp_para.lambda[n][m] = alpha / beta;
      ilp_para.delta2inv[n][m] = 1.0 / (delta * delta); //TODO: how faster?
      ilp_para.S[n][m] = S;
      ilp_para.rcutsq_ilp[n][m] = rcut_ilp * rcut_ilp;
      ilp_para.rcut_global[n][m] = rcut_global;
      float meV = 1e-3 * S;
      ilp_para.C[n][m] *= meV;
      ilp_para.C_6[n][m] *= meV;
      ilp_para.epsilon[n][m] *= meV;

      // TODO: ILP has taper function, check if necessary
      if (rc < rcut_global)
        rc = rcut_global;
    }
  }

  int max_neighbor_number = min(num_atoms, CUDA_MAX_NL);
  ilp_data.NN.resize(num_atoms);
  ilp_data.NL.resize(num_atoms * max_neighbor_number);
  ilp_data.cell_count.resize(num_atoms);
  ilp_data.cell_count_sum.resize(num_atoms);
  ilp_data.cell_contents.resize(num_atoms);

  // init ilp neighbor list
  ilp_data.ilp_NN.resize(num_atoms);
  ilp_data.ilp_NL.resize(num_atoms * MAX_ILP_NEIGHBOR);
  ilp_data.reduce_NL.resize(num_atoms * max_neighbor_number);

  ilp_data.f12x.resize(num_atoms * max_neighbor_number);
  ilp_data.f12y.resize(num_atoms * max_neighbor_number);
  ilp_data.f12z.resize(num_atoms * max_neighbor_number);

  ilp_data.f12x_ilp_neigh.resize(num_atoms * MAX_ILP_NEIGHBOR);
  ilp_data.f12y_ilp_neigh.resize(num_atoms * MAX_ILP_NEIGHBOR);
  ilp_data.f12z_ilp_neigh.resize(num_atoms * MAX_ILP_NEIGHBOR);

  // init constant cutoff coeff
  float h_tap_coeff[8] = \
    {1.0f, 0.0f, 0.0f, 0.0f, -35.0f, 84.0f, -70.0f, 20.0f};
  hipMemcpyToSymbol(HIP_SYMBOL(Tap_coeff), h_tap_coeff, 8 * sizeof(float));
  CUDA_CHECK_KERNEL
}

ILP::~ILP(void)
{
  // TODO
}

// calculate the long-range cutoff term
inline static __device__ double calc_Tap(const float r_ij, const float Rcutinv)
{
  float Tap, r;

  r = r_ij * Rcutinv;
  if (r >= 1.0f) {
    Tap = 0.0f;
  } else {
    Tap = Tap_coeff[7];
    for (int i = 6; i >= 0; --i) {
      Tap = Tap * r + Tap_coeff[i];
    }
  }

  return Tap;
}

// calculate the derivatives of long-range cutoff term
inline static __device__ double calc_dTap(const float r_ij, const float Rcut, const float Rcutinv)
{
  float dTap, r;
  
  r = r_ij * Rcutinv;
  if (r >= Rcut) {
    dTap = 0.0f;
  } else {
    dTap = 7.0f * Tap_coeff[7];
    for (int i = 6; i > 0; --i) {
      dTap = dTap * r + i * Tap_coeff[i];
    }
    dTap *= Rcutinv;
  }

  return dTap;
}

// create ILP neighbor list from main neighbor list to calculate normals
static __global__ void ILP_neighbor(
  const int number_of_particles,
  const int N1,
  const int N2,
  const Box box,
  const int *g_neighbor_number,
  const int *g_neighbor_list,
  const int *g_type,
  ILP_Para ilp_para,
  const double* __restrict__ g_x,
  const double* __restrict__ g_y,
  const double* __restrict__ g_z,
  int *ilp_neighbor_number,
  int *ilp_neighbor_list,
  const int *group_label)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1; // particle index

  if (n1 < N2) {
    int count = 0;
    int neighbor_number = g_neighbor_number[n1];
    int type1 = g_type[n1];
    double x1 = g_x[n1];
    double y1 = g_y[n1];
    double z1 = g_z[n1];

    for (int i1 = 0; i1 < neighbor_number; ++i1) {
      int n2 = g_neighbor_list[n1 + number_of_particles * i1];
      int type2 = g_type[n2];

      double x12 = g_x[n2] - x1;
      double y12 = g_y[n2] - y1;
      double z12 = g_z[n2] - z1;
      apply_mic(box, x12, y12, z12);
      double d12sq = x12 * x12 + y12 * y12 + z12 * z12;
      // TODO: use local memory to save rcutsq to reduce global read
      double rcutsq = ilp_para.rcutsq_ilp[type1][type2];


      if (group_label[n1] == group_label[n2] && d12sq < rcutsq && d12sq != 0) {
        ilp_neighbor_list[count++ * number_of_particles + n1] = n2;
      }
    }
    ilp_neighbor_number[n1] = count;

    if (count > MAX_ILP_NEIGHBOR) {
      // error, there are too many neighbors for some atoms, 
      printf("\n===== ILP neighbor number[%d] is greater than 3 =====\n", count);
      
      int nei1 = ilp_neighbor_list[0 * number_of_particles + n1];
      int nei2 = ilp_neighbor_list[1 * number_of_particles + n1];
      int nei3 = ilp_neighbor_list[2 * number_of_particles + n1];
      int nei4 = ilp_neighbor_list[3 * number_of_particles + n1];
      printf("===== n1[%d] nei1[%d] nei2 [%d] nei3[%d] nei4[%d] =====\n", n1, nei1, nei2, nei3, nei4);
      return;
      // please check your configuration
    }
  }
}

// calculate the normals and its derivatives
static __device__ void calc_normal(
  float (&vet)[3][3],
  int cont,
  float (&normal)[3],
  float (&dnormdri)[3][3],
  float (&dnormal)[3][3][3])
{
  int id, ip, m;
  float pv12[3], pv31[3], pv23[3], n1[3], dni[3];
  float dnn[3][3], dpvdri[3][3];
  float dn1[3][3][3], dpv12[3][3][3], dpv23[3][3][3], dpv31[3][3][3];

  float nninv, continv;

  // initialize the arrays
  for (id = 0; id < 3; id++) {
    pv12[id] = 0.0f;
    pv31[id] = 0.0f;
    pv23[id] = 0.0f;
    n1[id] = 0.0f;
    dni[id] = 0.0f;
    for (ip = 0; ip < 3; ip++) {
      dnn[ip][id] = 0.0f;
      dpvdri[ip][id] = 0.0f;
      for (m = 0; m < 3; m++) {
        dpv12[ip][id][m] = 0.0f;
        dpv31[ip][id][m] = 0.0f;
        dpv23[ip][id][m] = 0.0f;
        dn1[ip][id][m] = 0.0f;
      }
    }
  }

  if (cont <= 1) {
    normal[0] = 0.0;
    normal[1] = 0.0;
    normal[2] = 1.0;
    for (id = 0; id < 3; ++id) {
      for (ip = 0; ip < 3; ++ip) {
        dnormdri[id][ip] = 0.0;
        for (m = 0; m < 3; ++m) {
          dnormal[id][ip][m] = 0.0;
        }
      }
    }
  } else if (cont == 2) {
    pv12[0] = vet[0][1] * vet[1][2] - vet[1][1] * vet[0][2];
    pv12[1] = vet[0][2] * vet[1][0] - vet[1][2] * vet[0][0];
    pv12[2] = vet[0][0] * vet[1][1] - vet[1][0] * vet[0][1];
    // derivatives of pv12[0] to ri
    dpvdri[0][0] = 0.0f;
    dpvdri[0][1] = vet[0][2] - vet[1][2];
    dpvdri[0][2] = vet[1][1] - vet[0][1];
    // derivatives of pv12[1] to ri
    dpvdri[1][0] = vet[1][2] - vet[0][2];
    dpvdri[1][1] = 0.0f;
    dpvdri[1][2] = vet[0][0] - vet[1][0];
    // derivatives of pv12[2] to ri
    dpvdri[2][0] = vet[0][1] - vet[1][1];
    dpvdri[2][1] = vet[1][0] - vet[0][0];
    dpvdri[2][2] = 0.0f;

    dpv12[0][0][0] = 0.0f;
    dpv12[0][1][0] = vet[1][2];
    dpv12[0][2][0] = -vet[1][1];
    dpv12[1][0][0] = -vet[1][2];
    dpv12[1][1][0] = 0.0f;
    dpv12[1][2][0] = vet[1][0];
    dpv12[2][0][0] = vet[1][1];
    dpv12[2][1][0] = -vet[1][0];
    dpv12[2][2][0] = 0.0f;

    // derivatives respect to the second neighbor, atom l
    dpv12[0][0][1] = 0.0f;
    dpv12[0][1][1] = -vet[0][2];
    dpv12[0][2][1] = vet[0][1];
    dpv12[1][0][1] = vet[0][2];
    dpv12[1][1][1] = 0.0f;
    dpv12[1][2][1] = -vet[0][0];
    dpv12[2][0][1] = -vet[0][1];
    dpv12[2][1][1] = vet[0][0];
    dpv12[2][2][1] = 0.0f;

    // derivatives respect to the third neighbor, atom n
    // derivatives of pv12 to rn is zero
    for (id = 0; id < 3; id++) {
      for (ip = 0; ip < 3; ip++) { dpv12[id][ip][2] = 0.0f; }
    }

    n1[0] = pv12[0];
    n1[1] = pv12[1];
    n1[2] = pv12[2];
    // the magnitude of the normal vector
    // nn2 = n1[0] * n1[0] + n1[1] * n1[1] + n1[2] * n1[2];
    // nn = sqrt(nn2);
    // nninv = 1.0 / nn;
    nninv = rnorm3df(n1[0], n1[1], n1[2]);
    
    // TODO
    // if (nn == 0) error->one(FLERR, "The magnitude of the normal vector is zero");
    // the unit normal vector
    normal[0] = n1[0] * nninv;
    normal[1] = n1[1] * nninv;
    normal[2] = n1[2] * nninv;
    // derivatives of nn, dnn:3x1 vector
    dni[0] = (n1[0] * dpvdri[0][0] + n1[1] * dpvdri[1][0] + n1[2] * dpvdri[2][0]) * nninv;
    dni[1] = (n1[0] * dpvdri[0][1] + n1[1] * dpvdri[1][1] + n1[2] * dpvdri[2][1]) * nninv;
    dni[2] = (n1[0] * dpvdri[0][2] + n1[1] * dpvdri[1][2] + n1[2] * dpvdri[2][2]) * nninv;
    // derivatives of unit vector ni respect to ri, the result is 3x3 matrix
    for (id = 0; id < 3; id++) {
      for (ip = 0; ip < 3; ip++) {
        dnormdri[id][ip] = dpvdri[id][ip] * nninv - n1[id] * dni[ip] * nninv * nninv;
      }
    }
    // derivatives of non-normalized normal vector, dn1:3x3x3 array
    for (id = 0; id < 3; id++) {
      for (ip = 0; ip < 3; ip++) {
        for (m = 0; m < 3; m++) { dn1[id][ip][m] = dpv12[id][ip][m]; }
      }
    }
    // derivatives of nn, dnn:3x3 vector
    // dnn[id][m]: the derivative of nn respect to r[id][m], id,m=0,1,2
    // r[id][m]: the id's component of atom m
    for (m = 0; m < 3; m++) {
      for (id = 0; id < 3; id++) {
        dnn[id][m] = (n1[0] * dn1[0][id][m] + n1[1] * dn1[1][id][m] + n1[2] * dn1[2][id][m]) * nninv;
      }
    }
    // dnormal[id][ip][m][i]: the derivative of normal[id] respect to r[ip][m], id,ip=0,1,2
    // for atom m, which is a neighbor atom of atom i, m=0,jnum-1
    for (m = 0; m < 3; m++) {
      for (id = 0; id < 3; id++) {
        for (ip = 0; ip < 3; ip++) {
          dnormal[id][ip][m] = dn1[id][ip][m] * nninv - n1[id] * dnn[ip][m] * nninv * nninv;
        }
      }
    }
  } else if (cont == 3) {
    continv = 1.0 / cont;

    pv12[0] = vet[0][1] * vet[1][2] - vet[1][1] * vet[0][2];
    pv12[1] = vet[0][2] * vet[1][0] - vet[1][2] * vet[0][0];
    pv12[2] = vet[0][0] * vet[1][1] - vet[1][0] * vet[0][1];
    // derivatives respect to the first neighbor, atom k
    dpv12[0][0][0] = 0.0f;
    dpv12[0][1][0] = vet[1][2];
    dpv12[0][2][0] = -vet[1][1];
    dpv12[1][0][0] = -vet[1][2];
    dpv12[1][1][0] = 0.0f;
    dpv12[1][2][0] = vet[1][0];
    dpv12[2][0][0] = vet[1][1];
    dpv12[2][1][0] = -vet[1][0];
    dpv12[2][2][0] = 0.0f;
    // derivatives respect to the second neighbor, atom l
    dpv12[0][0][1] = 0.0f;
    dpv12[0][1][1] = -vet[0][2];
    dpv12[0][2][1] = vet[0][1];
    dpv12[1][0][1] = vet[0][2];
    dpv12[1][1][1] = 0.0f;
    dpv12[1][2][1] = -vet[0][0];
    dpv12[2][0][1] = -vet[0][1];
    dpv12[2][1][1] = vet[0][0];
    dpv12[2][2][1] = 0.0f;

    // derivatives respect to the third neighbor, atom n
    for (id = 0; id < 3; id++) {
      for (ip = 0; ip < 3; ip++) { dpv12[id][ip][2] = 0.0f; }
    }

    pv31[0] = vet[2][1] * vet[0][2] - vet[0][1] * vet[2][2];
    pv31[1] = vet[2][2] * vet[0][0] - vet[0][2] * vet[2][0];
    pv31[2] = vet[2][0] * vet[0][1] - vet[0][0] * vet[2][1];
    // derivatives respect to the first neighbor, atom k
    dpv31[0][0][0] = 0.0f;
    dpv31[0][1][0] = -vet[2][2];
    dpv31[0][2][0] = vet[2][1];
    dpv31[1][0][0] = vet[2][2];
    dpv31[1][1][0] = 0.0f;
    dpv31[1][2][0] = -vet[2][0];
    dpv31[2][0][0] = -vet[2][1];
    dpv31[2][1][0] = vet[2][0];
    dpv31[2][2][0] = 0.0f;
    // derivatives respect to the third neighbor, atom n
    dpv31[0][0][2] = 0.0f;
    dpv31[0][1][2] = vet[0][2];
    dpv31[0][2][2] = -vet[0][1];
    dpv31[1][0][2] = -vet[0][2];
    dpv31[1][1][2] = 0.0f;
    dpv31[1][2][2] = vet[0][0];
    dpv31[2][0][2] = vet[0][1];
    dpv31[2][1][2] = -vet[0][0];
    dpv31[2][2][2] = 0.0f;
    // derivatives respect to the second neighbor, atom l
    for (id = 0; id < 3; id++) {
      for (ip = 0; ip < 3; ip++) { dpv31[id][ip][1] = 0.0f; }
    }

    pv23[0] = vet[1][1] * vet[2][2] - vet[2][1] * vet[1][2];
    pv23[1] = vet[1][2] * vet[2][0] - vet[2][2] * vet[1][0];
    pv23[2] = vet[1][0] * vet[2][1] - vet[2][0] * vet[1][1];
    // derivatives respect to the second neighbor, atom k
    for (id = 0; id < 3; id++) {
      for (ip = 0; ip < 3; ip++) { dpv23[id][ip][0] = 0.0f; }
    }
    // derivatives respect to the second neighbor, atom l
    dpv23[0][0][1] = 0.0f;
    dpv23[0][1][1] = vet[2][2];
    dpv23[0][2][1] = -vet[2][1];
    dpv23[1][0][1] = -vet[2][2];
    dpv23[1][1][1] = 0.0f;
    dpv23[1][2][1] = vet[2][0];
    dpv23[2][0][1] = vet[2][1];
    dpv23[2][1][1] = -vet[2][0];
    dpv23[2][2][1] = 0.0f;
    // derivatives respect to the third neighbor, atom n
    dpv23[0][0][2] = 0.0f;
    dpv23[0][1][2] = -vet[1][2];
    dpv23[0][2][2] = vet[1][1];
    dpv23[1][0][2] = vet[1][2];
    dpv23[1][1][2] = 0.0f;
    dpv23[1][2][2] = -vet[1][0];
    dpv23[2][0][2] = -vet[1][1];
    dpv23[2][1][2] = vet[1][0];
    dpv23[2][2][2] = 0.0f;

    //############################################################################################
    // average the normal vectors by using the 3 neighboring planes
    n1[0] = (pv12[0] + pv31[0] + pv23[0]) * continv;
    n1[1] = (pv12[1] + pv31[1] + pv23[1]) * continv;
    n1[2] = (pv12[2] + pv31[2] + pv23[2]) * continv;
    // the magnitude of the normal vector
    // nn2 = n1[0] * n1[0] + n1[1] * n1[1] + n1[2] * n1[2];
    // nn = sqrt(nn2);

    // nninv = 1.0 / nn;
    nninv = rnorm3df(n1[0], n1[1], n1[2]);
    // TODO
    // if (nn == 0) error->one(FLERR, "The magnitude of the normal vector is zero");
    // the unit normal vector
    normal[0] = n1[0] * nninv;
    normal[1] = n1[1] * nninv;
    normal[2] = n1[2] * nninv;

    // for the central atoms, dnormdri is always zero
    for (id = 0; id < 3; id++) {
      for (ip = 0; ip < 3; ip++) { dnormdri[id][ip] = 0.0f; }
    }

    // derivatives of non-normalized normal vector, dn1:3x3x3 array
    for (id = 0; id < 3; id++) {
      for (ip = 0; ip < 3; ip++) {
        for (m = 0; m < 3; m++) {
          dn1[id][ip][m] = (dpv12[id][ip][m] + dpv23[id][ip][m] + dpv31[id][ip][m]) * continv;
        }
      }
    }
    // derivatives of nn, dnn:3x3 vector
    // dnn[id][m]: the derivative of nn respect to r[id][m], id,m=0,1,2
    // r[id][m]: the id's component of atom m
    for (m = 0; m < 3; m++) {
      for (id = 0; id < 3; id++) {
        dnn[id][m] = (n1[0] * dn1[0][id][m] + n1[1] * dn1[1][id][m] + n1[2] * dn1[2][id][m]) * nninv;
      }
    }
    // dnormal[id][ip][m][i]: the derivative of normal[id] respect to r[ip][m], id,ip=0,1,2
    // for atom m, which is a neighbor atom of atom i, m=0,jnum-1
    for (m = 0; m < 3; m++) {
      for (id = 0; id < 3; id++) {
        for (ip = 0; ip < 3; ip++) {
          dnormal[id][ip][m] = dn1[id][ip][m] * nninv - n1[id] * dnn[ip][m] * nninv * nninv;
        }
      }
    }
  } else {
    // TODO: error! too many neighbors for calculating normals
  }
}

// calculate the van der Waals force and energy
inline static __device__ void calc_vdW(
  float r,
  float rinv,
  float rsq,
  float d,
  float d_Seff,
  float C_6,
  float Tap,
  float dTap,
  float &p2_vdW,
  float &f2_vdW)
{
  float r2inv, r6inv, r8inv;
  float TSvdw, TSvdwinv, Vilp;
  float fpair, fsum;

  r2inv = 1.0f / rsq;
  r6inv = r2inv * r2inv * r2inv;
  r8inv = r2inv * r6inv;

  // TODO: use float
  // TSvdw = 1.0 + exp(-d_Seff * r + d);
  TSvdw = 1.0f + expf(-d_Seff * r + d);
  TSvdwinv = 1.0f / TSvdw;
  Vilp = -C_6 * r6inv * TSvdwinv;

  // derivatives
  // fpair = -6.0 * C_6 * r8inv * TSvdwinv + \
  //   C_6 * d_Seff * (TSvdw - 1.0) * TSvdwinv * TSvdwinv * r8inv * r;
  fpair = (-6.0f + d_Seff * (TSvdw - 1.0f) * TSvdwinv * r ) * C_6 * TSvdwinv * r8inv;
  fsum = fpair * Tap - Vilp * dTap * rinv;

  p2_vdW = Tap * Vilp;
  f2_vdW = fsum;
  
}



// force evaluation kernel
static __global__ void gpu_find_force(
  ILP_Para ilp_para,
  const int number_of_particles,
  const int N1,
  const int N2,
  const Box box,
  const int *g_neighbor_number,
  const int *g_neighbor_list,
  int *g_ilp_neighbor_number,
  int *g_ilp_neighbor_list,
  const int *group_label,
  const int *g_type,
  const double *__restrict__ g_x,
  const double *__restrict__ g_y,
  const double *__restrict__ g_z,
  double *g_fx,
  double *g_fy,
  double *g_fz,
  double *g_virial,
  double *g_potential,
  float *g_f12x,
  float *g_f12y,
  float *g_f12z,
  float *g_f12x_ilp_neigh,
  float *g_f12y_ilp_neigh,
  float *g_f12z_ilp_neigh)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1; // particle index
  float s_fx = 0.0f;                                   // force_x
  float s_fy = 0.0f;                                   // force_y
  float s_fz = 0.0f;                                   // force_z
  float s_pe = 0.0f;                                   // potential energy
  float s_sxx = 0.0f;                                  // virial_stress_xx
  float s_sxy = 0.0f;                                  // virial_stress_xy
  float s_sxz = 0.0f;                                  // virial_stress_xz
  float s_syx = 0.0f;                                  // virial_stress_yx
  float s_syy = 0.0f;                                  // virial_stress_yy
  float s_syz = 0.0f;                                  // virial_stress_yz
  float s_szx = 0.0f;                                  // virial_stress_zx
  float s_szy = 0.0f;                                  // virial_stress_zy
  float s_szz = 0.0f;                                  // virial_stress_zz

  float r = 0.0f;
  float rsq = 0.0f;
  float Rcut = 0.0f;

  if (n1 < N2) {
    double x12d, y12d, z12d;
    float x12f, y12f, z12f;
    int neighor_number = g_neighbor_number[n1];
    int type1 = g_type[n1];
    double x1 = g_x[n1];
    double y1 = g_y[n1];
    double z1 = g_z[n1];

    int index_ilp_vec[3] = {n1, n1 + number_of_particles, n1 + (number_of_particles << 1)};
    float fk_temp[9] = {0.0f};

    float delkix_half[3] = {0.0f, 0.0f, 0.0f};
    float delkiy_half[3] = {0.0f, 0.0f, 0.0f};
    float delkiz_half[3] = {0.0f, 0.0f, 0.0f};

    // calculate the normal
    int cont = 0;
    float normal[3];
    float dnormdri[3][3];
    float dnormal[3][3][3];

    float vet[3][3];
    int id, ip, m;
    for (id = 0; id < 3; ++id) {
      normal[id] = 0.0f;
      for (ip = 0; ip < 3; ++ip) {
        vet[id][ip] = 0.0f;
        dnormdri[id][ip] = 0.0f;
        for (m = 0; m < 3; ++m) {
          dnormal[id][ip][m] = 0.0f;
        }
      }
    }

    int ilp_neighbor_number = g_ilp_neighbor_number[n1];
    for (int i1 = 0; i1 < ilp_neighbor_number; ++i1) {
      int n2_ilp = g_ilp_neighbor_list[n1 + number_of_particles * i1];
      x12d = g_x[n2_ilp] - x1;
      y12d = g_y[n2_ilp] - y1;
      z12d = g_z[n2_ilp] - z1;
      apply_mic(box, x12d, y12d, z12d);
      vet[cont][0] = float(x12d);
      vet[cont][1] = float(y12d);
      vet[cont][2] = float(z12d);
      ++cont;

      delkix_half[i1] = float(x12d) * 0.5f;
      delkiy_half[i1] = float(y12d) * 0.5f;
      delkiz_half[i1] = float(z12d) * 0.5f;
    }

    calc_normal(vet, cont, normal, dnormdri, dnormal);

    // calculate energy and force
    double tt1,tt2,tt3;
    for (int i1 = 0; i1 < neighor_number; ++i1) {
      int index = n1 + number_of_particles * i1;
      int n2 = g_neighbor_list[index];
      int type2 = g_type[n2];

      // TODO shared double?
      tt1 = g_x[n2];
      tt2 = g_y[n2];
      tt3 = g_z[n2];
      x12d = tt1 - x1;
      y12d = tt2 - y1;
      z12d = tt3 - z1;
      // x12d = g_x[n2] - x1;
      // y12d = g_y[n2] - y1;
      // z12d = g_z[n2] - z1;
      apply_mic(box, x12d, y12d, z12d);

      // save x12, y12, z12 in float
      x12f = float(x12d);
      y12f = float(y12d);
      z12f = float(z12d);

      // calculate distance between atoms
      rsq = x12f * x12f + y12f * y12f + z12f * z12f;
      r = sqrtf(rsq);
      Rcut = ilp_para.rcut_global[type1][type2];
      // not in the same layer
      if (r >= Rcut || group_label[n1] == group_label[n2]) {
        continue;
      }

      // calc att
      float Tap, dTap, rinv;
      float Rcutinv = 1.0f / Rcut;
      rinv = 1.0f / r;
      Tap = calc_Tap(r, Rcutinv);
      dTap = calc_dTap(r, Rcut, Rcutinv);

      float p2_vdW, f2_vdW;
      calc_vdW(
        r,
        rinv,
        rsq,
        ilp_para.d[type1][type2],
        ilp_para.d_Seff[type1][type2],
        ilp_para.C_6[type1][type2],
        Tap,
        dTap,
        p2_vdW,
        f2_vdW);
      
      float f12x = -f2_vdW * x12f * 0.5f;
      float f12y = -f2_vdW * y12f * 0.5f;
      float f12z = -f2_vdW * z12f * 0.5f;
      float f21x = -f12x;
      float f21y = -f12y;
      float f21z = -f12z;

      s_fx += f12x - f21x;
      s_fy += f12y - f21y;
      s_fz += f12z - f21z;

      s_pe += p2_vdW * 0.5f;
      s_sxx += x12f * f21x;
      s_sxy += x12f * f21y;
      s_sxz += x12f * f21z;
      s_syx += y12f * f21x;
      s_syy += y12f * f21y;
      s_syz += y12f * f21z;
      s_szx += z12f * f21x;
      s_szy += z12f * f21y;
      s_szz += z12f * f21z;

      
      // calc rep
      float C = ilp_para.C[type1][type2];
      float lambda_ = ilp_para.lambda[type1][type2];
      float delta2inv = ilp_para.delta2inv[type1][type2];
      float epsilon = ilp_para.epsilon[type1][type2];
      float z0 = ilp_para.z0[type1][type2];
      // calc_rep
      float prodnorm1, rhosq1, rdsq1, exp0, exp1, frho1, Erep, Vilp;
      float fpair, fpair1, fsum, delx, dely, delz, fkcx, fkcy, fkcz;
      float dprodnorm1[3] = {0.0f, 0.0f, 0.0f};
      float fp1[3] = {0.0f, 0.0f, 0.0f};
      float fprod1[3] = {0.0f, 0.0f, 0.0f};
      float fk[3] = {0.0f, 0.0f, 0.0f};

      delx = -x12f;
      dely = -y12f;
      delz = -z12f;

      float delx_half = delx * 0.5f;
      float dely_half = dely * 0.5f;
      float delz_half = delz * 0.5f;

      // calculate the transverse distance
      prodnorm1 = normal[0] * delx + normal[1] * dely + normal[2] * delz;
      rhosq1 = rsq - prodnorm1 * prodnorm1;
      rdsq1 = rhosq1 * delta2inv;

      // store exponents
      // exp0 = exp(-lambda_ * (r - z0));
      // exp1 = exp(-rdsq1);
      // TODO: use float
      exp0 = expf(-lambda_ * (r - z0));
      exp1 = expf(-rdsq1);

      frho1 = exp1 * C;
      Erep = 0.5f * epsilon + frho1;
      Vilp = exp0 * Erep;

      // derivatives
      fpair = lambda_ * exp0 * rinv * Erep;
      fpair1 = 2.0f * exp0 * frho1 * delta2inv;
      fsum = fpair + fpair1;

      float prodnorm1_m_fpair1 = prodnorm1 * fpair1;
      float Vilp_m_dTap_m_rinv = Vilp * dTap * rinv;

      // derivatives of the product of rij and ni, the resutl is a vector
      dprodnorm1[0] = 
        dnormdri[0][0] * delx + dnormdri[1][0] * dely + dnormdri[2][0] * delz;
      dprodnorm1[1] = 
        dnormdri[0][1] * delx + dnormdri[1][1] * dely + dnormdri[2][1] * delz;
      dprodnorm1[2] = 
        dnormdri[0][2] * delx + dnormdri[1][2] * dely + dnormdri[2][2] * delz;
      // fp1[0] = prodnorm1 * normal[0] * fpair1;
      // fp1[1] = prodnorm1 * normal[1] * fpair1;
      // fp1[2] = prodnorm1 * normal[2] * fpair1;
      // fprod1[0] = prodnorm1 * dprodnorm1[0] * fpair1;
      // fprod1[1] = prodnorm1 * dprodnorm1[1] * fpair1;
      // fprod1[2] = prodnorm1 * dprodnorm1[2] * fpair1;
      fp1[0] = prodnorm1_m_fpair1 * normal[0];
      fp1[1] = prodnorm1_m_fpair1 * normal[1];
      fp1[2] = prodnorm1_m_fpair1 * normal[2];
      fprod1[0] = prodnorm1_m_fpair1 * dprodnorm1[0];
      fprod1[1] = prodnorm1_m_fpair1 * dprodnorm1[1];
      fprod1[2] = prodnorm1_m_fpair1 * dprodnorm1[2];

      // fkcx = (delx * fsum - fp1[0]) * Tap - Vilp * dTap * delx * rinv;
      // fkcy = (dely * fsum - fp1[1]) * Tap - Vilp * dTap * dely * rinv;
      // fkcz = (delz * fsum - fp1[2]) * Tap - Vilp * dTap * delz * rinv;
      fkcx = (delx * fsum - fp1[0]) * Tap - Vilp_m_dTap_m_rinv * delx;
      fkcy = (dely * fsum - fp1[1]) * Tap - Vilp_m_dTap_m_rinv * dely;
      fkcz = (delz * fsum - fp1[2]) * Tap - Vilp_m_dTap_m_rinv * delz;

      s_fx += fkcx - fprod1[0] * Tap;
      s_fy += fkcy - fprod1[1] * Tap;
      s_fz += fkcz - fprod1[2] * Tap;

      g_f12x[index] = fkcx;
      g_f12y[index] = fkcy;
      g_f12z[index] = fkcz;

      float minus_prodnorm1_m_fpair1_m_Tap = -prodnorm1 * fpair1 * Tap;
      for (int kk = 0; kk < ilp_neighbor_number; ++kk) {
        // int index_ilp = n1 + number_of_particles * kk;
        // int n2_ilp = g_ilp_neighbor_list[index_ilp];
        // if (n2_ilp_vec[kk] == n1) continue;
        // derivatives of the product of rij and ni respect to rk, k=0,1,2, where atom k is the neighbors of atom i
        dprodnorm1[0] = dnormal[0][0][kk] * delx + dnormal[1][0][kk] * dely +
            dnormal[2][0][kk] * delz;
        dprodnorm1[1] = dnormal[0][1][kk] * delx + dnormal[1][1][kk] * dely +
            dnormal[2][1][kk] * delz;
        dprodnorm1[2] = dnormal[0][2][kk] * delx + dnormal[1][2][kk] * dely +
            dnormal[2][2][kk] * delz;
        // fk[0] = (-prodnorm1 * dprodnorm1[0] * fpair1) * Tap;
        // fk[1] = (-prodnorm1 * dprodnorm1[1] * fpair1) * Tap;
        // fk[2] = (-prodnorm1 * dprodnorm1[2] * fpair1) * Tap;
        fk[0] = minus_prodnorm1_m_fpair1_m_Tap * dprodnorm1[0];
        fk[1] = minus_prodnorm1_m_fpair1_m_Tap * dprodnorm1[1];
        fk[2] = minus_prodnorm1_m_fpair1_m_Tap * dprodnorm1[2];

        // g_f12x_ilp_neigh[index_ilp_vec[kk]] += fk[0];
        // g_f12y_ilp_neigh[index_ilp_vec[kk]] += fk[1];
        // g_f12z_ilp_neigh[index_ilp_vec[kk]] += fk[2];
        fk_temp[kk] += fk[0];
        fk_temp[kk + 3] += fk[1];
        fk_temp[kk + 6] += fk[2];

        // delki[0] = g_x[n2_ilp] - x1;
        // delki[1] = g_y[n2_ilp] - y1;
        // delki[2] = g_z[n2_ilp] - z1;
        // apply_mic(box, delki[0], delki[1], delki[2]);

        // s_sxx += delki[0] * fk[0] * 0.5;
        // s_sxy += delki[0] * fk[1] * 0.5;
        // s_sxz += delki[0] * fk[2] * 0.5;
        // s_syx += delki[1] * fk[0] * 0.5;
        // s_syy += delki[1] * fk[1] * 0.5;
        // s_syz += delki[1] * fk[2] * 0.5;
        // s_szx += delki[2] * fk[0] * 0.5;
        // s_szy += delki[2] * fk[1] * 0.5;
        // s_szz += delki[2] * fk[2] * 0.5;

        s_sxx += delkix_half[kk] * fk[0];
        s_sxy += delkix_half[kk] * fk[1];
        s_sxz += delkix_half[kk] * fk[2];
        s_syx += delkiy_half[kk] * fk[0];
        s_syy += delkiy_half[kk] * fk[1];
        s_syz += delkiy_half[kk] * fk[2];
        s_szx += delkiz_half[kk] * fk[0];
        s_szy += delkiz_half[kk] * fk[1];
        s_szz += delkiz_half[kk] * fk[2];
      }
      s_pe += Tap * Vilp;
      s_sxx += delx_half * fkcx;
      s_sxy += delx_half * fkcy;
      s_sxz += delx_half * fkcz;
      s_syx += dely_half * fkcx;
      s_syy += dely_half * fkcy;
      s_syz += dely_half * fkcz;
      s_szx += delz_half * fkcx;
      s_szy += delz_half * fkcy;
      s_szz += delz_half * fkcz;
    }

    // save force
    g_fx[n1] += s_fx;
    g_fy[n1] += s_fy;
    g_fz[n1] += s_fz;
    g_f12x_ilp_neigh[index_ilp_vec[0]] = fk_temp[0];
    g_f12x_ilp_neigh[index_ilp_vec[1]] = fk_temp[1];
    g_f12x_ilp_neigh[index_ilp_vec[2]] = fk_temp[2];
    g_f12y_ilp_neigh[index_ilp_vec[0]] = fk_temp[3];
    g_f12y_ilp_neigh[index_ilp_vec[1]] = fk_temp[4];
    g_f12y_ilp_neigh[index_ilp_vec[2]] = fk_temp[5];
    g_f12z_ilp_neigh[index_ilp_vec[0]] = fk_temp[6];
    g_f12z_ilp_neigh[index_ilp_vec[1]] = fk_temp[7];
    g_f12z_ilp_neigh[index_ilp_vec[2]] = fk_temp[8];

    // save virial
    // xx xy xz    0 3 4
    // yx yy yz    6 1 5
    // zx zy zz    7 8 2
    g_virial[n1 + 0 * number_of_particles] += s_sxx;
    g_virial[n1 + 1 * number_of_particles] += s_syy;
    g_virial[n1 + 2 * number_of_particles] += s_szz;
    g_virial[n1 + 3 * number_of_particles] += s_sxy;
    g_virial[n1 + 4 * number_of_particles] += s_sxz;
    g_virial[n1 + 5 * number_of_particles] += s_syz;
    g_virial[n1 + 6 * number_of_particles] += s_syx;
    g_virial[n1 + 7 * number_of_particles] += s_szx;
    g_virial[n1 + 8 * number_of_particles] += s_szy;

    // save potential
    g_potential[n1] += s_pe;

  }
}

__global__ void build_reduce_neighbor_list(
  const int number_of_particles,
  const int N1,
  const int N2,
  const int *g_neighbor_number,
  const int *g_neighbor_list,
  int *g_reduce_neighbor_list)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1;
  if (N1 < N2) {
    int neighbor_number = g_neighbor_number[n1];
    int l, r, m, tmp_value;
    for (int i1 = 0; i1 < neighbor_number; ++i1) {
      int index = n1 + i1 * number_of_particles;
      int n2 = g_neighbor_list[index];

      l = 0;
      r = g_neighbor_number[n2];
      while (l < r) {
        m = (l + r) >> 1;
        tmp_value = g_neighbor_list[n2 + number_of_particles * m];
        if (tmp_value < n1) {
          l = m + 1;
        } else if (tmp_value > n1) {
          r = m - 1;
        } else {
          break;
        }
      }
      g_reduce_neighbor_list[index] = (l + r) >> 1;

    }
  }
}

__global__ void reduce_force_many_body(
  const int number_of_particles,
  const int N1,
  const int N2,
  const Box box,
  const int *g_neighbor_number,
  const int *g_neighbor_list,
  int *g_reduce_neighbor_list,
  int *g_ilp_neighbor_number,
  int *g_ilp_neighbor_list,
  const double *__restrict__ g_x,
  const double *__restrict__ g_y,
  const double *__restrict__ g_z,
  double *g_fx,
  double *g_fy,
  double *g_fz,
  double *g_virial,
  float *g_f12x,
  float *g_f12y,
  float *g_f12z,
  float *g_f12x_ilp_neigh,
  float *g_f12y_ilp_neigh,
  float *g_f12z_ilp_neigh)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1; // particle index
  float s_fx = 0.0f;                                   // force_x
  float s_fy = 0.0f;                                   // force_y
  float s_fz = 0.0f;                                   // force_z
  float s_sxx = 0.0f;                                  // virial_stress_xx
  float s_sxy = 0.0f;                                  // virial_stress_xy
  float s_sxz = 0.0f;                                  // virial_stress_xz
  float s_syx = 0.0f;                                  // virial_stress_yx
  float s_syy = 0.0f;                                  // virial_stress_yy
  float s_syz = 0.0f;                                  // virial_stress_yz
  float s_szx = 0.0f;                                  // virial_stress_zx
  float s_szy = 0.0f;                                  // virial_stress_zy
  float s_szz = 0.0f;                                  // virial_stress_zz


  if (n1 < N2) {
    double x12d, y12d, z12d;
    float x12f, y12f, z12f;
    int neighbor_number_1 = g_neighbor_number[n1];
    double x1 = g_x[n1];
    double y1 = g_y[n1];
    double z1 = g_z[n1];

    // calculate energy and force
    for (int i1 = 0; i1 < neighbor_number_1; ++i1) {
      int index = n1 + number_of_particles * i1;
      int n2 = g_neighbor_list[index];
      int neighor_number_2 = g_neighbor_number[n2];

      x12d = g_x[n2] - x1;
      y12d = g_y[n2] - y1;
      z12d = g_z[n2] - z1;
      apply_mic(box, x12d, y12d, z12d);
      x12f = float(x12d);
      y12f = float(y12d);
      z12f = float(z12d);

      // int offset = 0;
      // // for (int k = 0; k < neighor_number_2; ++k) {
      // //   if (n1 == g_neighbor_list[n2 + number_of_particles * k]) {
      // //     offset = k;
      // //     break;
      // //   }
      // // }
      // // TODO: binary search
      // int l = 0;
      // int r = neighor_number_2;
      // int m = 0;
      // int tmp_value = 0;
      // while (l < r) {
      //   m = (l + r) >> 1;
      //   tmp_value = g_neighbor_list[n2 + number_of_particles * m];
      //   if (tmp_value < n1) {
      //     l = m + 1;
      //   } else if (tmp_value > n1) {
      //     r = m - 1;
      //   } else {
      //     break;
      //   }
      // }
      // offset = (l + r) >> 1;
      // index = n2 + number_of_particles * offset;
      index = n2 + number_of_particles * g_reduce_neighbor_list[index];
      float f21x = g_f12x[index];
      float f21y = g_f12y[index];
      float f21z = g_f12z[index];

      s_fx -= f21x;
      s_fy -= f21y;
      s_fz -= f21z;

      // per-atom virial
      s_sxx += x12f * f21x * 0.5f;
      s_sxy += x12f * f21y * 0.5f;
      s_sxz += x12f * f21z * 0.5f;
      s_syx += y12f * f21x * 0.5f;
      s_syy += y12f * f21y * 0.5f;
      s_syz += y12f * f21z * 0.5f;
      s_szx += z12f * f21x * 0.5f;
      s_szy += z12f * f21y * 0.5f;
      s_szz += z12f * f21z * 0.5f;
    }

    int ilp_neighbor_number_1 = g_ilp_neighbor_number[n1];

    for (int i1 = 0; i1 < ilp_neighbor_number_1; ++i1) {
      int index = n1 + number_of_particles * i1;
      int n2 = g_ilp_neighbor_list[index];
      int ilp_neighor_number_2 = g_neighbor_number[n2];

      x12d = g_x[n2] - x1;
      y12d = g_y[n2] - y1;
      z12d = g_z[n2] - z1;
      apply_mic(box, x12d, y12d, z12d);
      x12f = float(x12d);
      y12f = float(y12d);
      z12f = float(z12d);

      int offset = 0;
      for (int k = 0; k < ilp_neighor_number_2; ++k) {
        if (n1 == g_ilp_neighbor_list[n2 + number_of_particles * k]) {
          offset = k;
          break;
        }
      }
      index = n2 + number_of_particles * offset;
      float f21x = g_f12x_ilp_neigh[index];
      float f21y = g_f12y_ilp_neigh[index];
      float f21z = g_f12z_ilp_neigh[index];

      s_fx += f21x;
      s_fy += f21y;
      s_fz += f21z;

      // per-atom virial
      s_sxx += -x12f * f21x * 0.5;
      s_sxy += -x12f * f21y * 0.5;
      s_sxz += -x12f * f21z * 0.5;
      s_syx += -y12f * f21x * 0.5;
      s_syy += -y12f * f21y * 0.5;
      s_syz += -y12f * f21z * 0.5;
      s_szx += -z12f * f21x * 0.5;
      s_szy += -z12f * f21y * 0.5;
      s_szz += -z12f * f21z * 0.5;
    }

    // save force
    g_fx[n1] += s_fx;
    g_fy[n1] += s_fy;
    g_fz[n1] += s_fz;

    // save virial
    // xx xy xz    0 3 4
    // yx yy yz    6 1 5
    // zx zy zz    7 8 2
    g_virial[n1 + 0 * number_of_particles] += s_sxx;
    g_virial[n1 + 1 * number_of_particles] += s_syy;
    g_virial[n1 + 2 * number_of_particles] += s_szz;
    g_virial[n1 + 3 * number_of_particles] += s_sxy;
    g_virial[n1 + 4 * number_of_particles] += s_sxz;
    g_virial[n1 + 5 * number_of_particles] += s_syz;
    g_virial[n1 + 6 * number_of_particles] += s_syx;
    g_virial[n1 + 7 * number_of_particles] += s_szx;
    g_virial[n1 + 8 * number_of_particles] += s_szy;
  }
  
}


void ILP::compute(
  Box &box,
  const GPU_Vector<int> &type,
  const GPU_Vector<double> &position_per_atom,
  GPU_Vector<double> &potential_per_atom,
  GPU_Vector<double> &force_per_atom,
  GPU_Vector<double> &virial_per_atom)
{
  // TODO
}


#define USE_FIXED_NEIGHBOR 1
#define UPDATE_TEMP 10
// find force and related quantities
void ILP::compute(
  Box &box,
  const GPU_Vector<int> &type,
  const GPU_Vector<double> &position_per_atom,
  GPU_Vector<double> &potential_per_atom,
  GPU_Vector<double> &force_per_atom,
  GPU_Vector<double> &virial_per_atom,
  std::vector<Group> &group)
{
  const int number_of_atoms = type.size();
  int grid_size = (N2 - N1 - 1) / BLOCK_SIZE_FORCE + 1;

// what's this??
#ifdef USE_FIXED_NEIGHBOR
  static int num_calls = 0;
#endif
#ifdef USE_FIXED_NEIGHBOR
  if (num_calls++ == 0) {
#endif
    find_neighbor(
      N1,
      N2,
      rc,
      box,
      type,
      position_per_atom,
      ilp_data.cell_count,
      ilp_data.cell_count_sum,
      ilp_data.cell_contents,
      ilp_data.NN,
      ilp_data.NL);

    build_reduce_neighbor_list<<<grid_size, BLOCK_SIZE_FORCE>>>(
      number_of_atoms,
      N1,
      N2,
      ilp_data.NN.data(),
      ilp_data.NL.data(),
      ilp_data.reduce_NL.data());
#ifdef USE_FIXED_NEIGHBOR
  }
  num_calls %= UPDATE_TEMP;
#endif

  const double* x = position_per_atom.data();
  const double* y = position_per_atom.data() + number_of_atoms;
  const double* z = position_per_atom.data() + number_of_atoms * 2;
  const int *NN = ilp_data.NN.data();
  const int *NL = ilp_data.NL.data();
  int *reduce_NL = ilp_data.reduce_NL.data();
  int *ilp_NL = ilp_data.ilp_NL.data();
  int *ilp_NN = ilp_data.ilp_NN.data();

  ilp_data.ilp_NL.fill(0);
  ilp_data.ilp_NN.fill(0);

  // find ILP neighbor list
  // TODO: assume the first group column is for ILP
  const int *group_label = group[0].label.data();
  ILP_neighbor<<<grid_size, BLOCK_SIZE_FORCE>>>(
    number_of_atoms, N1, N2, box, NN, NL, \
    type.data(), ilp_para, x, y, z, ilp_NN, \
    ilp_NL, group_label);
  CUDA_CHECK_KERNEL

  // initialize force of ilp neighbor temporary vector
  ilp_data.f12x_ilp_neigh.fill(0);
  ilp_data.f12y_ilp_neigh.fill(0);
  ilp_data.f12z_ilp_neigh.fill(0);
  ilp_data.f12x.fill(0);
  ilp_data.f12y.fill(0);
  ilp_data.f12z.fill(0);

  double *g_fx = force_per_atom.data();
  double *g_fy = force_per_atom.data() + number_of_atoms;
  double *g_fz = force_per_atom.data() + number_of_atoms * 2;
  double *g_virial = virial_per_atom.data();
  double *g_potential = potential_per_atom.data();
  float *g_f12x = ilp_data.f12x.data();
  float *g_f12y = ilp_data.f12y.data();
  float *g_f12z = ilp_data.f12z.data();
  float *g_f12x_ilp_neigh = ilp_data.f12x_ilp_neigh.data();
  float *g_f12y_ilp_neigh = ilp_data.f12y_ilp_neigh.data();
  float *g_f12z_ilp_neigh = ilp_data.f12z_ilp_neigh.data();

  gpu_find_force<<<grid_size, BLOCK_SIZE_FORCE>>>(
    ilp_para,
    number_of_atoms,
    N1,
    N2,
    box,
    NN,
    NL,
    ilp_NN,
    ilp_NL,
    group_label,
    type.data(),
    x,
    y,
    z,
    g_fx,
    g_fy,
    g_fz,
    g_virial,
    g_potential,
    g_f12x,
    g_f12y,
    g_f12z,
    g_f12x_ilp_neigh,
    g_f12y_ilp_neigh,
    g_f12z_ilp_neigh);
  CUDA_CHECK_KERNEL

  reduce_force_many_body<<<grid_size, BLOCK_SIZE_FORCE>>>(
    number_of_atoms,
    N1,
    N2,
    box,
    NN,
    NL,
    reduce_NL,
    ilp_NN,
    ilp_NL,
    x,
    y,
    z,
    g_fx,
    g_fy,
    g_fz,
    g_virial,
    g_f12x,
    g_f12y,
    g_f12z,
    g_f12x_ilp_neigh,
    g_f12y_ilp_neigh,
    g_f12z_ilp_neigh);
}
