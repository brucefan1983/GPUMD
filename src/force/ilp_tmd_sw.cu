#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan and GPUMD development team
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

/*----------------------------------------------------------------------------80
The class dealing with the interlayer potential(ILP) and SW.
TODO:
------------------------------------------------------------------------------*/

#include "ilp_tmd_sw.cuh"
#include "neighbor.cuh"
#include "utilities/error.cuh"
#include "utilities/common.cuh"
#include "utilities/gpu_macro.cuh"
#include <cstring>

#define BLOCK_SIZE_FORCE 128

// there are most 6 intra-layer neighbors for TMD
#define NNEI 6


ILP_TMD_SW::ILP_TMD_SW(FILE* fid_ilp, FILE* fid_sw, int num_types, int num_atoms)
{
  // read ILP TMD potential parameter
  printf("Use %d-element ILP potential with elements:\n", num_types);
  if (!(num_types >= 1 && num_types <= MAX_TYPE_ILP_TMD_SW)) {
    PRINT_INPUT_ERROR("Incorrect type number of ILP_TMD_SW parameters.\n");
  }
  for (int n = 0; n < num_types; ++n) {
    char atom_symbol[10];
    int count = fscanf(fid_ilp, "%s", atom_symbol);
    PRINT_SCANF_ERROR(count, 1, "Reading error for ILP_TMD_SW potential.");
    printf(" %s", atom_symbol);
  }
  printf("\n");

  // read parameters
  float beta, alpha, delta, epsilon, C, d, sR;
  float reff, C6, S, rcut_ilp, rcut_global;
  rc = 0.0;
  for (int n = 0; n < num_types; ++n) {
    for (int m = 0; m < num_types; ++m) {
      int count = fscanf(fid_ilp, "%f%f%f%f%f%f%f%f%f%f%f%f", \
      &beta, &alpha, &delta, &epsilon, &C, &d, &sR, &reff, &C6, &S, \
      &rcut_ilp, &rcut_global);
      PRINT_SCANF_ERROR(count, 12, "Reading error for ILP_TMD_SW potential.");

      ilp_para.C[n][m] = C;
      ilp_para.C_6[n][m] = C6;
      ilp_para.d[n][m] = d;
      ilp_para.d_Seff[n][m] = d / sR / reff;
      ilp_para.epsilon[n][m] = epsilon;
      ilp_para.z0[n][m] = beta;
      ilp_para.lambda[n][m] = alpha / beta;
      ilp_para.delta2inv[n][m] = 1.0 / (delta * delta);
      ilp_para.S[n][m] = S;
      ilp_para.rcutsq_ilp[n][m] = rcut_ilp * rcut_ilp;
      ilp_para.rcut_global[n][m] = rcut_global;
      float meV = 1e-3 * S;
      ilp_para.C[n][m] *= meV;
      ilp_para.C_6[n][m] *= meV;
      ilp_para.epsilon[n][m] *= meV;

      if (rc < rcut_global)
        rc = rcut_global;
    }
  }

  // read SW potential parameter
  if (num_types == 1) {
    initialize_sw_1985_1(fid_sw);
  }
  if (num_types == 2) {
    initialize_sw_1985_2(fid_sw);
  }
  if (num_types == 3) {
    initialize_sw_1985_3(fid_sw);
  }

  // initialize neighbor lists and some temp vectors
  int max_neighbor_number = min(num_atoms, CUDA_MAX_NL_TMD);
  ilp_data.NN.resize(num_atoms);
  ilp_data.NL.resize(num_atoms * max_neighbor_number);
  ilp_data.cell_count.resize(num_atoms);
  ilp_data.cell_count_sum.resize(num_atoms);
  ilp_data.cell_contents.resize(num_atoms);

  // init ilp neighbor list
  ilp_data.ilp_NN.resize(num_atoms);
  ilp_data.ilp_NL.resize(num_atoms * MAX_ILP_NEIGHBOR_TMD);
  ilp_data.reduce_NL.resize(num_atoms * max_neighbor_number);
  ilp_data.big_ilp_NN.resize(num_atoms);
  ilp_data.big_ilp_NL.resize(num_atoms * MAX_BIG_ILP_NEIGHBOR_TMD);

  ilp_data.f12x.resize(num_atoms * max_neighbor_number);
  ilp_data.f12y.resize(num_atoms * max_neighbor_number);
  ilp_data.f12z.resize(num_atoms * max_neighbor_number);

  ilp_data.f12x_ilp_neigh.resize(num_atoms * MAX_ILP_NEIGHBOR_TMD);
  ilp_data.f12y_ilp_neigh.resize(num_atoms * MAX_ILP_NEIGHBOR_TMD);
  ilp_data.f12z_ilp_neigh.resize(num_atoms * MAX_ILP_NEIGHBOR_TMD);

  // intialize sw neighbor list
  sw2_data.NN.resize(num_atoms);
  sw2_data.NL.resize(num_atoms * 1024); // the largest supported by CUDA
  sw2_data.cell_count.resize(num_atoms);
  sw2_data.cell_count_sum.resize(num_atoms);
  sw2_data.cell_contents.resize(num_atoms);

  // memory for the partial forces dU_i/dr_ij
  const int num_of_neighbors = MAX_SW_NEIGHBOR_NUM * num_atoms;
  sw2_data.f12x.resize(num_of_neighbors);
  sw2_data.f12y.resize(num_of_neighbors);
  sw2_data.f12z.resize(num_of_neighbors);

  // init constant cutoff coeff
  float h_tap_coeff[8] = \
    {1.0f, 0.0f, 0.0f, 0.0f, -35.0f, 84.0f, -70.0f, 20.0f};
  CHECK(gpuMemcpyToSymbol(Tap_coeff_tmd, h_tap_coeff, 8 * sizeof(float)));

  // set ilp_flag to 1
  ilp_flag = 1;
}

ILP_TMD_SW::~ILP_TMD_SW(void)
{
  // nothing
}

void ILP_TMD_SW::initialize_sw_1985_1(FILE* fid)
{
  printf("Use single-element Stillinger-Weber potential.\n");
  int count;
  double epsilon, lambda, A, B, a, gamma, sigma, cos0;
  count =
    fscanf(fid, "%lf%lf%lf%lf%lf%lf%lf%lf", &epsilon, &lambda, &A, &B, &a, &gamma, &sigma, &cos0);
  PRINT_SCANF_ERROR(count, 8, "Reading error for SW potential.");

  sw2_para.A[0][0] = epsilon * A;
  sw2_para.B[0][0] = B;
  sw2_para.a[0][0] = a;
  sw2_para.sigma[0][0] = sigma;
  sw2_para.gamma[0][0] = gamma;
  sw2_para.rc[0][0] = sigma * a;
  rc_sw = sw2_para.rc[0][0];
  sw2_para.lambda[0][0][0] = epsilon * lambda;
  sw2_para.cos0[0][0][0] = cos0;
}

void ILP_TMD_SW::initialize_sw_1985_2(FILE* fid)
{
  printf("Use two-element Stillinger-Weber potential.\n");
  int count;

  // 2-body parameters and the force cutoff
  double A[3], B[3], a[3], sigma[3], gamma[3];
  rc_sw = 0.0;
  for (int n = 0; n < 3; n++) {
    count = fscanf(fid, "%lf%lf%lf%lf%lf", &A[n], &B[n], &a[n], &sigma[n], &gamma[n]);
    PRINT_SCANF_ERROR(count, 5, "Reading error for SW potential.");
  }
  for (int n1 = 0; n1 < 2; n1++)
    for (int n2 = 0; n2 < 2; n2++) {
      sw2_para.A[n1][n2] = A[n1 + n2];
      sw2_para.B[n1][n2] = B[n1 + n2];
      sw2_para.a[n1][n2] = a[n1 + n2];
      sw2_para.sigma[n1][n2] = sigma[n1 + n2];
      sw2_para.gamma[n1][n2] = gamma[n1 + n2];
      sw2_para.rc[n1][n2] = sigma[n1 + n2] * a[n1 + n2];
      if (rc_sw < sw2_para.rc[n1][n2])
        rc_sw = sw2_para.rc[n1][n2];
    }

  // 3-body parameters
  double lambda, cos0;
  for (int n1 = 0; n1 < 2; n1++)
    for (int n2 = 0; n2 < 2; n2++)
      for (int n3 = 0; n3 < 2; n3++) {
        count = fscanf(fid, "%lf%lf", &lambda, &cos0);
        PRINT_SCANF_ERROR(count, 2, "Reading error for SW potential.");
        sw2_para.lambda[n1][n2][n3] = lambda;
        sw2_para.cos0[n1][n2][n3] = cos0;
      }
}

void ILP_TMD_SW::initialize_sw_1985_3(FILE* fid)
{
  printf("Use three-element Stillinger-Weber potential.\n");
  int count;

  // 2-body parameters and the force cutoff
  double A, B, a, sigma, gamma;
  rc_sw = 0.0;
  for (int n1 = 0; n1 < 3; n1++)
    for (int n2 = 0; n2 < 3; n2++) {
      count = fscanf(fid, "%lf%lf%lf%lf%lf", &A, &B, &a, &sigma, &gamma);
      PRINT_SCANF_ERROR(count, 5, "Reading error for SW potential.");
      sw2_para.A[n1][n2] = A;
      sw2_para.B[n1][n2] = B;
      sw2_para.a[n1][n2] = a;
      sw2_para.sigma[n1][n2] = sigma;
      sw2_para.gamma[n1][n2] = gamma;
      sw2_para.rc[n1][n2] = sigma * a;
      if (rc_sw < sw2_para.rc[n1][n2])
        rc_sw = sw2_para.rc[n1][n2];
    }

  // 3-body parameters
  double lambda, cos0;
  for (int n1 = 0; n1 < 3; n1++) {
    for (int n2 = 0; n2 < 3; n2++) {
      for (int n3 = 0; n3 < 3; n3++) {
        count = fscanf(fid, "%lf%lf", &lambda, &cos0);
        PRINT_SCANF_ERROR(count, 2, "Reading error for SW potential.");
        sw2_para.lambda[n1][n2][n3] = lambda;
        sw2_para.cos0[n1][n2][n3] = cos0;
      }
    }
  }
}

static __device__ __forceinline__ float calc_Tap(const float r_ij, const float Rcutinv)
{
  float Tap, r;

  r = r_ij * Rcutinv;
  if (r >= 1.0f) {
    Tap = 0.0f;
  } else {
    Tap = Tap_coeff_tmd[7];
    for (int i = 6; i >= 0; --i) {
      Tap = Tap * r + Tap_coeff_tmd[i];
    }
  }

  return Tap;
}

// calculate the derivatives of long-range cutoff term
static __device__ __forceinline__ float calc_dTap(const float r_ij, const float Rcut, const float Rcutinv)
{
  float dTap, r;
  
  r = r_ij * Rcutinv;
  if (r >= Rcut) {
    dTap = 0.0f;
  } else {
    dTap = 7.0f * Tap_coeff_tmd[7];
    for (int i = 6; i > 0; --i) {
      dTap = dTap * r + i * Tap_coeff_tmd[i];
    }
    dTap *= Rcutinv;
  }

  return dTap;
}

// create ILP neighbor list from main neighbor list to calculate normals
static __global__ void ILP_neighbor(
  const int number_of_particles,
  const int N1,
  const int N2,
  const Box box,
  const int *g_neighbor_number,
  const int *g_neighbor_list,
  const int *g_type,
  ILP_TMD_Para ilp_para,
  const double* __restrict__ g_x,
  const double* __restrict__ g_y,
  const double* __restrict__ g_z,
  int *ilp_neighbor_number,
  int *ilp_neighbor_list,
  const int *group_label)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1; // particle index

  if (n1 < N2) {
    // TMD
    int neighptr[10], check[10], neighsort[10];
    for (int ll = 0; ll < 10; ++ll) {
      neighptr[ll] = -1;
      neighsort[ll] = -1;
      check[ll] = -1;
    }

    int count = 0;
    int neighbor_number = g_neighbor_number[n1];
    int type1 = g_type[n1];
    double x1 = g_x[n1];
    double y1 = g_y[n1];
    double z1 = g_z[n1];

    for (int i1 = 0; i1 < neighbor_number; ++i1) {
      int n2 = g_neighbor_list[n1 + number_of_particles * i1];
      int type2 = g_type[n2];

      double x12 = g_x[n2] - x1;
      double y12 = g_y[n2] - y1;
      double z12 = g_z[n2] - z1;
      apply_mic(box, x12, y12, z12);
      double d12sq = x12 * x12 + y12 * y12 + z12 * z12;
      double rcutsq = ilp_para.rcutsq_ilp[type1][type2];


      if (group_label[n1] == group_label[n2] && d12sq < rcutsq && type1 == type2 && d12sq != 0) {
        // ilp_neighbor_list[count++ * number_of_particles + n1] = n2;
        neighptr[count++] = n2;
      }
    }

    // TMD
    for (int ll = 0; ll < count; ++ll) {
      neighsort[ll] = neighptr[ll];
      check[ll] = neighptr[ll];
    }

    // TMD
    if (count == NNEI) {
      neighsort[0] = neighptr[0];
      check[0] = -1;
    } else if (count < NNEI && count > 0) {
      for (int jj = 0; jj < count; ++jj) {
        int j = neighptr[jj];
        int jtype = g_type[j];
        int count_temp = 0;
        for (int ll = 0; ll < count; ++ll) {
          int l = neighptr[ll];
          int ltype = g_type[l];
          if (l == j) continue;
          double deljx = g_x[l] - g_x[j];
          double deljy = g_y[l] - g_y[j];
          double deljz = g_z[l] - g_z[j];
          apply_mic(box, deljx, deljy, deljz);
          double rsqlj = deljx * deljx + deljy * deljy + deljz * deljz;
          if (rsqlj != 0 && rsqlj < ilp_para.rcutsq_ilp[ltype][jtype]) {
            ++count_temp;
          }

        }
        if (count_temp == 1) {
          neighsort[0] = neighptr[jj];
          check[jj] = -1;
          break;
        }
      }
    } else if (count > NNEI) {
      printf("ERROR in ILP NEIGHBOR LIST\n");
      printf("\n===== ILP neighbor number[%d] is greater than 6 =====\n", count);
      return;
    }

    // TMD
    // sort the order of neighbors of atom n1
    for (int jj = 0; jj < count; ++jj) {
      int j = neighsort[jj];
      int jtype = g_type[j];
      int ll = 0;
      while (ll < count) {
        int l = neighptr[ll];
        if (check[ll] == -1) {
          ++ll;
          continue;
        }
        int ltype = g_type[l];
        double deljx = g_x[l] - g_x[j];
        double deljy = g_y[l] - g_y[j];
        double deljz = g_z[l] - g_z[j];
        apply_mic(box, deljx, deljy, deljz);
        double rsqlj = deljx * deljx + deljy * deljy + deljz * deljz;
        
        if (abs(rsqlj) >= 1e-6 && rsqlj < ilp_para.rcutsq_ilp[ltype][jtype]) {
          neighsort[jj + 1] = l;
          check[ll] = -1;
          break;
        }
        ++ll;
      }
    }
    ilp_neighbor_number[n1] = count;
    for (int jj = 0; jj < count; ++jj) {
      ilp_neighbor_list[jj * number_of_particles + n1] = neighsort[jj];
    }
  }
}

// modulo func to change atom index
static __device__ __forceinline__ int modulo(int k, int range)
{
  return (k + range) % range;
}

// calculate the normals and its derivatives
static __device__ void calc_normal(
  float (&vect)[NNEI][3],
  int cont,
  float (&normal)[3],
  float (&dnormdri)[3][3],
  float (&dnormal)[3][NNEI][3])
{
  int id, ip, m;
  float  dni[3];
  float  dnn[3][3], dpvdri[3][3];
  float Nave[3], pvet[NNEI][3], dpvet1[NNEI][3][3], dpvet2[NNEI][3][3], dNave[3][NNEI][3];

  float nninv;

  // initialize the arrays
  for (id = 0; id < 3; id++) {
    dni[id] = 0.0f;

    Nave[id] = 0.0f;
    for (ip = 0; ip < 3; ip++) {
      dpvdri[ip][id] = 0.0f;
      for (m = 0; m < NNEI; m++) {
        dnn[m][id] = 0.0f;
        pvet[m][id] = 0.0f;
        dpvet1[m][ip][id] = 0.0f;
        dpvet2[m][ip][id] = 0.0f;
        dNave[id][m][ip] = 0.0f;
      }
    }
  }

  if (cont <= 1) {
    normal[0] = 0.0f;
    normal[1] = 0.0f;
    normal[2] = 1.0f;
    for (id = 0; id < 3; ++id) {
      for (ip = 0; ip < 3; ++ip) {
        dnormdri[id][ip] = 0.0f;
        for (m = 0; m < NNEI; ++m) {
          dnormal[id][m][ip] = 0.0f;
        }
      }
    }
  } else if (cont > 1 && cont < NNEI) {
    for (int k = 0; k < cont - 1; ++k) {
      for (ip = 0; ip < 3; ++ip) {
        pvet[k][ip] = vect[k][modulo(ip + 1, 3)] * vect[k + 1][modulo(ip + 2, 3)] -
                vect[k][modulo(ip + 2, 3)] * vect[k + 1][modulo(ip + 1, 3)];
      }
      // dpvet1[k][l][ip]: the derivatve of the k (=0,...cont-1)th Nik respect to the ip component of atom l
      // derivatives respect to atom l
      // dNik,x/drl
      dpvet1[k][0][0] = 0.0f;
      dpvet1[k][0][1] = vect[modulo(k + 1, NNEI)][2];
      dpvet1[k][0][2] = -vect[modulo(k + 1, NNEI)][1];
      // dNik,y/drl
      dpvet1[k][1][0] = -vect[modulo(k + 1, NNEI)][2];
      dpvet1[k][1][1] = 0.0f;
      dpvet1[k][1][2] = vect[modulo(k + 1, NNEI)][0];
      // dNik,z/drl
      dpvet1[k][2][0] = vect[modulo(k + 1, NNEI)][1];
      dpvet1[k][2][1] = -vect[modulo(k + 1, NNEI)][0];
      dpvet1[k][2][2] = 0.0f;

      // dpvet2[k][l][ip]: the derivatve of the k (=0,...cont-1)th Nik respect to the ip component of atom l+1
      // derivatives respect to atom l+1
      // dNik,x/drl+1
      dpvet2[k][0][0] = 0.0f;
      dpvet2[k][0][1] = -vect[modulo(k, NNEI)][2];
      dpvet2[k][0][2] = vect[modulo(k, NNEI)][1];
      // dNik,y/drl+1
      dpvet2[k][1][0] = vect[modulo(k, NNEI)][2];
      dpvet2[k][1][1] = 0.0f;
      dpvet2[k][1][2] = -vect[modulo(k, NNEI)][0];
      // dNik,z/drl+1
      dpvet2[k][2][0] = -vect[modulo(k, NNEI)][1];
      dpvet2[k][2][1] = vect[modulo(k, NNEI)][0];
      dpvet2[k][2][2] = 0.0f;
    }

    // average the normal vectors by using the NNEI neighboring planes
    for (ip = 0; ip < 3; ip++) {
      Nave[ip] = 0.0f;
      for (int k = 0; k < cont - 1; k++) {
        Nave[ip] += pvet[k][ip];
      }
      Nave[ip] /= (cont - 1);
    }
    nninv = rnorm3df(Nave[0], Nave[1], Nave[2]);
    
    // the unit normal vector
    normal[0] = Nave[0] * nninv;
    normal[1] = Nave[1] * nninv;
    normal[2] = Nave[2] * nninv;

    // derivatives of non-normalized normal vector, dNave:3xcontx3 array
    // dNave[id][m][ip]: the derivatve of the id component of Nave respect to the ip component of atom m
    for (id = 0; id < 3; id++) {
      for (ip = 0; ip < 3; ip++) {
        for (m = 0; m < cont; m++) {
          if (m == 0) {
            dNave[id][m][ip] = dpvet1[m][id][ip] / (cont - 1);
          } else if (m == cont - 1) {
            dNave[id][m][ip] = dpvet2[m - 1][id][ip] / (cont - 1);
          } else {    // sum of the derivatives of the mth and (m-1)th normal vector respect to the atom m
            dNave[id][m][ip] = (dpvet1[m][id][ip] + dpvet2[m - 1][id][ip]) / (cont - 1);
          }
        }
      }
    }
    // derivatives of nn, dnn:contx3 vector
    // dnn[m][id]: the derivative of nn respect to r[m][id], m=0,...NNEI-1; id=0,1,2
    // r[m][id]: the id's component of atom m
    for (m = 0; m < cont; m++) {
      for (id = 0; id < 3; id++) {
        dnn[m][id] = (Nave[0] * dNave[0][m][id] + Nave[1] * dNave[1][m][id] +
                      Nave[2] * dNave[2][m][id]) * nninv;
      }
    }
    // dnormal[i][id][m][ip]: the derivative of normal[i][id] respect to r[m][ip], id,ip=0,1,2.
    // for atom m, which is a neighbor atom of atom i, m = 0,...,NNEI-1
    for (m = 0; m < cont; m++) {
      for (id = 0; id < 3; id++) {
        for (ip = 0; ip < 3; ip++) {
          dnormal[id][m][ip] = dNave[id][m][ip] * nninv - Nave[id] * dnn[m][ip] * nninv * nninv;
        }
      }
    }
    // Calculte dNave/dri, defined as dpvdri
    for (id = 0; id < 3; id++) {
      for (ip = 0; ip < 3; ip++) {
        dpvdri[id][ip] = 0.0;
        for (int k = 0; k < cont; k++) {
          dpvdri[id][ip] -= dNave[id][k][ip];
        }
      }
    }

    // derivatives of nn, dnn:3x1 vector
    dni[0] = (Nave[0] * dpvdri[0][0] + Nave[1] * dpvdri[1][0] + Nave[2] * dpvdri[2][0]) * nninv;
    dni[1] = (Nave[0] * dpvdri[0][1] + Nave[1] * dpvdri[1][1] + Nave[2] * dpvdri[2][1]) * nninv;
    dni[2] = (Nave[0] * dpvdri[0][2] + Nave[1] * dpvdri[1][2] + Nave[2] * dpvdri[2][2]) * nninv;
    // derivatives of unit vector ni respect to ri, the result is 3x3 matrix
    for (id = 0; id < 3; id++) {
      for (ip = 0; ip < 3; ip++) {
        dnormdri[id][ip] = dpvdri[id][ip] * nninv - Nave[id] * dni[ip] * nninv * nninv;
      }
    }
  } else if (cont == NNEI) {
    // derivatives of Ni[l] respect to the NNEI neighbors
    for (int k = 0; k < NNEI; ++k) {
      for (ip = 0; ip < 3; ++ip) {
        pvet[k][ip] = vect[modulo(k, NNEI)][modulo(ip + 1, 3)] *
                vect[modulo(k + 1, NNEI)][modulo(ip + 2, 3)] -
            vect[modulo(k, NNEI)][modulo(ip + 2, 3)] *
                vect[modulo(k + 1, NNEI)][modulo(ip + 1, 3)];
      }
      // dpvet1[k][l][ip]: the derivatve of the k (=0,...cont-1)th Nik respect to the ip component of atom l
      // derivatives respect to atom l
      // dNik,x/drl
      dpvet1[k][0][0] = 0.0f;
      dpvet1[k][0][1] = vect[modulo(k + 1, NNEI)][2];
      dpvet1[k][0][2] = -vect[modulo(k + 1, NNEI)][1];
      // dNik,y/drl
      dpvet1[k][1][0] = -vect[modulo(k + 1, NNEI)][2];
      dpvet1[k][1][1] = 0.0f;
      dpvet1[k][1][2] = vect[modulo(k + 1, NNEI)][0];
      // dNik,z/drl
      dpvet1[k][2][0] = vect[modulo(k + 1, NNEI)][1];
      dpvet1[k][2][1] = -vect[modulo(k + 1, NNEI)][0];
      dpvet1[k][2][2] = 0.0f;

      // dpvet2[k][l][ip]: the derivatve of the k (=0,...cont-1)th Nik respect to the ip component of atom l+1
      // derivatives respect to atom l+1
      // dNik,x/drl+1
      dpvet2[k][0][0] = 0.0f;
      dpvet2[k][0][1] = -vect[modulo(k, NNEI)][2];
      dpvet2[k][0][2] = vect[modulo(k, NNEI)][1];
      // dNik,y/drl+1
      dpvet2[k][1][0] = vect[modulo(k, NNEI)][2];
      dpvet2[k][1][1] = 0.0f;
      dpvet2[k][1][2] = -vect[modulo(k, NNEI)][0];
      // dNik,z/drl+1
      dpvet2[k][2][0] = -vect[modulo(k, NNEI)][1];
      dpvet2[k][2][1] = vect[modulo(k, NNEI)][0];
      dpvet2[k][2][2] = 0.0f;
    }

    // average the normal vectors by using the NNEI neighboring planes
    for (ip = 0; ip < 3; ++ip) {
      Nave[ip] = 0.0f;
      for (int k = 0; k < NNEI; ++k) {
        Nave[ip] += pvet[k][ip];
      }
      Nave[ip] /= NNEI;
    }
    // the magnitude of the normal vector
    // nn2 = Nave[0] * Nave[0] + Nave[1] * Nave[1] + Nave[2] * Nave[2];
    nninv = rnorm3df(Nave[0], Nave[1], Nave[2]);
    // the unit normal vector
    normal[0] = Nave[0] * nninv;
    normal[1] = Nave[1] * nninv;
    normal[2] = Nave[2] * nninv;

    // for the central atoms, dnormdri is always zero
    for (id = 0; id < 3; ++id) {
      for (ip = 0; ip < 3; ++ip) {
        dnormdri[id][ip] = 0.0f;
      }
    }

    // derivatives of non-normalized normal vector, dNave:3xNNEIx3 array
    // dNave[id][m][ip]: the derivatve of the id component of Nave respect to the ip component of atom m
    for (id = 0; id < 3; ++id) {
      for (ip = 0; ip < 3; ++ip) {
        for (
            m = 0; m < NNEI;
            ++m) {    // sum of the derivatives of the mth and (m-1)th normal vector respect to the atom m
          dNave[id][m][ip] =
              (dpvet1[modulo(m, NNEI)][id][ip] + dpvet2[modulo(m - 1, NNEI)][id][ip]) / NNEI;
        }
      }
    }
    // derivatives of nn, dnn:NNEIx3 vector
    // dnn[m][id]: the derivative of nn respect to r[m][id], m=0,...NNEI-1; id=0,1,2
    // r[m][id]: the id's component of atom m
    for (m = 0; m < NNEI; ++m) {
      for (id = 0; id < 3; ++id) {
        dnn[m][id] =
            (Nave[0] * dNave[0][m][id] + Nave[1] * dNave[1][m][id] + Nave[2] * dNave[2][m][id]) *
            nninv;
      }
    }
    // dnormal[i][id][m][ip]: the derivative of normal[i][id] respect to r[m][ip], id,ip=0,1,2.
    // for atom m, which is a neighbor atom of atom i, m = 0,...,NNEI-1
    for (m = 0; m < NNEI; ++m) {
      for (id = 0; id < 3; ++id) {
        for (ip = 0; ip < 3; ++ip) {
          dnormal[id][m][ip] = dNave[id][m][ip] * nninv - Nave[id] * dnn[m][ip] * nninv * nninv;
        }
      }
    }
  } else {
    printf("\n===== ILP neighbor number[%d] is greater than 6 =====\n", cont);
    return;
  }
}

// calculate the van der Waals force and energy
static __device__ void calc_vdW(
  float r,
  float rinv,
  float rsq,
  float d,
  float d_Seff,
  float C_6,
  float Tap,
  float dTap,
  float &p2_vdW,
  float &f2_vdW)
{
  float r2inv, r6inv, r8inv;
  double TSvdw, TSvdwinv_double;
  float Vilp, TSvdwinv_float;
  float fpair, fsum;

  r2inv = 1.0f / rsq;
  r6inv = r2inv * r2inv * r2inv;
  r8inv = r2inv * r6inv;

  // TSvdw = 1.0 + exp(-d_Seff * r + d);
  // use double to avoid inf from exp function
  TSvdw = 1.0 + exp((double) (-d_Seff * r + d));
  TSvdwinv_double = 1.0 / TSvdw;
  TSvdwinv_float = (float) TSvdwinv_double;
  Vilp = -C_6 * r6inv * TSvdwinv_float;

  // derivatives
  // fpair = -6.0 * C_6 * r8inv * TSvdwinv + \
  //   C_6 * d_Seff * (TSvdw - 1.0) * TSvdwinv * TSvdwinv * r8inv * r;
  fpair = (-6.0f + d_Seff * (1.0f - TSvdwinv_float) * r ) * C_6 * TSvdwinv_float * r8inv;
  fsum = fpair * Tap - Vilp * dTap * rinv;

  p2_vdW = Tap * Vilp;
  f2_vdW = fsum;
  
}

// force evaluation kernel
static __global__ void gpu_find_force(
  ILP_TMD_Para ilp_para,
  const int number_of_particles,
  const int N1,
  const int N2,
  const Box box,
  const int *g_neighbor_number,
  const int *g_neighbor_list,
  int *g_ilp_neighbor_number,
  int *g_ilp_neighbor_list,
  const int *group_label,
  const int *g_type,
  const double *__restrict__ g_x,
  const double *__restrict__ g_y,
  const double *__restrict__ g_z,
  double *g_fx,
  double *g_fy,
  double *g_fz,
  double *g_virial,
  double *g_potential,
  float *g_f12x,
  float *g_f12y,
  float *g_f12z,
  float *g_f12x_ilp_neigh,
  float *g_f12y_ilp_neigh,
  float *g_f12z_ilp_neigh)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1; // particle index
  float s_fx = 0.0f;                                   // force_x
  float s_fy = 0.0f;                                   // force_y
  float s_fz = 0.0f;                                   // force_z
  float s_pe = 0.0f;                                   // potential energy
  float s_sxx = 0.0f;                                  // virial_stress_xx
  float s_sxy = 0.0f;                                  // virial_stress_xy
  float s_sxz = 0.0f;                                  // virial_stress_xz
  float s_syx = 0.0f;                                  // virial_stress_yx
  float s_syy = 0.0f;                                  // virial_stress_yy
  float s_syz = 0.0f;                                  // virial_stress_yz
  float s_szx = 0.0f;                                  // virial_stress_zx
  float s_szy = 0.0f;                                  // virial_stress_zy
  float s_szz = 0.0f;                                  // virial_stress_zz

  float r = 0.0f;
  float rsq = 0.0f;
  float Rcut = 0.0f;

  if (n1 < N2) {
    double x12d, y12d, z12d;
    float x12f, y12f, z12f;
    int neighor_number = g_neighbor_number[n1];
    int type1 = g_type[n1];
    double x1 = g_x[n1];
    double y1 = g_y[n1];
    double z1 = g_z[n1];

    float delkix_half[NNEI] = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f};
    float delkiy_half[NNEI] = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f};
    float delkiz_half[NNEI] = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f};

    // calculate the normal
    int cont = 0;
    float normal[3];
    float dnormdri[3][3];
    float dnormal[3][NNEI][3];

    float vet[NNEI][3];
    int id, ip, m;
    for (id = 0; id < 3; ++id) {
      normal[id] = 0.0f;
      for (ip = 0; ip < 3; ++ip) {
        dnormdri[ip][id] = 0.0f;
        for (m = 0; m < NNEI; ++m) {
          dnormal[id][m][ip] = 0.0f;
          vet[m][id] = 0.0f;
        }
      }
    }

    int ilp_neighbor_number = g_ilp_neighbor_number[n1];
    for (int i1 = 0; i1 < ilp_neighbor_number; ++i1) {
      int n2_ilp = g_ilp_neighbor_list[n1 + number_of_particles * i1];
      x12d = g_x[n2_ilp] - x1;
      y12d = g_y[n2_ilp] - y1;
      z12d = g_z[n2_ilp] - z1;
      apply_mic(box, x12d, y12d, z12d);
      vet[cont][0] = float(x12d);
      vet[cont][1] = float(y12d);
      vet[cont][2] = float(z12d);
      ++cont;

      delkix_half[i1] = float(x12d) * 0.5f;
      delkiy_half[i1] = float(y12d) * 0.5f;
      delkiz_half[i1] = float(z12d) * 0.5f;
    }
    
    calc_normal(vet, cont, normal, dnormdri, dnormal);

    // calculate energy and force
    double tt1,tt2,tt3;
    for (int i1 = 0; i1 < neighor_number; ++i1) {
      int index = n1 + number_of_particles * i1;
      int n2 = g_neighbor_list[index];
      int type2 = g_type[n2];

      tt1 = g_x[n2];
      tt2 = g_y[n2];
      tt3 = g_z[n2];
      x12d = tt1 - x1;
      y12d = tt2 - y1;
      z12d = tt3 - z1;
      apply_mic(box, x12d, y12d, z12d);

      // save x12, y12, z12 in float
      x12f = float(x12d);
      y12f = float(y12d);
      z12f = float(z12d);

      // calculate distance between atoms
      rsq = x12f * x12f + y12f * y12f + z12f * z12f;
      r = sqrtf(rsq);
      Rcut = ilp_para.rcut_global[type1][type2];

      if (r >= Rcut) {
        continue;
      }

      // calc att
      float Tap, dTap, rinv;
      float Rcutinv = 1.0f / Rcut;
      rinv = 1.0f / r;
      Tap = calc_Tap(r, Rcutinv);
      dTap = calc_dTap(r, Rcut, Rcutinv);

      float p2_vdW, f2_vdW;
      calc_vdW(
        r,
        rinv,
        rsq,
        ilp_para.d[type1][type2],
        ilp_para.d_Seff[type1][type2],
        ilp_para.C_6[type1][type2],
        Tap,
        dTap,
        p2_vdW,
        f2_vdW);
      
      float f12x = -f2_vdW * x12f * 0.5f;
      float f12y = -f2_vdW * y12f * 0.5f;
      float f12z = -f2_vdW * z12f * 0.5f;
      float f21x = -f12x;
      float f21y = -f12y;
      float f21z = -f12z;

      s_fx += f12x - f21x;
      s_fy += f12y - f21y;
      s_fz += f12z - f21z;

      s_pe += p2_vdW * 0.5f;
      s_sxx += x12f * f21x;
      s_sxy += x12f * f21y;
      s_sxz += x12f * f21z;
      s_syx += y12f * f21x;
      s_syy += y12f * f21y;
      s_syz += y12f * f21z;
      s_szx += z12f * f21x;
      s_szy += z12f * f21y;
      s_szz += z12f * f21z;

      
      // calc rep
      float C = ilp_para.C[type1][type2];
      float lambda_ = ilp_para.lambda[type1][type2];
      float delta2inv = ilp_para.delta2inv[type1][type2];
      float epsilon = ilp_para.epsilon[type1][type2];
      float z0 = ilp_para.z0[type1][type2];
      // calc_rep
      float prodnorm1, rhosq1, rdsq1, exp0, exp1, frho1, Erep, Vilp;
      float fpair, fpair1, fsum, delx, dely, delz, fkcx, fkcy, fkcz;
      float dprodnorm1[3] = {0.0f, 0.0f, 0.0f};
      float fp1[3] = {0.0f, 0.0f, 0.0f};
      float fprod1[3] = {0.0f, 0.0f, 0.0f};
      float fk[3] = {0.0f, 0.0f, 0.0f};

      delx = -x12f;
      dely = -y12f;
      delz = -z12f;

      float delx_half = delx * 0.5f;
      float dely_half = dely * 0.5f;
      float delz_half = delz * 0.5f;

      // calculate the transverse distance
      prodnorm1 = normal[0] * delx + normal[1] * dely + normal[2] * delz;
      rhosq1 = rsq - prodnorm1 * prodnorm1;
      rdsq1 = rhosq1 * delta2inv;

      // store exponents
      // exp0 = exp(-lambda_ * (r - z0));
      // exp1 = exp(-rdsq1);
      exp0 = expf(-lambda_ * (r - z0));
      exp1 = expf(-rdsq1);

      frho1 = exp1 * C;
      Erep = 0.5f * epsilon + frho1;
      Vilp = exp0 * Erep;

      // derivatives
      fpair = lambda_ * exp0 * rinv * Erep;
      fpair1 = 2.0f * exp0 * frho1 * delta2inv;
      fsum = fpair + fpair1;

      float prodnorm1_m_fpair1 = prodnorm1 * fpair1;
      float Vilp_m_dTap_m_rinv = Vilp * dTap * rinv;

      // derivatives of the product of rij and ni, the resutl is a vector
      dprodnorm1[0] = 
        dnormdri[0][0] * delx + dnormdri[1][0] * dely + dnormdri[2][0] * delz;
      dprodnorm1[1] = 
        dnormdri[0][1] * delx + dnormdri[1][1] * dely + dnormdri[2][1] * delz;
      dprodnorm1[2] = 
        dnormdri[0][2] * delx + dnormdri[1][2] * dely + dnormdri[2][2] * delz;
      // fp1[0] = prodnorm1 * normal[0] * fpair1;
      // fp1[1] = prodnorm1 * normal[1] * fpair1;
      // fp1[2] = prodnorm1 * normal[2] * fpair1;
      // fprod1[0] = prodnorm1 * dprodnorm1[0] * fpair1;
      // fprod1[1] = prodnorm1 * dprodnorm1[1] * fpair1;
      // fprod1[2] = prodnorm1 * dprodnorm1[2] * fpair1;
      fp1[0] = prodnorm1_m_fpair1 * normal[0];
      fp1[1] = prodnorm1_m_fpair1 * normal[1];
      fp1[2] = prodnorm1_m_fpair1 * normal[2];
      fprod1[0] = prodnorm1_m_fpair1 * dprodnorm1[0];
      fprod1[1] = prodnorm1_m_fpair1 * dprodnorm1[1];
      fprod1[2] = prodnorm1_m_fpair1 * dprodnorm1[2];

      // fkcx = (delx * fsum - fp1[0]) * Tap - Vilp * dTap * delx * rinv;
      // fkcy = (dely * fsum - fp1[1]) * Tap - Vilp * dTap * dely * rinv;
      // fkcz = (delz * fsum - fp1[2]) * Tap - Vilp * dTap * delz * rinv;
      fkcx = (delx * fsum - fp1[0]) * Tap - Vilp_m_dTap_m_rinv * delx;
      fkcy = (dely * fsum - fp1[1]) * Tap - Vilp_m_dTap_m_rinv * dely;
      fkcz = (delz * fsum - fp1[2]) * Tap - Vilp_m_dTap_m_rinv * delz;

      s_fx += fkcx - fprod1[0] * Tap;
      s_fy += fkcy - fprod1[1] * Tap;
      s_fz += fkcz - fprod1[2] * Tap;

      g_f12x[index] = fkcx;
      g_f12y[index] = fkcy;
      g_f12z[index] = fkcz;

      float minus_prodnorm1_m_fpair1_m_Tap = -prodnorm1 * fpair1 * Tap;
      for (int kk = 0; kk < ilp_neighbor_number; ++kk) {
      // for (int kk = 0; kk < 0; ++kk) {
        // int index_ilp = n1 + number_of_particles * kk;
        // int n2_ilp = g_ilp_neighbor_list[index_ilp];
        // derivatives of the product of rij and ni respect to rk, k=0,1,2, where atom k is the neighbors of atom i
        dprodnorm1[0] = dnormal[0][kk][0] * delx + dnormal[1][kk][0] * dely +
            dnormal[2][kk][0] * delz;
        dprodnorm1[1] = dnormal[0][kk][1] * delx + dnormal[1][kk][1] * dely +
            dnormal[2][kk][1] * delz;
        dprodnorm1[2] = dnormal[0][kk][2] * delx + dnormal[1][kk][2] * dely +
            dnormal[2][kk][2] * delz;
        // fk[0] = (-prodnorm1 * dprodnorm1[0] * fpair1) * Tap;
        // fk[1] = (-prodnorm1 * dprodnorm1[1] * fpair1) * Tap;
        // fk[2] = (-prodnorm1 * dprodnorm1[2] * fpair1) * Tap;
        fk[0] = minus_prodnorm1_m_fpair1_m_Tap * dprodnorm1[0];
        fk[1] = minus_prodnorm1_m_fpair1_m_Tap * dprodnorm1[1];
        fk[2] = minus_prodnorm1_m_fpair1_m_Tap * dprodnorm1[2];

        g_f12x_ilp_neigh[n1 + number_of_particles * kk] += fk[0];
        g_f12y_ilp_neigh[n1 + number_of_particles * kk] += fk[1];
        g_f12z_ilp_neigh[n1 + number_of_particles * kk] += fk[2];

        // delki[0] = g_x[n2_ilp] - x1;
        // delki[1] = g_y[n2_ilp] - y1;
        // delki[2] = g_z[n2_ilp] - z1;
        // apply_mic(box, delki[0], delki[1], delki[2]);

        // s_sxx += delki[0] * fk[0] * 0.5;
        // s_sxy += delki[0] * fk[1] * 0.5;
        // s_sxz += delki[0] * fk[2] * 0.5;
        // s_syx += delki[1] * fk[0] * 0.5;
        // s_syy += delki[1] * fk[1] * 0.5;
        // s_syz += delki[1] * fk[2] * 0.5;
        // s_szx += delki[2] * fk[0] * 0.5;
        // s_szy += delki[2] * fk[1] * 0.5;
        // s_szz += delki[2] * fk[2] * 0.5;
        s_sxx += delkix_half[kk] * fk[0];
        s_sxy += delkix_half[kk] * fk[1];
        s_sxz += delkix_half[kk] * fk[2];
        s_syx += delkiy_half[kk] * fk[0];
        s_syy += delkiy_half[kk] * fk[1];
        s_syz += delkiy_half[kk] * fk[2];
        s_szx += delkiz_half[kk] * fk[0];
        s_szy += delkiz_half[kk] * fk[1];
        s_szz += delkiz_half[kk] * fk[2];
      }
      s_pe += Tap * Vilp;
      s_sxx += delx_half * fkcx;
      s_sxy += delx_half * fkcy;
      s_sxz += delx_half * fkcz;
      s_syx += dely_half * fkcx;
      s_syy += dely_half * fkcy;
      s_syz += dely_half * fkcz;
      s_szx += delz_half * fkcx;
      s_szy += delz_half * fkcy;
      s_szz += delz_half * fkcz;
    }

    // save force
    g_fx[n1] += s_fx;
    g_fy[n1] += s_fy;
    g_fz[n1] += s_fz;

    // save virial
    // xx xy xz    0 3 4
    // yx yy yz    6 1 5
    // zx zy zz    7 8 2
    g_virial[n1 + 0 * number_of_particles] += s_sxx;
    g_virial[n1 + 1 * number_of_particles] += s_syy;
    g_virial[n1 + 2 * number_of_particles] += s_szz;
    g_virial[n1 + 3 * number_of_particles] += s_sxy;
    g_virial[n1 + 4 * number_of_particles] += s_sxz;
    g_virial[n1 + 5 * number_of_particles] += s_syz;
    g_virial[n1 + 6 * number_of_particles] += s_syx;
    g_virial[n1 + 7 * number_of_particles] += s_szx;
    g_virial[n1 + 8 * number_of_particles] += s_szy;

    // save potential
    g_potential[n1] += s_pe;

  }
}

// build a neighbor list for reducing force
static __global__ void build_reduce_neighbor_list(
  const int number_of_particles,
  const int N1,
  const int N2,
  const int *g_neighbor_number,
  const int *g_neighbor_list,
  int *g_reduce_neighbor_list)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1;
  if (N1 < N2) {
    int neighbor_number = g_neighbor_number[n1];
    int l, r, m, tmp_value;
    for (int i1 = 0; i1 < neighbor_number; ++i1) {
      int index = n1 + i1 * number_of_particles;
      int n2 = g_neighbor_list[index];

      l = 0;
      r = g_neighbor_number[n2];
      while (l < r) {
        m = (l + r) >> 1;
        tmp_value = g_neighbor_list[n2 + number_of_particles * m];
        if (tmp_value < n1) {
          l = m + 1;
        } else if (tmp_value > n1) {
          r = m - 1;
        } else {
          break;
        }
      }
      g_reduce_neighbor_list[index] = (l + r) >> 1;
    }
  }
}

// reduce the rep force
static __global__ void reduce_force_many_body(
  const int number_of_particles,
  const int N1,
  const int N2,
  const Box box,
  const int *g_neighbor_number,
  const int *g_neighbor_list,
  int *g_reduce_neighbor_list,
  int *g_ilp_neighbor_number,
  int *g_ilp_neighbor_list,
  const double *__restrict__ g_x,
  const double *__restrict__ g_y,
  const double *__restrict__ g_z,
  double *g_fx,
  double *g_fy,
  double *g_fz,
  double *g_virial,
  float *g_f12x,
  float *g_f12y,
  float *g_f12z,
  float *g_f12x_ilp_neigh,
  float *g_f12y_ilp_neigh,
  float *g_f12z_ilp_neigh)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1; // particle index
  float s_fx = 0.0f;                                   // force_x
  float s_fy = 0.0f;                                   // force_y
  float s_fz = 0.0f;                                   // force_z
  float s_sxx = 0.0f;                                  // virial_stress_xx
  float s_sxy = 0.0f;                                  // virial_stress_xy
  float s_sxz = 0.0f;                                  // virial_stress_xz
  float s_syx = 0.0f;                                  // virial_stress_yx
  float s_syy = 0.0f;                                  // virial_stress_yy
  float s_syz = 0.0f;                                  // virial_stress_yz
  float s_szx = 0.0f;                                  // virial_stress_zx
  float s_szy = 0.0f;                                  // virial_stress_zy
  float s_szz = 0.0f;                                  // virial_stress_zz


  if (n1 < N2) {
    double x12d, y12d, z12d;
    float x12f, y12f, z12f;
    int neighbor_number_1 = g_neighbor_number[n1];
    double x1 = g_x[n1];
    double y1 = g_y[n1];
    double z1 = g_z[n1];

    // calculate energy and force
    for (int i1 = 0; i1 < neighbor_number_1; ++i1) {
      int index = n1 + number_of_particles * i1;
      int n2 = g_neighbor_list[index];

      x12d = g_x[n2] - x1;
      y12d = g_y[n2] - y1;
      z12d = g_z[n2] - z1;
      apply_mic(box, x12d, y12d, z12d);
      x12f = float(x12d);
      y12f = float(y12d);
      z12f = float(z12d);

      index = n2 + number_of_particles * g_reduce_neighbor_list[index];
      float f21x = g_f12x[index];
      float f21y = g_f12y[index];
      float f21z = g_f12z[index];

      s_fx -= f21x;
      s_fy -= f21y;
      s_fz -= f21z;

      // per-atom virial
      s_sxx += x12f * f21x * 0.5f;
      s_sxy += x12f * f21y * 0.5f;
      s_sxz += x12f * f21z * 0.5f;
      s_syx += y12f * f21x * 0.5f;
      s_syy += y12f * f21y * 0.5f;
      s_syz += y12f * f21z * 0.5f;
      s_szx += z12f * f21x * 0.5f;
      s_szy += z12f * f21y * 0.5f;
      s_szz += z12f * f21z * 0.5f;
    }

    int ilp_neighbor_number_1 = g_ilp_neighbor_number[n1];

    for (int i1 = 0; i1 < ilp_neighbor_number_1; ++i1) {
      int index = n1 + number_of_particles * i1;
      int n2 = g_ilp_neighbor_list[index];
      int ilp_neighor_number_2 = g_ilp_neighbor_number[n2];

      x12d = g_x[n2] - x1;
      y12d = g_y[n2] - y1;
      z12d = g_z[n2] - z1;
      apply_mic(box, x12d, y12d, z12d);
      x12f = float(x12d);
      y12f = float(y12d);
      z12f = float(z12d);

      int offset = 0;
      for (int k = 0; k < ilp_neighor_number_2; ++k) {
        if (n1 == g_ilp_neighbor_list[n2 + number_of_particles * k]) {
          offset = k;
          break;
        }
      }
      index = n2 + number_of_particles * offset;
      float f21x = g_f12x_ilp_neigh[index];
      float f21y = g_f12y_ilp_neigh[index];
      float f21z = g_f12z_ilp_neigh[index];

      s_fx += f21x;
      s_fy += f21y;
      s_fz += f21z;

      // per-atom virial
      s_sxx += -x12f * f21x * 0.5f;
      s_sxy += -x12f * f21y * 0.5f;
      s_sxz += -x12f * f21z * 0.5f;
      s_syx += -y12f * f21x * 0.5f;
      s_syy += -y12f * f21y * 0.5f;
      s_syz += -y12f * f21z * 0.5f;
      s_szx += -z12f * f21x * 0.5f;
      s_szy += -z12f * f21y * 0.5f;
      s_szz += -z12f * f21z * 0.5f;
    }

    // save force
    g_fx[n1] += s_fx;
    g_fy[n1] += s_fy;
    g_fz[n1] += s_fz;

    // save virial
    // xx xy xz    0 3 4
    // yx yy yz    6 1 5
    // zx zy zz    7 8 2
    g_virial[n1 + 0 * number_of_particles] += s_sxx;
    g_virial[n1 + 1 * number_of_particles] += s_syy;
    g_virial[n1 + 2 * number_of_particles] += s_szz;
    g_virial[n1 + 3 * number_of_particles] += s_sxy;
    g_virial[n1 + 4 * number_of_particles] += s_sxz;
    g_virial[n1 + 5 * number_of_particles] += s_syz;
    g_virial[n1 + 6 * number_of_particles] += s_syx;
    g_virial[n1 + 7 * number_of_particles] += s_szx;
    g_virial[n1 + 8 * number_of_particles] += s_szy;
  }
}

// SW term
// two-body part of the SW potential
static __device__ void find_p2_and_f2(
  double sigma, double a, double B, double epsilon_times_A, double d12, double& p2, double& f2)
{
  double r12 = d12 / sigma;
  double B_over_r12power4 = B / (r12 * r12 * r12 * r12);
  double exp_factor = epsilon_times_A * exp(1.0 / (r12 - a));
  p2 = exp_factor * (B_over_r12power4 - 1.0);
  f2 = -p2 / ((r12 - a) * (r12 - a)) - exp_factor * 4.0 * B_over_r12power4 / r12;
  f2 /= (sigma * d12);
}

// find the partial forces dU_i/dr_ij of SW potential
static __global__ void gpu_find_force_sw3_partial(
  const int number_of_particles,
  const int N1,
  const int N2,
  const Box box,
  const SW2_Para sw3,
  const int* g_neighbor_number,
  const int* g_neighbor_list,
  const int* g_type,
  const double* __restrict__ g_x,
  const double* __restrict__ g_y,
  const double* __restrict__ g_z,
  double* g_potential,
  double* g_f12x,
  double* g_f12y,
  double* g_f12z)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1; // particle index
  if (n1 >= N1 && n1 < N2) {
    int neighbor_number = g_neighbor_number[n1];
    int type1 = g_type[n1];
    double x1 = g_x[n1];
    double y1 = g_y[n1];
    double z1 = g_z[n1];
    double potential_energy = 0.0;

    for (int i1 = 0; i1 < neighbor_number; ++i1) {
      int index = i1 * number_of_particles + n1;
      int n2 = g_neighbor_list[index];
      int type2 = g_type[n2];
      double x12 = g_x[n2] - x1;
      double y12 = g_y[n2] - y1;
      double z12 = g_z[n2] - z1;
      apply_mic(box, x12, y12, z12);
      double d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12);
      double d12inv = 1.0 / d12;
      if (d12 >= sw3.rc[type1][type2]) {
        continue;
      }

      double gamma12 = sw3.gamma[type1][type2];
      double sigma12 = sw3.sigma[type1][type2];
      double a12 = sw3.a[type1][type2];
      double tmp = gamma12 / (sigma12 * (d12 / sigma12 - a12) * (d12 / sigma12 - a12));
      double p2, f2;
      find_p2_and_f2(sigma12, a12, sw3.B[type1][type2], sw3.A[type1][type2], d12, p2, f2);

      // treat the two-body part in the same way as the many-body part
      double f12x = f2 * x12 * 0.5;
      double f12y = f2 * y12 * 0.5;
      double f12z = f2 * z12 * 0.5;
      // accumulate potential energy
      potential_energy += p2 * 0.5;

      // three-body part
      for (int i2 = 0; i2 < neighbor_number; ++i2) {
        int n3 = g_neighbor_list[n1 + number_of_particles * i2];
        if (n3 == n2) {
          continue;
        }
        int type3 = g_type[n3];
        double x13 = g_x[n3] - x1;
        double y13 = g_y[n3] - y1;
        double z13 = g_z[n3] - z1;
        apply_mic(box, x13, y13, z13);
        double d13 = sqrt(x13 * x13 + y13 * y13 + z13 * z13);
        if (d13 >= sw3.rc[type1][type3]) {
          continue;
        }

        double cos0 = sw3.cos0[type1][type2][type3];
        double lambda = sw3.lambda[type1][type2][type3];
        double exp123 = d13 / sw3.sigma[type1][type3] - sw3.a[type1][type3];
        exp123 = sw3.gamma[type1][type3] / exp123;
        exp123 = exp(gamma12 / (d12 / sigma12 - a12) + exp123);
        double one_over_d12d13 = 1.0 / (d12 * d13);
        double cos123 = (x12 * x13 + y12 * y13 + z12 * z13) * one_over_d12d13;
        double cos123_over_d12d12 = cos123 * d12inv * d12inv;
        // cos123 - cos0
        double delta_cos = cos123 - cos0;

        // modification to (cos123 - cos0)
        double abs_delta_cos = fabs(delta_cos);
        if (abs_delta_cos >= DELTA2) {
          delta_cos = 0.0;
        } else if (abs_delta_cos < DELTA2 && abs_delta_cos > DELTA1) {
          double factor = 0.5 + 0.5 * cos(PI * (abs_delta_cos - DELTA1) / (DELTA2 - DELTA1));
          delta_cos *= factor;
        }

        // double tmp1 = exp123 * (cos123 - cos0) * lambda;
        // double tmp2 = tmp * (cos123 - cos0) * d12inv;
        double tmp1 = exp123 * delta_cos * lambda;
        double tmp2 = tmp * delta_cos * d12inv;

        // accumulate potential energy
        // potential_energy += (cos123 - cos0) * tmp1 * 0.5;
        // double tmp_e = (cos123 - cos0) * tmp1 * 0.5;
        double tmp_e = delta_cos * tmp1 * 0.5;
        potential_energy += tmp_e;

        double cos_d = x13 * one_over_d12d13 - x12 * cos123_over_d12d12;
        f12x += tmp1 * (2.0 * cos_d - tmp2 * x12);

        cos_d = y13 * one_over_d12d13 - y12 * cos123_over_d12d12;
        f12y += tmp1 * (2.0 * cos_d - tmp2 * y12);

        cos_d = z13 * one_over_d12d13 - z12 * cos123_over_d12d12;
        f12z += tmp1 * (2.0 * cos_d - tmp2 * z12);
      }
      g_f12x[index] = f12x;
      g_f12y[index] = f12y;
      g_f12z[index] = f12z;
    }
    // save potential
    g_potential[n1] += potential_energy;
  }
}

// define the pure virtual func
void ILP_TMD_SW::compute(
  Box &box,
  const GPU_Vector<int> &type,
  const GPU_Vector<double> &position_per_atom,
  GPU_Vector<double> &potential_per_atom,
  GPU_Vector<double> &force_per_atom,
  GPU_Vector<double> &virial_per_atom)
{
  // nothing
}

#define USE_FIXED_NEIGHBOR 1
#define UPDATE_TEMP 10
#define BIG_ILP_CUTOFF_SQUARE 50.0
// find force and related quantities
void ILP_TMD_SW::compute_ilp(
  Box &box,
  const GPU_Vector<int> &type,
  const GPU_Vector<double> &position_per_atom,
  GPU_Vector<double> &potential_per_atom,
  GPU_Vector<double> &force_per_atom,
  GPU_Vector<double> &virial_per_atom,
  std::vector<Group> &group)
{
  const int number_of_atoms = type.size();
  int grid_size = (N2 - N1 - 1) / BLOCK_SIZE_FORCE + 1;

  // assume the first group column is for ILP
  const int *group_label = group[0].label.data();

#ifdef USE_FIXED_NEIGHBOR
  static int num_calls = 0;
#endif
#ifdef USE_FIXED_NEIGHBOR
  if (num_calls++ == 0) {
#endif
    find_neighbor_ilp(
      N1,
      N2,
      rc,
      BIG_ILP_CUTOFF_SQUARE,
      box,
      group_label,
      type,
      position_per_atom,
      ilp_data.cell_count,
      ilp_data.cell_count_sum,
      ilp_data.cell_contents,
      ilp_data.NN,
      ilp_data.NL,
      ilp_data.big_ilp_NN,
      ilp_data.big_ilp_NL);
    
    find_neighbor_SW(
      N1,
      N2,
      rc_sw,
      box,
      group_label,
      type,
      position_per_atom,
      sw2_data.cell_count,
      sw2_data.cell_count_sum,
      sw2_data.cell_contents,
      sw2_data.NN,
      sw2_data.NL
    );

    build_reduce_neighbor_list<<<grid_size, BLOCK_SIZE_FORCE>>>(
      number_of_atoms,
      N1,
      N2,
      ilp_data.NN.data(),
      ilp_data.NL.data(),
      ilp_data.reduce_NL.data());
#ifdef USE_FIXED_NEIGHBOR
  }
  num_calls %= UPDATE_TEMP;
#endif

  const double* x = position_per_atom.data();
  const double* y = position_per_atom.data() + number_of_atoms;
  const double* z = position_per_atom.data() + number_of_atoms * 2;
  const int *NN = ilp_data.NN.data();
  const int *NL = ilp_data.NL.data();
  const int* big_ilp_NN = ilp_data.big_ilp_NN.data();
  const int* big_ilp_NL = ilp_data.big_ilp_NL.data();
  int *reduce_NL = ilp_data.reduce_NL.data();
  int *ilp_NL = ilp_data.ilp_NL.data();
  int *ilp_NN = ilp_data.ilp_NN.data();

  const int* NN_sw = sw2_data.NN.data();
  const int* NL_sw = sw2_data.NL.data();

  ilp_data.ilp_NL.fill(0);
  ilp_data.ilp_NN.fill(0);

  // find ILP neighbor list
  ILP_neighbor<<<grid_size, BLOCK_SIZE_FORCE>>>(
    number_of_atoms, N1, N2, box, big_ilp_NN, big_ilp_NL, \
    type.data(), ilp_para, x, y, z, ilp_NN, \
    ilp_NL, group[1].label.data());
  GPU_CHECK_KERNEL

  // initialize force of ilp neighbor temporary vector
  ilp_data.f12x_ilp_neigh.fill(0);
  ilp_data.f12y_ilp_neigh.fill(0);
  ilp_data.f12z_ilp_neigh.fill(0);
  ilp_data.f12x.fill(0);
  ilp_data.f12y.fill(0);
  ilp_data.f12z.fill(0);

  sw2_data.f12x.fill(0);
  sw2_data.f12y.fill(0);
  sw2_data.f12z.fill(0);

  double *g_fx = force_per_atom.data();
  double *g_fy = force_per_atom.data() + number_of_atoms;
  double *g_fz = force_per_atom.data() + number_of_atoms * 2;
  double *g_virial = virial_per_atom.data();
  double *g_potential = potential_per_atom.data();
  float *g_f12x = ilp_data.f12x.data();
  float *g_f12y = ilp_data.f12y.data();
  float *g_f12z = ilp_data.f12z.data();
  float *g_f12x_ilp_neigh = ilp_data.f12x_ilp_neigh.data();
  float *g_f12y_ilp_neigh = ilp_data.f12y_ilp_neigh.data();
  float *g_f12z_ilp_neigh = ilp_data.f12z_ilp_neigh.data();

  gpu_find_force<<<grid_size, BLOCK_SIZE_FORCE>>>(
    ilp_para,
    number_of_atoms,
    N1,
    N2,
    box,
    NN,
    NL,
    ilp_NN,
    ilp_NL,
    group_label,
    type.data(),
    x,
    y,
    z,
    g_fx,
    g_fy,
    g_fz,
    g_virial,
    g_potential,
    g_f12x,
    g_f12y,
    g_f12z,
    g_f12x_ilp_neigh,
    g_f12y_ilp_neigh,
    g_f12z_ilp_neigh);
  GPU_CHECK_KERNEL

  reduce_force_many_body<<<grid_size, BLOCK_SIZE_FORCE>>>(
    number_of_atoms,
    N1,
    N2,
    box,
    NN,
    NL,
    reduce_NL,
    ilp_NN,
    ilp_NL,
    x,
    y,
    z,
    g_fx,
    g_fy,
    g_fz,
    g_virial,
    g_f12x,
    g_f12y,
    g_f12z,
    g_f12x_ilp_neigh,
    g_f12y_ilp_neigh,
    g_f12z_ilp_neigh);
    GPU_CHECK_KERNEL

  // step 1: calculate the partial forces
  gpu_find_force_sw3_partial<<<grid_size, BLOCK_SIZE_FORCE>>>(
    number_of_atoms, N1, N2, box, sw2_para, sw2_data.NN.data(), sw2_data.NL.data(),
    type.data(), position_per_atom.data(), position_per_atom.data() + number_of_atoms,
    position_per_atom.data() + number_of_atoms * 2, potential_per_atom.data(), sw2_data.f12x.data(),
    sw2_data.f12y.data(), sw2_data.f12z.data());
  GPU_CHECK_KERNEL

  // step 2: calculate force and related quantities
  find_properties_many_body(
    box, sw2_data.NN.data(), sw2_data.NL.data(), sw2_data.f12x.data(),
    sw2_data.f12y.data(), sw2_data.f12z.data(), position_per_atom, force_per_atom, virial_per_atom);
}