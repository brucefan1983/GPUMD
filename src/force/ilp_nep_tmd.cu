#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan and GPUMD development team
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

/*----------------------------------------------------------------------------80
The class dealing with the interlayer potential(ILP) and neuroevolution
potential (NEP) for transition metal dichalcogenide (TMD) system.
TODO:
------------------------------------------------------------------------------*/

#include "ilp_nep_tmd.cuh"
#include "neighbor.cuh"
#include "utilities/error.cuh"
#include "utilities/common.cuh"
#include "utilities/gpu_macro.cuh"
#include "utilities/nep_utilities.cuh"
#include <fstream>
#include <iostream>
#include <string>
#include <vector>
#include <cstring>

#define BLOCK_SIZE_FORCE 128

// there are most 3 intra-layer neighbors for TMDs
#define NNEI 6

const std::string ELEMENTS[NUM_ELEMENTS] = {
  "H",  "He", "Li", "Be", "B",  "C",  "N",  "O",  "F",  "Ne", "Na", "Mg", "Al", "Si", "P",  "S",
  "Cl", "Ar", "K",  "Ca", "Sc", "Ti", "V",  "Cr", "Mn", "Fe", "Co", "Ni", "Cu", "Zn", "Ga", "Ge",
  "As", "Se", "Br", "Kr", "Rb", "Sr", "Y",  "Zr", "Nb", "Mo", "Tc", "Ru", "Rh", "Pd", "Ag", "Cd",
  "In", "Sn", "Sb", "Te", "I",  "Xe", "Cs", "Ba", "La", "Ce", "Pr", "Nd", "Pm", "Sm", "Eu", "Gd",
  "Tb", "Dy", "Ho", "Er", "Tm", "Yb", "Lu", "Hf", "Ta", "W",  "Re", "Os", "Ir", "Pt", "Au", "Hg",
  "Tl", "Pb", "Bi", "Po", "At", "Rn", "Fr", "Ra", "Ac", "Th", "Pa", "U",  "Np", "Pu"};

ILP_NEP_TMD::ILP_NEP_TMD(FILE* fid_ilp, const char* file_nep, int num_types, int num_atoms)
{
  // read ILP TMD potential parameter
  printf("Use %d-element ILP potential with elements:\n", num_types);
  if (!(num_types >= 1 && num_types <= IN_MAX_TYPE_ILP_TMD)) {
    PRINT_INPUT_ERROR("Incorrect type number of ILP_NEP_TMD parameters.\n");
  }
  std::vector<std::string> ilp_elements(num_types);
  for (int n = 0; n < num_types; ++n) {
    char atom_symbol[10];
    int count = fscanf(fid_ilp, "%s", atom_symbol);
    PRINT_SCANF_ERROR(count, 1, "Reading error for ILP_NEP_TMD potential.");
    printf(" %s", atom_symbol);
    
    ilp_elements[n] = atom_symbol;
  }
  printf("\n");

  // read ILP group method
  PRINT_SCANF_ERROR(fscanf(fid_ilp, "%d", &ilp_group_method), 1, 
  "Reading error for ILP group method of molecular layer.");
  printf("Use group method %d to identify molecule for ILP.\n", ilp_group_method);
  PRINT_SCANF_ERROR(fscanf(fid_ilp, "%d", &ilp_sub_group_method), 1, 
  "Reading error for ILP group method of sublayer.");
  printf("Use group method %d to identify molecule(sublayer) for ILP.\n", ilp_sub_group_method);

  // read parameters
  float beta, alpha, delta, epsilon, C, d, sR;
  float reff, C6, S, rcut_ilp, rcut_global;
  ilp_rc = 0.0;
  for (int n = 0; n < num_types; ++n) {
    for (int m = 0; m < num_types; ++m) {
      int count = fscanf(fid_ilp, "%f%f%f%f%f%f%f%f%f%f%f%f", \
      &beta, &alpha, &delta, &epsilon, &C, &d, &sR, &reff, &C6, &S, \
      &rcut_ilp, &rcut_global);
      PRINT_SCANF_ERROR(count, 12, "Reading error for ILP_NEP_TMD potential.");

      ilp_para.C[n][m] = C;
      ilp_para.C_6[n][m] = C6;
      ilp_para.d[n][m] = d;
      ilp_para.d_Seff[n][m] = d / sR / reff;
      ilp_para.epsilon[n][m] = epsilon;
      ilp_para.z0[n][m] = beta;
      ilp_para.lambda[n][m] = alpha / beta;
      ilp_para.delta2inv[n][m] = 1.0 / (delta * delta);
      ilp_para.S[n][m] = S;
      ilp_para.rcutsq_ilp[n][m] = rcut_ilp * rcut_ilp;
      ilp_para.rcut_global[n][m] = rcut_global;
      float meV = 1e-3 * S;
      ilp_para.C[n][m] *= meV;
      ilp_para.C_6[n][m] *= meV;
      ilp_para.epsilon[n][m] *= meV;

      if (rc_ilp < rcut_global)
        rc_ilp = rcut_global;
    }
  }


  // initialize neighbor lists and some temp vectors
  int max_neighbor_number = min(num_atoms, IN_CUDA_MAX_NL_TMD);
  ilp_data.NN.resize(num_atoms);
  ilp_data.NL.resize(num_atoms * max_neighbor_number);
  ilp_data.cell_count.resize(num_atoms);
  ilp_data.cell_count_sum.resize(num_atoms);
  ilp_data.cell_contents.resize(num_atoms);

  // init ilp neighbor list
  ilp_data.ilp_NN.resize(num_atoms);
  ilp_data.ilp_NL.resize(num_atoms * IN_MAX_ILP_NEIGHBOR_TMD);
  ilp_data.reduce_NL.resize(num_atoms * max_neighbor_number);
  ilp_data.big_ilp_NN.resize(num_atoms);
  ilp_data.big_ilp_NL.resize(num_atoms * IN_MAX_BIG_ILP_NEIGHBOR_TMD);

  ilp_data.f12x.resize(num_atoms * max_neighbor_number);
  ilp_data.f12y.resize(num_atoms * max_neighbor_number);
  ilp_data.f12z.resize(num_atoms * max_neighbor_number);

  ilp_data.f12x_ilp_neigh.resize(num_atoms * IN_MAX_ILP_NEIGHBOR_TMD);
  ilp_data.f12y_ilp_neigh.resize(num_atoms * IN_MAX_ILP_NEIGHBOR_TMD);
  ilp_data.f12z_ilp_neigh.resize(num_atoms * IN_MAX_ILP_NEIGHBOR_TMD);


  // init constant cutoff coeff
  float h_tap_coeff[8] = \
    {1.0f, 0.0f, 0.0f, 0.0f, -35.0f, 84.0f, -70.0f, 20.0f};
  CHECK(gpuMemcpyToSymbol(IN_Tap_coeff_tmd, h_tap_coeff, 8 * sizeof(float)));

  // set ilp_flag to 1
  ilp_flag = 1;


  std::ifstream input(file_nep);
  if (!input.is_open()) {
    std::cout << "Failed to open " << file_nep << std::endl;
    exit(1);
  }

  // nep3 1 C
  std::vector<std::string> tokens = get_tokens(input);
  if (tokens.size() < 3) {
    std::cout << "The first line of nep.txt should have at least 3 items." << std::endl;
    exit(1);
  }
  if (tokens[0] == "nep3") {
    paramb.version = 3;
  } else if (tokens[0] == "nep4") {
    paramb.version = 4;
  } else if (tokens[0] == "nep5") {
    paramb.version = 5;
  } else {
    std::cout << tokens[0]
              << " is an unsupported NEP model. We only support NEP3 and NEP4 models now."
              << std::endl;
    exit(1);
  }
  paramb.num_types = get_int_from_token(tokens[1], __FILE__, __LINE__);
  if (tokens.size() != 2 + paramb.num_types) {
    std::cout << "The first line of nep.txt should have " << paramb.num_types << " atom symbols."
              << std::endl;
    exit(1);
  }

  if (paramb.num_types == 1) {
    printf("Use the NEP%d potential with %d atom type.\n", paramb.version, paramb.num_types);
  } else {
    printf("Use the NEP%d potential with %d atom types.\n", paramb.version, paramb.num_types);
  }

  for (int n = 0; n < paramb.num_types; ++n) {
    if (tokens[2 + n] != ilp_elements[n] || paramb.num_types != num_types) {
      std::cout << "ILP and NEP potential files must have the same element list\n"
                << "Element list size of ILP is " << num_types << " NEP is " << paramb.num_types << std::endl
                << "Element " << n << " in ILP is " << ilp_elements[n] << " in NEP is " << tokens[2 + n] << std::endl;
      exit(1);
    }
    int atomic_number = 0;
    for (int m = 0; m < NUM_ELEMENTS; ++m) {
      if (tokens[2 + n] == ELEMENTS[m]) {
        atomic_number = m + 1;
        break;
      }
    }
    paramb.atomic_numbers[n] = atomic_number - 1;
    printf("    type %d (%s with Z = %d).\n", n, tokens[2 + n].c_str(), atomic_number);
  }


  // cutoff 4.2 3.7 80 47 1
  tokens = get_tokens(input);
  if (tokens.size() != 5 && tokens.size() != 8) {
    std::cout << "This line should be cutoff rc_radial rc_angular MN_radial MN_angular "
                 "[radial_factor] [angular_factor] [zbl_factor].\n";
    exit(1);
  }
  paramb.rc_radial = get_double_from_token(tokens[1], __FILE__, __LINE__);
  paramb.rc_angular = get_double_from_token(tokens[2], __FILE__, __LINE__);
  printf("    radial cutoff = %g A.\n", paramb.rc_radial);
  printf("    angular cutoff = %g A.\n", paramb.rc_angular);

  int MN_radial = get_int_from_token(tokens[3], __FILE__, __LINE__);
  int MN_angular = get_int_from_token(tokens[4], __FILE__, __LINE__);
  printf("    MN_radial = %d.\n", MN_radial);
  if (MN_radial > 819) {
    std::cout << "The maximum number of neighbors exceeds 819. Please reduce this value."
              << std::endl;
    exit(1);
  }
  paramb.MN_radial = int(ceil(MN_radial * 1.25));
  paramb.MN_angular = int(ceil(MN_angular * 1.25));
  printf("    enlarged MN_radial = %d.\n", paramb.MN_radial);
  printf("    enlarged MN_angular = %d.\n", paramb.MN_angular);

  if (tokens.size() == 8) {
    paramb.typewise_cutoff_radial_factor = get_double_from_token(tokens[5], __FILE__, __LINE__);
    paramb.typewise_cutoff_angular_factor = get_double_from_token(tokens[6], __FILE__, __LINE__);
    paramb.typewise_cutoff_zbl_factor = get_double_from_token(tokens[7], __FILE__, __LINE__);
    if (paramb.typewise_cutoff_radial_factor > 0.0f) {
      paramb.use_typewise_cutoff = true;
    }
    if (paramb.typewise_cutoff_zbl_factor > 0.0f) {
      paramb.use_typewise_cutoff_zbl = true;
    }
  }
#ifdef USE_TABLE
  if (paramb.use_typewise_cutoff) {
    PRINT_INPUT_ERROR("Cannot use tabulated radial functions with typewise cutoff.");
  }
#endif

  // n_max 10 8
  tokens = get_tokens(input);
  if (tokens.size() != 3) {
    std::cout << "This line should be n_max n_max_radial n_max_angular." << std::endl;
    exit(1);
  }
  paramb.n_max_radial = get_int_from_token(tokens[1], __FILE__, __LINE__);
  paramb.n_max_angular = get_int_from_token(tokens[2], __FILE__, __LINE__);
  printf("    n_max_radial = %d.\n", paramb.n_max_radial);
  printf("    n_max_angular = %d.\n", paramb.n_max_angular);

  // basis_size 10 8
  tokens = get_tokens(input);
  if (tokens.size() != 3) {
    std::cout << "This line should be basis_size basis_size_radial basis_size_angular."
              << std::endl;
    exit(1);
  }
  paramb.basis_size_radial = get_int_from_token(tokens[1], __FILE__, __LINE__);
  paramb.basis_size_angular = get_int_from_token(tokens[2], __FILE__, __LINE__);
  printf("    basis_size_radial = %d.\n", paramb.basis_size_radial);
  printf("    basis_size_angular = %d.\n", paramb.basis_size_angular);

  // l_max
  tokens = get_tokens(input);
  if (tokens.size() != 4) {
    std::cout << "This line should be l_max l_max_3body l_max_4body l_max_5body." << std::endl;
    exit(1);
  }

  paramb.L_max = get_int_from_token(tokens[1], __FILE__, __LINE__);
  printf("    l_max_3body = %d.\n", paramb.L_max);
  paramb.num_L = paramb.L_max;

  int L_max_4body = get_int_from_token(tokens[2], __FILE__, __LINE__);
  int L_max_5body = get_int_from_token(tokens[3], __FILE__, __LINE__);
  printf("    l_max_4body = %d.\n", L_max_4body);
  printf("    l_max_5body = %d.\n", L_max_5body);
  if (L_max_4body == 2) {
    paramb.num_L += 1;
  }
  if (L_max_5body == 1) {
    paramb.num_L += 1;
  }

  paramb.dim_angular = (paramb.n_max_angular + 1) * paramb.num_L;

  // ANN
  tokens = get_tokens(input);
  if (tokens.size() != 3) {
    std::cout << "This line should be ANN num_neurons 0." << std::endl;
    exit(1);
  }
  annmb.num_neurons1 = get_int_from_token(tokens[1], __FILE__, __LINE__);
  annmb.dim = (paramb.n_max_radial + 1) + paramb.dim_angular;
  nep_model_type = paramb.model_type;
  if (paramb.model_type == 3) {
    annmb.dim += 1;
  }
  printf("    ANN = %d-%d-1.\n", annmb.dim, annmb.num_neurons1);

  // calculated parameters:
  rc = paramb.rc_radial; // largest cutoff
  paramb.rcinv_radial = 1.0f / paramb.rc_radial;
  paramb.rcinv_angular = 1.0f / paramb.rc_angular;
  paramb.num_types_sq = paramb.num_types * paramb.num_types;

  if (paramb.version == 3) {
    annmb.num_para_ann = (annmb.dim + 2) * annmb.num_neurons1 + 1;
  } else if (paramb.version == 4) {
    annmb.num_para_ann = (annmb.dim + 2) * annmb.num_neurons1 * paramb.num_types + 1;
  } else {
    annmb.num_para_ann = ((annmb.dim + 2) * annmb.num_neurons1 + 1) * paramb.num_types + 1;
  }
  if (paramb.model_type == 2) {
    // Polarizability models have twice as many parameters
    annmb.num_para_ann *= 2;
  }
  printf("    number of neural network parameters = %d.\n", annmb.num_para_ann);
  int num_para_descriptor =
    paramb.num_types_sq * ((paramb.n_max_radial + 1) * (paramb.basis_size_radial + 1) +
                           (paramb.n_max_angular + 1) * (paramb.basis_size_angular + 1));
  printf("    number of descriptor parameters = %d.\n", num_para_descriptor);
  annmb.num_para = annmb.num_para_ann + num_para_descriptor;
  printf("    total number of parameters = %d.\n", annmb.num_para);

  paramb.num_c_radial =
    paramb.num_types_sq * (paramb.n_max_radial + 1) * (paramb.basis_size_radial + 1);

  // NN and descriptor parameters
  std::vector<float> parameters(annmb.num_para);
  for (int n = 0; n < annmb.num_para; ++n) {
    tokens = get_tokens(input);
    parameters[n] = get_double_from_token(tokens[0], __FILE__, __LINE__);
  }
  nep_data.parameters.resize(annmb.num_para);
  nep_data.parameters.copy_from_host(parameters.data());
  update_potential(nep_data.parameters.data(), annmb);
  for (int d = 0; d < annmb.dim; ++d) {
    tokens = get_tokens(input);
    paramb.q_scaler[d] = get_double_from_token(tokens[0], __FILE__, __LINE__);
  }


  nep_data.f12x.resize(num_atoms * paramb.MN_angular);
  nep_data.f12y.resize(num_atoms * paramb.MN_angular);
  nep_data.f12z.resize(num_atoms * paramb.MN_angular);
  nep_data.NN_radial.resize(num_atoms);
  nep_data.NL_radial.resize(num_atoms * paramb.MN_radial);
  nep_data.NN_angular.resize(num_atoms);
  nep_data.NL_angular.resize(num_atoms * paramb.MN_angular);
  nep_data.Fp.resize(num_atoms * annmb.dim);
  nep_data.sum_fxyz.resize(num_atoms * (paramb.n_max_angular + 1) * NUM_OF_ABC);
  nep_data.cell_count.resize(num_atoms);
  nep_data.cell_count_sum.resize(num_atoms);
  nep_data.cell_contents.resize(num_atoms);
  nep_data.cpu_NN_radial.resize(num_atoms);
  nep_data.cpu_NN_angular.resize(num_atoms);

#ifdef USE_TABLE
  construct_table(parameters.data());
  printf("    use tabulated radial functions to speed up.\n");
#endif

}

ILP_NEP_TMD::~ILP_NEP_TMD(void)
{
  // nothing
}

// ILP term
static __device__ __forceinline__ float calc_Tap(const float r_ij, const float Rcutinv)
{
  float Tap, r;

  r = r_ij * Rcutinv;
  if (r >= 1.0f) {
    Tap = 0.0f;
  } else {
    Tap = IN_Tap_coeff_tmd[7];
    for (int i = 6; i >= 0; --i) {
      Tap = Tap * r + IN_Tap_coeff_tmd[i];
    }
  }

  return Tap;
}

// calculate the derivatives of long-range cutoff term
static __device__ __forceinline__ float calc_dTap(const float r_ij, const float Rcut, const float Rcutinv)
{
  float dTap, r;
  
  r = r_ij * Rcutinv;
  if (r >= 1.0f) {
    dTap = 0.0f;
  } else {
    dTap = 7.0f * IN_Tap_coeff_tmd[7];
    for (int i = 6; i > 0; --i) {
      dTap = dTap * r + i * IN_Tap_coeff_tmd[i];
    }
    dTap *= Rcutinv;
  }

  return dTap;
}

// create ILP neighbor list from main neighbor list to calculate normals
static __global__ void ILP_neighbor(
  const int number_of_particles,
  const int N1,
  const int N2,
  const Box box,
  const int *g_neighbor_number,
  const int *g_neighbor_list,
  const int *g_type,
  IN_ILP_TMD_Para ilp_para,
  const double* __restrict__ g_x,
  const double* __restrict__ g_y,
  const double* __restrict__ g_z,
  int *ilp_neighbor_number,
  int *ilp_neighbor_list,
  const int *group_label)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1; // particle index

  if (n1 < N2) {
    // TMD
    int neighptr[10], check[10], neighsort[10];
    for (int ll = 0; ll < 10; ++ll) {
      neighptr[ll] = -1;
      neighsort[ll] = -1;
      check[ll] = -1;
    }

    int count = 0;
    int neighbor_number = g_neighbor_number[n1];
    int type1 = g_type[n1];
    double x1 = g_x[n1];
    double y1 = g_y[n1];
    double z1 = g_z[n1];

    for (int i1 = 0; i1 < neighbor_number; ++i1) {
      int n2 = g_neighbor_list[n1 + number_of_particles * i1];
      int type2 = g_type[n2];

      double x12 = g_x[n2] - x1;
      double y12 = g_y[n2] - y1;
      double z12 = g_z[n2] - z1;
      apply_mic(box, x12, y12, z12);
      double d12sq = x12 * x12 + y12 * y12 + z12 * z12;
      double rcutsq = ilp_para.rcutsq_ilp[type1][type2];


      if (group_label[n1] == group_label[n2] && d12sq < rcutsq && type1 == type2 && d12sq != 0) {
        // ilp_neighbor_list[count++ * number_of_particles + n1] = n2;
        neighptr[count++] = n2;
      }
    }

    // TMD
    for (int ll = 0; ll < count; ++ll) {
      neighsort[ll] = neighptr[ll];
      check[ll] = neighptr[ll];
    }

    // TMD
    if (count == NNEI) {
      neighsort[0] = neighptr[0];
      check[0] = -1;
    } else if (count < NNEI && count > 0) {
      for (int jj = 0; jj < count; ++jj) {
        int j = neighptr[jj];
        int jtype = g_type[j];
        int count_temp = 0;
        for (int ll = 0; ll < count; ++ll) {
          int l = neighptr[ll];
          int ltype = g_type[l];
          if (l == j) continue;
          double deljx = g_x[l] - g_x[j];
          double deljy = g_y[l] - g_y[j];
          double deljz = g_z[l] - g_z[j];
          apply_mic(box, deljx, deljy, deljz);
          double rsqlj = deljx * deljx + deljy * deljy + deljz * deljz;
          if (rsqlj != 0 && rsqlj < ilp_para.rcutsq_ilp[ltype][jtype]) {
            ++count_temp;
          }

        }
        if (count_temp == 1) {
          neighsort[0] = neighptr[jj];
          check[jj] = -1;
          break;
        }
      }
    } else if (count > NNEI) {
      printf("ERROR in ILP NEIGHBOR LIST\n");
      printf("\n===== ILP neighbor number[%d] is greater than 6 =====\n", count);
      return;
    }

    // TMD
    // sort the order of neighbors of atom n1
    for (int jj = 0; jj < count; ++jj) {
      int j = neighsort[jj];
      int jtype = g_type[j];
      int ll = 0;
      while (ll < count) {
        int l = neighptr[ll];
        if (check[ll] == -1) {
          ++ll;
          continue;
        }
        int ltype = g_type[l];
        double deljx = g_x[l] - g_x[j];
        double deljy = g_y[l] - g_y[j];
        double deljz = g_z[l] - g_z[j];
        apply_mic(box, deljx, deljy, deljz);
        double rsqlj = deljx * deljx + deljy * deljy + deljz * deljz;
        
        if (abs(rsqlj) >= 1e-6 && rsqlj < ilp_para.rcutsq_ilp[ltype][jtype]) {
          neighsort[jj + 1] = l;
          check[ll] = -1;
          break;
        }
        ++ll;
      }
    }
    ilp_neighbor_number[n1] = count;
    for (int jj = 0; jj < count; ++jj) {
      ilp_neighbor_list[jj * number_of_particles + n1] = neighsort[jj];
    }
  }
}

// modulo func to change atom index
static __device__ __forceinline__ int modulo(int k, int range)
{
  return (k + range) % range;
}

// calculate the normals and its derivatives
static __device__ void calc_normal(
  float (&vect)[NNEI][3],
  int cont,
  float (&normal)[3],
  float (&dnormdri)[3][3],
  float (&dnormal)[3][NNEI][3])
{
  int id, ip, m;
  float  dni[3];
  float  dnn[3][3], dpvdri[3][3];
  float Nave[3], pvet[NNEI][3], dpvet1[NNEI][3][3], dpvet2[NNEI][3][3], dNave[3][NNEI][3];

  float nninv;

  // initialize the arrays
  for (id = 0; id < 3; id++) {
    dni[id] = 0.0f;

    Nave[id] = 0.0f;
    for (ip = 0; ip < 3; ip++) {
      dpvdri[ip][id] = 0.0f;
      for (m = 0; m < NNEI; m++) {
        dnn[m][id] = 0.0f;
        pvet[m][id] = 0.0f;
        dpvet1[m][ip][id] = 0.0f;
        dpvet2[m][ip][id] = 0.0f;
        dNave[id][m][ip] = 0.0f;
      }
    }
  }

  if (cont <= 1) {
    normal[0] = 0.0f;
    normal[1] = 0.0f;
    normal[2] = 1.0f;
    for (id = 0; id < 3; ++id) {
      for (ip = 0; ip < 3; ++ip) {
        dnormdri[id][ip] = 0.0f;
        for (m = 0; m < NNEI; ++m) {
          dnormal[id][m][ip] = 0.0f;
        }
      }
    }
  } else if (cont > 1 && cont < NNEI) {
    for (int k = 0; k < cont - 1; ++k) {
      for (ip = 0; ip < 3; ++ip) {
        pvet[k][ip] = vect[k][modulo(ip + 1, 3)] * vect[k + 1][modulo(ip + 2, 3)] -
                vect[k][modulo(ip + 2, 3)] * vect[k + 1][modulo(ip + 1, 3)];
      }
      // dpvet1[k][l][ip]: the derivatve of the k (=0,...cont-1)th Nik respect to the ip component of atom l
      // derivatives respect to atom l
      // dNik,x/drl
      dpvet1[k][0][0] = 0.0f;
      dpvet1[k][0][1] = vect[modulo(k + 1, NNEI)][2];
      dpvet1[k][0][2] = -vect[modulo(k + 1, NNEI)][1];
      // dNik,y/drl
      dpvet1[k][1][0] = -vect[modulo(k + 1, NNEI)][2];
      dpvet1[k][1][1] = 0.0f;
      dpvet1[k][1][2] = vect[modulo(k + 1, NNEI)][0];
      // dNik,z/drl
      dpvet1[k][2][0] = vect[modulo(k + 1, NNEI)][1];
      dpvet1[k][2][1] = -vect[modulo(k + 1, NNEI)][0];
      dpvet1[k][2][2] = 0.0f;

      // dpvet2[k][l][ip]: the derivatve of the k (=0,...cont-1)th Nik respect to the ip component of atom l+1
      // derivatives respect to atom l+1
      // dNik,x/drl+1
      dpvet2[k][0][0] = 0.0f;
      dpvet2[k][0][1] = -vect[modulo(k, NNEI)][2];
      dpvet2[k][0][2] = vect[modulo(k, NNEI)][1];
      // dNik,y/drl+1
      dpvet2[k][1][0] = vect[modulo(k, NNEI)][2];
      dpvet2[k][1][1] = 0.0f;
      dpvet2[k][1][2] = -vect[modulo(k, NNEI)][0];
      // dNik,z/drl+1
      dpvet2[k][2][0] = -vect[modulo(k, NNEI)][1];
      dpvet2[k][2][1] = vect[modulo(k, NNEI)][0];
      dpvet2[k][2][2] = 0.0f;
    }

    // average the normal vectors by using the NNEI neighboring planes
    for (ip = 0; ip < 3; ip++) {
      Nave[ip] = 0.0f;
      for (int k = 0; k < cont - 1; k++) {
        Nave[ip] += pvet[k][ip];
      }
      Nave[ip] /= (cont - 1);
    }
    nninv = rnorm3df(Nave[0], Nave[1], Nave[2]);
    
    // the unit normal vector
    normal[0] = Nave[0] * nninv;
    normal[1] = Nave[1] * nninv;
    normal[2] = Nave[2] * nninv;

    // derivatives of non-normalized normal vector, dNave:3xcontx3 array
    // dNave[id][m][ip]: the derivatve of the id component of Nave respect to the ip component of atom m
    for (id = 0; id < 3; id++) {
      for (ip = 0; ip < 3; ip++) {
        for (m = 0; m < cont; m++) {
          if (m == 0) {
            dNave[id][m][ip] = dpvet1[m][id][ip] / (cont - 1);
          } else if (m == cont - 1) {
            dNave[id][m][ip] = dpvet2[m - 1][id][ip] / (cont - 1);
          } else {    // sum of the derivatives of the mth and (m-1)th normal vector respect to the atom m
            dNave[id][m][ip] = (dpvet1[m][id][ip] + dpvet2[m - 1][id][ip]) / (cont - 1);
          }
        }
      }
    }
    // derivatives of nn, dnn:contx3 vector
    // dnn[m][id]: the derivative of nn respect to r[m][id], m=0,...NNEI-1; id=0,1,2
    // r[m][id]: the id's component of atom m
    for (m = 0; m < cont; m++) {
      for (id = 0; id < 3; id++) {
        dnn[m][id] = (Nave[0] * dNave[0][m][id] + Nave[1] * dNave[1][m][id] +
                      Nave[2] * dNave[2][m][id]) * nninv;
      }
    }
    // dnormal[i][id][m][ip]: the derivative of normal[i][id] respect to r[m][ip], id,ip=0,1,2.
    // for atom m, which is a neighbor atom of atom i, m = 0,...,NNEI-1
    for (m = 0; m < cont; m++) {
      for (id = 0; id < 3; id++) {
        for (ip = 0; ip < 3; ip++) {
          dnormal[id][m][ip] = dNave[id][m][ip] * nninv - Nave[id] * dnn[m][ip] * nninv * nninv;
        }
      }
    }
    // Calculte dNave/dri, defined as dpvdri
    for (id = 0; id < 3; id++) {
      for (ip = 0; ip < 3; ip++) {
        dpvdri[id][ip] = 0.0;
        for (int k = 0; k < cont; k++) {
          dpvdri[id][ip] -= dNave[id][k][ip];
        }
      }
    }

    // derivatives of nn, dnn:3x1 vector
    dni[0] = (Nave[0] * dpvdri[0][0] + Nave[1] * dpvdri[1][0] + Nave[2] * dpvdri[2][0]) * nninv;
    dni[1] = (Nave[0] * dpvdri[0][1] + Nave[1] * dpvdri[1][1] + Nave[2] * dpvdri[2][1]) * nninv;
    dni[2] = (Nave[0] * dpvdri[0][2] + Nave[1] * dpvdri[1][2] + Nave[2] * dpvdri[2][2]) * nninv;
    // derivatives of unit vector ni respect to ri, the result is 3x3 matrix
    for (id = 0; id < 3; id++) {
      for (ip = 0; ip < 3; ip++) {
        dnormdri[id][ip] = dpvdri[id][ip] * nninv - Nave[id] * dni[ip] * nninv * nninv;
      }
    }
  } else if (cont == NNEI) {
    // derivatives of Ni[l] respect to the NNEI neighbors
    for (int k = 0; k < NNEI; ++k) {
      for (ip = 0; ip < 3; ++ip) {
        pvet[k][ip] = vect[modulo(k, NNEI)][modulo(ip + 1, 3)] *
                vect[modulo(k + 1, NNEI)][modulo(ip + 2, 3)] -
            vect[modulo(k, NNEI)][modulo(ip + 2, 3)] *
                vect[modulo(k + 1, NNEI)][modulo(ip + 1, 3)];
      }
      // dpvet1[k][l][ip]: the derivatve of the k (=0,...cont-1)th Nik respect to the ip component of atom l
      // derivatives respect to atom l
      // dNik,x/drl
      dpvet1[k][0][0] = 0.0f;
      dpvet1[k][0][1] = vect[modulo(k + 1, NNEI)][2];
      dpvet1[k][0][2] = -vect[modulo(k + 1, NNEI)][1];
      // dNik,y/drl
      dpvet1[k][1][0] = -vect[modulo(k + 1, NNEI)][2];
      dpvet1[k][1][1] = 0.0f;
      dpvet1[k][1][2] = vect[modulo(k + 1, NNEI)][0];
      // dNik,z/drl
      dpvet1[k][2][0] = vect[modulo(k + 1, NNEI)][1];
      dpvet1[k][2][1] = -vect[modulo(k + 1, NNEI)][0];
      dpvet1[k][2][2] = 0.0f;

      // dpvet2[k][l][ip]: the derivatve of the k (=0,...cont-1)th Nik respect to the ip component of atom l+1
      // derivatives respect to atom l+1
      // dNik,x/drl+1
      dpvet2[k][0][0] = 0.0f;
      dpvet2[k][0][1] = -vect[modulo(k, NNEI)][2];
      dpvet2[k][0][2] = vect[modulo(k, NNEI)][1];
      // dNik,y/drl+1
      dpvet2[k][1][0] = vect[modulo(k, NNEI)][2];
      dpvet2[k][1][1] = 0.0f;
      dpvet2[k][1][2] = -vect[modulo(k, NNEI)][0];
      // dNik,z/drl+1
      dpvet2[k][2][0] = -vect[modulo(k, NNEI)][1];
      dpvet2[k][2][1] = vect[modulo(k, NNEI)][0];
      dpvet2[k][2][2] = 0.0f;
    }

    // average the normal vectors by using the NNEI neighboring planes
    for (ip = 0; ip < 3; ++ip) {
      Nave[ip] = 0.0f;
      for (int k = 0; k < NNEI; ++k) {
        Nave[ip] += pvet[k][ip];
      }
      Nave[ip] /= NNEI;
    }
    // the magnitude of the normal vector
    // nn2 = Nave[0] * Nave[0] + Nave[1] * Nave[1] + Nave[2] * Nave[2];
    nninv = rnorm3df(Nave[0], Nave[1], Nave[2]);
    // the unit normal vector
    normal[0] = Nave[0] * nninv;
    normal[1] = Nave[1] * nninv;
    normal[2] = Nave[2] * nninv;

    // for the central atoms, dnormdri is always zero
    for (id = 0; id < 3; ++id) {
      for (ip = 0; ip < 3; ++ip) {
        dnormdri[id][ip] = 0.0f;
      }
    }

    // derivatives of non-normalized normal vector, dNave:3xNNEIx3 array
    // dNave[id][m][ip]: the derivatve of the id component of Nave respect to the ip component of atom m
    for (id = 0; id < 3; ++id) {
      for (ip = 0; ip < 3; ++ip) {
        for (
            m = 0; m < NNEI;
            ++m) {    // sum of the derivatives of the mth and (m-1)th normal vector respect to the atom m
          dNave[id][m][ip] =
              (dpvet1[modulo(m, NNEI)][id][ip] + dpvet2[modulo(m - 1, NNEI)][id][ip]) / NNEI;
        }
      }
    }
    // derivatives of nn, dnn:NNEIx3 vector
    // dnn[m][id]: the derivative of nn respect to r[m][id], m=0,...NNEI-1; id=0,1,2
    // r[m][id]: the id's component of atom m
    for (m = 0; m < NNEI; ++m) {
      for (id = 0; id < 3; ++id) {
        dnn[m][id] =
            (Nave[0] * dNave[0][m][id] + Nave[1] * dNave[1][m][id] + Nave[2] * dNave[2][m][id]) *
            nninv;
      }
    }
    // dnormal[i][id][m][ip]: the derivative of normal[i][id] respect to r[m][ip], id,ip=0,1,2.
    // for atom m, which is a neighbor atom of atom i, m = 0,...,NNEI-1
    for (m = 0; m < NNEI; ++m) {
      for (id = 0; id < 3; ++id) {
        for (ip = 0; ip < 3; ++ip) {
          dnormal[id][m][ip] = dNave[id][m][ip] * nninv - Nave[id] * dnn[m][ip] * nninv * nninv;
        }
      }
    }
  } else {
    printf("\n===== ILP neighbor number[%d] is greater than 6 =====\n", cont);
    return;
  }
}

// calculate the van der Waals force and energy
static __device__ void calc_vdW(
  float r,
  float rinv,
  float rsq,
  float d,
  float d_Seff,
  float C_6,
  float Tap,
  float dTap,
  float &p2_vdW,
  float &f2_vdW)
{
  float r2inv, r6inv, r8inv;
  float TSvdw, TSvdwinv, Vilp;
  float fpair, fsum;

  r2inv = 1.0f / rsq;
  r6inv = r2inv * r2inv * r2inv;
  r8inv = r2inv * r6inv;

  // TSvdw = 1.0 + exp(-d_Seff * r + d);
  TSvdw = 1.0f + expf(-d_Seff * r + d);
  TSvdwinv = 1.0f / TSvdw;
  Vilp = -C_6 * r6inv * TSvdwinv;

  // derivatives
  // fpair = -6.0 * C_6 * r8inv * TSvdwinv + \
  //   C_6 * d_Seff * (TSvdw - 1.0) * TSvdwinv * TSvdwinv * r8inv * r;
  fpair = (-6.0f + d_Seff * (TSvdw - 1.0f) * TSvdwinv * r ) * C_6 * TSvdwinv * r8inv;
  fsum = fpair * Tap - Vilp * dTap * rinv;

  p2_vdW = Tap * Vilp;
  f2_vdW = fsum;
  
}

// force evaluation kernel
static __global__ void gpu_find_force(
  IN_ILP_TMD_Para ilp_para,
  const int number_of_particles,
  const int N1,
  const int N2,
  const Box box,
  const int *g_neighbor_number,
  const int *g_neighbor_list,
  int *g_ilp_neighbor_number,
  int *g_ilp_neighbor_list,
  const int *group_label,
  const int *g_type,
  const double *__restrict__ g_x,
  const double *__restrict__ g_y,
  const double *__restrict__ g_z,
  double *g_fx,
  double *g_fy,
  double *g_fz,
  double *g_virial,
  double *g_potential,
  float *g_f12x,
  float *g_f12y,
  float *g_f12z,
  float *g_f12x_ilp_neigh,
  float *g_f12y_ilp_neigh,
  float *g_f12z_ilp_neigh)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1; // particle index
  float s_fx = 0.0f;                                   // force_x
  float s_fy = 0.0f;                                   // force_y
  float s_fz = 0.0f;                                   // force_z
  float s_pe = 0.0f;                                   // potential energy
  float s_sxx = 0.0f;                                  // virial_stress_xx
  float s_sxy = 0.0f;                                  // virial_stress_xy
  float s_sxz = 0.0f;                                  // virial_stress_xz
  float s_syx = 0.0f;                                  // virial_stress_yx
  float s_syy = 0.0f;                                  // virial_stress_yy
  float s_syz = 0.0f;                                  // virial_stress_yz
  float s_szx = 0.0f;                                  // virial_stress_zx
  float s_szy = 0.0f;                                  // virial_stress_zy
  float s_szz = 0.0f;                                  // virial_stress_zz

  float r = 0.0f;
  float rsq = 0.0f;
  float Rcut = 0.0f;

  if (n1 < N2) {
    double x12d, y12d, z12d;
    float x12f, y12f, z12f;
    int neighor_number = g_neighbor_number[n1];
    int type1 = g_type[n1];
    double x1 = g_x[n1];
    double y1 = g_y[n1];
    double z1 = g_z[n1];

    float delkix_half[NNEI] = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f};
    float delkiy_half[NNEI] = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f};
    float delkiz_half[NNEI] = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f};

    // calculate the normal
    int cont = 0;
    float normal[3];
    float dnormdri[3][3];
    float dnormal[3][NNEI][3];

    float vet[NNEI][3];
    int id, ip, m;
    for (id = 0; id < 3; ++id) {
      normal[id] = 0.0f;
      for (ip = 0; ip < 3; ++ip) {
        dnormdri[ip][id] = 0.0f;
        for (m = 0; m < NNEI; ++m) {
          dnormal[id][m][ip] = 0.0f;
          vet[m][id] = 0.0f;
        }
      }
    }

    int ilp_neighbor_number = g_ilp_neighbor_number[n1];
    for (int i1 = 0; i1 < ilp_neighbor_number; ++i1) {
      int n2_ilp = g_ilp_neighbor_list[n1 + number_of_particles * i1];
      x12d = g_x[n2_ilp] - x1;
      y12d = g_y[n2_ilp] - y1;
      z12d = g_z[n2_ilp] - z1;
      apply_mic(box, x12d, y12d, z12d);
      vet[cont][0] = float(x12d);
      vet[cont][1] = float(y12d);
      vet[cont][2] = float(z12d);
      ++cont;

      delkix_half[i1] = float(x12d) * 0.5f;
      delkiy_half[i1] = float(y12d) * 0.5f;
      delkiz_half[i1] = float(z12d) * 0.5f;
    }
    
    calc_normal(vet, cont, normal, dnormdri, dnormal);

    // calculate energy and force
    double tt1,tt2,tt3;
    for (int i1 = 0; i1 < neighor_number; ++i1) {
      int index = n1 + number_of_particles * i1;
      int n2 = g_neighbor_list[index];
      int type2 = g_type[n2];

      tt1 = g_x[n2];
      tt2 = g_y[n2];
      tt3 = g_z[n2];
      x12d = tt1 - x1;
      y12d = tt2 - y1;
      z12d = tt3 - z1;
      apply_mic(box, x12d, y12d, z12d);

      // save x12, y12, z12 in float
      x12f = float(x12d);
      y12f = float(y12d);
      z12f = float(z12d);

      // calculate distance between atoms
      rsq = x12f * x12f + y12f * y12f + z12f * z12f;
      r = sqrtf(rsq);
      Rcut = ilp_para.rcut_global[type1][type2];

      if (r >= Rcut) {
        continue;
      }

      // calc att
      float Tap, dTap, rinv;
      float Rcutinv = 1.0f / Rcut;
      rinv = 1.0f / r;
      Tap = calc_Tap(r, Rcutinv);
      dTap = calc_dTap(r, Rcut, Rcutinv);

      float p2_vdW, f2_vdW;
      calc_vdW(
        r,
        rinv,
        rsq,
        ilp_para.d[type1][type2],
        ilp_para.d_Seff[type1][type2],
        ilp_para.C_6[type1][type2],
        Tap,
        dTap,
        p2_vdW,
        f2_vdW);
      
      float f12x = -f2_vdW * x12f * 0.5f;
      float f12y = -f2_vdW * y12f * 0.5f;
      float f12z = -f2_vdW * z12f * 0.5f;
      float f21x = -f12x;
      float f21y = -f12y;
      float f21z = -f12z;

      s_fx += f12x - f21x;
      s_fy += f12y - f21y;
      s_fz += f12z - f21z;

      s_pe += p2_vdW * 0.5f;
      s_sxx += x12f * f21x;
      s_sxy += x12f * f21y;
      s_sxz += x12f * f21z;
      s_syx += y12f * f21x;
      s_syy += y12f * f21y;
      s_syz += y12f * f21z;
      s_szx += z12f * f21x;
      s_szy += z12f * f21y;
      s_szz += z12f * f21z;

      
      // calc rep
      float C = ilp_para.C[type1][type2];
      float lambda_ = ilp_para.lambda[type1][type2];
      float delta2inv = ilp_para.delta2inv[type1][type2];
      float epsilon = ilp_para.epsilon[type1][type2];
      float z0 = ilp_para.z0[type1][type2];
      // calc_rep
      float prodnorm1, rhosq1, rdsq1, exp0, exp1, frho1, Erep, Vilp;
      float fpair, fpair1, fsum, delx, dely, delz, fkcx, fkcy, fkcz;
      float dprodnorm1[3] = {0.0f, 0.0f, 0.0f};
      float fp1[3] = {0.0f, 0.0f, 0.0f};
      float fprod1[3] = {0.0f, 0.0f, 0.0f};
      float fk[3] = {0.0f, 0.0f, 0.0f};

      delx = -x12f;
      dely = -y12f;
      delz = -z12f;

      float delx_half = delx * 0.5f;
      float dely_half = dely * 0.5f;
      float delz_half = delz * 0.5f;

      // calculate the transverse distance
      prodnorm1 = normal[0] * delx + normal[1] * dely + normal[2] * delz;
      rhosq1 = rsq - prodnorm1 * prodnorm1;
      rdsq1 = rhosq1 * delta2inv;

      // store exponents
      // exp0 = exp(-lambda_ * (r - z0));
      // exp1 = exp(-rdsq1);
      exp0 = expf(-lambda_ * (r - z0));
      exp1 = expf(-rdsq1);

      frho1 = exp1 * C;
      Erep = 0.5f * epsilon + frho1;
      Vilp = exp0 * Erep;

      // derivatives
      fpair = lambda_ * exp0 * rinv * Erep;
      fpair1 = 2.0f * exp0 * frho1 * delta2inv;
      fsum = fpair + fpair1;

      float prodnorm1_m_fpair1 = prodnorm1 * fpair1;
      float Vilp_m_dTap_m_rinv = Vilp * dTap * rinv;

      // derivatives of the product of rij and ni, the resutl is a vector
      dprodnorm1[0] = 
        dnormdri[0][0] * delx + dnormdri[1][0] * dely + dnormdri[2][0] * delz;
      dprodnorm1[1] = 
        dnormdri[0][1] * delx + dnormdri[1][1] * dely + dnormdri[2][1] * delz;
      dprodnorm1[2] = 
        dnormdri[0][2] * delx + dnormdri[1][2] * dely + dnormdri[2][2] * delz;
      // fp1[0] = prodnorm1 * normal[0] * fpair1;
      // fp1[1] = prodnorm1 * normal[1] * fpair1;
      // fp1[2] = prodnorm1 * normal[2] * fpair1;
      // fprod1[0] = prodnorm1 * dprodnorm1[0] * fpair1;
      // fprod1[1] = prodnorm1 * dprodnorm1[1] * fpair1;
      // fprod1[2] = prodnorm1 * dprodnorm1[2] * fpair1;
      fp1[0] = prodnorm1_m_fpair1 * normal[0];
      fp1[1] = prodnorm1_m_fpair1 * normal[1];
      fp1[2] = prodnorm1_m_fpair1 * normal[2];
      fprod1[0] = prodnorm1_m_fpair1 * dprodnorm1[0];
      fprod1[1] = prodnorm1_m_fpair1 * dprodnorm1[1];
      fprod1[2] = prodnorm1_m_fpair1 * dprodnorm1[2];

      // fkcx = (delx * fsum - fp1[0]) * Tap - Vilp * dTap * delx * rinv;
      // fkcy = (dely * fsum - fp1[1]) * Tap - Vilp * dTap * dely * rinv;
      // fkcz = (delz * fsum - fp1[2]) * Tap - Vilp * dTap * delz * rinv;
      fkcx = (delx * fsum - fp1[0]) * Tap - Vilp_m_dTap_m_rinv * delx;
      fkcy = (dely * fsum - fp1[1]) * Tap - Vilp_m_dTap_m_rinv * dely;
      fkcz = (delz * fsum - fp1[2]) * Tap - Vilp_m_dTap_m_rinv * delz;

      s_fx += fkcx - fprod1[0] * Tap;
      s_fy += fkcy - fprod1[1] * Tap;
      s_fz += fkcz - fprod1[2] * Tap;

      g_f12x[index] = fkcx;
      g_f12y[index] = fkcy;
      g_f12z[index] = fkcz;

      float minus_prodnorm1_m_fpair1_m_Tap = -prodnorm1 * fpair1 * Tap;
      for (int kk = 0; kk < ilp_neighbor_number; ++kk) {
      // for (int kk = 0; kk < 0; ++kk) {
        // int index_ilp = n1 + number_of_particles * kk;
        // int n2_ilp = g_ilp_neighbor_list[index_ilp];
        // derivatives of the product of rij and ni respect to rk, k=0,1,2, where atom k is the neighbors of atom i
        dprodnorm1[0] = dnormal[0][kk][0] * delx + dnormal[1][kk][0] * dely +
            dnormal[2][kk][0] * delz;
        dprodnorm1[1] = dnormal[0][kk][1] * delx + dnormal[1][kk][1] * dely +
            dnormal[2][kk][1] * delz;
        dprodnorm1[2] = dnormal[0][kk][2] * delx + dnormal[1][kk][2] * dely +
            dnormal[2][kk][2] * delz;
        // fk[0] = (-prodnorm1 * dprodnorm1[0] * fpair1) * Tap;
        // fk[1] = (-prodnorm1 * dprodnorm1[1] * fpair1) * Tap;
        // fk[2] = (-prodnorm1 * dprodnorm1[2] * fpair1) * Tap;
        fk[0] = minus_prodnorm1_m_fpair1_m_Tap * dprodnorm1[0];
        fk[1] = minus_prodnorm1_m_fpair1_m_Tap * dprodnorm1[1];
        fk[2] = minus_prodnorm1_m_fpair1_m_Tap * dprodnorm1[2];

        g_f12x_ilp_neigh[n1 + number_of_particles * kk] += fk[0];
        g_f12y_ilp_neigh[n1 + number_of_particles * kk] += fk[1];
        g_f12z_ilp_neigh[n1 + number_of_particles * kk] += fk[2];

        // delki[0] = g_x[n2_ilp] - x1;
        // delki[1] = g_y[n2_ilp] - y1;
        // delki[2] = g_z[n2_ilp] - z1;
        // apply_mic(box, delki[0], delki[1], delki[2]);

        // s_sxx += delki[0] * fk[0] * 0.5;
        // s_sxy += delki[0] * fk[1] * 0.5;
        // s_sxz += delki[0] * fk[2] * 0.5;
        // s_syx += delki[1] * fk[0] * 0.5;
        // s_syy += delki[1] * fk[1] * 0.5;
        // s_syz += delki[1] * fk[2] * 0.5;
        // s_szx += delki[2] * fk[0] * 0.5;
        // s_szy += delki[2] * fk[1] * 0.5;
        // s_szz += delki[2] * fk[2] * 0.5;
        s_sxx += delkix_half[kk] * fk[0];
        s_sxy += delkix_half[kk] * fk[1];
        s_sxz += delkix_half[kk] * fk[2];
        s_syx += delkiy_half[kk] * fk[0];
        s_syy += delkiy_half[kk] * fk[1];
        s_syz += delkiy_half[kk] * fk[2];
        s_szx += delkiz_half[kk] * fk[0];
        s_szy += delkiz_half[kk] * fk[1];
        s_szz += delkiz_half[kk] * fk[2];
      }
      s_pe += Tap * Vilp;
      s_sxx += delx_half * fkcx;
      s_sxy += delx_half * fkcy;
      s_sxz += delx_half * fkcz;
      s_syx += dely_half * fkcx;
      s_syy += dely_half * fkcy;
      s_syz += dely_half * fkcz;
      s_szx += delz_half * fkcx;
      s_szy += delz_half * fkcy;
      s_szz += delz_half * fkcz;
    }

    // save force
    g_fx[n1] += s_fx;
    g_fy[n1] += s_fy;
    g_fz[n1] += s_fz;

    // save virial
    // xx xy xz    0 3 4
    // yx yy yz    6 1 5
    // zx zy zz    7 8 2
    g_virial[n1 + 0 * number_of_particles] += s_sxx;
    g_virial[n1 + 1 * number_of_particles] += s_syy;
    g_virial[n1 + 2 * number_of_particles] += s_szz;
    g_virial[n1 + 3 * number_of_particles] += s_sxy;
    g_virial[n1 + 4 * number_of_particles] += s_sxz;
    g_virial[n1 + 5 * number_of_particles] += s_syz;
    g_virial[n1 + 6 * number_of_particles] += s_syx;
    g_virial[n1 + 7 * number_of_particles] += s_szx;
    g_virial[n1 + 8 * number_of_particles] += s_szy;

    // save potential
    g_potential[n1] += s_pe;

  }
}

// build a neighbor list for reducing force
static __global__ void build_reduce_neighbor_list(
  const int number_of_particles,
  const int N1,
  const int N2,
  const int *g_neighbor_number,
  const int *g_neighbor_list,
  int *g_reduce_neighbor_list)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1;
  if (N1 < N2) {
    int neighbor_number = g_neighbor_number[n1];
    int l, r, m, tmp_value;
    for (int i1 = 0; i1 < neighbor_number; ++i1) {
      int index = n1 + i1 * number_of_particles;
      int n2 = g_neighbor_list[index];

      l = 0;
      r = g_neighbor_number[n2];
      while (l < r) {
        m = (l + r) >> 1;
        tmp_value = g_neighbor_list[n2 + number_of_particles * m];
        if (tmp_value < n1) {
          l = m + 1;
        } else if (tmp_value > n1) {
          r = m - 1;
        } else {
          break;
        }
      }
      g_reduce_neighbor_list[index] = (l + r) >> 1;
    }
  }
}

// reduce the rep force
static __global__ void reduce_force_many_body(
  const int number_of_particles,
  const int N1,
  const int N2,
  const Box box,
  const int *g_neighbor_number,
  const int *g_neighbor_list,
  int *g_reduce_neighbor_list,
  int *g_ilp_neighbor_number,
  int *g_ilp_neighbor_list,
  const double *__restrict__ g_x,
  const double *__restrict__ g_y,
  const double *__restrict__ g_z,
  double *g_fx,
  double *g_fy,
  double *g_fz,
  double *g_virial,
  float *g_f12x,
  float *g_f12y,
  float *g_f12z,
  float *g_f12x_ilp_neigh,
  float *g_f12y_ilp_neigh,
  float *g_f12z_ilp_neigh)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1; // particle index
  float s_fx = 0.0f;                                   // force_x
  float s_fy = 0.0f;                                   // force_y
  float s_fz = 0.0f;                                   // force_z
  float s_sxx = 0.0f;                                  // virial_stress_xx
  float s_sxy = 0.0f;                                  // virial_stress_xy
  float s_sxz = 0.0f;                                  // virial_stress_xz
  float s_syx = 0.0f;                                  // virial_stress_yx
  float s_syy = 0.0f;                                  // virial_stress_yy
  float s_syz = 0.0f;                                  // virial_stress_yz
  float s_szx = 0.0f;                                  // virial_stress_zx
  float s_szy = 0.0f;                                  // virial_stress_zy
  float s_szz = 0.0f;                                  // virial_stress_zz


  if (n1 < N2) {
    double x12d, y12d, z12d;
    float x12f, y12f, z12f;
    int neighbor_number_1 = g_neighbor_number[n1];
    double x1 = g_x[n1];
    double y1 = g_y[n1];
    double z1 = g_z[n1];

    // calculate energy and force
    for (int i1 = 0; i1 < neighbor_number_1; ++i1) {
      int index = n1 + number_of_particles * i1;
      int n2 = g_neighbor_list[index];

      x12d = g_x[n2] - x1;
      y12d = g_y[n2] - y1;
      z12d = g_z[n2] - z1;
      apply_mic(box, x12d, y12d, z12d);
      x12f = float(x12d);
      y12f = float(y12d);
      z12f = float(z12d);

      index = n2 + number_of_particles * g_reduce_neighbor_list[index];
      float f21x = g_f12x[index];
      float f21y = g_f12y[index];
      float f21z = g_f12z[index];

      s_fx -= f21x;
      s_fy -= f21y;
      s_fz -= f21z;

      // per-atom virial
      s_sxx += x12f * f21x * 0.5f;
      s_sxy += x12f * f21y * 0.5f;
      s_sxz += x12f * f21z * 0.5f;
      s_syx += y12f * f21x * 0.5f;
      s_syy += y12f * f21y * 0.5f;
      s_syz += y12f * f21z * 0.5f;
      s_szx += z12f * f21x * 0.5f;
      s_szy += z12f * f21y * 0.5f;
      s_szz += z12f * f21z * 0.5f;
    }

    int ilp_neighbor_number_1 = g_ilp_neighbor_number[n1];

    for (int i1 = 0; i1 < ilp_neighbor_number_1; ++i1) {
      int index = n1 + number_of_particles * i1;
      int n2 = g_ilp_neighbor_list[index];
      int ilp_neighor_number_2 = g_ilp_neighbor_number[n2];

      x12d = g_x[n2] - x1;
      y12d = g_y[n2] - y1;
      z12d = g_z[n2] - z1;
      apply_mic(box, x12d, y12d, z12d);
      x12f = float(x12d);
      y12f = float(y12d);
      z12f = float(z12d);

      int offset = 0;
      for (int k = 0; k < ilp_neighor_number_2; ++k) {
        if (n1 == g_ilp_neighbor_list[n2 + number_of_particles * k]) {
          offset = k;
          break;
        }
      }
      index = n2 + number_of_particles * offset;
      float f21x = g_f12x_ilp_neigh[index];
      float f21y = g_f12y_ilp_neigh[index];
      float f21z = g_f12z_ilp_neigh[index];

      s_fx += f21x;
      s_fy += f21y;
      s_fz += f21z;

      // per-atom virial
      s_sxx += -x12f * f21x * 0.5f;
      s_sxy += -x12f * f21y * 0.5f;
      s_sxz += -x12f * f21z * 0.5f;
      s_syx += -y12f * f21x * 0.5f;
      s_syy += -y12f * f21y * 0.5f;
      s_syz += -y12f * f21z * 0.5f;
      s_szx += -z12f * f21x * 0.5f;
      s_szy += -z12f * f21y * 0.5f;
      s_szz += -z12f * f21z * 0.5f;
    }

    // save force
    g_fx[n1] += s_fx;
    g_fy[n1] += s_fy;
    g_fz[n1] += s_fz;

    // save virial
    // xx xy xz    0 3 4
    // yx yy yz    6 1 5
    // zx zy zz    7 8 2
    g_virial[n1 + 0 * number_of_particles] += s_sxx;
    g_virial[n1 + 1 * number_of_particles] += s_syy;
    g_virial[n1 + 2 * number_of_particles] += s_szz;
    g_virial[n1 + 3 * number_of_particles] += s_sxy;
    g_virial[n1 + 4 * number_of_particles] += s_sxz;
    g_virial[n1 + 5 * number_of_particles] += s_syz;
    g_virial[n1 + 6 * number_of_particles] += s_syx;
    g_virial[n1 + 7 * number_of_particles] += s_szx;
    g_virial[n1 + 8 * number_of_particles] += s_szy;
  }
}



void ILP_NEP_TMD::update_potential(float* parameters, ANN& ann)
{
  float* pointer = parameters;
  for (int t = 0; t < paramb.num_types; ++t) {
    if (t > 0 && paramb.version == 3) { // Use the same set of NN parameters for NEP3
      pointer -= (ann.dim + 2) * ann.num_neurons1;
    }
    ann.w0[t] = pointer;
    pointer += ann.num_neurons1 * ann.dim;
    ann.b0[t] = pointer;
    pointer += ann.num_neurons1;
    ann.w1[t] = pointer;
    pointer += ann.num_neurons1;
    if (paramb.version == 5) {
      pointer += 1; // one extra bias for NEP5 stored in ann.w1[t]
    }
  }
  ann.b1 = pointer;
  pointer += 1;


  ann.c = pointer;
}

#ifdef USE_TABLE
void ILP_NEP_TMD::construct_table(float* parameters)
{
  nep_data.gn_radial.resize(table_length * paramb.num_types_sq * (paramb.n_max_radial + 1));
  nep_data.gnp_radial.resize(table_length * paramb.num_types_sq * (paramb.n_max_radial + 1));
  nep_data.gn_angular.resize(table_length * paramb.num_types_sq * (paramb.n_max_angular + 1));
  nep_data.gnp_angular.resize(table_length * paramb.num_types_sq * (paramb.n_max_angular + 1));
  std::vector<float> gn_radial(table_length * paramb.num_types_sq * (paramb.n_max_radial + 1));
  std::vector<float> gnp_radial(table_length * paramb.num_types_sq * (paramb.n_max_radial + 1));
  std::vector<float> gn_angular(table_length * paramb.num_types_sq * (paramb.n_max_angular + 1));
  std::vector<float> gnp_angular(table_length * paramb.num_types_sq * (paramb.n_max_angular + 1));
  float* c_pointer = parameters + annmb.num_para_ann;
  construct_table_radial_or_angular(
    paramb.num_types,
    paramb.num_types_sq,
    paramb.n_max_radial,
    paramb.basis_size_radial,
    paramb.rc_radial,
    paramb.rcinv_radial,
    c_pointer,
    gn_radial.data(),
    gnp_radial.data());
  construct_table_radial_or_angular(
    paramb.num_types,
    paramb.num_types_sq,
    paramb.n_max_angular,
    paramb.basis_size_angular,
    paramb.rc_angular,
    paramb.rcinv_angular,
    c_pointer + paramb.num_c_radial,
    gn_angular.data(),
    gnp_angular.data());
  nep_data.gn_radial.copy_from_host(gn_radial.data());
  nep_data.gnp_radial.copy_from_host(gnp_radial.data());
  nep_data.gn_angular.copy_from_host(gn_angular.data());
  nep_data.gnp_angular.copy_from_host(gnp_angular.data());
}
#endif

static __global__ void find_neighbor_list_nep(
  ILP_NEP_TMD::ParaMB paramb,
  const int N,
  const int N1,
  const int N2,
  const int nx,
  const int ny,
  const int nz,
  const Box box,
  const int* g_type,
  const int* __restrict__ g_cell_count,
  const int* __restrict__ g_cell_count_sum,
  const int* __restrict__ g_cell_contents,
  const double* __restrict__ g_x,
  const double* __restrict__ g_y,
  const double* __restrict__ g_z,
  int* g_NN_radial,
  int* g_NL_radial,
  int* g_NN_angular,
  int* g_NL_angular)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1;
  if (n1 >= N2) {
    return;
  }

  double x1 = g_x[n1];
  double y1 = g_y[n1];
  double z1 = g_z[n1];
  int t1 = g_type[n1];
  int count_radial = 0;
  int count_angular = 0;

  int cell_id;
  int cell_id_x;
  int cell_id_y;
  int cell_id_z;
  find_cell_id(
    box,
    x1,
    y1,
    z1,
    2.0f * paramb.rcinv_radial,
    nx,
    ny,
    nz,
    cell_id_x,
    cell_id_y,
    cell_id_z,
    cell_id);

  const int z_lim = box.pbc_z ? 2 : 0;
  const int y_lim = box.pbc_y ? 2 : 0;
  const int x_lim = box.pbc_x ? 2 : 0;

  for (int zz = -z_lim; zz <= z_lim; ++zz) {
    for (int yy = -y_lim; yy <= y_lim; ++yy) {
      for (int xx = -x_lim; xx <= x_lim; ++xx) {
        int neighbor_cell = cell_id + zz * nx * ny + yy * nx + xx;
        if (cell_id_x + xx < 0)
          neighbor_cell += nx;
        if (cell_id_x + xx >= nx)
          neighbor_cell -= nx;
        if (cell_id_y + yy < 0)
          neighbor_cell += ny * nx;
        if (cell_id_y + yy >= ny)
          neighbor_cell -= ny * nx;
        if (cell_id_z + zz < 0)
          neighbor_cell += nz * ny * nx;
        if (cell_id_z + zz >= nz)
          neighbor_cell -= nz * ny * nx;

        const int num_atoms_neighbor_cell = g_cell_count[neighbor_cell];
        const int num_atoms_previous_cells = g_cell_count_sum[neighbor_cell];

        for (int m = 0; m < num_atoms_neighbor_cell; ++m) {
          const int n2 = g_cell_contents[num_atoms_previous_cells + m];

          if (n2 < N1 || n2 >= N2 || n1 == n2) {
            continue;
          }

          double x12double = g_x[n2] - x1;
          double y12double = g_y[n2] - y1;
          double z12double = g_z[n2] - z1;
          apply_mic(box, x12double, y12double, z12double);
          float x12 = float(x12double), y12 = float(y12double), z12 = float(z12double);
          float d12_square = x12 * x12 + y12 * y12 + z12 * z12;

          int t2 = g_type[n2];
          float rc_radial = paramb.rc_radial;
          float rc_angular = paramb.rc_angular;
          if (paramb.use_typewise_cutoff) {
            int z1 = paramb.atomic_numbers[t1];
            int z2 = paramb.atomic_numbers[t2];
            rc_radial = min(
              (COVALENT_RADIUS[z1] + COVALENT_RADIUS[z2]) * paramb.typewise_cutoff_radial_factor,
              rc_radial);
            rc_angular = min(
              (COVALENT_RADIUS[z1] + COVALENT_RADIUS[z2]) * paramb.typewise_cutoff_angular_factor,
              rc_angular);
          }

          if (d12_square >= rc_radial * rc_radial) {
            continue;
          }

          g_NL_radial[count_radial++ * N + n1] = n2;

          if (d12_square < rc_angular * rc_angular) {
            g_NL_angular[count_angular++ * N + n1] = n2;
          }
        }
      }
    }
  }

  g_NN_radial[n1] = count_radial;
  g_NN_angular[n1] = count_angular;
}

static __global__ void find_descriptor(
  ILP_NEP_TMD::ParaMB paramb,
  ILP_NEP_TMD::ANN annmb,
  const int N,
  const int N1,
  const int N2,
  const Box box,
  const int* g_NN,
  const int* g_NL,
  const int* g_NN_angular,
  const int* g_NL_angular,
  const int* __restrict__ g_type,
  const double* __restrict__ g_x,
  const double* __restrict__ g_y,
  const double* __restrict__ g_z,
#ifdef USE_TABLE
  const float* __restrict__ g_gn_radial,
  const float* __restrict__ g_gn_angular,
#endif
  double* g_pe,
  float* g_Fp,
  double* g_virial,
  float* g_sum_fxyz)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1;
  if (n1 < N2) {
    int t1 = g_type[n1];
    double x1 = g_x[n1];
    double y1 = g_y[n1];
    double z1 = g_z[n1];
    float q[MAX_DIM] = {0.0f};

    // get radial descriptors
    for (int i1 = 0; i1 < g_NN[n1]; ++i1) {
      int n2 = g_NL[n1 + N * i1];
      double x12double = g_x[n2] - x1;
      double y12double = g_y[n2] - y1;
      double z12double = g_z[n2] - z1;
      apply_mic(box, x12double, y12double, z12double);
      float x12 = float(x12double), y12 = float(y12double), z12 = float(z12double);
      float d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12);

#ifdef USE_TABLE
      int index_left, index_right;
      float weight_left, weight_right;
      find_index_and_weight(
        d12 * paramb.rcinv_radial, index_left, index_right, weight_left, weight_right);
      int t12 = t1 * paramb.num_types + g_type[n2];
      for (int n = 0; n <= paramb.n_max_radial; ++n) {
        q[n] +=
          g_gn_radial[(index_left * paramb.num_types_sq + t12) * (paramb.n_max_radial + 1) + n] *
            weight_left +
          g_gn_radial[(index_right * paramb.num_types_sq + t12) * (paramb.n_max_radial + 1) + n] *
            weight_right;
      }
#else
      float fc12;
      int t2 = g_type[n2];
      float rc = paramb.rc_radial;
      if (paramb.use_typewise_cutoff) {
        rc = min(
          (COVALENT_RADIUS[paramb.atomic_numbers[t1]] +
           COVALENT_RADIUS[paramb.atomic_numbers[t2]]) *
            paramb.typewise_cutoff_radial_factor,
          rc);
      }
      float rcinv = 1.0f / rc;
      find_fc(rc, rcinv, d12, fc12);
      float fn12[MAX_NUM_N];

      find_fn(paramb.basis_size_radial, rcinv, d12, fc12, fn12);
      for (int n = 0; n <= paramb.n_max_radial; ++n) {
        float gn12 = 0.0f;
        for (int k = 0; k <= paramb.basis_size_radial; ++k) {
          int c_index = (n * (paramb.basis_size_radial + 1) + k) * paramb.num_types_sq;
          c_index += t1 * paramb.num_types + t2;
          gn12 += fn12[k] * annmb.c[c_index];
        }
        q[n] += gn12;
      }
#endif
    }

    // get angular descriptors
    for (int n = 0; n <= paramb.n_max_angular; ++n) {
      float s[NUM_OF_ABC] = {0.0f};
      for (int i1 = 0; i1 < g_NN_angular[n1]; ++i1) {
        int n2 = g_NL_angular[n1 + N * i1];
        double x12double = g_x[n2] - x1;
        double y12double = g_y[n2] - y1;
        double z12double = g_z[n2] - z1;
        apply_mic(box, x12double, y12double, z12double);
        float x12 = float(x12double), y12 = float(y12double), z12 = float(z12double);
        float d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12);
#ifdef USE_TABLE
        int index_left, index_right;
        float weight_left, weight_right;
        find_index_and_weight(
          d12 * paramb.rcinv_angular, index_left, index_right, weight_left, weight_right);
        int t12 = t1 * paramb.num_types + g_type[n2];
        float gn12 =
          g_gn_angular[(index_left * paramb.num_types_sq + t12) * (paramb.n_max_angular + 1) + n] *
            weight_left +
          g_gn_angular[(index_right * paramb.num_types_sq + t12) * (paramb.n_max_angular + 1) + n] *
            weight_right;
        accumulate_s(paramb.L_max, d12, x12, y12, z12, gn12, s);
#else
        float fc12;
        int t2 = g_type[n2];
        float rc = paramb.rc_angular;
        if (paramb.use_typewise_cutoff) {
          rc = min(
            (COVALENT_RADIUS[paramb.atomic_numbers[t1]] +
             COVALENT_RADIUS[paramb.atomic_numbers[t2]]) *
              paramb.typewise_cutoff_angular_factor,
            rc);
        }
        float rcinv = 1.0f / rc;
        find_fc(rc, rcinv, d12, fc12);
        float fn12[MAX_NUM_N];
        find_fn(paramb.basis_size_angular, rcinv, d12, fc12, fn12);
        float gn12 = 0.0f;
        for (int k = 0; k <= paramb.basis_size_angular; ++k) {
          int c_index = (n * (paramb.basis_size_angular + 1) + k) * paramb.num_types_sq;
          c_index += t1 * paramb.num_types + t2 + paramb.num_c_radial;
          gn12 += fn12[k] * annmb.c[c_index];
        }
        accumulate_s(paramb.L_max, d12, x12, y12, z12, gn12, s);
#endif
      }
      find_q(paramb.L_max, paramb.num_L, paramb.n_max_angular + 1, n, s, q + (paramb.n_max_radial + 1));
      for (int abc = 0; abc < NUM_OF_ABC; ++abc) {
        g_sum_fxyz[(n * NUM_OF_ABC + abc) * N + n1] = s[abc];
      }
    }

    // nomalize descriptor
    for (int d = 0; d < annmb.dim; ++d) {
      q[d] = q[d] * paramb.q_scaler[d];
    }

    // get energy and energy gradient
    float F = 0.0f, Fp[MAX_DIM] = {0.0f};


    if (paramb.version == 5) {
      apply_ann_one_layer_nep5(
        annmb.dim,
        annmb.num_neurons1,
        annmb.w0[t1],
        annmb.b0[t1],
        annmb.w1[t1],
        annmb.b1,
        q,
        F,
        Fp);
    } else {
      apply_ann_one_layer(
        annmb.dim,
        annmb.num_neurons1,
        annmb.w0[t1],
        annmb.b0[t1],
        annmb.w1[t1],
        annmb.b1,
        q,
        F,
        Fp);
    }
    g_pe[n1] += F;

    for (int d = 0; d < annmb.dim; ++d) {
      g_Fp[d * N + n1] = Fp[d] * paramb.q_scaler[d];
    }
  }
}

static __global__ void find_force_radial(
  ILP_NEP_TMD::ParaMB paramb,
  ILP_NEP_TMD::ANN annmb,
  const int N,
  const int N1,
  const int N2,
  const Box box,
  const int* g_NN,
  const int* g_NL,
  const int* __restrict__ g_type,
  const double* __restrict__ g_x,
  const double* __restrict__ g_y,
  const double* __restrict__ g_z,
  const float* __restrict__ g_Fp,
#ifdef USE_TABLE
  const float* __restrict__ g_gnp_radial,
#endif
  double* g_fx,
  double* g_fy,
  double* g_fz,
  double* g_virial)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1;
  if (n1 < N2) {
    int t1 = g_type[n1];
    float s_fx = 0.0f;
    float s_fy = 0.0f;
    float s_fz = 0.0f;
    float s_sxx = 0.0f;
    float s_sxy = 0.0f;
    float s_sxz = 0.0f;
    float s_syx = 0.0f;
    float s_syy = 0.0f;
    float s_syz = 0.0f;
    float s_szx = 0.0f;
    float s_szy = 0.0f;
    float s_szz = 0.0f;
    double x1 = g_x[n1];
    double y1 = g_y[n1];
    double z1 = g_z[n1];
    for (int i1 = 0; i1 < g_NN[n1]; ++i1) {
      int n2 = g_NL[n1 + N * i1];
      int t2 = g_type[n2];
      double x12double = g_x[n2] - x1;
      double y12double = g_y[n2] - y1;
      double z12double = g_z[n2] - z1;
      apply_mic(box, x12double, y12double, z12double);
      float r12[3] = {float(x12double), float(y12double), float(z12double)};
      float d12 = sqrt(r12[0] * r12[0] + r12[1] * r12[1] + r12[2] * r12[2]);
      float d12inv = 1.0f / d12;
      float f12[3] = {0.0f};
      float f21[3] = {0.0f};
#ifdef USE_TABLE
      int index_left, index_right;
      float weight_left, weight_right;
      find_index_and_weight(
        d12 * paramb.rcinv_radial, index_left, index_right, weight_left, weight_right);
      int t12 = t1 * paramb.num_types + t2;
      int t21 = t2 * paramb.num_types + t1;
      for (int n = 0; n <= paramb.n_max_radial; ++n) {
        float gnp12 =
          g_gnp_radial[(index_left * paramb.num_types_sq + t12) * (paramb.n_max_radial + 1) + n] *
            weight_left +
          g_gnp_radial[(index_right * paramb.num_types_sq + t12) * (paramb.n_max_radial + 1) + n] *
            weight_right;
        float gnp21 =
          g_gnp_radial[(index_left * paramb.num_types_sq + t21) * (paramb.n_max_radial + 1) + n] *
            weight_left +
          g_gnp_radial[(index_right * paramb.num_types_sq + t21) * (paramb.n_max_radial + 1) + n] *
            weight_right;
        float tmp12 = g_Fp[n1 + n * N] * gnp12 * d12inv;
        float tmp21 = g_Fp[n2 + n * N] * gnp21 * d12inv;
        for (int d = 0; d < 3; ++d) {
          f12[d] += tmp12 * r12[d];
          f21[d] -= tmp21 * r12[d];
        }
      }
#else
      float fc12, fcp12;
      float rc = paramb.rc_radial;
      if (paramb.use_typewise_cutoff) {
        rc = min(
          (COVALENT_RADIUS[paramb.atomic_numbers[t1]] +
           COVALENT_RADIUS[paramb.atomic_numbers[t2]]) *
            paramb.typewise_cutoff_radial_factor,
          rc);
      }
      float rcinv = 1.0f / rc;
      find_fc_and_fcp(rc, rcinv, d12, fc12, fcp12);
      float fn12[MAX_NUM_N];
      float fnp12[MAX_NUM_N];
      find_fn_and_fnp(paramb.basis_size_radial, rcinv, d12, fc12, fcp12, fn12, fnp12);
      for (int n = 0; n <= paramb.n_max_radial; ++n) {
        float gnp12 = 0.0f;
        float gnp21 = 0.0f;
        for (int k = 0; k <= paramb.basis_size_radial; ++k) {
          int c_index = (n * (paramb.basis_size_radial + 1) + k) * paramb.num_types_sq;
          gnp12 += fnp12[k] * annmb.c[c_index + t1 * paramb.num_types + t2];
          gnp21 += fnp12[k] * annmb.c[c_index + t2 * paramb.num_types + t1];
        }
        float tmp12 = g_Fp[n1 + n * N] * gnp12 * d12inv;
        float tmp21 = g_Fp[n2 + n * N] * gnp21 * d12inv;
        for (int d = 0; d < 3; ++d) {
          f12[d] += tmp12 * r12[d];
          f21[d] -= tmp21 * r12[d];
        }
      }
#endif
      s_fx += f12[0] - f21[0];
      s_fy += f12[1] - f21[1];
      s_fz += f12[2] - f21[2];

      s_sxx += r12[0] * f21[0];
      s_syy += r12[1] * f21[1];
      s_szz += r12[2] * f21[2];
      s_sxy += r12[0] * f21[1];
      s_sxz += r12[0] * f21[2];
      s_syx += r12[1] * f21[0];
      s_syz += r12[1] * f21[2];
      s_szx += r12[2] * f21[0];
      s_szy += r12[2] * f21[1];
    }
    g_fx[n1] += s_fx;
    g_fy[n1] += s_fy;
    g_fz[n1] += s_fz;
    // save virial
    // xx xy xz    0 3 4
    // yx yy yz    6 1 5
    // zx zy zz    7 8 2
    g_virial[n1 + 0 * N] += s_sxx;
    g_virial[n1 + 1 * N] += s_syy;
    g_virial[n1 + 2 * N] += s_szz;
    g_virial[n1 + 3 * N] += s_sxy;
    g_virial[n1 + 4 * N] += s_sxz;
    g_virial[n1 + 5 * N] += s_syz;
    g_virial[n1 + 6 * N] += s_syx;
    g_virial[n1 + 7 * N] += s_szx;
    g_virial[n1 + 8 * N] += s_szy;
  }
}

static __global__ void find_partial_force_angular(
  ILP_NEP_TMD::ParaMB paramb,
  ILP_NEP_TMD::ANN annmb,
  const int N,
  const int N1,
  const int N2,
  const Box box,
  const int* g_NN_angular,
  const int* g_NL_angular,
  const int* __restrict__ g_type,
  const double* __restrict__ g_x,
  const double* __restrict__ g_y,
  const double* __restrict__ g_z,
  const float* __restrict__ g_Fp,
  const float* __restrict__ g_sum_fxyz,
#ifdef USE_TABLE
  const float* __restrict__ g_gn_angular,
  const float* __restrict__ g_gnp_angular,
#endif
  float* g_f12x,
  float* g_f12y,
  float* g_f12z)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1;
  if (n1 < N2) {

    float Fp[MAX_DIM_ANGULAR] = {0.0f};
    float sum_fxyz[NUM_OF_ABC * MAX_NUM_N];
    for (int d = 0; d < paramb.dim_angular; ++d) {
      Fp[d] = g_Fp[(paramb.n_max_radial + 1 + d) * N + n1];
    }
    for (int d = 0; d < (paramb.n_max_angular + 1) * NUM_OF_ABC; ++d) {
      sum_fxyz[d] = g_sum_fxyz[d * N + n1];
    }

    int t1 = g_type[n1];
    double x1 = g_x[n1];
    double y1 = g_y[n1];
    double z1 = g_z[n1];
    for (int i1 = 0; i1 < g_NN_angular[n1]; ++i1) {
      int index = i1 * N + n1;
      int n2 = g_NL_angular[n1 + N * i1];
      double x12double = g_x[n2] - x1;
      double y12double = g_y[n2] - y1;
      double z12double = g_z[n2] - z1;
      apply_mic(box, x12double, y12double, z12double);
      float r12[3] = {float(x12double), float(y12double), float(z12double)};
      float d12 = sqrt(r12[0] * r12[0] + r12[1] * r12[1] + r12[2] * r12[2]);
      float f12[3] = {0.0f};
#ifdef USE_TABLE
      int index_left, index_right;
      float weight_left, weight_right;
      find_index_and_weight(
        d12 * paramb.rcinv_angular, index_left, index_right, weight_left, weight_right);
      int t12 = t1 * paramb.num_types + g_type[n2];
      for (int n = 0; n <= paramb.n_max_angular; ++n) {
        int index_left_all =
          (index_left * paramb.num_types_sq + t12) * (paramb.n_max_angular + 1) + n;
        int index_right_all =
          (index_right * paramb.num_types_sq + t12) * (paramb.n_max_angular + 1) + n;
        float gn12 =
          g_gn_angular[index_left_all] * weight_left + g_gn_angular[index_right_all] * weight_right;
        float gnp12 = g_gnp_angular[index_left_all] * weight_left +
                      g_gnp_angular[index_right_all] * weight_right;
        accumulate_f12(paramb.L_max, paramb.num_L, n, paramb.n_max_angular + 1, d12, r12, gn12, gnp12, Fp, sum_fxyz, f12);
      }
#else
      float fc12, fcp12;
      int t2 = g_type[n2];
      float rc = paramb.rc_angular;
      if (paramb.use_typewise_cutoff) {
        rc = min(
          (COVALENT_RADIUS[paramb.atomic_numbers[t1]] +
           COVALENT_RADIUS[paramb.atomic_numbers[t2]]) *
            paramb.typewise_cutoff_angular_factor,
          rc);
      }
      float rcinv = 1.0f / rc;
      find_fc_and_fcp(rc, rcinv, d12, fc12, fcp12);

      float fn12[MAX_NUM_N];
      float fnp12[MAX_NUM_N];
      find_fn_and_fnp(paramb.basis_size_angular, rcinv, d12, fc12, fcp12, fn12, fnp12);
      for (int n = 0; n <= paramb.n_max_angular; ++n) {
        float gn12 = 0.0f;
        float gnp12 = 0.0f;
        for (int k = 0; k <= paramb.basis_size_angular; ++k) {
          int c_index = (n * (paramb.basis_size_angular + 1) + k) * paramb.num_types_sq;
          c_index += t1 * paramb.num_types + t2 + paramb.num_c_radial;
          gn12 += fn12[k] * annmb.c[c_index];
          gnp12 += fnp12[k] * annmb.c[c_index];
        }
        accumulate_f12(paramb.L_max, paramb.num_L, n, paramb.n_max_angular + 1, d12, r12, gn12, gnp12, Fp, sum_fxyz, f12);
      }
#endif
      g_f12x[index] = f12[0];
      g_f12y[index] = f12[1];
      g_f12z[index] = f12[2];
    }
  }
}


//#define USE_FIXED_NEIGHBOR 1
#define UPDATE_TEMP 1
#define BIG_ILP_CUTOFF_SQUARE 50.0
// find force and related quantities
void ILP_NEP_TMD::compute_ilp(
  Box &box,
  const GPU_Vector<int> &type,
  const GPU_Vector<double> &position_per_atom,
  GPU_Vector<double> &potential_per_atom,
  GPU_Vector<double> &force_per_atom,
  GPU_Vector<double> &virial_per_atom,
  std::vector<Group> &group)
{
  const int number_of_atoms = type.size();
  int grid_size = (N2 - N1 - 1) / BLOCK_SIZE_FORCE + 1;

  // TODO: assume the first group column is for ILP
  const int *group_label_ilp = group[ilp_group_method].label.data();
  const int *sublayer_group_label_ilp = group[ilp_sub_group_method].label.data();

#ifdef USE_FIXED_NEIGHBOR
  static int num_calls = 0;
  if (num_calls++ == 0) {
#endif
    find_neighbor_ilp(
      N1,
      N2,
      rc_ilp,
      BIG_ILP_CUTOFF_SQUARE,
      box,
      group_label_ilp,
      type,
      position_per_atom,
      ilp_data.cell_count,
      ilp_data.cell_count_sum,
      ilp_data.cell_contents,
      ilp_data.NN,
      ilp_data.NL,
      ilp_data.big_ilp_NN,
      ilp_data.big_ilp_NL);

    build_reduce_neighbor_list<<<grid_size, BLOCK_SIZE_FORCE>>>(
      number_of_atoms,
      N1,
      N2,
      ilp_data.NN.data(),
      ilp_data.NL.data(),
      ilp_data.reduce_NL.data());
#ifdef USE_FIXED_NEIGHBOR
  }
  num_calls %= UPDATE_TEMP;
#endif

  const double* x = position_per_atom.data();
  const double* y = position_per_atom.data() + number_of_atoms;
  const double* z = position_per_atom.data() + number_of_atoms * 2;
  const int *NN = ilp_data.NN.data();
  const int *NL = ilp_data.NL.data();
  const int* big_ilp_NN = ilp_data.big_ilp_NN.data();
  const int* big_ilp_NL = ilp_data.big_ilp_NL.data();
  int *reduce_NL = ilp_data.reduce_NL.data();
  int *ilp_NL = ilp_data.ilp_NL.data();
  int *ilp_NN = ilp_data.ilp_NN.data();

  ilp_data.ilp_NL.fill(0);
  ilp_data.ilp_NN.fill(0);

  // find ILP neighbor list
  ILP_neighbor<<<grid_size, BLOCK_SIZE_FORCE>>>(
    number_of_atoms, N1, N2, box, big_ilp_NN, big_ilp_NL, \
    type.data(), ilp_para, x, y, z, ilp_NN, \
    ilp_NL, sublayer_group_label_ilp);
  GPU_CHECK_KERNEL

  // initialize force of ilp neighbor temporary vector
  ilp_data.f12x_ilp_neigh.fill(0);
  ilp_data.f12y_ilp_neigh.fill(0);
  ilp_data.f12z_ilp_neigh.fill(0);
  ilp_data.f12x.fill(0);
  ilp_data.f12y.fill(0);
  ilp_data.f12z.fill(0);

  double *g_fx = force_per_atom.data();
  double *g_fy = force_per_atom.data() + number_of_atoms;
  double *g_fz = force_per_atom.data() + number_of_atoms * 2;
  double *g_virial = virial_per_atom.data();
  double *g_potential = potential_per_atom.data();
  float *g_f12x = ilp_data.f12x.data();
  float *g_f12y = ilp_data.f12y.data();
  float *g_f12z = ilp_data.f12z.data();
  float *g_f12x_ilp_neigh = ilp_data.f12x_ilp_neigh.data();
  float *g_f12y_ilp_neigh = ilp_data.f12y_ilp_neigh.data();
  float *g_f12z_ilp_neigh = ilp_data.f12z_ilp_neigh.data();

  gpu_find_force<<<grid_size, BLOCK_SIZE_FORCE>>>(
    ilp_para,
    number_of_atoms,
    N1,
    N2,
    box,
    NN,
    NL,
    ilp_NN,
    ilp_NL,
    group_label_ilp,
    type.data(),
    x,
    y,
    z,
    g_fx,
    g_fy,
    g_fz,
    g_virial,
    g_potential,
    g_f12x,
    g_f12y,
    g_f12z,
    g_f12x_ilp_neigh,
    g_f12y_ilp_neigh,
    g_f12z_ilp_neigh);
  GPU_CHECK_KERNEL

  reduce_force_many_body<<<grid_size, BLOCK_SIZE_FORCE>>>(
    number_of_atoms,
    N1,
    N2,
    box,
    NN,
    NL,
    reduce_NL,
    ilp_NN,
    ilp_NL,
    x,
    y,
    z,
    g_fx,
    g_fy,
    g_fz,
    g_virial,
    g_f12x,
    g_f12y,
    g_f12z,
    g_f12x_ilp_neigh,
    g_f12y_ilp_neigh,
    g_f12z_ilp_neigh);
  GPU_CHECK_KERNEL



  // NEP term
  const int BLOCK_SIZE_nep = 64;
  const int N = type.size();
  const int grid_size_nep = (N2 - N1 - 1) / BLOCK_SIZE_nep + 1;

  const double rc_cell_list = 0.5 * rc;
  nep_data.f12x.fill(0);
  nep_data.f12y.fill(0);
  nep_data.f12z.fill(0);

  int num_bins[3];
  box.get_num_bins(rc_cell_list, num_bins);

  find_cell_list(
    rc_cell_list,
    num_bins,
    box,
    position_per_atom,
    nep_data.cell_count,
    nep_data.cell_count_sum,
    nep_data.cell_contents);

  find_neighbor_list_nep<<<grid_size_nep, BLOCK_SIZE_nep>>>(
    paramb,
    N,
    N1,
    N2,
    num_bins[0],
    num_bins[1],
    num_bins[2],
    box,
    type.data(),
    nep_data.cell_count.data(),
    nep_data.cell_count_sum.data(),
    nep_data.cell_contents.data(),
    position_per_atom.data(),
    position_per_atom.data() + N,
    position_per_atom.data() + N * 2,
    nep_data.NN_radial.data(),
    nep_data.NL_radial.data(),
    nep_data.NN_angular.data(),
    nep_data.NL_angular.data());
  GPU_CHECK_KERNEL

  static int num_calls_nei = 0;
  if (num_calls_nei++ % 1000 == 0) {
    nep_data.NN_radial.copy_to_host(nep_data.cpu_NN_radial.data());
    nep_data.NN_angular.copy_to_host(nep_data.cpu_NN_angular.data());
    int radial_actual = 0;
    int angular_actual = 0;
    for (int n = 0; n < N; ++n) {
      if (radial_actual < nep_data.cpu_NN_radial[n]) {
        radial_actual = nep_data.cpu_NN_radial[n];
      }
      if (angular_actual < nep_data.cpu_NN_angular[n]) {
        angular_actual = nep_data.cpu_NN_angular[n];
      }
    }
    std::ofstream output_file("neighbor.out", std::ios_base::app);
    output_file << "Neighbor info at step " << num_calls_nei - 1 << ": "
                << "radial(max=" << paramb.MN_radial << ",actual=" << radial_actual
                << "), angular(max=" << paramb.MN_angular << ",actual=" << angular_actual << ")."
                << std::endl;
    output_file.close();
  }

  gpu_sort_neighbor_list<<<N, paramb.MN_radial, paramb.MN_radial * sizeof(int)>>>(
    N, nep_data.NN_radial.data(), nep_data.NL_radial.data());
  GPU_CHECK_KERNEL

  gpu_sort_neighbor_list<<<N, paramb.MN_angular, paramb.MN_angular * sizeof(int)>>>(
    N, nep_data.NN_angular.data(), nep_data.NL_angular.data());
  GPU_CHECK_KERNEL

  find_descriptor<<<grid_size_nep, BLOCK_SIZE_nep>>>(
    paramb,
    annmb,
    N,
    N1,
    N2,
    box,
    nep_data.NN_radial.data(),
    nep_data.NL_radial.data(),
    nep_data.NN_angular.data(),
    nep_data.NL_angular.data(),
    type.data(),
    position_per_atom.data(),
    position_per_atom.data() + N,
    position_per_atom.data() + N * 2,
#ifdef USE_TABLE
    nep_data.gn_radial.data(),
    nep_data.gn_angular.data(),
#endif
    potential_per_atom.data(),
    nep_data.Fp.data(),
    virial_per_atom.data(),
    nep_data.sum_fxyz.data());
  GPU_CHECK_KERNEL

  bool is_dipole = paramb.model_type == 1;
  find_force_radial<<<grid_size_nep, BLOCK_SIZE_nep>>>(
    paramb,
    annmb,
    N,
    N1,
    N2,
    box,
    nep_data.NN_radial.data(),
    nep_data.NL_radial.data(),
    type.data(),
    position_per_atom.data(),
    position_per_atom.data() + N,
    position_per_atom.data() + N * 2,
    nep_data.Fp.data(),
#ifdef USE_TABLE
    nep_data.gnp_radial.data(),
#endif
    force_per_atom.data(),
    force_per_atom.data() + N,
    force_per_atom.data() + N * 2,
    virial_per_atom.data());
  GPU_CHECK_KERNEL

  find_partial_force_angular<<<grid_size_nep, BLOCK_SIZE_nep>>>(
    paramb,
    annmb,
    N,
    N1,
    N2,
    box,
    nep_data.NN_angular.data(),
    nep_data.NL_angular.data(),
    type.data(),
    position_per_atom.data(),
    position_per_atom.data() + N,
    position_per_atom.data() + N * 2,
    nep_data.Fp.data(),
    nep_data.sum_fxyz.data(),
#ifdef USE_TABLE
    nep_data.gn_angular.data(),
    nep_data.gnp_angular.data(),
#endif
    nep_data.f12x.data(),
    nep_data.f12y.data(),
    nep_data.f12z.data());
  GPU_CHECK_KERNEL

  find_properties_many_body(
    box,
    nep_data.NN_angular.data(),
    nep_data.NL_angular.data(),
    nep_data.f12x.data(),
    nep_data.f12y.data(),
    nep_data.f12z.data(),
    is_dipole,
    position_per_atom,
    force_per_atom,
    virial_per_atom);
  GPU_CHECK_KERNEL
}