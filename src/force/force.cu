#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

/*----------------------------------------------------------------------------80
The driver class calculating force and related quantities.
------------------------------------------------------------------------------*/

#include "eam.cuh"
#include "fcp.cuh"
#include "force.cuh"
#include "lj.cuh"
#include "nep.cuh"
#include "potential.cuh"
#include "rebo_mos2.cuh"
#include "ri.cuh"
#include "sw.cuh"
#include "tersoff1988.cuh"
#include "tersoff1989.cuh"
#include "tersoff_mini.cuh"
#include "utilities/error.cuh"
#include "utilities/read_file.cuh"
#include "vashishta.cuh"
#include <vector>

#define BLOCK_SIZE 128

Force::Force(void)
{
  num_of_potentials = 0;
  rc_max = 0.0;
  group_method = -1;
}

void Force::parse_potential(
  char** param,
  int num_param,
  char* input_dir,
  const Box& box,
  const Neighbor& neighbor,
  const std::vector<int>& cpu_type,
  const std::vector<int>& cpu_type_size)
{
  // check for at least the file path
  if (num_param < 2) {
    PRINT_INPUT_ERROR("potential should have at least 1 parameter.\n");
  }
  strcpy(file_potential[num_of_potentials], param[1]);

  // open file to check number of types used in potential
  char potential_name[20];
  FILE* fid_potential = my_fopen(file_potential[num_of_potentials], "r");
  int count = fscanf(fid_potential, "%s", potential_name);
  PRINT_SCANF_ERROR(count, 1, "Reading error for number of types.");
  int num_types = get_number_of_types(fid_potential);
  fclose(fid_potential);

  if (strcmp(potential_name, "lj") == 0) {
    is_lj[num_of_potentials] = true;
    if (num_param == 3) {
      if (!is_valid_int(param[2], &group_method)) {
        PRINT_INPUT_ERROR("Group method for LJ potential should be an integer.\n");
      }
    }
    atom_begin[num_of_potentials] = 0;
    atom_end[num_of_potentials] = num_types - 1;
  } else {
    is_lj[num_of_potentials] = false;
    if (num_param != num_types + 2) {
      PRINT_INPUT_ERROR("potential has incorrect number of types defined.\n");
    }

    std::vector<int> atom_type(num_types);

    for (int i = 0; i < num_types; i++) {
      if (!is_valid_int(param[i + 2], &atom_type[i])) {
        PRINT_INPUT_ERROR("type should be an integer.\n");
      }
      if (i != 0 && atom_type[i] < atom_type[i - 1]) {
        PRINT_INPUT_ERROR("potential types must be in ascending order.\n");
      }
    }
    atom_begin[num_of_potentials] = atom_type[0];
    atom_end[num_of_potentials] = atom_type[num_types - 1];

    if (atom_type[num_types - 1] - atom_type[0] + 1 > num_types) {
      PRINT_INPUT_ERROR("Error: types for one potential must be contiguous.\n");
    }
  }

  num_of_potentials++;

  add_potential(input_dir, box, neighbor, cpu_type, cpu_type_size);
}

int Force::get_number_of_types(FILE* fid_potential)
{
  int num_of_types;
  int count = fscanf(fid_potential, "%d", &num_of_types);
  PRINT_SCANF_ERROR(count, 1, "Reading error for number of types.");
  return num_of_types;
}

void Force::initialize_potential(
  char* input_dir,
  const Box& box,
  const Neighbor& neighbor,
  const std::vector<int>& cpu_type_size,
  const int m)
{
  const int number_of_atoms = neighbor.NN.size();
  FILE* fid_potential = my_fopen(file_potential[m], "r");
  char potential_name[20];
  int count = fscanf(fid_potential, "%s", potential_name);
  if (count != 1) {
    PRINT_INPUT_ERROR("reading error for potential file.");
  }

  int num_types = get_number_of_types(fid_potential);

  // determine the potential
  if (strcmp(potential_name, "tersoff_1989") == 0) {
    potential[m].reset(new Tersoff1989(fid_potential, num_types, neighbor));
  } else if (strcmp(potential_name, "tersoff_1988") == 0) {
    potential[m].reset(new Tersoff1988(fid_potential, num_types, neighbor));
  } else if (strcmp(potential_name, "tersoff_mini") == 0) {
    potential[m].reset(new Tersoff_mini(fid_potential, num_types, neighbor));
  } else if (strcmp(potential_name, "sw_1985") == 0) {
    potential[m].reset(new SW2(fid_potential, num_types, neighbor));
  } else if (strcmp(potential_name, "rebo_mos2") == 0) {
    potential[m].reset(new REBO_MOS(neighbor));
  } else if (strcmp(potential_name, "eam_zhou_2004") == 0) {
    potential[m].reset(new EAM(fid_potential, potential_name, num_types, number_of_atoms));
  } else if (strcmp(potential_name, "eam_dai_2006") == 0) {
    potential[m].reset(new EAM(fid_potential, potential_name, num_types, number_of_atoms));
  } else if (strcmp(potential_name, "vashishta") == 0) {
    potential[m].reset(new Vashishta(fid_potential, neighbor));
  } else if (strcmp(potential_name, "fcp") == 0) {
    potential[m].reset(new FCP(fid_potential, input_dir, number_of_atoms, box));
  } else if (strcmp(potential_name, "nep") == 0) {
    potential[m].reset(new NEP2(fid_potential, input_dir, num_types, neighbor));
  } else if (strcmp(potential_name, "lj") == 0) {
    potential[m].reset(new LJ(fid_potential, num_types));
  } else if (strcmp(potential_name, "ri") == 0) {
    potential[m].reset(new RI(fid_potential));
  } else {
    PRINT_INPUT_ERROR("illegal potential model.\n");
  }

  potential[m]->N1 = 0;
  potential[m]->N2 = 0;

  for (int n = 0; n < atom_begin[m]; ++n) {
    potential[m]->N1 += cpu_type_size[n];
  }
  for (int n = 0; n <= atom_end[m]; ++n) {
    potential[m]->N2 += cpu_type_size[n];
  }

  printf(
    "    applies to atoms [%d, %d) from type %d to type %d.\n", potential[m]->N1, potential[m]->N2,
    atom_begin[m], atom_end[m]);

  fclose(fid_potential);
}

void Force::add_potential(
  char* input_dir,
  const Box& box,
  const Neighbor& neighbor,
  const std::vector<int>& cpu_type,
  const std::vector<int>& cpu_type_size)
{
  int m = num_of_potentials - 1; // current potential ID
  initialize_potential(input_dir, box, neighbor, cpu_type_size, m);

  if (rc_max < potential[m]->rc)
    rc_max = potential[m]->rc;

  // check the atom types in xyz.in
  for (int n = potential[m]->N1; n < potential[m]->N2; ++n) {
    if (cpu_type[n] < atom_begin[m] || cpu_type[n] > atom_end[m]) {
      printf("ERROR: type for potential # %d not from %d to %d.", m, atom_begin[m], atom_end[m]);
      exit(1);
    }
  }
  type_shift_[m] = atom_begin[m];
}

// Construct the local neighbor list from the global one (Kernel)
static __global__ void gpu_find_neighbor_local(
  const bool use_group,
  int* group_label,
  Box box,
  int type_begin,
  int type_end,
  int* type,
  int N,
  int N1,
  int N2,
  double cutoff_square,
  int* NN,
  int* NL,
  int* NN_local,
  int* NL_local,
  const double* __restrict__ x,
  const double* __restrict__ y,
  const double* __restrict__ z)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1;
  int count = 0;
  int layer_n1;

  if (n1 >= N1 && n1 < N2) {
    int neighbor_number = NN[n1];
    if (use_group)
      layer_n1 = group_label[n1];
    double x1 = x[n1];
    double y1 = y[n1];
    double z1 = z[n1];
    for (int i1 = 0; i1 < neighbor_number; ++i1) {
      int n2 = NL[n1 + N * i1];

      if (use_group) {
        if (layer_n1 == group_label[n2])
          continue;
      }

      // only include neighbors with the correct types
      int type_n2 = type[n2];
      if (type_n2 < type_begin || type_n2 > type_end)
        continue;

      double x12 = x[n2] - x1;
      double y12 = y[n2] - y1;
      double z12 = z[n2] - z1;
      apply_mic(box, x12, y12, z12);
      double distance_square = x12 * x12 + y12 * y12 + z12 * z12;
      if (distance_square < cutoff_square) {
        NL_local[count * N + n1] = n2;
        ++count;
      }
    }
    NN_local[n1] = count;
  }
}

// Construct the local neighbor list from the global one (Wrapper)
void Force::find_neighbor_local(
  const int m,
  std::vector<Group>& group,
  GPU_Vector<int>& atom_type,
  const GPU_Vector<double>& position_per_atom,
  const Box& box,
  Neighbor& neighbor)
{
  const int number_of_atoms = neighbor.NN.size();
  int grid_size = (potential[m]->N2 - potential[m]->N1 - 1) / 128 + 1;

  const bool use_group = is_lj[m] && (group_method > -1);
  int* group_label = nullptr;
  if (use_group)
    group_label = group[group_method].label.data();

  gpu_find_neighbor_local<<<grid_size, 128>>>(
    use_group, group_label, box, atom_begin[m], atom_end[m], atom_type.data(), number_of_atoms,
    potential[m]->N1, potential[m]->N2, potential[m]->rc * potential[m]->rc, neighbor.NN.data(),
    neighbor.NL.data(), neighbor.NN_local.data(), neighbor.NL_local.data(),
    position_per_atom.data(), position_per_atom.data() + number_of_atoms,
    position_per_atom.data() + number_of_atoms * 2);
  CUDA_CHECK_KERNEL
}

static __global__ void gpu_add_driving_force(
  int N,
  double fe_x,
  double fe_y,
  double fe_z,
  double* g_sxx,
  double* g_sxy,
  double* g_sxz,
  double* g_syx,
  double* g_syy,
  double* g_syz,
  double* g_szx,
  double* g_szy,
  double* g_szz,
  double* g_fx,
  double* g_fy,
  double* g_fz)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) {
    g_fx[i] += fe_x * g_sxx[i] + fe_y * g_syx[i] + fe_z * g_szx[i];
    g_fy[i] += fe_x * g_sxy[i] + fe_y * g_syy[i] + fe_z * g_szy[i];
    g_fz[i] += fe_x * g_sxz[i] + fe_y * g_syz[i] + fe_z * g_szz[i];
  }
}

// get the total force
static __global__ void gpu_sum_force(int N, double* g_fx, double* g_fy, double* g_fz, double* g_f)
{
  //<<<3, 1024>>>
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int number_of_patches = (N - 1) / 1024 + 1;
  __shared__ double s_f[1024];
  double f = 0.0;

  switch (bid) {
    case 0:
      for (int patch = 0; patch < number_of_patches; ++patch) {
        int n = tid + patch * 1024;
        if (n < N)
          f += g_fx[n];
      }
      break;
    case 1:
      for (int patch = 0; patch < number_of_patches; ++patch) {
        int n = tid + patch * 1024;
        if (n < N)
          f += g_fy[n];
      }
      break;
    case 2:
      for (int patch = 0; patch < number_of_patches; ++patch) {
        int n = tid + patch * 1024;
        if (n < N)
          f += g_fz[n];
      }
      break;
  }
  s_f[tid] = f;
  __syncthreads();

#pragma unroll
  for (int offset = blockDim.x >> 1; offset > 32; offset >>= 1) {
    if (tid < offset) {
      s_f[tid] += s_f[tid + offset];
    }
    __syncthreads();
  }
  for (int offset = 32; offset > 0; offset >>= 1) {
    if (tid < offset) {
      s_f[tid] += s_f[tid + offset];
    }
    __syncwarp();
  }

  if (tid == 0) {
    g_f[bid] = s_f[0];
  }
}

// correct the total force
static __global__ void
gpu_correct_force(int N, double one_over_N, double* g_fx, double* g_fy, double* g_fz, double* g_f)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) {
    g_fx[i] -= g_f[0] * one_over_N;
    g_fy[i] -= g_f[1] * one_over_N;
    g_fz[i] -= g_f[2] * one_over_N;
  }
}

static __global__ void initialize_properties(
  int N, double* g_fx, double* g_fy, double* g_fz, double* g_pe, double* g_virial)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x;
  if (n1 < N) {
    g_fx[n1] = 0.0;
    g_fy[n1] = 0.0;
    g_fz[n1] = 0.0;
    g_pe[n1] = 0.0;
    g_virial[n1 + 0 * N] = 0.0;
    g_virial[n1 + 1 * N] = 0.0;
    g_virial[n1 + 2 * N] = 0.0;
    g_virial[n1 + 3 * N] = 0.0;
    g_virial[n1 + 4 * N] = 0.0;
    g_virial[n1 + 5 * N] = 0.0;
    g_virial[n1 + 6 * N] = 0.0;
    g_virial[n1 + 7 * N] = 0.0;
    g_virial[n1 + 8 * N] = 0.0;
  }
}

void Force::set_hnemd_parameters(
  const bool compute_hnemd,
  const double hnemd_fe_x,
  const double hnemd_fe_y,
  const double hnemd_fe_z)
{
  compute_hnemd_ = compute_hnemd;
  if (compute_hnemd) {
    hnemd_fe_[0] = hnemd_fe_x;
    hnemd_fe_[1] = hnemd_fe_y;
    hnemd_fe_[2] = hnemd_fe_z;
  }
}

void Force::compute(
  const Box& box,
  const GPU_Vector<double>& position_per_atom,
  GPU_Vector<int>& type,
  std::vector<Group>& group,
  Neighbor& neighbor,
  GPU_Vector<double>& potential_per_atom,
  GPU_Vector<double>& force_per_atom,
  GPU_Vector<double>& virial_per_atom)
{
  const int number_of_atoms = type.size();

  initialize_properties<<<(number_of_atoms - 1) / 128 + 1, 128>>>(
    number_of_atoms, force_per_atom.data(), force_per_atom.data() + number_of_atoms,
    force_per_atom.data() + number_of_atoms * 2, potential_per_atom.data(), virial_per_atom.data());
  CUDA_CHECK_KERNEL

  for (int m = 0; m < num_of_potentials; m++) {
    // first build a local neighbor list
#ifndef USE_FCP // the FCP does not use a neighbor list at all
    find_neighbor_local(m, group, type, position_per_atom, box, neighbor);
#endif
    // and then calculate the forces and related quantities
    potential[m]->compute(
      type_shift_[m], box, neighbor, type, position_per_atom, potential_per_atom, force_per_atom,
      virial_per_atom);
  }

  if (compute_hnemd_) {
    // the virial tensor:
    // xx xy xz    0 3 4
    // yx yy yz    6 1 5
    // zx zy zz    7 8 2
    gpu_add_driving_force<<<(number_of_atoms - 1) / 128 + 1, 128>>>(
      number_of_atoms, hnemd_fe_[0], hnemd_fe_[1], hnemd_fe_[2],
      virial_per_atom.data() + 0 * number_of_atoms, virial_per_atom.data() + 3 * number_of_atoms,
      virial_per_atom.data() + 4 * number_of_atoms, virial_per_atom.data() + 6 * number_of_atoms,
      virial_per_atom.data() + 1 * number_of_atoms, virial_per_atom.data() + 5 * number_of_atoms,
      virial_per_atom.data() + 7 * number_of_atoms, virial_per_atom.data() + 8 * number_of_atoms,
      virial_per_atom.data() + 2 * number_of_atoms, force_per_atom.data(),
      force_per_atom.data() + number_of_atoms, force_per_atom.data() + 2 * number_of_atoms);

    GPU_Vector<double> ftot(3); // total force vector of the system

    gpu_sum_force<<<3, 1024>>>(
      number_of_atoms, force_per_atom.data(), force_per_atom.data() + number_of_atoms,
      force_per_atom.data() + 2 * number_of_atoms, ftot.data());
    CUDA_CHECK_KERNEL

    gpu_correct_force<<<(number_of_atoms - 1) / 128 + 1, 128>>>(
      number_of_atoms, 1.0 / number_of_atoms, force_per_atom.data(),
      force_per_atom.data() + number_of_atoms, force_per_atom.data() + 2 * number_of_atoms,
      ftot.data());
    CUDA_CHECK_KERNEL
  }

  // always correct the force when using the FCP potential
#ifdef USE_FCP
  if (!compute_hnemd_) {
    GPU_Vector<double> ftot(3); // total force vector of the system
    gpu_sum_force<<<3, 1024>>>(
      number_of_atoms, force_per_atom.data(), force_per_atom.data() + number_of_atoms,
      force_per_atom.data() + 2 * number_of_atoms, ftot.data());
    CUDA_CHECK_KERNEL

    gpu_correct_force<<<(number_of_atoms - 1) / 128 + 1, 128>>>(
      number_of_atoms, 1.0 / number_of_atoms, force_per_atom.data(),
      force_per_atom.data() + number_of_atoms, force_per_atom.data() + 2 * number_of_atoms,
      ftot.data());
    CUDA_CHECK_KERNEL
  }
#endif
}
