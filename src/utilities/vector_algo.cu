#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

#include "vector_algo.cuh"

__global__ void gpu_multiply(const int size, double a, double* b, double* c)
{
  int n = blockDim.x * blockIdx.x + threadIdx.x;
  if (n < size)
    c[n] = b[n] * a;
}

__global__ void gpu_vector_sum(const int size, double* a, double* b, double* c)
{
  int n = blockDim.x * blockIdx.x + threadIdx.x;
  if (n < size)
    c[n] = a[n] + b[n];
}

__global__ void gpu_pairwise_product(const int size, double* a, double* b, double* c)
{
  int n = blockDim.x * blockIdx.x + threadIdx.x;
  if (n < size)
    c[n] = a[n] * b[n];
}

void pairwise_product(GPU_Vector<double>& a, GPU_Vector<double>& b, GPU_Vector<double>& c)
{
  int size = a.size();
  gpu_pairwise_product<<<(size - 1) / 128 + 1, 128>>>(size, a.data(), b.data(), c.data());
}

__global__ void gpu_sum(const int size, double* a, double* result)
{
  int number_of_patches = (size - 1) / 1024 + 1;
  int tid = threadIdx.x;
  int n, patch;
  __shared__ double data[1024];
  data[tid] = 0.0;
  for (patch = 0; patch < number_of_patches; ++patch) {
    n = tid + patch * 1024;
    if (n < size)
      data[tid] += a[n];
  }
  __syncthreads();
  for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1) {
    if (tid < offset) {
      data[tid] += data[tid + offset];
    }
    __syncthreads();
  }
  if (tid == 0)
    *result = data[0];
}

double sum(GPU_Vector<double>& a)
{
  double ret;
  GPU_Vector<double> result(1);
  gpu_sum<<<1, 1024>>>(a.size(), a.data(), result.data());
  result.copy_to_host(&ret);
  return ret;
}

double dot(GPU_Vector<double>& a, GPU_Vector<double>& b)
{
  GPU_Vector<double> temp(a.size());
  pairwise_product(a, b, temp);
  CHECK(hipDeviceSynchronize());
  return sum(temp);
}

void scalar_multiply(const double& a, GPU_Vector<double>& b, GPU_Vector<double>& c)
{
  int size = b.size();
  gpu_multiply<<<(size - 1) / 128 + 1, 128>>>(size, a, b.data(), c.data());
}

void vector_sum(GPU_Vector<double>& a, GPU_Vector<double>& b, GPU_Vector<double>& c)
{
  int size = a.size();
  gpu_vector_sum<<<(size - 1) / 128 + 1, 128>>>(size, a.data(), b.data(), c.data());
}