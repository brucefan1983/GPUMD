#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan and GPUMD development team
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

/*----------------------------------------------------------------------------80
The neuroevolution potential (NEP)
Ref: Zheyong Fan et al., Neuroevolution machine learning potentials:
Combining high accuracy and low cost in atomistic simulations and application to
heat transport, Phys. Rev. B. 104, 104309 (2021).
------------------------------------------------------------------------------*/

#include "nep_energy.cuh"
#include "utilities/common.cuh"
#include "utilities/error.cuh"
#include "utilities/gpu_macro.cuh"
#include "utilities/nep_utilities.cuh"
#include <fstream>
#include <iostream>
#include <string>
#include <vector>
#include <cstring>

const std::string ELEMENTS[NUM_ELEMENTS] = {
  "H",  "He", "Li", "Be", "B",  "C",  "N",  "O",  "F",  "Ne", "Na", "Mg", "Al", "Si", "P",  "S",
  "Cl", "Ar", "K",  "Ca", "Sc", "Ti", "V",  "Cr", "Mn", "Fe", "Co", "Ni", "Cu", "Zn", "Ga", "Ge",
  "As", "Se", "Br", "Kr", "Rb", "Sr", "Y",  "Zr", "Nb", "Mo", "Tc", "Ru", "Rh", "Pd", "Ag", "Cd",
  "In", "Sn", "Sb", "Te", "I",  "Xe", "Cs", "Ba", "La", "Ce", "Pr", "Nd", "Pm", "Sm", "Eu", "Gd",
  "Tb", "Dy", "Ho", "Er", "Tm", "Yb", "Lu", "Hf", "Ta", "W",  "Re", "Os", "Ir", "Pt", "Au", "Hg",
  "Tl", "Pb", "Bi", "Po", "At", "Rn", "Fr", "Ra", "Ac", "Th", "Pa", "U",  "Np", "Pu"};

void NEP_Energy::initialize(const char* file_potential)
{

  std::ifstream input(file_potential);
  if (!input.is_open()) {
    std::cout << "Failed to open " << file_potential << std::endl;
    exit(1);
  }

  // nep3 1 C
  std::vector<std::string> tokens = get_tokens(input);
  if (tokens.size() < 3) {
    std::cout << "The first line of nep.txt should have at least 3 items." << std::endl;
    exit(1);
  }
  if (tokens[0] == "nep3") {
    paramb.version = 3;
    zbl.enabled = false;
  } else if (tokens[0] == "nep3_zbl") {
    paramb.version = 3;
    zbl.enabled = true;
  } else if (tokens[0] == "nep4") {
    paramb.version = 4;
    zbl.enabled = false;
  } else if (tokens[0] == "nep4_zbl") {
    paramb.version = 4;
    zbl.enabled = true;
  } else if (tokens[0] == "nep5") {
    paramb.version = 5;
    zbl.enabled = false;
  } else if (tokens[0] == "nep5_zbl") {
    paramb.version = 5;
    zbl.enabled = true;
  } else {
    std::cout << tokens[0]
              << " is an unsupported NEP model. We only support NEP3 and NEP4 models now."
              << std::endl;
    exit(1);
  }
  paramb.num_types = get_int_from_token(tokens[1], __FILE__, __LINE__);
  if (tokens.size() != 2 + paramb.num_types) {
    std::cout << "The first line of nep.txt should have " << paramb.num_types << " atom symbols."
              << std::endl;
    exit(1);
  }

  if (paramb.num_types == 1) {
    printf("    Use the NEP%d potential with %d atom type.\n", paramb.version, paramb.num_types);
  } else {
    printf("    Use the NEP%d potential with %d atom types.\n", paramb.version, paramb.num_types);
  }

  for (int n = 0; n < paramb.num_types; ++n) {
    int atomic_number = 0;
    for (int m = 0; m < NUM_ELEMENTS; ++m) {
      if (tokens[2 + n] == ELEMENTS[m]) {
        atomic_number = m + 1;
        break;
      }
    }
    zbl.atomic_numbers[n] = atomic_number;
    paramb.atomic_numbers[n] = atomic_number - 1;
    printf("        type %d (%s with Z = %d).\n", n, tokens[2 + n].c_str(), zbl.atomic_numbers[n]);
  }

  // zbl 0.7 1.4
  if (zbl.enabled) {
    tokens = get_tokens(input);
    if (tokens.size() != 3) {
      std::cout << "This line should be zbl rc_inner rc_outer." << std::endl;
      exit(1);
    }
    zbl.rc_inner = get_double_from_token(tokens[1], __FILE__, __LINE__);
    zbl.rc_outer = get_double_from_token(tokens[2], __FILE__, __LINE__);
    if (zbl.rc_inner == 0 && zbl.rc_outer == 0) {
      zbl.flexibled = true;
      printf("        has the flexible ZBL potential\n");
    } else {
      printf(
        "        has the universal ZBL with inner cutoff %g A and outer cutoff %g A.\n",
        zbl.rc_inner,
        zbl.rc_outer);
    }
  }

  // cutoff 4.2 3.7 80 47
  tokens = get_tokens(input);
  if (tokens.size() != 5 && tokens.size() != 8) {
    std::cout << "This line should be cutoff rc_radial rc_angular MN_radial MN_angular "
                 "[radial_factor] [angular_factor] [zbl_factor].\n";
    exit(1);
  }
  paramb.rc_radial = get_double_from_token(tokens[1], __FILE__, __LINE__);
  paramb.rc_angular = get_double_from_token(tokens[2], __FILE__, __LINE__);
  printf("        radial cutoff = %g A.\n", paramb.rc_radial);
  printf("        angular cutoff = %g A.\n", paramb.rc_angular);

  int MN_radial = get_int_from_token(tokens[3], __FILE__, __LINE__);
  int MN_angular = get_int_from_token(tokens[4], __FILE__, __LINE__);
  printf("        MN_radial = %d.\n", MN_radial);
  printf("        MN_angular = %d.\n", MN_angular);
  paramb.MN_radial = int(ceil(MN_radial * 1.25));
  paramb.MN_angular = int(ceil(MN_angular * 1.25));
  printf("        enlarged MN_radial = %d.\n", paramb.MN_radial);
  printf("        enlarged MN_angular = %d.\n", paramb.MN_angular);

  if (tokens.size() == 8) {
    paramb.typewise_cutoff_radial_factor = get_double_from_token(tokens[5], __FILE__, __LINE__);
    paramb.typewise_cutoff_angular_factor = get_double_from_token(tokens[6], __FILE__, __LINE__);
    paramb.typewise_cutoff_zbl_factor = get_double_from_token(tokens[7], __FILE__, __LINE__);
    if (paramb.typewise_cutoff_radial_factor > 0.0f) {
      paramb.use_typewise_cutoff = true;
    }
    if (paramb.typewise_cutoff_zbl_factor > 0.0f) {
      paramb.use_typewise_cutoff_zbl = true;
    }
  }

  // n_max 10 8
  tokens = get_tokens(input);
  if (tokens.size() != 3) {
    std::cout << "This line should be n_max n_max_radial n_max_angular." << std::endl;
    exit(1);
  }
  paramb.n_max_radial = get_int_from_token(tokens[1], __FILE__, __LINE__);
  paramb.n_max_angular = get_int_from_token(tokens[2], __FILE__, __LINE__);
  printf("        n_max_radial = %d.\n", paramb.n_max_radial);
  printf("        n_max_angular = %d.\n", paramb.n_max_angular);

  // basis_size 10 8
  tokens = get_tokens(input);
  if (tokens.size() != 3) {
    std::cout << "This line should be basis_size basis_size_radial basis_size_angular."
              << std::endl;
    exit(1);
  }
  paramb.basis_size_radial = get_int_from_token(tokens[1], __FILE__, __LINE__);
  paramb.basis_size_angular = get_int_from_token(tokens[2], __FILE__, __LINE__);
  printf("        basis_size_radial = %d.\n", paramb.basis_size_radial);
  printf("        basis_size_angular = %d.\n", paramb.basis_size_angular);

  // l_max
  tokens = get_tokens(input);
  if (tokens.size() != 4) {
    std::cout << "This line should be l_max l_max_3body l_max_4body l_max_5body." << std::endl;
    exit(1);
  }

  paramb.L_max = get_int_from_token(tokens[1], __FILE__, __LINE__);
  printf("        l_max_3body = %d.\n", paramb.L_max);
  paramb.num_L = paramb.L_max;

  int L_max_4body = get_int_from_token(tokens[2], __FILE__, __LINE__);
  int L_max_5body = get_int_from_token(tokens[3], __FILE__, __LINE__);
  printf("        l_max_4body = %d.\n", L_max_4body);
  printf("        l_max_5body = %d.\n", L_max_5body);
  if (L_max_4body == 2) {
    paramb.num_L += 1;
  }
  if (L_max_5body == 1) {
    paramb.num_L += 1;
  }

  paramb.dim_angular = (paramb.n_max_angular + 1) * paramb.num_L;

  // ANN
  tokens = get_tokens(input);
  if (tokens.size() != 3) {
    std::cout << "This line should be ANN num_neurons 0." << std::endl;
    exit(1);
  }
  annmb.num_neurons1 = get_int_from_token(tokens[1], __FILE__, __LINE__);
  annmb.dim = (paramb.n_max_radial + 1) + paramb.dim_angular;
  printf("        ANN = %d-%d-1.\n", annmb.dim, annmb.num_neurons1);

  // calculated parameters:
  paramb.rcinv_radial = 1.0f / paramb.rc_radial;
  paramb.rcinv_angular = 1.0f / paramb.rc_angular;
  paramb.num_types_sq = paramb.num_types * paramb.num_types;

  if (paramb.version == 3) {
    annmb.num_para = (annmb.dim + 2) * annmb.num_neurons1 + 1;
  } else if (paramb.version == 4) {
    annmb.num_para = (annmb.dim + 2) * annmb.num_neurons1 * paramb.num_types + 1;
  } else {
    annmb.num_para = ((annmb.dim + 2) * annmb.num_neurons1 + 1) * paramb.num_types + 1;
  }

  printf("        number of neural network parameters = %d.\n", annmb.num_para);
  int num_para_descriptor =
    paramb.num_types_sq * ((paramb.n_max_radial + 1) * (paramb.basis_size_radial + 1) +
                           (paramb.n_max_angular + 1) * (paramb.basis_size_angular + 1));
  printf("        number of descriptor parameters = %d.\n", num_para_descriptor);
  annmb.num_para += num_para_descriptor;
  printf("        total number of parameters = %d.\n", annmb.num_para);

  paramb.num_c_radial =
    paramb.num_types_sq * (paramb.n_max_radial + 1) * (paramb.basis_size_radial + 1);

  // NN and descriptor parameters
  std::vector<float> parameters(annmb.num_para);
  for (int n = 0; n < annmb.num_para; ++n) {
    tokens = get_tokens(input);
    parameters[n] = get_double_from_token(tokens[0], __FILE__, __LINE__);
  }
  nep_parameters.resize(annmb.num_para);
  nep_parameters.copy_from_host(parameters.data());
  update_potential(nep_parameters.data(), annmb);
  for (int d = 0; d < annmb.dim; ++d) {
    tokens = get_tokens(input);
    paramb.q_scaler[d] = get_double_from_token(tokens[0], __FILE__, __LINE__);
  }

  // flexible zbl potential parameters
  if (zbl.flexibled) {
    int num_type_zbl = (paramb.num_types * (paramb.num_types + 1)) / 2;
    for (int d = 0; d < 10 * num_type_zbl; ++d) {
      tokens = get_tokens(input);
      zbl.para[d] = get_double_from_token(tokens[0], __FILE__, __LINE__);
    }
    zbl.num_types = paramb.num_types;
  }
}

NEP_Energy::NEP_Energy(void)
{
  // nothing
}

NEP_Energy::~NEP_Energy(void)
{
  // nothing
}

void NEP_Energy::update_potential(float* parameters, ANN& ann)
{
  float* pointer = parameters;
  for (int t = 0; t < paramb.num_types; ++t) {
    if (t > 0 && paramb.version == 3) { // Use the same set of NN parameters for NEP3
      pointer -= (ann.dim + 2) * ann.num_neurons1;
    }
    ann.w0[t] = pointer;
    pointer += ann.num_neurons1 * ann.dim;
    ann.b0[t] = pointer;
    pointer += ann.num_neurons1;
    ann.w1[t] = pointer;
    pointer += ann.num_neurons1;
    if (paramb.version == 5) {
      pointer += 1; // one extra bias for NEP5 stored in ann.w1[t]
    }
  }
  ann.b1 = pointer;
  ann.c = ann.b1 + 1;
}

static __global__ void find_energy_nep(
  NEP_Energy::ParaMB paramb,
  NEP_Energy::ANN annmb,
  const int N,
  const int* g_NN_radial,
  const int* g_NN_angular,
  const int* __restrict__ g_type,
  const int* __restrict__ g_t2_radial,
  const int* __restrict__ g_t2_angular,
  const float* __restrict__ g_x12_radial,
  const float* __restrict__ g_y12_radial,
  const float* __restrict__ g_z12_radial,
  const float* __restrict__ g_x12_angular,
  const float* __restrict__ g_y12_angular,
  const float* __restrict__ g_z12_angular,
  float* g_pe)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x;
  if (n1 < N) {
    int t1 = g_type[n1];
    float q[MAX_DIM] = {0.0f};

    // get radial descriptors
    for (int i1 = 0; i1 < g_NN_radial[n1]; ++i1) {
      int index = i1 * N + n1;
      float r12[3] = {g_x12_radial[index], g_y12_radial[index], g_z12_radial[index]};
      float d12 = sqrt(r12[0] * r12[0] + r12[1] * r12[1] + r12[2] * r12[2]);
      float fc12;
      int t2 = g_t2_radial[index];
      double rc = paramb.rc_radial;
      double rcinv = paramb.rcinv_radial;
      if (paramb.use_typewise_cutoff) {
        rc = min(
          (COVALENT_RADIUS[paramb.atomic_numbers[t1]] +
           COVALENT_RADIUS[paramb.atomic_numbers[t2]]) *
            paramb.typewise_cutoff_radial_factor,
          rc);
        rcinv = 1.0f / rc;
      }
      find_fc(rc, rcinv, d12, fc12);

      float fn12[MAX_NUM_N];
      find_fn(paramb.basis_size_radial, rcinv, d12, fc12, fn12);
      for (int n = 0; n <= paramb.n_max_radial; ++n) {
        float gn12 = 0.0f;
        for (int k = 0; k <= paramb.basis_size_radial; ++k) {
          int c_index = (n * (paramb.basis_size_radial + 1) + k) * paramb.num_types_sq;
          c_index += t1 * paramb.num_types + t2;
          gn12 += fn12[k] * annmb.c[c_index];
        }
        q[n] += gn12;
      }
    }

    // get angular descriptors
    for (int n = 0; n <= paramb.n_max_angular; ++n) {
      float s[NUM_OF_ABC] = {0.0f};
      for (int i1 = 0; i1 < g_NN_angular[n1]; ++i1) {
        int index = i1 * N + n1;
        float r12[3] = {g_x12_angular[index], g_y12_angular[index], g_z12_angular[index]};
        float d12 = sqrt(r12[0] * r12[0] + r12[1] * r12[1] + r12[2] * r12[2]);
        float fc12;
        int t2 = g_t2_angular[index];
        double rc = paramb.rc_angular;
        double rcinv = paramb.rcinv_angular;
        if (paramb.use_typewise_cutoff) {
          rc = min(
            (COVALENT_RADIUS[paramb.atomic_numbers[t1]] +
             COVALENT_RADIUS[paramb.atomic_numbers[t2]]) *
              paramb.typewise_cutoff_angular_factor,
            rc);
          rcinv = 1.0f / rc;
        }
        find_fc(rc, rcinv, d12, fc12);

        float fn12[MAX_NUM_N];
        find_fn(paramb.basis_size_angular, rcinv, d12, fc12, fn12);
        float gn12 = 0.0f;
        for (int k = 0; k <= paramb.basis_size_angular; ++k) {
          int c_index = (n * (paramb.basis_size_angular + 1) + k) * paramb.num_types_sq;
          c_index += t1 * paramb.num_types + t2 + paramb.num_c_radial;
          gn12 += fn12[k] * annmb.c[c_index];
        }
        accumulate_s(paramb.L_max, d12, r12[0], r12[1], r12[2], gn12, s);
      }
      find_q(paramb.L_max, paramb.num_L, paramb.n_max_angular + 1, n, s, q + (paramb.n_max_radial + 1));
    }

    // nomalize descriptor
    for (int d = 0; d < annmb.dim; ++d) {
      q[d] = q[d] * paramb.q_scaler[d];
    }

    // get energy and energy gradient
    float F = 0.0f, Fp[MAX_DIM] = {0.0f};
    if (paramb.version == 5) {
      apply_ann_one_layer_nep5(
        annmb.dim, annmb.num_neurons1, annmb.w0[t1], annmb.b0[t1], annmb.w1[t1], annmb.b1, q, F, Fp);
    } else {
      apply_ann_one_layer(
        annmb.dim, annmb.num_neurons1, annmb.w0[t1], annmb.b0[t1], annmb.w1[t1], annmb.b1, q, F, Fp);
    }
    g_pe[n1] = F;
  }
}

static __global__ void find_energy_zbl(
  const int N,
  const NEP_Energy::ParaMB paramb,
  const NEP_Energy::ZBL zbl,
  const int* g_NN,
  const int* __restrict__ g_type,
  const int* g_t2_angular,
  const float* __restrict__ g_x12,
  const float* __restrict__ g_y12,
  const float* __restrict__ g_z12,
  float* g_pe)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x;
  if (n1 < N) {
    float s_pe = 0.0f;
    int type1 = g_type[n1];
    int zi = zbl.atomic_numbers[type1];
    float pow_zi = pow(float(zi), 0.23f);
    for (int i1 = 0; i1 < g_NN[n1]; ++i1) {
      int index = i1 * N + n1;
      float r12[3] = {g_x12[index], g_y12[index], g_z12[index]};
      float d12 = sqrt(r12[0] * r12[0] + r12[1] * r12[1] + r12[2] * r12[2]);
      float d12inv = 1.0f / d12;
      float f, fp;
      int type2 = g_t2_angular[index];
      int zj = zbl.atomic_numbers[type2];
      float a_inv = (pow_zi + pow(float(zj), 0.23f)) * 2.134563f;
      float zizj = K_C_SP * zi * zj;
      if (zbl.flexibled) {
        int t1, t2;
        if (type1 < type2) {
          t1 = type1;
          t2 = type2;
        } else {
          t1 = type2;
          t2 = type1;
        }
        int zbl_index = t1 * zbl.num_types - (t1 * (t1 - 1)) / 2 + (t2 - t1);
        float ZBL_para[10];
        for (int i = 0; i < 10; ++i) {
          ZBL_para[i] = zbl.para[10 * zbl_index + i];
        }
        find_f_and_fp_zbl(ZBL_para, zizj, a_inv, d12, d12inv, f, fp);
      } else {
        float rc_inner = zbl.rc_inner;
        float rc_outer = zbl.rc_outer;
        if (paramb.use_typewise_cutoff_zbl) {
          // zi and zj start from 1, so need to minus 1 here
          rc_outer = min(
            (COVALENT_RADIUS[zi - 1] + COVALENT_RADIUS[zj - 1]) * paramb.typewise_cutoff_zbl_factor,
            rc_outer);
          rc_inner = rc_outer * 0.5f;
        }
        find_f_and_fp_zbl(zizj, a_inv, rc_inner, rc_outer, d12, d12inv, f, fp);
      }
      s_pe += f * 0.5f;
    }
    g_pe[n1] += s_pe;
  }
}

void NEP_Energy::find_energy(
  const int N,
  const int* g_NN_radial,
  const int* g_NN_angular,
  const int* g_type,
  const int* g_t2_radial,
  const int* g_t2_angular,
  const float* g_x12_radial,
  const float* g_y12_radial,
  const float* g_z12_radial,
  const float* g_x12_angular,
  const float* g_y12_angular,
  const float* g_z12_angular,
  float* g_pe)
{
  find_energy_nep<<<(N - 1) / 64 + 1, 64>>>(
    paramb,
    annmb,
    N,
    g_NN_radial,
    g_NN_angular,
    g_type,
    g_t2_radial,
    g_t2_angular,
    g_x12_radial,
    g_y12_radial,
    g_z12_radial,
    g_x12_angular,
    g_y12_angular,
    g_z12_angular,
    g_pe);
  GPU_CHECK_KERNEL

  if (zbl.enabled) {
    find_energy_zbl<<<(N - 1) / 64 + 1, 64>>>(
      N,
      paramb,
      zbl,
      g_NN_angular,
      g_type,
      g_t2_angular,
      g_x12_angular,
      g_y12_angular,
      g_z12_angular,
      g_pe);
    GPU_CHECK_KERNEL
  }
}
