#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

/*----------------------------------------------------------------------------80
The neuroevolution potential (NEP)
Ref: Zheyong Fan et al., Neuroevolution machine learning potentials:
Combining high accuracy and low cost in atomistic simulations and application to
heat transport, Phys. Rev. B. 104, 104309 (2021).
------------------------------------------------------------------------------*/

#include "nep_energy.cuh"
#include "utilities/common.cuh"
#include "utilities/error.cuh"
#include "utilities/nep_utilities.cuh"
#include <fstream>
#include <iostream>
#include <string>
#include <vector>

const std::string ELEMENTS[NUM_ELEMENTS] = {
  "H",  "He", "Li", "Be", "B",  "C",  "N",  "O",  "F",  "Ne", "Na", "Mg", "Al", "Si", "P",
  "S",  "Cl", "Ar", "K",  "Ca", "Sc", "Ti", "V",  "Cr", "Mn", "Fe", "Co", "Ni", "Cu", "Zn",
  "Ga", "Ge", "As", "Se", "Br", "Kr", "Rb", "Sr", "Y",  "Zr", "Nb", "Mo", "Tc", "Ru", "Rh",
  "Pd", "Ag", "Cd", "In", "Sn", "Sb", "Te", "I",  "Xe", "Cs", "Ba", "La", "Ce", "Pr", "Nd",
  "Pm", "Sm", "Eu", "Gd", "Tb", "Dy", "Ho", "Er", "Tm", "Yb", "Lu", "Hf", "Ta", "W",  "Re",
  "Os", "Ir", "Pt", "Au", "Hg", "Tl", "Pb", "Bi", "Po", "At", "Rn", "Fr", "Ra", "Ac", "Th",
  "Pa", "U",  "Np", "Pu", "Am", "Cm", "Bk", "Cf", "Es", "Fm", "Md", "No", "Lr"};

void NEP_Energy::initialize(const char* file_potential)
{

  std::ifstream input(file_potential);
  if (!input.is_open()) {
    std::cout << "Failed to open " << file_potential << std::endl;
    exit(1);
  }

  // nep3 1 C
  std::vector<std::string> tokens = get_tokens(input);
  if (tokens.size() < 3) {
    std::cout << "The first line of nep.txt should have at least 3 items." << std::endl;
    exit(1);
  }
  if (tokens[0] == "nep") {
    paramb.version = 2;
    zbl.enabled = false;
  } else if (tokens[0] == "nep3") {
    paramb.version = 3;
    zbl.enabled = false;
  } else if (tokens[0] == "nep_zbl") {
    paramb.version = 2;
    zbl.enabled = true;
  } else if (tokens[0] == "nep3_zbl") {
    paramb.version = 3;
    zbl.enabled = true;
  } else if (tokens[0] == "nep4") {
    paramb.version = 4;
    zbl.enabled = false;
  } else if (tokens[0] == "nep4_zbl") {
    paramb.version = 4;
    zbl.enabled = true;
  }
  paramb.num_types = get_int_from_token(tokens[1], __FILE__, __LINE__);
  if (tokens.size() != 2 + paramb.num_types) {
    std::cout << "The first line of nep.txt should have " << paramb.num_types << " atom symbols."
              << std::endl;
    exit(1);
  }

  if (paramb.num_types == 1) {
    printf("    Use the NEP%d potential with %d atom type.\n", paramb.version, paramb.num_types);
  } else {
    printf("    Use the NEP%d potential with %d atom types.\n", paramb.version, paramb.num_types);
  }

  for (int n = 0; n < paramb.num_types; ++n) {
    int atomic_number = 0;
    for (int m = 0; m < NUM_ELEMENTS; ++m) {
      if (tokens[2 + n] == ELEMENTS[m]) {
        atomic_number = m + 1;
        break;
      }
    }
    zbl.atomic_numbers[n] = atomic_number;
    printf("        type %d (%s with Z = %g).\n", n, tokens[2 + n].c_str(), zbl.atomic_numbers[n]);
  }

  // zbl 0.7 1.4
  if (zbl.enabled) {
    tokens = get_tokens(input);
    if (tokens.size() != 3) {
      std::cout << "This line should be zbl rc_inner rc_outer." << std::endl;
      exit(1);
    }
    zbl.rc_inner = get_float_from_token(tokens[1], __FILE__, __LINE__);
    zbl.rc_outer = get_float_from_token(tokens[2], __FILE__, __LINE__);
    if (zbl.rc_inner == 0 && zbl.rc_outer == 0) {
      zbl.flexibled = true;
      printf("        has the flexible ZBL potential\n");
    } else {
      printf(
        "        has the universal ZBL with inner cutoff %g A and outer cutoff %g A.\n",
        zbl.rc_inner,
        zbl.rc_outer);
    }
  }

  // cutoff 4.2 3.7 80 47
  tokens = get_tokens(input);
  if (tokens.size() != 3 && tokens.size() != 5) {
    std::cout << "This line should be cutoff rc_radial rc_angular [MN_radial] [MN_angular].\n";
    exit(1);
  }
  paramb.rc_radial = get_float_from_token(tokens[1], __FILE__, __LINE__);
  paramb.rc_angular = get_float_from_token(tokens[2], __FILE__, __LINE__);
  printf("        radial cutoff = %g A.\n", paramb.rc_radial);
  printf("        angular cutoff = %g A.\n", paramb.rc_angular);

  paramb.MN_radial = 500;
  paramb.MN_angular = 100;

  if (tokens.size() == 5) {
    int MN_radial = get_int_from_token(tokens[3], __FILE__, __LINE__);
    int MN_angular = get_int_from_token(tokens[4], __FILE__, __LINE__);
    printf("        MN_radial = %d.\n", MN_radial);
    printf("        MN_angular = %d.\n", MN_angular);
    paramb.MN_radial = int(ceil(MN_radial * 1.25));
    paramb.MN_angular = int(ceil(MN_angular * 1.25));
    printf("        enlarged MN_radial = %d.\n", paramb.MN_radial);
    printf("        enlarged MN_angular = %d.\n", paramb.MN_angular);
  }

  // n_max 10 8
  tokens = get_tokens(input);
  if (tokens.size() != 3) {
    std::cout << "This line should be n_max n_max_radial n_max_angular." << std::endl;
    exit(1);
  }
  paramb.n_max_radial = get_int_from_token(tokens[1], __FILE__, __LINE__);
  paramb.n_max_angular = get_int_from_token(tokens[2], __FILE__, __LINE__);
  printf("        n_max_radial = %d.\n", paramb.n_max_radial);
  printf("        n_max_angular = %d.\n", paramb.n_max_angular);

  // basis_size 10 8
  if (paramb.version >= 3) {
    tokens = get_tokens(input);
    if (tokens.size() != 3) {
      std::cout << "This line should be basis_size basis_size_radial basis_size_angular."
                << std::endl;
      exit(1);
    }
    paramb.basis_size_radial = get_int_from_token(tokens[1], __FILE__, __LINE__);
    paramb.basis_size_angular = get_int_from_token(tokens[2], __FILE__, __LINE__);
    printf("        basis_size_radial = %d.\n", paramb.basis_size_radial);
    printf("        basis_size_angular = %d.\n", paramb.basis_size_angular);
  }

  // l_max
  tokens = get_tokens(input);
  if (paramb.version == 2) {
    if (tokens.size() != 2) {
      std::cout << "This line should be l_max l_max_3body." << std::endl;
      exit(1);
    }
  } else {
    if (tokens.size() != 4) {
      std::cout << "This line should be l_max l_max_3body l_max_4body l_max_5body." << std::endl;
      exit(1);
    }
  }

  paramb.L_max = get_int_from_token(tokens[1], __FILE__, __LINE__);
  printf("        l_max_3body = %d.\n", paramb.L_max);
  paramb.num_L = paramb.L_max;

  if (paramb.version >= 3) {
    int L_max_4body = get_int_from_token(tokens[2], __FILE__, __LINE__);
    int L_max_5body = get_int_from_token(tokens[3], __FILE__, __LINE__);
    printf("        l_max_4body = %d.\n", L_max_4body);
    printf("        l_max_5body = %d.\n", L_max_5body);
    if (L_max_4body == 2) {
      paramb.num_L += 1;
    }
    if (L_max_5body == 1) {
      paramb.num_L += 1;
    }
  }

  paramb.dim_angular = (paramb.n_max_angular + 1) * paramb.num_L;

  // ANN
  tokens = get_tokens(input);
  if (tokens.size() != 3) {
    std::cout << "This line should be ANN num_neurons 0." << std::endl;
    exit(1);
  }
  annmb.num_neurons1 = get_int_from_token(tokens[1], __FILE__, __LINE__);
  annmb.dim = (paramb.n_max_radial + 1) + paramb.dim_angular;
  printf("        ANN = %d-%d-1.\n", annmb.dim, annmb.num_neurons1);

  // calculated parameters:
  paramb.rcinv_radial = 1.0f / paramb.rc_radial;
  paramb.rcinv_angular = 1.0f / paramb.rc_angular;
  paramb.num_types_sq = paramb.num_types * paramb.num_types;

  annmb.num_para =
    (annmb.dim + 2) * annmb.num_neurons1 * (paramb.version == 4 ? paramb.num_types : 1) + 1;
  printf("        number of neural network parameters = %d.\n", annmb.num_para);
  int num_para_descriptor =
    paramb.num_types_sq * ((paramb.n_max_radial + 1) * (paramb.basis_size_radial + 1) +
                           (paramb.n_max_angular + 1) * (paramb.basis_size_angular + 1));
  if (paramb.version == 2) {
    num_para_descriptor =
      (paramb.num_types == 1)
        ? 0
        : paramb.num_types_sq * (paramb.n_max_radial + paramb.n_max_angular + 2);
  }
  printf("        number of descriptor parameters = %d.\n", num_para_descriptor);
  annmb.num_para += num_para_descriptor;
  printf("        total number of parameters = %d.\n", annmb.num_para);

  paramb.num_c_radial =
    paramb.num_types_sq * (paramb.n_max_radial + 1) * (paramb.basis_size_radial + 1);
  if (paramb.version == 2) {
    paramb.num_c_radial =
      (paramb.num_types == 1) ? 0 : paramb.num_types_sq * (paramb.n_max_radial + 1);
  }

  // NN and descriptor parameters
  std::vector<float> parameters(annmb.num_para);
  for (int n = 0; n < annmb.num_para; ++n) {
    tokens = get_tokens(input);
    parameters[n] = get_float_from_token(tokens[0], __FILE__, __LINE__);
  }
  nep_parameters.resize(annmb.num_para);
  nep_parameters.copy_from_host(parameters.data());
  update_potential(nep_parameters.data(), annmb);
  for (int d = 0; d < annmb.dim; ++d) {
    tokens = get_tokens(input);
    paramb.q_scaler[d] = get_float_from_token(tokens[0], __FILE__, __LINE__);
  }

  // flexible zbl potential parameters
  if (zbl.flexibled) {
    int num_type_zbl = (paramb.num_types * (paramb.num_types + 1)) / 2;
    for (int d = 0; d < num_type_zbl; ++d) {
      tokens = get_tokens(input);
      zbl.rc_flexible_inner[d] = get_float_from_token(tokens[0], __FILE__, __LINE__);
    }
    for (int d = 0; d < num_type_zbl; ++d) {
      tokens = get_tokens(input);
      zbl.rc_flexible_outer[d] = get_float_from_token(tokens[0], __FILE__, __LINE__);
    }
    for (int d = 0; d < 6 * num_type_zbl; ++d) {
      tokens = get_tokens(input);
      zbl.para[d] = get_float_from_token(tokens[0], __FILE__, __LINE__);
    }
    zbl.num_types = paramb.num_types;
  }
}

NEP_Energy::NEP_Energy(void)
{
  // nothing
}

NEP_Energy::~NEP_Energy(void)
{
  // nothing
}

void NEP_Energy::update_potential(float* parameters, ANN& ann)
{
  float* pointer = parameters;
  for (int t = 0; t < paramb.num_types; ++t) {
    if (t > 0 && paramb.version != 4) { // Use the same set of NN parameters for NEP2 and NEP3
      pointer -= (ann.dim + 2) * ann.num_neurons1;
    }
    ann.w0[t] = pointer;
    pointer += ann.num_neurons1 * ann.dim;
    ann.b0[t] = pointer;
    pointer += ann.num_neurons1;
    ann.w1[t] = pointer;
    pointer += ann.num_neurons1;
  }
  ann.b1 = pointer;
  ann.c = ann.b1 + 1;
}

static __global__ void find_energy_nep(
  NEP_Energy::ParaMB paramb,
  NEP_Energy::ANN annmb,
  const int N,
  const int* g_NN_radial,
  const int* g_NN_angular,
  const int* __restrict__ g_type,
  const int* __restrict__ g_t2_radial,
  const int* __restrict__ g_t2_angular,
  const float* __restrict__ g_x12_radial,
  const float* __restrict__ g_y12_radial,
  const float* __restrict__ g_z12_radial,
  const float* __restrict__ g_x12_angular,
  const float* __restrict__ g_y12_angular,
  const float* __restrict__ g_z12_angular,
  float* g_pe)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x;
  if (n1 < N) {
    int t1 = g_type[n1];
    float q[MAX_DIM] = {0.0f};

    // get radial descriptors
    for (int i1 = 0; i1 < g_NN_radial[n1]; ++i1) {
      int index = i1 * N + n1;
      float r12[3] = {g_x12_radial[index], g_y12_radial[index], g_z12_radial[index]};
      float d12 = sqrt(r12[0] * r12[0] + r12[1] * r12[1] + r12[2] * r12[2]);
      float fc12;
      find_fc(paramb.rc_radial, paramb.rcinv_radial, d12, fc12);
      int t2 = g_t2_radial[index];
      float fn12[MAX_NUM_N];
      if (paramb.version == 2) {
        find_fn(paramb.n_max_radial, paramb.rcinv_radial, d12, fc12, fn12);
        for (int n = 0; n <= paramb.n_max_radial; ++n) {
          float c = (paramb.num_types == 1)
                      ? 1.0f
                      : annmb.c[(n * paramb.num_types + t1) * paramb.num_types + t2];
          q[n] += fn12[n] * c;
        }
      } else {
        find_fn(paramb.basis_size_radial, paramb.rcinv_radial, d12, fc12, fn12);
        for (int n = 0; n <= paramb.n_max_radial; ++n) {
          float gn12 = 0.0f;
          for (int k = 0; k <= paramb.basis_size_radial; ++k) {
            int c_index = (n * (paramb.basis_size_radial + 1) + k) * paramb.num_types_sq;
            c_index += t1 * paramb.num_types + t2;
            gn12 += fn12[k] * annmb.c[c_index];
          }
          q[n] += gn12;
        }
      }
    }

    // get angular descriptors
    for (int n = 0; n <= paramb.n_max_angular; ++n) {
      float s[NUM_OF_ABC] = {0.0f};
      for (int i1 = 0; i1 < g_NN_angular[n1]; ++i1) {
        int index = i1 * N + n1;
        float r12[3] = {g_x12_angular[index], g_y12_angular[index], g_z12_angular[index]};
        float d12 = sqrt(r12[0] * r12[0] + r12[1] * r12[1] + r12[2] * r12[2]);
        float fc12;
        find_fc(paramb.rc_angular, paramb.rcinv_angular, d12, fc12);
        int t2 = g_t2_angular[index];
        if (paramb.version == 2) {
          float fn;
          find_fn(n, paramb.rcinv_angular, d12, fc12, fn);
          fn *=
            (paramb.num_types == 1)
              ? 1.0f
              : annmb.c
                  [((paramb.n_max_radial + 1 + n) * paramb.num_types + t1) * paramb.num_types + t2];
          accumulate_s(d12, r12[0], r12[1], r12[2], fn, s);
        } else {
          float fn12[MAX_NUM_N];
          find_fn(paramb.basis_size_angular, paramb.rcinv_angular, d12, fc12, fn12);
          float gn12 = 0.0f;
          for (int k = 0; k <= paramb.basis_size_angular; ++k) {
            int c_index = (n * (paramb.basis_size_angular + 1) + k) * paramb.num_types_sq;
            c_index += t1 * paramb.num_types + t2 + paramb.num_c_radial;
            gn12 += fn12[k] * annmb.c[c_index];
          }
          accumulate_s(d12, r12[0], r12[1], r12[2], gn12, s);
        }
      }
      if (paramb.num_L == paramb.L_max) {
        find_q(paramb.n_max_angular + 1, n, s, q + (paramb.n_max_radial + 1));
      } else if (paramb.num_L == paramb.L_max + 1) {
        find_q_with_4body(paramb.n_max_angular + 1, n, s, q + (paramb.n_max_radial + 1));
      } else {
        find_q_with_5body(paramb.n_max_angular + 1, n, s, q + (paramb.n_max_radial + 1));
      }
    }

    // nomalize descriptor
    for (int d = 0; d < annmb.dim; ++d) {
      q[d] = q[d] * paramb.q_scaler[d];
    }

    // get energy and energy gradient
    float F = 0.0f, Fp[MAX_DIM] = {0.0f};
    apply_ann_one_layer(
      annmb.dim, annmb.num_neurons1, annmb.w0[t1], annmb.b0[t1], annmb.w1[t1], annmb.b1, q, F, Fp);
    g_pe[n1] = F;
  }
}

static __global__ void find_energy_zbl(
  const int N,
  const NEP_Energy::ZBL zbl,
  const int* g_NN,
  const int* __restrict__ g_type,
  const int* g_t2_angular,
  const float* __restrict__ g_x12,
  const float* __restrict__ g_y12,
  const float* __restrict__ g_z12,
  float* g_pe)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x;
  if (n1 < N) {
    float s_pe = 0.0f;
    int type1 = g_type[n1];
    float zi = zbl.atomic_numbers[type1];
    float pow_zi = pow(zi, 0.23f);
    for (int i1 = 0; i1 < g_NN[n1]; ++i1) {
      int index = i1 * N + n1;
      float r12[3] = {g_x12[index], g_y12[index], g_z12[index]};
      float d12 = sqrt(r12[0] * r12[0] + r12[1] * r12[1] + r12[2] * r12[2]);
      float d12inv = 1.0f / d12;
      float f, fp;
      int type2 = g_t2_angular[index];
      float zj = zbl.atomic_numbers[type2];
      float a_inv = (pow_zi + pow(zj, 0.23f)) * 2.134563f;
      float zizj = K_C_SP * zi * zj;
      if (zbl.flexibled) {
        int t1, t2;
        if (type1 < type2) {
          t1 = type1;
          t2 = type2;
        } else {
          t1 = type2;
          t2 = type1;
        }
        int zbl_index = t1 * zbl.num_types - (t1 * (t1 - 1)) / 2 + (t2 - t1);
        float rc_inner = zbl.rc_flexible_inner[zbl_index];
        float rc_outer = zbl.rc_flexible_outer[zbl_index];
        float ZBL_para[6];
        for (int i = 0; i < 6; ++i) {
          ZBL_para[i] = zbl.para[6 * zbl_index + i];
        }
        find_f_and_fp_zbl(ZBL_para, zizj, a_inv, rc_inner, rc_outer, d12, d12inv, f, fp);
      } else {
        find_f_and_fp_zbl(zizj, a_inv, zbl.rc_inner, zbl.rc_outer, d12, d12inv, f, fp);
      }
      s_pe += f * 0.5f;
    }
    g_pe[n1] += s_pe;
  }
}

void NEP_Energy::find_energy(
  const int N,
  const int* g_NN_radial,
  const int* g_NN_angular,
  const int* g_type,
  const int* g_t2_radial,
  const int* g_t2_angular,
  const float* g_x12_radial,
  const float* g_y12_radial,
  const float* g_z12_radial,
  const float* g_x12_angular,
  const float* g_y12_angular,
  const float* g_z12_angular,
  float* g_pe)
{
  find_energy_nep<<<(N - 1) / 64 + 1, 64>>>(
    paramb,
    annmb,
    N,
    g_NN_radial,
    g_NN_angular,
    g_type,
    g_t2_radial,
    g_t2_angular,
    g_x12_radial,
    g_y12_radial,
    g_z12_radial,
    g_x12_angular,
    g_y12_angular,
    g_z12_angular,
    g_pe);
  CUDA_CHECK_KERNEL

  if (zbl.enabled) {
    find_energy_zbl<<<(N - 1) / 64 + 1, 64>>>(
      N,
      zbl,
      g_NN_angular,
      g_type,
      g_t2_angular,
      g_x12_angular,
      g_y12_angular,
      g_z12_angular,
      g_pe);
    CUDA_CHECK_KERNEL
  }
}
