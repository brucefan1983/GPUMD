/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

/*----------------------------------------------------------------------------80
The class defining the simulation model.
------------------------------------------------------------------------------*/

#include "atom.cuh"
#include "box.cuh"
#include "group.cuh"
#include "read_xyz.cuh"
#include "utilities/common.cuh"
#include "utilities/error.cuh"
#include <algorithm>
#include <cctype>
#include <fstream>
#include <iostream>
#include <map>
#include <sstream>
#include <string>

const std::map<std::string, double> MASS_TABLE{
  {"H", 1.0080000000},
  {"He", 4.0026020000},
  {"Li", 6.9400000000},
  {"Be", 9.0121831000},
  {"B", 10.8100000000},
  {"C", 12.0110000000},
  {"N", 14.0070000000},
  {"O", 15.9990000000},
  {"F", 18.9984031630},
  {"Ne", 20.1797000000},
  {"Na", 22.9897692800},
  {"Mg", 24.3050000000},
  {"Al", 26.9815385000},
  {"Si", 28.0850000000},
  {"P", 30.9737619980},
  {"S", 32.0600000000},
  {"Cl", 35.4500000000},
  {"Ar", 39.9480000000},
  {"K", 39.0983000000},
  {"Ca", 40.0780000000},
  {"Sc", 44.9559080000},
  {"Ti", 47.8670000000},
  {"V", 50.9415000000},
  {"Cr", 51.9961000000},
  {"Mn", 54.9380440000},
  {"Fe", 55.8450000000},
  {"Co", 58.9331940000},
  {"Ni", 58.6934000000},
  {"Cu", 63.5460000000},
  {"Zn", 65.3800000000},
  {"Ga", 69.7230000000},
  {"Ge", 72.6300000000},
  {"As", 74.9215950000},
  {"Se", 78.9710000000},
  {"Br", 79.9040000000},
  {"Kr", 83.7980000000},
  {"Rb", 85.4678000000},
  {"Sr", 87.6200000000},
  {"Y", 88.9058400000},
  {"Zr", 91.2240000000},
  {"Nb", 92.9063700000},
  {"Mo", 95.9500000000},
  {"Tc", 98},
  {"Ru", 101.0700000000},
  {"Rh", 102.9055000000},
  {"Pd", 106.4200000000},
  {"Ag", 107.8682000000},
  {"Cd", 112.4140000000},
  {"In", 114.8180000000},
  {"Sn", 118.7100000000},
  {"Sb", 121.7600000000},
  {"Te", 127.6000000000},
  {"I", 126.9044700000},
  {"Xe", 131.2930000000},
  {"Cs", 132.9054519600},
  {"Ba", 137.3270000000},
  {"La", 138.9054700000},
  {"Ce", 140.1160000000},
  {"Pr", 140.9076600000},
  {"Nd", 144.2420000000},
  {"Pm", 145},
  {"Sm", 150.3600000000},
  {"Eu", 151.9640000000},
  {"Gd", 157.2500000000},
  {"Tb", 158.9253500000},
  {"Dy", 162.5000000000},
  {"Ho", 164.9303300000},
  {"Er", 167.2590000000},
  {"Tm", 168.9342200000},
  {"Yb", 173.0450000000},
  {"Lu", 174.9668000000},
  {"Hf", 178.4900000000},
  {"Ta", 180.9478800000},
  {"W", 183.8400000000},
  {"Re", 186.2070000000},
  {"Os", 190.2300000000},
  {"Ir", 192.2170000000},
  {"Pt", 195.0840000000},
  {"Au", 196.9665690000},
  {"Hg", 200.5920000000},
  {"Tl", 204.3800000000},
  {"Pb", 207.2000000000},
  {"Bi", 208.9804000000},
  {"Po", 210},
  {"At", 210},
  {"Rn", 222},
  {"Fr", 223},
  {"Ra", 226},
  {"Ac", 227},
  {"Th", 232.0377000000},
  {"Pa", 231.0358800000},
  {"U", 238.0289100000},
  {"Np", 237},
  {"Pu", 244},
  {"Am", 243},
  {"Cm", 247},
  {"Bk", 247},
  {"Cf", 251},
  {"Es", 252},
  {"Fm", 257},
  {"Md", 258},
  {"No", 259},
  {"Lr", 262}};

static bool need_triclinic()
{
  std::ifstream input_run("run.in");
  if (!input_run.is_open()) {
    PRINT_INPUT_ERROR("Cannot open run.in.");
  }
  bool triclinic = false;
  std::string line;
  while (std::getline(input_run, line)) {
    std::vector<std::string> tokens = get_tokens(line);
    if (tokens.size() != 0) {
      if (tokens[0] == "compute_elastic") {
        triclinic = true;
      }
      if (tokens[0] == "change_box" && tokens.size() == 7) {
        triclinic = true;
      }
      if (tokens[0] == "ensemble" && tokens.size() >= 18) {
        triclinic = true;
      }
    }
  }

  input_run.close();
  return triclinic;
}

static void read_xyz_line_1(std::ifstream& input, int& N)
{
  std::vector<std::string> tokens = get_tokens(input);
  if (tokens.size() != 1) {
    PRINT_INPUT_ERROR("The first line for the xyz file should have one value.");
  }
  N = get_int_from_token(tokens[0], __FILE__, __LINE__);
  if (N < 2) {
    PRINT_INPUT_ERROR("Number of atoms should >= 2.");
  } else {
    printf("Number of atoms is %d.\n", N);
  }
}

static void read_xyz_line_2(
  std::ifstream& input,
  Box& box,
  int& has_velocity_in_xyz,
  bool& has_mass,
  int& num_columns,
  int* property_offset,
  std::vector<Group>& group)
{
  std::vector<std::string> tokens = get_tokens_without_unwanted_spaces(input);
  for (auto& token : tokens) {
    std::transform(
      token.begin(), token.end(), token.begin(), [](unsigned char c) { return std::tolower(c); });
  }

  box.pbc_x = box.pbc_y = box.pbc_z = 1; // default is periodic
  for (int n = 0; n < tokens.size(); ++n) {
    const std::string tmp_string = "pbc=";
    if (tokens[n].substr(0, tmp_string.length()) == tmp_string) {
      if (tokens[n].back() == 't') {
        box.pbc_x = 1;
      } else if (tokens[n].back() == 'f') {
        box.pbc_x = 0;
      } else {
        PRINT_INPUT_ERROR("periodic boundary in x direction should be T or F.");
      }
      if (tokens[n + 1] == "t") {
        box.pbc_y = 1;
      } else if (tokens[n + 1] == "f") {
        box.pbc_y = 0;
      } else {
        PRINT_INPUT_ERROR("periodic boundary in y direction should be T or F.");
      }
      if (tokens[n + 2].front() == 't') {
        box.pbc_z = 1;
      } else if (tokens[n + 2].front() == 'f') {
        box.pbc_z = 0;
      } else {
        PRINT_INPUT_ERROR("periodic boundary in z direction should be T or F.");
      }
    }
  }
  printf("Use %s boundary conditions along x.\n", (box.pbc_x == 1) ? "periodic" : "free");
  printf("Use %s boundary conditions along y.\n", (box.pbc_y == 1) ? "periodic" : "free");
  printf("Use %s boundary conditions along z.\n", (box.pbc_z == 1) ? "periodic" : "free");

  // box matrix
  bool has_lattice_in_exyz = false;
  for (int n = 0; n < tokens.size(); ++n) {
    const std::string lattice_string = "lattice=";
    if (tokens[n].substr(0, lattice_string.length()) == lattice_string) {
      has_lattice_in_exyz = true;
      const int transpose_index[9] = {0, 3, 6, 1, 4, 7, 2, 5, 8};
      for (int m = 0; m < 9; ++m) {
        box.cpu_h[transpose_index[m]] = get_double_from_token(
          tokens[n + m].substr(
            (m == 0) ? (lattice_string.length() + 1) : 0,
            (m == 8) ? (tokens[n + m].length() - 1) : tokens[n + m].length()),
          __FILE__,
          __LINE__);
      }
    }
  }
  if (!has_lattice_in_exyz) {
    PRINT_INPUT_ERROR("'lattice' is missing in the second line of the model file.");
  } else {

    if (
      !need_triclinic() && box.cpu_h[1] == 0 && box.cpu_h[2] == 0 && box.cpu_h[3] == 0 &&
      box.cpu_h[5] == 0 && box.cpu_h[6] == 0 && box.cpu_h[7] == 0) {
      box.triclinic = 0;
    } else {
      box.triclinic = 1;
    }

    (box.triclinic == 0) ? printf("Use orthogonal box.\n") : printf("Use triclinic box.\n");

    if (box.triclinic == 1) {
      printf("Box matrix h = [a, b, c] is\n");
      for (int d1 = 0; d1 < 3; ++d1) {
        for (int d2 = 0; d2 < 3; ++d2) {
          printf("%20.10e", box.cpu_h[d1 * 3 + d2]);
        }
        printf("\n");
      }

      box.get_inverse();

      printf("Inverse box matrix g = inv(h) is\n");
      for (int d1 = 0; d1 < 3; ++d1) {
        for (int d2 = 0; d2 < 3; ++d2) {
          printf("%20.10e", box.cpu_h[9 + d1 * 3 + d2]);
        }
        printf("\n");
      }
    } else {
      box.cpu_h[1] = box.cpu_h[4];
      box.cpu_h[2] = box.cpu_h[8];
      box.cpu_h[3] = box.cpu_h[0] * 0.5;
      box.cpu_h[4] = box.cpu_h[1] * 0.5;
      box.cpu_h[5] = box.cpu_h[2] * 0.5;
      if (box.cpu_h[0] <= 0) {
        PRINT_INPUT_ERROR("Box length in x direction <= 0.");
      }
      if (box.cpu_h[1] <= 0) {
        PRINT_INPUT_ERROR("Box length in y direction <= 0.");
      }
      if (box.cpu_h[2] <= 0) {
        PRINT_INPUT_ERROR("Box length in z direction <= 0.");
      }
      printf("Box lengths are\n");
      printf("    Lx = %20.10e A\n", box.cpu_h[0]);
      printf("    Ly = %20.10e A\n", box.cpu_h[1]);
      printf("    Lz = %20.10e A\n", box.cpu_h[2]);
    }
  }

  // properties
  std::string property_name[5] = {"species", "pos", "mass", "vel", "group"};
  int property_position[5] = {-1, -1, -1, -1, -1}; // species,pos,mass,vel,group
  for (int n = 0; n < tokens.size(); ++n) {
    const std::string properties_string = "properties=";
    if (tokens[n].substr(0, properties_string.length()) == properties_string) {
      std::string line = tokens[n].substr(properties_string.length(), tokens[n].length());
      for (auto& letter : line) {
        if (letter == ':') {
          letter = ' ';
        }
      }
      std::vector<std::string> sub_tokens = get_tokens(line);
      for (int k = 0; k < sub_tokens.size() / 3; ++k) {
        for (int prop = 0; prop < 5; ++prop) {
          if (sub_tokens[k * 3] == property_name[prop]) {
            property_position[prop] = k;
          }
        }
      }

      if (property_position[3] < 0) {
        has_velocity_in_xyz = 0;
        printf("Do not specify initial velocities here.\n");
      } else {
        has_velocity_in_xyz = 1;
        printf("Specify initial velocities here.\n");
      }

      if (property_position[4] < 0) {
        group.resize(0);
        printf("Have no grouping method.\n");
      } else {
        int num_of_grouping_methods =
          get_int_from_token(sub_tokens[property_position[4] * 3 + 2], __FILE__, __LINE__);
        group.resize(num_of_grouping_methods);
        printf("Have %d grouping method(s).\n", num_of_grouping_methods);
      }

      for (int k = 0; k < sub_tokens.size() / 3; ++k) {
        const int tmp_length = get_int_from_token(sub_tokens[k * 3 + 2], __FILE__, __LINE__);
        for (int prop = 0; prop < 5; ++prop) {
          if (k < property_position[prop]) {
            property_offset[prop] += tmp_length;
          }
        }
        num_columns += tmp_length;
      }
    }
  }

  if (property_position[0] < 0) {
    PRINT_INPUT_ERROR("'species' or 'properties' is missing in the model file.");
  }
  if (property_position[1] < 0) {
    PRINT_INPUT_ERROR("'pos' or 'properties' is missing in the model file.");
  }
  if (property_position[2] < 0) {
    has_mass = false;
  } else {
    has_mass = true;
  }
}

void read_xyz_in_line_3(
  std::ifstream& input,
  const int N,
  const int has_velocity_in_xyz,
  const bool has_mass,
  const int num_columns,
  const int* property_offset,
  int& number_of_types,
  std::vector<std::string>& atom_symbols,
  std::vector<std::string>& cpu_atom_symbol,
  std::vector<int>& cpu_type,
  std::vector<double>& cpu_mass,
  std::vector<double>& cpu_position_per_atom,
  std::vector<double>& cpu_velocity_per_atom,
  std::vector<Group>& group)
{
  cpu_atom_symbol.resize(N);
  cpu_type.resize(N);
  cpu_mass.resize(N);
  cpu_position_per_atom.resize(N * 3);
  cpu_velocity_per_atom.resize(N * 3);
  number_of_types = atom_symbols.size();

  for (int m = 0; m < group.size(); ++m) {
    group[m].cpu_label.resize(N);
    group[m].number = 0;
  }

  for (int n = 0; n < N; n++) {
    std::vector<std::string> tokens = get_tokens(input);
    if (tokens.size() != num_columns) {
      PRINT_INPUT_ERROR("number of columns does not match properties.\n");
    }

    cpu_atom_symbol[n] = tokens[property_offset[0]];

    bool is_allowed_element = false;
    for (int t = 0; t < number_of_types; ++t) {
      if (cpu_atom_symbol[n] == atom_symbols[t]) {
        cpu_type[n] = t;
        is_allowed_element = true;
      }
    }
    if (!is_allowed_element) {
      PRINT_INPUT_ERROR("There is atom in model.xyz that is not allowed in the used potential.\n");
    }

    for (int d = 0; d < 3; ++d) {
      cpu_position_per_atom[n + N * d] =
        get_double_from_token(tokens[property_offset[1] + d], __FILE__, __LINE__);
    }

    if (has_mass) {
      cpu_mass[n] = get_double_from_token(tokens[property_offset[2]], __FILE__, __LINE__);
      if (cpu_mass[n] <= 0) {
        PRINT_INPUT_ERROR("Atom mass should > 0.");
      }
    } else {
      cpu_mass[n] = MASS_TABLE.at(cpu_atom_symbol[n]);
    }

    if (has_velocity_in_xyz) {
      const double A_per_fs_to_natural = TIME_UNIT_CONVERSION;
      for (int d = 0; d < 3; ++d) {
        cpu_velocity_per_atom[n + N * d] =
          get_double_from_token(tokens[property_offset[3] + d], __FILE__, __LINE__) *
          A_per_fs_to_natural;
      }
    }

    for (int m = 0; m < group.size(); ++m) {
      group[m].cpu_label[n] =
        get_int_from_token(tokens[property_offset[4] + m], __FILE__, __LINE__);
      if (group[m].cpu_label[n] < 0 || group[m].cpu_label[n] >= N) {
        PRINT_INPUT_ERROR("Group label should >= 0 and < N.");
      }
      if ((group[m].cpu_label[n] + 1) > group[m].number) {
        group[m].number = group[m].cpu_label[n] + 1;
      }
    }
  }
}

void find_type_size(
  const int N,
  const int number_of_types,
  const std::vector<int>& cpu_type,
  std::vector<int>& cpu_type_size)
{
  cpu_type_size.resize(number_of_types);

  if (number_of_types == 1) {
    printf("There is only one atom type.\n");
  } else {
    printf("There are %d atom types.\n", number_of_types);
  }

  for (int m = 0; m < number_of_types; m++) {
    cpu_type_size[m] = 0;
  }
  for (int n = 0; n < N; n++) {
    cpu_type_size[cpu_type[n]]++;
  }
  for (int m = 0; m < number_of_types; m++) {
    printf("    %d atoms of type %d.\n", cpu_type_size[m], m);
  }
}

static std::string get_filename_potential()
{
  std::ifstream input_run("run.in");
  if (!input_run.is_open()) {
    PRINT_INPUT_ERROR("No run.in.");
  }

  std::string line;
  std::string filename_potential;
  while (std::getline(input_run, line)) {
    std::vector<std::string> tokens = get_tokens(line);
    if (tokens.size() >= 2) {
      if (tokens[0] == "potential") {
        filename_potential = tokens[1];
      }
    }
  }
  input_run.close();
  if (filename_potential.size() == 0) {
    PRINT_INPUT_ERROR("There is no 'potential' keyword in run.in.");
  } else {
    return filename_potential;
  }
}

static std::vector<std::string> get_atom_symbols(std::string& filename_potential)
{
  std::ifstream input_potential(filename_potential);
  if (!input_potential.is_open()) {
    std::cout << "Error: cannot open " + filename_potential << std::endl;
    exit(1);
  }

  std::vector<std::string> tokens = get_tokens(input_potential);
  if (tokens.size() < 3) {
    std::cout << "The first line of the potential file should have at least 3 items." << std::endl;
    exit(1);
  }

  int number_of_types = get_int_from_token(tokens[1], __FILE__, __LINE__);
  if (tokens.size() != 2 + number_of_types) {
    std::cout << "The first line of the potential file should have " << number_of_types
              << " atom symbols." << std::endl;
    exit(1);
  }

  std::vector<std::string> atom_symbols(number_of_types);
  for (int n = 0; n < number_of_types; ++n) {
    atom_symbols[n] = tokens[2 + n];
  }

  input_potential.close();
  return atom_symbols;
}

void initialize_position(
  int& has_velocity_in_xyz, int& number_of_types, Box& box, std::vector<Group>& group, Atom& atom)
{
  std::string filename("model.xyz");
  std::ifstream input(filename);

  if (!input.is_open()) {
    PRINT_INPUT_ERROR("Failed to open model.xyz.");
  }

  std::vector<std::string> atom_symbols;
  auto filename_potential = get_filename_potential();
  atom_symbols = get_atom_symbols(filename_potential);

  read_xyz_line_1(input, atom.number_of_atoms);
  int property_offset[5] = {0, 0, 0, 0, 0}; // species,pos,mass,vel,group
  int num_columns = 0;
  bool has_mass = true;
  read_xyz_line_2(input, box, has_velocity_in_xyz, has_mass, num_columns, property_offset, group);

  read_xyz_in_line_3(
    input,
    atom.number_of_atoms,
    has_velocity_in_xyz,
    has_mass,
    num_columns,
    property_offset,
    number_of_types,
    atom_symbols,
    atom.cpu_atom_symbol,
    atom.cpu_type,
    atom.cpu_mass,
    atom.cpu_position_per_atom,
    atom.cpu_velocity_per_atom,
    group);

  input.close();

  for (int m = 0; m < group.size(); ++m) {
    group[m].find_size(atom.number_of_atoms, m);
    group[m].find_contents(atom.number_of_atoms);
  }

  find_type_size(atom.number_of_atoms, number_of_types, atom.cpu_type, atom.cpu_type_size);
}

void allocate_memory_gpu(std::vector<Group>& group, Atom& atom, GPU_Vector<double>& thermo)
{
  const int N = atom.number_of_atoms;
  atom.type.resize(N);
  atom.type.copy_from_host(atom.cpu_type.data());
  for (int m = 0; m < group.size(); ++m) {
    group[m].label.resize(N);
    group[m].size.resize(group[m].number);
    group[m].size_sum.resize(group[m].number);
    group[m].contents.resize(N);
    group[m].label.copy_from_host(group[m].cpu_label.data());
    group[m].size.copy_from_host(group[m].cpu_size.data());
    group[m].size_sum.copy_from_host(group[m].cpu_size_sum.data());
    group[m].contents.copy_from_host(group[m].cpu_contents.data());
  }
  atom.mass.resize(N);
  atom.mass.copy_from_host(atom.cpu_mass.data());
  atom.position_per_atom.resize(N * 3);
  atom.unwrapped_position.resize(N * 3);
  atom.position_temp.resize(N * 3);
  atom.position_per_atom.copy_from_host(atom.cpu_position_per_atom.data());
  atom.unwrapped_position.copy_from_host(atom.cpu_position_per_atom.data());
  atom.velocity_per_atom.resize(N * 3);
  atom.force_per_atom.resize(N * 3, 0);
  atom.virial_per_atom.resize(N * 9);
  atom.potential_per_atom.resize(N);
  atom.heat_per_atom.resize(N * 5);
  thermo.resize(12);
}
