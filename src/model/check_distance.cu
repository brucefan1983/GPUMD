#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan and GPUMD development team
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

/*----------------------------------------------------------------------------80
Calculate the distance between any two atoms in the model.xyz file.
------------------------------------------------------------------------------*/

#include "atom.cuh"
#include "box.cuh"
#include "check_distance.cuh"
#include "utilities/error.cuh"
#include <cmath>

void applyMicOne(double& x12)
{
  if (x12 < -0.5) {
    x12 += 1.0;
  } else if (x12 > +0.5) {
    x12 -= 1.0;
  }
}

void applyMic(const Box& box, double& x12, double& y12, double& z12)
{

  double sx = box.cpu_h[9] * x12 + box.cpu_h[10] * y12 + box.cpu_h[11] * z12;
  double sy = box.cpu_h[12] * x12 + box.cpu_h[13] * y12 + box.cpu_h[14] * z12;
  double sz = box.cpu_h[15] * x12 + box.cpu_h[16] * y12 + box.cpu_h[17] * z12;

  if (box.pbc_x)
    applyMicOne(sx);
  if (box.pbc_y)
    applyMicOne(sy);
  if (box.pbc_z)
    applyMicOne(sz);

  x12 = box.cpu_h[0] * sx + box.cpu_h[1] * sy + box.cpu_h[2] * sz;
  y12 = box.cpu_h[3] * sx + box.cpu_h[4] * sy + box.cpu_h[5] * sz;
  z12 = box.cpu_h[6] * sx + box.cpu_h[7] * sy + box.cpu_h[8] * sz;
}

void findCell(
  const Box& box, const double* thickness, const double* r, const int* numCells, int* cell)
{
  double s[3];
  s[0] = box.cpu_h[9] * r[0] + box.cpu_h[10] * r[1] + box.cpu_h[11] * r[2];
  s[1] = box.cpu_h[12] * r[0] + box.cpu_h[13] * r[1] + box.cpu_h[14] * r[2];
  s[2] = box.cpu_h[15] * r[0] + box.cpu_h[16] * r[1] + box.cpu_h[17] * r[2];
  for (int d = 0; d < 3; ++d) {
    cell[d] = floor(s[d] * thickness[d] * 0.2);
    if (cell[d] < 0)
      cell[d] += numCells[d];
    if (cell[d] >= numCells[d])
      cell[d] -= numCells[d];
  }
  cell[3] = cell[0] + numCells[0] * (cell[1] + numCells[1] * cell[2]);
}

// just check the atom pairs whose distance less than this value
#define MIN_DISTANCE_LIMIT 2.0

void calculate_min_atomic_distance(const Atom& atom, const Box& box)
{
  const int N = atom.number_of_atoms;
  const double* pos = atom.cpu_position_per_atom.data();

  double min_dist_sq = MIN_DISTANCE_LIMIT * MIN_DISTANCE_LIMIT;
  int min_n1 = -1, min_n2 = -1;

  double thickness[3];
  thickness[0] =
    sqrt(box.cpu_h[0] * box.cpu_h[0] + box.cpu_h[3] * box.cpu_h[3] + box.cpu_h[6] * box.cpu_h[6]);
  thickness[1] =
    sqrt(box.cpu_h[1] * box.cpu_h[1] + box.cpu_h[4] * box.cpu_h[4] + box.cpu_h[7] * box.cpu_h[7]);
  thickness[2] =
    sqrt(box.cpu_h[2] * box.cpu_h[2] + box.cpu_h[5] * box.cpu_h[5] + box.cpu_h[8] * box.cpu_h[8]);

  int cell[4], numCells[4];
  numCells[0] = std::max(1, static_cast<int>(ceil(thickness[0] * 0.2)));
  numCells[1] = std::max(1, static_cast<int>(ceil(thickness[1] * 0.2)));
  numCells[2] = std::max(1, static_cast<int>(ceil(thickness[2] * 0.2)));
  numCells[3] = numCells[0] * numCells[1] * numCells[2];

  std::vector<int> cellContents(N, 0);
  std::vector<int> cellCount(numCells[3], 0);
  std::vector<int> cellCountSum(numCells[3], 0);

  for (int n = 0; n < N; ++n) {
    const double r[3] = {pos[n], pos[n + N], pos[n + 2 * N]};
    findCell(box, thickness, r, numCells, cell);
    ++cellCount[cell[3]];
  }

  for (int i = 1; i < numCells[3]; ++i) {
    cellCountSum[i] = cellCountSum[i - 1] + cellCount[i - 1];
  }
  std::fill(cellCount.begin(), cellCount.end(), 0);
  for (int n = 0; n < N; ++n) {
    const double r[3] = {pos[n], pos[n + N], pos[n + 2 * N]};
    findCell(box, thickness, r, numCells, cell);
    cellContents[cellCountSum[cell[3]] + cellCount[cell[3]]] = n;
    ++cellCount[cell[3]];
  }

  for (int n1 = 0; n1 < N; ++n1) {
    const double r1[3] = {pos[n1], pos[n1 + N], pos[n1 + 2 * N]};
    findCell(box, thickness, r1, numCells, cell);
    for (int k = -1; k <= 1; ++k) {
      for (int j = -1; j <= 1; ++j) {
        for (int i = -1; i <= 1; ++i) {
          int neighborCell = cell[3] + (k * numCells[1] + j) * numCells[0] + i;
          if (cell[0] + i < 0)
            neighborCell += numCells[0];
          if (cell[0] + i >= numCells[0])
            neighborCell -= numCells[0];
          if (cell[1] + j < 0)
            neighborCell += numCells[1] * numCells[0];
          if (cell[1] + j >= numCells[1])
            neighborCell -= numCells[1] * numCells[0];
          if (cell[2] + k < 0)
            neighborCell += numCells[3];
          if (cell[2] + k >= numCells[2])
            neighborCell -= numCells[3];
          for (int m = 0; m < cellCount[neighborCell]; ++m) {
            const int n2 = cellContents[cellCountSum[neighborCell] + m];
            if (n1 < n2) {
              double x12 = pos[n2] - r1[0];
              double y12 = pos[n2 + N] - r1[1];
              double z12 = pos[n2 + 2 * N] - r1[2];
              applyMic(box, x12, y12, z12);
              if (fabs(x12) > MIN_DISTANCE_LIMIT || fabs(y12) > MIN_DISTANCE_LIMIT || 
                  fabs(z12) > MIN_DISTANCE_LIMIT) {
                continue;
              }
              const double dist = x12 * x12 + y12 * y12 + z12 * z12;

              if (dist < min_dist_sq) {
                min_dist_sq = dist;
                min_n1 = n1;
                min_n2 = n2;
              }
            }
          }
        }
      }
    }
  }
  double min_distance = sqrt(min_dist_sq);

  if (min_n1 == -1) {
    printf("Minimum distance is larger than %f Å\n", MIN_DISTANCE_LIMIT);
  }
  else {
    printf(
      "Minimum distance between atoms %d (%s) and %d (%s): %f Å\n",
      min_n1,
      atom.cpu_atom_symbol[min_n1].c_str(),
      min_n2,
      atom.cpu_atom_symbol[min_n2].c_str(),
      min_distance);
  }
  
}
