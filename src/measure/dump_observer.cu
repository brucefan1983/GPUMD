#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan and GPUMD development team
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details. You should have received a copy of the GNU General
   Public License along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

/*-----------------------------------------------------------------------------------------------100
Dump energy/force/virial with all loaded potentials at a given interval.
--------------------------------------------------------------------------------------------------*/

#include "dump_observer.cuh"
#include "model/box.cuh"
#include "parse_utilities.cuh"
#include "utilities/common.cuh"
#include "utilities/error.cuh"
#include "utilities/gpu_macro.cuh"
#include "utilities/gpu_vector.cuh"
#include "utilities/read_file.cuh"
#include <iostream>
#include <vector>
#include <cstring>

static __global__ void gpu_sum(const int N, const double* g_data, double* g_data_sum)
{
  int number_of_rounds = (N - 1) / 1024 + 1;
  __shared__ double s_data[1024];
  s_data[threadIdx.x] = 0.0;
  for (int round = 0; round < number_of_rounds; ++round) {
    int n = threadIdx.x + round * 1024;
    if (n < N) {
      s_data[threadIdx.x] += g_data[n + blockIdx.x * N];
    }
  }
  __syncthreads();
  for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1) {
    if (threadIdx.x < offset) {
      s_data[threadIdx.x] += s_data[threadIdx.x + offset];
    }
    __syncthreads();
  }
  if (threadIdx.x == 0) {
    g_data_sum[blockIdx.x] = s_data[0];
  }
}

static __global__ void initialize_properties(
  int N, double* g_fx, double* g_fy, double* g_fz, double* g_pe, double* g_virial)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x;
  if (n1 < N) {
    g_fx[n1] = 0.0;
    g_fy[n1] = 0.0;
    g_fz[n1] = 0.0;
    g_pe[n1] = 0.0;
    g_virial[n1 + 0 * N] = 0.0;
    g_virial[n1 + 1 * N] = 0.0;
    g_virial[n1 + 2 * N] = 0.0;
    g_virial[n1 + 3 * N] = 0.0;
    g_virial[n1 + 4 * N] = 0.0;
    g_virial[n1 + 5 * N] = 0.0;
    g_virial[n1 + 6 * N] = 0.0;
    g_virial[n1 + 7 * N] = 0.0;
    g_virial[n1 + 8 * N] = 0.0;
  }
}

Dump_Observer::Dump_Observer(const char** param, int num_param)
{
  parse(param, num_param);
  property_name = "dump_observer";
}

void Dump_Observer::parse(const char** param, int num_param)
{
  dump_ = true;
  printf("Dump observer.\n");

  if (num_param != 6) {
    PRINT_INPUT_ERROR("dump_observer should have 5 parameters.");
  }
  mode_ = param[1];
  if (mode_.compare("observe") != 0 && mode_.compare("average") != 0) {
    PRINT_INPUT_ERROR("observer mode should be 'observe' or 'average'");
  }
  if (!is_valid_int(param[2], &dump_interval_thermo_)) {
    PRINT_INPUT_ERROR("dump interval thermo should be an integer.");
  }
  if (dump_interval_thermo_ <= 0) {
    PRINT_INPUT_ERROR("dump interval thermo should > 0.");
  }
  if (!is_valid_int(param[3], &dump_interval_exyz_)) {
    PRINT_INPUT_ERROR("dump interval exyz should be an integer.");
  }
  if (dump_interval_exyz_ <= 0) {
    PRINT_INPUT_ERROR("dump interval exyz should > 0.");
  }

  printf("    .out every %d steps.\n", dump_interval_thermo_);
  printf("    .exyz every %d steps.\n", dump_interval_exyz_);

  if (!is_valid_int(param[4], &has_velocity_)) {
    PRINT_INPUT_ERROR("has_velocity should be an integer.");
  }
  if (has_velocity_ == 0) {
    printf("    without velocity data.\n");
  } else {
    printf("    with velocity data.\n");
  }

  if (!is_valid_int(param[5], &has_force_)) {
    PRINT_INPUT_ERROR("has_force should be an integer.");
  }
  if (has_force_ == 0) {
    printf("    without force data.\n");
  } else {
    printf("    with force data.\n");
  }

  if (mode_.compare("observe") == 0) {
    printf(
      "    evaluate all potentials, dumping .out every %d and .exyz every %d steps.\n",
      dump_interval_thermo_,
      dump_interval_exyz_);
  } else if (mode_.compare("average") == 0) {
    printf(
      "    use the average potential in the molecular dynamics run, and dump .out every %d and "
      ".exyz every %d steps.\n",
      dump_interval_thermo_,
      dump_interval_exyz_);
  }
}

void Dump_Observer::preprocess(
  const int number_of_steps,
  const double time_step,
  Integrate& integrate,
  std::vector<Group>& group,
  Atom& atom,
  Box& box,
  Force& force)
{
  // Setup a dump_exyz with the dump_interval for dump_observer.
  force.set_multiple_potentials_mode(mode_);
  if (dump_) {
    const int number_of_files = (mode_.compare("observe") == 0) ? force.potentials.size() : 1;
    for (int i = 0; i < number_of_files; i++) {
      const std::string file_number = (number_of_files == 1) ? "" : std::to_string(i);
      std::string exyz_filename = "observer" + file_number + ".xyz";
      exyz_files_.push_back(my_fopen(exyz_filename.c_str(), "a"));
      std::string thermo_filename = "observer" + file_number + ".out";
      thermo_files_.push_back(my_fopen(thermo_filename.c_str(), "a"));
    }
    gpu_total_virial_.resize(6);
    cpu_total_virial_.resize(6);
    if (has_force_) {
      cpu_force_per_atom_.resize(atom.number_of_atoms * 3);
    }
  }
}

void Dump_Observer::process(
  const int number_of_steps,
  int step,
  const int fixed_group,
  const int move_group,
  const double global_time,
  const double temperature,
  Integrate& integrate,
  Box& box,
  std::vector<Group>& group,
  GPU_Vector<double>& thermo,
  Atom& atom,
  Force& force)
{
  // Only run if should dump, since forces have to be recomputed with each potential.
  if (!dump_)
    return;
  if (((step + 1) % dump_interval_thermo_ != 0) & ((step + 1) % dump_interval_exyz_ != 0))
    return;

  int number_of_atoms_fixed = (fixed_group < 0) ? 0 : group[0].cpu_size[fixed_group];
  number_of_atoms_fixed += (move_group < 0) ? 0 : group[0].cpu_size[move_group];

  if (mode_.compare("observe") == 0) {
    // If observing, calculate properties with all potentials.
    const int number_of_potentials = force.potentials.size();
    const int number_of_atoms = atom.type.size();
    // Loop backwards over files to evaluate the main potential last, keeping it's properties intact
    for (int potential_index = number_of_potentials - 1; potential_index >= 0; potential_index--) {
      // Set potential/force/virials to zero
      initialize_properties<<<(number_of_atoms - 1) / 128 + 1, 128>>>(
        number_of_atoms,
        atom.force_per_atom.data(),
        atom.force_per_atom.data() + number_of_atoms,
        atom.force_per_atom.data() + number_of_atoms * 2,
        atom.potential_per_atom.data(),
        atom.virial_per_atom.data());
      GPU_CHECK_KERNEL
      // Compute new potential properties
      force.potentials[potential_index]->compute(
        box,
        atom.type,
        atom.position_per_atom,
        atom.potential_per_atom,
        atom.force_per_atom,
        atom.virial_per_atom);
      integrate.ensemble->find_thermo(
        false,
        box.get_volume(),
        group,
        atom.mass,
        atom.potential_per_atom,
        atom.velocity_per_atom,
        atom.virial_per_atom,
        thermo);
      // Write properties
      write_exyz(
        step,
        global_time,
        box,
        atom.cpu_atom_symbol,
        atom.cpu_type,
        atom.position_per_atom,
        atom.cpu_position_per_atom,
        atom.velocity_per_atom,
        atom.cpu_velocity_per_atom,
        atom.force_per_atom,
        atom.virial_per_atom,
        thermo,
        potential_index);
      write_thermo(step, number_of_atoms, number_of_atoms_fixed, box, thermo, potential_index);
    }
  } else if (mode_.compare("average") == 0) {
    // If average, dump already computed properties to file.
    const int number_of_atoms = atom.type.size();
    write_exyz(
      step,
      global_time,
      box,
      atom.cpu_atom_symbol,
      atom.cpu_type,
      atom.position_per_atom,
      atom.cpu_position_per_atom,
      atom.velocity_per_atom,
      atom.cpu_velocity_per_atom,
      atom.force_per_atom,
      atom.virial_per_atom,
      thermo,
      0);
    write_thermo(step, number_of_atoms, number_of_atoms_fixed, box, thermo, 0);
  } else {
    PRINT_INPUT_ERROR("Invalid observer mode.\n");
  }
}

void Dump_Observer::output_line2(
  const double time,
  const Box& box,
  const std::vector<std::string>& cpu_atom_symbol,
  GPU_Vector<double>& virial_per_atom,
  GPU_Vector<double>& gpu_thermo,
  FILE* fid_)
{
  // time
  fprintf(fid_, "Time=%.8f", time * TIME_UNIT_CONVERSION); // output time is in units of fs

  // PBC
  fprintf(
    fid_, " pbc=\"%c %c %c\"", box.pbc_x ? 'T' : 'F', box.pbc_y ? 'T' : 'F', box.pbc_z ? 'T' : 'F');

  // box
  fprintf(
    fid_,
    " Lattice=\"%.8f %.8f %.8f %.8f %.8f %.8f %.8f %.8f %.8f\"",
    box.cpu_h[0],
    box.cpu_h[3],
    box.cpu_h[6],
    box.cpu_h[1],
    box.cpu_h[4],
    box.cpu_h[7],
    box.cpu_h[2],
    box.cpu_h[5],
    box.cpu_h[8]);

  // energy and virial (symmetric tensor) in eV, and stress (symmetric tensor) in eV/A^3
  double cpu_thermo[8];
  gpu_thermo.copy_to_host(cpu_thermo, 8);
  const int N = virial_per_atom.size() / 9;
  gpu_sum<<<6, 1024>>>(N, virial_per_atom.data(), gpu_total_virial_.data());
  gpu_total_virial_.copy_to_host(cpu_total_virial_.data());

  fprintf(fid_, " energy=%.8f", cpu_thermo[1]);
  fprintf(
    fid_,
    " virial=\"%.8f %.8f %.8f %.8f %.8f %.8f %.8f %.8f %.8f\"",
    cpu_total_virial_[0],
    cpu_total_virial_[3],
    cpu_total_virial_[4],
    cpu_total_virial_[3],
    cpu_total_virial_[1],
    cpu_total_virial_[5],
    cpu_total_virial_[4],
    cpu_total_virial_[5],
    cpu_total_virial_[2]);
  fprintf(
    fid_,
    " stress=\"%.8f %.8f %.8f %.8f %.8f %.8f %.8f %.8f %.8f\"",
    cpu_thermo[2],
    cpu_thermo[5],
    cpu_thermo[6],
    cpu_thermo[5],
    cpu_thermo[3],
    cpu_thermo[7],
    cpu_thermo[6],
    cpu_thermo[7],
    cpu_thermo[4]);

  // Properties
  fprintf(fid_, " Properties=species:S:1:pos:R:3");

  if (has_velocity_) {
    fprintf(fid_, ":vel:R:3");
  }
  if (has_force_) {
    fprintf(fid_, ":forces:R:3");
  }

  // Over
  fprintf(fid_, "\n");
}

void Dump_Observer::write_exyz(
  const int step,
  const double global_time,
  const Box& box,
  const std::vector<std::string>& cpu_atom_symbol,
  const std::vector<int>& cpu_type,
  GPU_Vector<double>& position_per_atom,
  std::vector<double>& cpu_position_per_atom,
  GPU_Vector<double>& velocity_per_atom,
  std::vector<double>& cpu_velocity_per_atom,
  GPU_Vector<double>& force_per_atom,
  GPU_Vector<double>& virial_per_atom,
  GPU_Vector<double>& gpu_thermo,
  const int file_index)
{
  if (!dump_)
    return;
  if ((step + 1) % dump_interval_exyz_ != 0)
    return;

  const int num_atoms_total = position_per_atom.size() / 3;
  FILE* fid_ = exyz_files_[file_index];
  position_per_atom.copy_to_host(cpu_position_per_atom.data());
  if (has_velocity_) {
    velocity_per_atom.copy_to_host(cpu_velocity_per_atom.data());
  }
  if (has_force_) {
    force_per_atom.copy_to_host(cpu_force_per_atom_.data());
  }

  // line 1
  fprintf(fid_, "%d\n", num_atoms_total);

  // line 2
  output_line2(global_time, box, cpu_atom_symbol, virial_per_atom, gpu_thermo, fid_);

  // other lines
  for (int n = 0; n < num_atoms_total; n++) {
    fprintf(fid_, "%s", cpu_atom_symbol[n].c_str());
    for (int d = 0; d < 3; ++d) {
      fprintf(fid_, " %.8f", cpu_position_per_atom[n + num_atoms_total * d]);
    }
    if (has_velocity_) {
      const double natural_to_A_per_fs = 1.0 / TIME_UNIT_CONVERSION;
      for (int d = 0; d < 3; ++d) {
        fprintf(
          fid_, " %.8f", cpu_velocity_per_atom[n + num_atoms_total * d] * natural_to_A_per_fs);
      }
    }
    if (has_force_) {
      for (int d = 0; d < 3; ++d) {
        fprintf(fid_, " %.8f", cpu_force_per_atom_[n + num_atoms_total * d]);
      }
    }
    fprintf(fid_, "\n");
  }

  fflush(fid_);
}

void Dump_Observer::write_thermo(
  const int step,
  const int number_of_atoms,
  const int number_of_atoms_fixed,
  const Box& box,
  GPU_Vector<double>& gpu_thermo,
  const int file_index)
{
  if (!dump_)
    return;
  if ((step + 1) % dump_interval_thermo_ != 0)
    return;

  FILE* fid_ = thermo_files_[file_index];
  double thermo[8];
  gpu_thermo.copy_to_host(thermo, 8);

  const int number_of_atoms_moving = number_of_atoms - number_of_atoms_fixed;
  double energy_kin = 1.5 * number_of_atoms_moving * K_B * thermo[0];

  // stress components are in Voigt notation: xx, yy, zz, yz, xz, xy
  fprintf(
    fid_,
    "%20.10e%20.10e%20.10e%20.10e%20.10e%20.10e%20.10e%20.10e%20.10e",
    thermo[0],
    energy_kin,
    thermo[1],
    thermo[2] * PRESSURE_UNIT_CONVERSION,
    thermo[3] * PRESSURE_UNIT_CONVERSION,
    thermo[4] * PRESSURE_UNIT_CONVERSION,
    thermo[7] * PRESSURE_UNIT_CONVERSION,
    thermo[6] * PRESSURE_UNIT_CONVERSION,
    thermo[5] * PRESSURE_UNIT_CONVERSION);

  fprintf(
    fid_,
    "%20.10e%20.10e%20.10e%20.10e%20.10e%20.10e%20.10e%20.10e%20.10e\n",
    box.cpu_h[0],
    box.cpu_h[3],
    box.cpu_h[6],
    box.cpu_h[1],
    box.cpu_h[4],
    box.cpu_h[7],
    box.cpu_h[2],
    box.cpu_h[5],
    box.cpu_h[8]);
  fflush(fid_);
}

void Dump_Observer::postprocess(
  Atom& atom,
  Box& box,
  Integrate& integrate,
  const int number_of_steps,
  const double time_step,
  const double temperature)
{
  for (int i = 0; i < exyz_files_.size(); i++) {
    fclose(exyz_files_[i]);
  }
  for (int i = 0; i < thermo_files_.size(); i++) {
    fclose(thermo_files_[i]);
  }
  dump_ = false;
}
