#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan and GPUMD development team
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

/*----------------------------------------------------------------------------80
Calculate the stress autocorrelation function and viscosity.
------------------------------------------------------------------------------*/

#include "utilities/common.cuh"
#include "utilities/gpu_macro.cuh"
#include "utilities/read_file.cuh"
#include "viscosity.cuh"
#include <vector>
#include <cstring>

#define NUM_OF_COMPONENTS 9

void Viscosity::preprocess(
  const int number_of_steps,
  const double time_step,
  Integrate& integrate,
  std::vector<Group>& group,
  Atom& atom,
  Box& box,
  Force& force)
{
  if (compute) {
    int number_of_frames = number_of_steps / sample_interval;
    stress_all.resize(NUM_OF_COMPONENTS * number_of_frames);
  }
}

static __global__ void gpu_sum_stress(
  const int N,
  const int Nd,
  const int nd,
  const double* g_mass,
  const double* g_velocity,
  const double* g_virial,
  double* g_stress_all)
{
  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  const int number_of_rounds = (N - 1) / 1024 + 1;

  __shared__ double s_data[1024];
  s_data[tid] = 0.0;

  for (int round = 0; round < number_of_rounds; ++round) {
    const int n = tid + round * 1024;
    if (n < N) {
      // the virial tensor:
      // xx xy xz    0 3 4
      // yx yy yz    6 1 5
      // zx zy zz    7 8 2
      int a[NUM_OF_COMPONENTS] = {0, 1, 2, 0, 0, 1, 1, 2, 2};
      int b[NUM_OF_COMPONENTS] = {0, 1, 2, 1, 2, 2, 0, 0, 1};
      s_data[tid] += g_mass[n] * g_velocity[n + N * a[bid]] * g_velocity[n + N * b[bid]] +
                     g_virial[n + N * blockIdx.x];
    }
  }

  __syncthreads();
  for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1) {
    if (tid < offset) {
      s_data[tid] += s_data[tid + offset];
    }
    __syncthreads();
  }
  if (tid == 0) {
    g_stress_all[nd + Nd * blockIdx.x] = s_data[0];
  }
}

void Viscosity::process(
  const int number_of_steps,
  int step,
  const int fixed_group,
  const int move_group,
  const double global_time,
  const double temperature,
  Integrate& integrate,
  Box& box,
  std::vector<Group>& group,
  GPU_Vector<double>& thermo,
  Atom& atom,
  Force& force)
{
  if (!compute)
    return;
  if ((step + 1) % sample_interval != 0)
    return;

  const int N = atom.number_of_atoms;

  int nd = (step + 1) / sample_interval - 1;
  int Nd = number_of_steps / sample_interval;
  gpu_sum_stress<<<NUM_OF_COMPONENTS, 1024>>>(
    N, Nd, nd, atom.mass.data(), atom.velocity_per_atom.data(), atom.virial_per_atom.data(), stress_all.data());
  GPU_CHECK_KERNEL
}

static __global__ void gpu_correct_stress(const int Nd, double* g_stress_all)
{
  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  const int number_of_rounds = (Nd - 1) / 1024 + 1;

  __shared__ double s_data[1024];
  s_data[tid] = 0.0;

  for (int round = 0; round < number_of_rounds; ++round) {
    const int n = tid + round * 1024;
    if (n < Nd) {
      s_data[tid] += g_stress_all[n + bid * Nd];
    }
  }

  __syncthreads();
  for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1) {
    if (tid < offset) {
      s_data[tid] += s_data[tid + offset];
    }
    __syncthreads();
  }

  for (int round = 0; round < number_of_rounds; ++round) {
    const int n = tid + round * 1024;
    if (n < Nd) {
      g_stress_all[n + bid * Nd] -= s_data[0] / Nd;
    }
  }
}

static __global__ void
gpu_find_correlation(const int Nc, const int Nd, const double* g_stress, double* g_correlation)
{
  __shared__ double s_correlation[NUM_OF_COMPONENTS][128];

  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int number_of_rounds = (Nd - 1) / 128 + 1;
  int number_of_data = Nd - bid;

  for (int k = 0; k < NUM_OF_COMPONENTS; ++k) {
    s_correlation[k][tid] = 0.0;
  }

  for (int round = 0; round < number_of_rounds; ++round) {
    int index = tid + round * 128;
    if (index + bid < Nd) {
      for (int k = 0; k < NUM_OF_COMPONENTS; ++k) {
        s_correlation[k][tid] += g_stress[index + Nd * k] * g_stress[index + bid + Nd * k];
      }
    }
  }
  __syncthreads();

  for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1) {
    if (tid < offset) {
      for (int k = 0; k < NUM_OF_COMPONENTS; ++k) {
        s_correlation[k][tid] += s_correlation[k][tid + offset];
      }
    }
    __syncthreads();
  }

  if (tid == 0) {
    for (int k = 0; k < NUM_OF_COMPONENTS; ++k) {
      g_correlation[bid + Nc * k] = s_correlation[k][0] / number_of_data;
    }
  }
}

static void
find_viscosity(const int Nc, const double factor, const double* correlation, double* viscosity)
{
  for (int k = 0; k < NUM_OF_COMPONENTS; k++) {
    for (int nc = 1; nc < Nc; nc++) {
      const int index = Nc * k + nc;
      viscosity[index] =
        viscosity[index - 1] + (correlation[index - 1] + correlation[index]) * factor;
    }
  }
}

void Viscosity::postprocess(
  Atom& atom,
  Box& box,
  Integrate& integrate,
  const int number_of_steps,
  const double time_step,
  const double temperature)
{
  if (!compute)
    return;
  print_line_1();
  printf("Start to calculate viscosity.\n");

  const int Nd = number_of_steps / sample_interval;
  const double dt = time_step * sample_interval;
  const double dt_in_ps = dt * TIME_UNIT_CONVERSION / 1000.0; // ps

  std::vector<double> viscosity(Nc * NUM_OF_COMPONENTS, 0.0);
  GPU_Vector<double> correlation_gpu(Nc * NUM_OF_COMPONENTS);
  std::vector<double> correlation_cpu(Nc * NUM_OF_COMPONENTS);

  gpu_correct_stress<<<NUM_OF_COMPONENTS, 1024>>>(Nd, stress_all.data());
  GPU_CHECK_KERNEL
  gpu_find_correlation<<<Nc, 128>>>(Nc, Nd, stress_all.data(), correlation_gpu.data());
  GPU_CHECK_KERNEL

  correlation_gpu.copy_to_host(correlation_cpu.data());

  double factor = dt * 0.5 / (K_B * temperature * box.get_volume());
  factor *= PRESSURE_UNIT_CONVERSION * TIME_UNIT_CONVERSION * 1.0e-6; // Pa s

  find_viscosity(Nc, factor, correlation_cpu.data(), viscosity.data());

  FILE* fid = fopen("viscosity.out", "a");
  for (int nc = 0; nc < Nc; nc++) {
    fprintf(fid, "%25.15e", nc * dt_in_ps);
    for (int m = 0; m < NUM_OF_COMPONENTS; m++) {
      fprintf(fid, "%25.15e", correlation_cpu[Nc * m + nc]);
    }
    for (int m = 0; m < NUM_OF_COMPONENTS; m++) {
      fprintf(fid, "%25.15e", viscosity[Nc * m + nc]);
    }
    fprintf(fid, "\n");
  }
  fflush(fid);
  fclose(fid);

  printf("Viscosity is calculated.\n");
  print_line_2();

  compute = 0;
}

Viscosity::Viscosity(const char** param, int num_param)
{
  parse(param, num_param);
  property_name = "compute_viscosity";
}

void Viscosity::parse(const char** param, int num_param)
{
  compute = 1;

  printf("Compute Viscosity.\n");

  if (num_param != 3) {
    PRINT_INPUT_ERROR("compute_viscosity should have 2 parameters.\n");
  }

  if (!is_valid_int(param[1], &sample_interval)) {
    PRINT_INPUT_ERROR("sample interval for viscosity should be an integer number.\n");
  }
  printf("    sample interval is %d.\n", sample_interval);

  if (!is_valid_int(param[2], &Nc)) {
    PRINT_INPUT_ERROR("Nc for viscosity should be an integer number.\n");
  }
  printf("    Nc is %d\n", Nc);
}
