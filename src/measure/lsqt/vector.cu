#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

#include "utilities/error.cuh"
#include "vector.cuh"
#include <string.h>    // memcpy
#define BLOCK_SIZE 512 // optimized

__global__ void gpu_set_zero(int number_of_elements, real* g_state_real, real* g_state_imag)
{
  int n = blockIdx.x * blockDim.x + threadIdx.x;
  if (n < number_of_elements) {
    g_state_real[n] = 0;
    g_state_imag[n] = 0;
  }
}

void Vector::initialize_gpu(int n)
{
  this->n = n;
  array_size = n * sizeof(real);
  CHECK(hipMalloc((void**)&real_part, array_size));
  CHECK(hipMalloc((void**)&imag_part, array_size));
}

Vector::Vector(int n)
{
  initialize_gpu(n);
  gpu_set_zero<<<(n - 1) / BLOCK_SIZE + 1, BLOCK_SIZE>>>(n, real_part, imag_part);
  CHECK(hipGetLastError());
}

__global__ void gpu_copy_state(int N, real* in_real, real* in_imag, real* out_real, real* out_imag)
{
  int n = blockIdx.x * blockDim.x + threadIdx.x;
  if (n < N) {
    out_real[n] = in_real[n];
    out_imag[n] = in_imag[n];
  }
}

Vector::Vector(Vector& original)
{
  // Just teach myself: one can access private members of another instance
  // of the class from within the class
  initialize_gpu(original.n);
  gpu_copy_state<<<(n - 1) / BLOCK_SIZE + 1, BLOCK_SIZE>>>(
    n, original.real_part, original.imag_part, real_part, imag_part);
  CHECK(hipGetLastError());
}

Vector::~Vector()
{
  CHECK(hipFree(real_part));
  CHECK(hipFree(imag_part));
}

__global__ void gpu_add_state(int n, real* in_real, real* in_imag, real* out_real, real* out_imag)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    out_real[i] += in_real[i];
    out_imag[i] += in_imag[i];
  }
}

void Vector::add(Vector& other)
{
  gpu_add_state<<<(n - 1) / BLOCK_SIZE + 1, BLOCK_SIZE>>>(
    n, other.real_part, other.imag_part, real_part, imag_part);
  CHECK(hipGetLastError());
}

void Vector::copy(Vector& other)
{
  gpu_copy_state<<<(n - 1) / BLOCK_SIZE + 1, BLOCK_SIZE>>>(
    n, other.real_part, other.imag_part, real_part, imag_part);
  CHECK(hipGetLastError());
}

__global__ void gpu_apply_sz(int n, real* in_real, real* in_imag, real* out_real, real* out_imag)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    if (i % 2 == 0) {
      out_real[i] = in_real[i];
      out_imag[i] = in_imag[i];
    } else {
      out_real[i] = -in_real[i];
      out_imag[i] = -in_imag[i];
    }
  }
}

void Vector::apply_sz(Vector& other)
{
  gpu_apply_sz<<<(n - 1) / BLOCK_SIZE + 1, BLOCK_SIZE>>>(
    n, other.real_part, other.imag_part, real_part, imag_part);
  CHECK(hipGetLastError());
}

void Vector::copy_from_host(real* other_real, real* other_imag)
{
  CHECK(hipMemcpy(real_part, other_real, array_size, hipMemcpyHostToDevice));
  CHECK(hipMemcpy(imag_part, other_imag, array_size, hipMemcpyHostToDevice));
}

void Vector::copy_to_host(real* target_real, real* target_imag)
{
  CHECK(hipMemcpy(target_real, real_part, array_size, hipMemcpyDeviceToHost));
  CHECK(hipMemcpy(target_imag, imag_part, array_size, hipMemcpyDeviceToHost));
}

void Vector::swap(Vector& other)
{
  real* tmp_real = real_part;
  real* tmp_imag = imag_part;
  real_part = other.real_part, imag_part = other.imag_part;
  other.real_part = tmp_real;
  other.imag_part = tmp_imag;
}

__device__ void warp_reduce(volatile real* s, int t)
{
  s[t] += s[t + 32];
  s[t] += s[t + 16];
  s[t] += s[t + 8];
  s[t] += s[t + 4];
  s[t] += s[t + 2];
  s[t] += s[t + 1];
}

__global__ void gpu_find_inner_product_1(
  int number_of_atoms,
  real* g_final_state_real,
  real* g_final_state_imag,
  real* g_random_state_real,
  real* g_random_state_imag,
  real* g_inner_product_real,
  real* g_inner_product_imag,
  int g_offset)
{
  int tid = threadIdx.x;
  int n = blockIdx.x * blockDim.x + tid;
  int m;
  real a, b, c, d;
  __shared__ real s_data_real[BLOCK_SIZE];
  __shared__ real s_data_imag[BLOCK_SIZE];
  s_data_real[tid] = 0.0;
  s_data_imag[tid] = 0.0;

  if (n < number_of_atoms) {
    a = g_final_state_real[n];
    b = g_final_state_imag[n];
    c = g_random_state_real[n];
    d = g_random_state_imag[n];
    s_data_real[tid] = (a * c + b * d);
    s_data_imag[tid] = (b * c - a * d);
  }
  __syncthreads();

  if (tid < 256) {
    m = tid + 256;
    s_data_real[tid] += s_data_real[m];
    s_data_imag[tid] += s_data_imag[m];
  }
  __syncthreads();
  if (tid < 128) {
    m = tid + 128;
    s_data_real[tid] += s_data_real[m];
    s_data_imag[tid] += s_data_imag[m];
  }
  __syncthreads();
  if (tid < 64) {
    m = tid + 64;
    s_data_real[tid] += s_data_real[m];
    s_data_imag[tid] += s_data_imag[m];
  }
  __syncthreads();
  if (tid < 32) {
    warp_reduce(s_data_real, tid);
    warp_reduce(s_data_imag, tid);
  }
  if (tid == 0) {
    g_inner_product_real[blockIdx.x + g_offset] = s_data_real[0];
    g_inner_product_imag[blockIdx.x + g_offset] = s_data_imag[0];
  }
}

void Vector::inner_product_1(int number_of_atoms, Vector& other, Vector& target, int offset)
{
  int grid_size = (number_of_atoms - 1) / BLOCK_SIZE + 1;
  gpu_find_inner_product_1<<<grid_size, BLOCK_SIZE>>>(
    number_of_atoms,
    real_part,
    imag_part,
    other.real_part,
    other.imag_part,
    target.real_part,
    target.imag_part,
    offset);
  CHECK(hipGetLastError());
}

__global__ void gpu_find_inner_product_2(
  int number_of_atoms,
  real* g_inner_product_1_real,
  real* g_inner_product_1_imag,
  real* g_inner_product_2_real,
  real* g_inner_product_2_imag)
{
  //<<<para.number_of_energy_points, BLOCK_SIZE)>>>
  int tid = threadIdx.x;
  int patch, n, m;

  __shared__ real s_data_real[BLOCK_SIZE];
  __shared__ real s_data_imag[BLOCK_SIZE];
  s_data_real[tid] = 0.0;
  s_data_imag[tid] = 0.0;
  int number_of_blocks = (number_of_atoms - 1) / BLOCK_SIZE + 1;
  int number_of_patches = (number_of_blocks - 1) / BLOCK_SIZE + 1;

  for (patch = 0; patch < number_of_patches; ++patch) {
    n = tid + patch * BLOCK_SIZE;
    if (n < number_of_blocks) {
      m = blockIdx.x * number_of_blocks + n;
      s_data_real[tid] += g_inner_product_1_real[m];
      s_data_imag[tid] += g_inner_product_1_imag[m];
    }
  }
  __syncthreads();

  if (tid < 256) {
    m = tid + 256;
    s_data_real[tid] += s_data_real[m];
    s_data_imag[tid] += s_data_imag[m];
  }
  __syncthreads();
  if (tid < 128) {
    m = tid + 128;
    s_data_real[tid] += s_data_real[m];
    s_data_imag[tid] += s_data_imag[m];
  }
  __syncthreads();
  if (tid < 64) {
    m = tid + 64;
    s_data_real[tid] += s_data_real[m];
    s_data_imag[tid] += s_data_imag[m];
  }
  __syncthreads();
  if (tid < 32) {
    warp_reduce(s_data_real, tid);
    warp_reduce(s_data_imag, tid);
  }
  if (tid == 0) {
    g_inner_product_2_real[blockIdx.x] = s_data_real[0];
    g_inner_product_2_imag[blockIdx.x] = s_data_imag[0];
  }
}

void Vector::inner_product_2(int number_of_atoms, int number_of_moments, Vector& target)
{
  gpu_find_inner_product_2<<<number_of_moments, BLOCK_SIZE>>>(
    number_of_atoms, real_part, imag_part, target.real_part, target.imag_part);
  CHECK(hipGetLastError());
}
