#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

#include "model.cuh"
#include "vector.cuh"
#include <chrono>
#include <fstream>
#include <iostream>
#include <random>
#include <sstream>

#define PI 3.141592653589793

Model::Model(std::string input_dir)
{
#ifdef DEBUG
  // use the same seed for different runs
  generator = std::mt19937(12345678);
#else
  // use different seeds for different runs
  generator = std::mt19937(std::chrono::system_clock::now().time_since_epoch().count());
#endif

  // determine the input directory
  this->input_dir = input_dir;

  // read in para.in
  initialize_parameters();

  initialize_model_general();

  // always need to read in energies
  initialize_energy();

  // only read in time steps when needed
  if (requires_time)
    initialize_time();
}

Model::~Model()
{
  // other memory will be freed when constructing the Hamiltonian
  delete[] energy;
  if (requires_time)
    delete[] time_step;
}

// This function is called by the lsqt function in the lsqt.cu file
// It initializes a random vector
void Model::initialize_state(Vector& random_state)
{
  std::uniform_real_distribution<real> phase(0, 2 * PI);
  real* random_state_real = new real[number_of_atoms];
  real* random_state_imag = new real[number_of_atoms];

  for (int n = 0; n < number_of_atoms; ++n) {
    real random_phase = phase(generator);
    random_state_real[n] = cos(random_phase);
    random_state_imag[n] = sin(random_phase);
  }

  random_state.copy_from_host(random_state_real, random_state_imag);
  delete[] random_state_real;
  delete[] random_state_imag;
}

void Model::print_started_reading(std::string filename)
{
  std::cout << std::endl;
  std::cout << "===========================================================";
  std::cout << std::endl;
  std::cout << "Started reading " + filename << std::endl;
  std::cout << std::endl;
}

void Model::print_finished_reading(std::string filename)
{
  std::cout << std::endl;
  std::cout << "Finished reading " + filename << std::endl;
  std::cout << "===========================================================";
  std::cout << std::endl << std::endl;
}

void Model::verify_parameters()
{
  // determine whether or not we need to read in time steps
  if (calculate_vac || calculate_msd || calculate_spin)
    requires_time = true;

  std::cout << "- Use general model" << std::endl;
  if (anderson.has_disorder) {
    std::cout << "Error: General model does not allowed to add "
              << "Anderson disorder" << std::endl;
    exit(1);
  }
  if (has_vacancy_disorder) {
    std::cout << "Error: General model does not allowed to add "
              << "vacancy disorder" << std::endl;
    exit(1);
  }
  if (charge.has) {
    std::cout << "Error: General model does not allowed to add "
              << "charged impurities" << std::endl;
    exit(1);
  }

  std::cout << "- DOS will be calculated" << std::endl;

  if (calculate_vac0)
    std::cout << "- VAC0 will be calculated" << std::endl;
  else
    std::cout << "- VAC0 will not be calculated" << std::endl;

  if (calculate_vac)
    std::cout << "- VAC will be calculated" << std::endl;
  else
    std::cout << "- VAC will not be calculated" << std::endl;

  if (calculate_msd)
    std::cout << "- MSD will be calculated" << std::endl;
  else
    std::cout << "- MSD will not be calculated" << std::endl;

  if (calculate_spin)
    std::cout << "- spin polarization will be calculated" << std::endl;
  else
    std::cout << "- spin polarization will not be calculated" << std::endl;

  if (calculate_spin && calculate_vac0) {
    std::cout << "Error: spin and VAC0 cannot be calculated together" << std::endl;
    exit(1);
  }

  if (calculate_spin && calculate_vac) {
    std::cout << "Error: spin and VAC cannot be calculated together" << std::endl;
    exit(1);
  }

  if (calculate_spin && calculate_msd) {
    std::cout << "Error: spin and MSD cannot be calculated together" << std::endl;
    exit(1);
  }

  std::cout << "- Number of random vectors is " << number_of_random_vectors << std::endl;
  if (number_of_random_vectors <= 0) {
    std::cout << "Error: Number of random vectors should > 0" << std::endl;
    exit(1);
  }

  std::cout << "- Number of moments is " << number_of_moments << std::endl;
  if (number_of_moments <= 0) {
    std::cout << "Error: Number of moments should > 0" << std::endl;
    exit(1);
  }

  std::cout << "- Energy maximum is " << energy_max << std::endl;
  if (energy_max <= 0) {
    std::cout << "Error: Energy maximum should > 0" << std::endl;
    exit(1);
  }
}

void Model::initialize_parameters()
{
  std::string filename = input_dir + "/para.in";
  std::ifstream input(filename);
  if (!input.is_open()) {
    std::cout << "Error: cannot open " + filename << std::endl;
    exit(1);
  }
  print_started_reading(filename);

  std::string line;
  while (std::getline(input, line)) {
    std::stringstream ss(line);
    std::string token;
    ss >> token;
    if (token == "")
      continue;
    if (token == "anderson_disorder") {
      anderson.has_disorder = true;
      ss >> anderson.disorder_strength;
    } else if (token == "charged_impurity") {
      charge.has = true;
      ss >> charge.Ni;
      ss >> charge.W;
      ss >> charge.xi;
    } else if (token == "vacancy_disorder") {
      has_vacancy_disorder = true;
      ss >> number_of_vacancies;
    } else if (token == "calculate_vac0") {
      calculate_vac0 = true;
    } else if (token == "calculate_vac") {
      calculate_vac = true;
    } else if (token == "calculate_msd") {
      calculate_msd = true;
    } else if (token == "calculate_spin") {
      calculate_spin = true;
    } else if (token == "calculate_ldos") {
      calculate_ldos = true;
    } else if (token == "number_of_random_vectors") {
      ss >> number_of_random_vectors;
    } else if (token == "number_of_moments") {
      ss >> number_of_moments;
    } else if (token == "energy_max") {
      ss >> energy_max;
    } else {
      std::cout << "Error: Unknown identifier in " << input_dir + "/para.in: " + line << std::endl;
      std::cout << "Valid keywords include: " << std::endl
                << "--model" << std::endl
                << "--anderson_disorder" << std::endl
                << "--charged_impurity" << std::endl
                << "--vacancy_disorder" << std::endl
                << "--calculate_vac0" << std::endl
                << "--calculate_vac" << std::endl
                << "--calculate_msd" << std::endl
                << "--calculate_spin" << std::endl
                << "--calculate_ldos" << std::endl
                << "--number_of_random_vectors" << std::endl
                << "--number_of_moments" << std::endl
                << "--energy_max" << std::endl;
      exit(1);
    }
  }
  input.close();
  verify_parameters();
  print_finished_reading(filename);
}

void Model::initialize_energy()
{
  std::string filename = input_dir + "/energy.in";
  std::ifstream input(filename);
  if (!input.is_open()) {
    std::cout << "Error: cannot open " + filename << std::endl;
    exit(1);
  }

  print_started_reading(filename);

  input >> number_of_energy_points;
  std::cout << "- number of energy points = " << number_of_energy_points << std::endl;
  energy = new real[number_of_energy_points];

  for (int n = 0; n < number_of_energy_points; ++n) {
    input >> energy[n];
  }

  input.close();

  print_finished_reading(filename);
}

void Model::initialize_time()
{
  std::string filename = input_dir + "/time_step.in";
  std::ifstream input(filename);

  if (!input.is_open()) {
    std::cout << "Error: cannot open " + filename << std::endl;
    exit(1);
  }
  print_started_reading(filename);

  input >> number_of_steps_correlation;
  std::cout << "- number of time steps = " << number_of_steps_correlation << std::endl;
  time_step = new real[number_of_steps_correlation];

  for (int n = 0; n < number_of_steps_correlation; ++n) {
    input >> time_step[n];
  }

  input.close();
  print_finished_reading(filename);
}
