#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

#include "hamiltonian.cuh"
#include "model.cuh"
#include "utilities/error.cuh"
#include "vector.cuh"
#include <string.h>    // memcpy
#define BLOCK_SIZE 512 // optimized

void Hamiltonian::initialize_gpu(Model& model)
{
  n = model.number_of_atoms;
  max_neighbor = model.max_neighbor;
  energy_max = model.energy_max;
  grid_size = (model.number_of_atoms - 1) / BLOCK_SIZE + 1;

  CHECK(hipMalloc((void**)&neighbor_number, sizeof(int) * n));
  CHECK(hipMalloc((void**)&neighbor_list, sizeof(int) * model.number_of_pairs));
  CHECK(hipMalloc((void**)&potential, sizeof(real) * n));
  CHECK(hipMalloc((void**)&hopping_real, sizeof(real) * model.number_of_pairs));
  CHECK(hipMalloc((void**)&hopping_imag, sizeof(real) * model.number_of_pairs));
  CHECK(hipMalloc((void**)&xx, sizeof(real) * model.number_of_pairs));
}

Hamiltonian::Hamiltonian(Model& model) { initialize_gpu(model); }

Hamiltonian::~Hamiltonian()
{
  CHECK(hipFree(neighbor_number));
  CHECK(hipFree(neighbor_list));
  CHECK(hipFree(potential));
  CHECK(hipFree(hopping_real));
  CHECK(hipFree(hopping_imag));
  CHECK(hipFree(xx));
}

__global__ void gpu_apply_hamiltonian(
  int number_of_atoms,
  real energy_max,
  int* g_neighbor_number,
  int* g_neighbor_list,
  real* g_potential,
  real* g_hopping_real,
  real* g_hopping_imag,
  real* g_state_in_real,
  real* g_state_in_imag,
  real* g_state_out_real,
  real* g_state_out_imag)
{
  int n = blockIdx.x * blockDim.x + threadIdx.x;
  if (n < number_of_atoms) {
    real temp_real = g_potential[n] * g_state_in_real[n]; // on-site
    real temp_imag = g_potential[n] * g_state_in_imag[n]; // on-site

    for (int m = 0; m < g_neighbor_number[n]; ++m) {
      int index_1 = m * number_of_atoms + n;
      int index_2 = g_neighbor_list[index_1];
      real a = g_hopping_real[index_1];
      real b = g_hopping_imag[index_1];
      real c = g_state_in_real[index_2];
      real d = g_state_in_imag[index_2];
      temp_real += a * c - b * d; // hopping
      temp_imag += a * d + b * c; // hopping
    }
    temp_real /= energy_max; // scale
    temp_imag /= energy_max; // scale
    g_state_out_real[n] = temp_real;
    g_state_out_imag[n] = temp_imag;
  }
}

// |output> = H |input>
void Hamiltonian::apply(Vector& input, Vector& output)
{
  gpu_apply_hamiltonian<<<grid_size, BLOCK_SIZE>>>(
    n,
    energy_max,
    neighbor_number,
    neighbor_list,
    potential,
    hopping_real,
    hopping_imag,
    input.real_part,
    input.imag_part,
    output.real_part,
    output.imag_part);
  CHECK(hipGetLastError());
}

__global__ void gpu_apply_commutator(
  int number_of_atoms,
  real energy_max,
  int* g_neighbor_number,
  int* g_neighbor_list,
  real* g_hopping_real,
  real* g_hopping_imag,
  real* g_xx,
  real* g_state_in_real,
  real* g_state_in_imag,
  real* g_state_out_real,
  real* g_state_out_imag)
{
  int n = blockIdx.x * blockDim.x + threadIdx.x;
  if (n < number_of_atoms) {
    real temp_real = 0.0;
    real temp_imag = 0.0;
    for (int m = 0; m < g_neighbor_number[n]; ++m) {
      int index_1 = m * number_of_atoms + n;
      int index_2 = g_neighbor_list[index_1];
      real a = g_hopping_real[index_1];
      real b = g_hopping_imag[index_1];
      real c = g_state_in_real[index_2];
      real d = g_state_in_imag[index_2];
      real xx = g_xx[index_1];
      temp_real -= (a * c - b * d) * xx;
      temp_imag -= (a * d + b * c) * xx;
    }
    g_state_out_real[n] = temp_real / energy_max; // scale
    g_state_out_imag[n] = temp_imag / energy_max; // scale
  }
}

// |output> = [X, H] |input>
void Hamiltonian::apply_commutator(Vector& input, Vector& output)
{
  gpu_apply_commutator<<<grid_size, BLOCK_SIZE>>>(
    n,
    energy_max,
    neighbor_number,
    neighbor_list,
    hopping_real,
    hopping_imag,
    xx,
    input.real_part,
    input.imag_part,
    output.real_part,
    output.imag_part);
  CHECK(hipGetLastError());
}

__global__ void gpu_apply_current(
  int number_of_atoms,
  int* g_neighbor_number,
  int* g_neighbor_list,
  real* g_hopping_real,
  real* g_hopping_imag,
  real* g_xx,
  real* g_state_in_real,
  real* g_state_in_imag,
  real* g_state_out_real,
  real* g_state_out_imag)
{
  int n = blockIdx.x * blockDim.x + threadIdx.x;
  if (n < number_of_atoms) {
    real temp_real = 0.0;
    real temp_imag = 0.0;
    for (int m = 0; m < g_neighbor_number[n]; ++m) {
      int index_1 = m * number_of_atoms + n;
      int index_2 = g_neighbor_list[index_1];
      real a = g_hopping_real[index_1];
      real b = g_hopping_imag[index_1];
      real c = g_state_in_real[index_2];
      real d = g_state_in_imag[index_2];
      temp_real += (a * c - b * d) * g_xx[index_1];
      temp_imag += (a * d + b * c) * g_xx[index_1];
    }
    g_state_out_real[n] = +temp_imag;
    g_state_out_imag[n] = -temp_real;
  }
}

// |output> = V |input>
void Hamiltonian::apply_current(Vector& input, Vector& output)
{
  gpu_apply_current<<<grid_size, BLOCK_SIZE>>>(
    n,
    neighbor_number,
    neighbor_list,
    hopping_real,
    hopping_imag,
    xx,
    input.real_part,
    input.imag_part,
    output.real_part,
    output.imag_part);
  CHECK(hipGetLastError());
}

// Kernel which calculates the two first terms of time evolution as described by
// Eq. (36) in [Comput. Phys. Commun.185, 28 (2014)].
__global__ void gpu_chebyshev_01(
  int number_of_atoms,
  real* g_state_0_real,
  real* g_state_0_imag,
  real* g_state_1_real,
  real* g_state_1_imag,
  real* g_state_real,
  real* g_state_imag,
  real b0,
  real b1,
  int direction)
{
  int n = blockIdx.x * blockDim.x + threadIdx.x;
  if (n < number_of_atoms) {
    real bessel_0 = b0;
    real bessel_1 = b1 * direction;
    g_state_real[n] = bessel_0 * g_state_0_real[n] + bessel_1 * g_state_1_imag[n];
    g_state_imag[n] = bessel_0 * g_state_0_imag[n] - bessel_1 * g_state_1_real[n];
  }
}

// Wrapper for the kernel above
void Hamiltonian::chebyshev_01(
  Vector& state_0, Vector& state_1, Vector& state, real bessel_0, real bessel_1, int direction)
{
  gpu_chebyshev_01<<<grid_size, BLOCK_SIZE>>>(
    n,
    state_0.real_part,
    state_0.imag_part,
    state_1.real_part,
    state_1.imag_part,
    state.real_part,
    state.imag_part,
    bessel_0,
    bessel_1,
    direction);
  CHECK(hipGetLastError());
}

// Kernel for calculating further terms of Eq. (36)
// in [Comput. Phys. Commun.185, 28 (2014)].
__global__ void gpu_chebyshev_2(
  int number_of_atoms,
  real energy_max,
  int* g_neighbor_number,
  int* g_neighbor_list,
  real* g_potential,
  real* g_hopping_real,
  real* g_hopping_imag,
  real* g_state_0_real,
  real* g_state_0_imag,
  real* g_state_1_real,
  real* g_state_1_imag,
  real* g_state_2_real,
  real* g_state_2_imag,
  real* g_state_real,
  real* g_state_imag,
  real bessel_m,
  int label)
{
  int n = blockIdx.x * blockDim.x + threadIdx.x;
  if (n < number_of_atoms) {
    real temp_real = g_potential[n] * g_state_1_real[n]; // on-site
    real temp_imag = g_potential[n] * g_state_1_imag[n]; // on-site

    for (int m = 0; m < g_neighbor_number[n]; ++m) {
      int index_1 = m * number_of_atoms + n;
      int index_2 = g_neighbor_list[index_1];
      real a = g_hopping_real[index_1];
      real b = g_hopping_imag[index_1];
      real c = g_state_1_real[index_2];
      real d = g_state_1_imag[index_2];
      temp_real += a * c - b * d; // hopping
      temp_imag += a * d + b * c; // hopping
    }
    temp_real /= energy_max; // scale
    temp_imag /= energy_max; // scale

    temp_real = 2.0 * temp_real - g_state_0_real[n];
    temp_imag = 2.0 * temp_imag - g_state_0_imag[n];
    switch (label) {
      case 1: {
        g_state_real[n] += bessel_m * temp_real;
        g_state_imag[n] += bessel_m * temp_imag;
        break;
      }
      case 2: {
        g_state_real[n] -= bessel_m * temp_real;
        g_state_imag[n] -= bessel_m * temp_imag;
        break;
      }
      case 3: {
        g_state_real[n] += bessel_m * temp_imag;
        g_state_imag[n] -= bessel_m * temp_real;
        break;
      }
      case 4: {
        g_state_real[n] -= bessel_m * temp_imag;
        g_state_imag[n] += bessel_m * temp_real;
        break;
      }
    }
    g_state_2_real[n] = temp_real;
    g_state_2_imag[n] = temp_imag;
  }
}

// Wrapper for the kernel above
void Hamiltonian::chebyshev_2(
  Vector& state_0, Vector& state_1, Vector& state_2, Vector& state, real bessel_m, int label)
{
  gpu_chebyshev_2<<<grid_size, BLOCK_SIZE>>>(
    n,
    energy_max,
    neighbor_number,
    neighbor_list,
    potential,
    hopping_real,
    hopping_imag,
    state_0.real_part,
    state_0.imag_part,
    state_1.real_part,
    state_1.imag_part,
    state_2.real_part,
    state_2.imag_part,
    state.real_part,
    state.imag_part,
    bessel_m,
    label);
  CHECK(hipGetLastError());
}

// Kernel which calculates the two first terms of commutator [X, U(dt)]
// Corresponds to Eq. (37) in [Comput. Phys. Commun.185, 28 (2014)].
__global__ void gpu_chebyshev_1x(
  int number_of_atoms,
  real* g_state_1x_real,
  real* g_state_1x_imag,
  real* g_state_real,
  real* g_state_imag,
  real g_bessel_1)
{
  int n = blockIdx.x * blockDim.x + threadIdx.x;
  if (n < number_of_atoms) {
    real b1 = g_bessel_1;
    g_state_real[n] = +b1 * g_state_1x_imag[n];
    g_state_imag[n] = -b1 * g_state_1x_real[n];
  }
}

// Wrapper for kernel above
void Hamiltonian::chebyshev_1x(Vector& input, Vector& output, real bessel_1)
{
  gpu_chebyshev_1x<<<grid_size, BLOCK_SIZE>>>(
    n, input.real_part, input.imag_part, output.real_part, output.imag_part, bessel_1);
  CHECK(hipGetLastError());
}

// Kernel which calculates the further terms of [X, U(dt)]
__global__ void gpu_chebyshev_2x(
  int number_of_atoms,
  real energy_max,
  int* g_neighbor_number,
  int* g_neighbor_list,
  real* g_potential,
  real* g_hopping_real,
  real* g_hopping_imag,
  real* g_xx,
  real* g_state_0_real,
  real* g_state_0_imag,
  real* g_state_0x_real,
  real* g_state_0x_imag,
  real* g_state_1_real,
  real* g_state_1_imag,
  real* g_state_1x_real,
  real* g_state_1x_imag,
  real* g_state_2_real,
  real* g_state_2_imag,
  real* g_state_2x_real,
  real* g_state_2x_imag,
  real* g_state_real,
  real* g_state_imag,
  real g_bessel_m,
  int g_label)
{
  int n = blockIdx.x * blockDim.x + threadIdx.x;
  if (n < number_of_atoms) {
    real temp_real = g_potential[n] * g_state_1_real[n];    // on-site
    real temp_imag = g_potential[n] * g_state_1_imag[n];    // on-site
    real temp_x_real = g_potential[n] * g_state_1x_real[n]; // on-site
    real temp_x_imag = g_potential[n] * g_state_1x_imag[n]; // on-site

    for (int m = 0; m < g_neighbor_number[n]; ++m) {
      int index_1 = m * number_of_atoms + n;
      int index_2 = g_neighbor_list[index_1];

      real a = g_hopping_real[index_1];
      real b = g_hopping_imag[index_1];
      real c = g_state_1_real[index_2];
      real d = g_state_1_imag[index_2];
      temp_real += a * c - b * d; // hopping
      temp_imag += a * d + b * c; // hopping

      real cx = g_state_1x_real[index_2];
      real dx = g_state_1x_imag[index_2];
      temp_x_real += a * cx - b * dx; // hopping
      temp_x_imag += a * dx + b * cx; // hopping

      real xx = g_xx[index_1];
      temp_x_real -= (a * c - b * d) * xx; // hopping
      temp_x_imag -= (a * d + b * c) * xx; // hopping
    }

    temp_real /= energy_max; // scale
    temp_imag /= energy_max; // scale
    temp_real = 2.0 * temp_real - g_state_0_real[n];
    temp_imag = 2.0 * temp_imag - g_state_0_imag[n];
    g_state_2_real[n] = temp_real;
    g_state_2_imag[n] = temp_imag;

    temp_x_real /= energy_max; // scale
    temp_x_imag /= energy_max; // scale
    temp_x_real = 2.0 * temp_x_real - g_state_0x_real[n];
    temp_x_imag = 2.0 * temp_x_imag - g_state_0x_imag[n];
    g_state_2x_real[n] = temp_x_real;
    g_state_2x_imag[n] = temp_x_imag;

    real bessel_m = g_bessel_m;
    switch (g_label) {
      case 1: {
        g_state_real[n] += bessel_m * temp_x_real;
        g_state_imag[n] += bessel_m * temp_x_imag;
        break;
      }
      case 2: {
        g_state_real[n] -= bessel_m * temp_x_real;
        g_state_imag[n] -= bessel_m * temp_x_imag;
        break;
      }
      case 3: {
        g_state_real[n] += bessel_m * temp_x_imag;
        g_state_imag[n] -= bessel_m * temp_x_real;
        break;
      }
      case 4: {
        g_state_real[n] -= bessel_m * temp_x_imag;
        g_state_imag[n] += bessel_m * temp_x_real;
        break;
      }
    }
  }
}

// Wrapper for the kernel above
void Hamiltonian::chebyshev_2x(
  Vector& state_0,
  Vector& state_0x,
  Vector& state_1,
  Vector& state_1x,
  Vector& state_2,
  Vector& state_2x,
  Vector& state,
  real bessel_m,
  int label)
{
  gpu_chebyshev_2x<<<grid_size, BLOCK_SIZE>>>(
    n,
    energy_max,
    neighbor_number,
    neighbor_list,
    potential,
    hopping_real,
    hopping_imag,
    xx,
    state_0.real_part,
    state_0.imag_part,
    state_0x.real_part,
    state_0x.imag_part,
    state_1.real_part,
    state_1.imag_part,
    state_1x.real_part,
    state_1x.imag_part,
    state_2.real_part,
    state_2.imag_part,
    state_2x.real_part,
    state_2x.imag_part,
    state.real_part,
    state.imag_part,
    bessel_m,
    label);
  CHECK(hipGetLastError());
}

// Kernel for doing the Chebyshev iteration phi_2 = 2 * H * phi_1 - phi_0.
__global__ void gpu_kernel_polynomial(
  int number_of_atoms,
  real energy_max,
  int* g_neighbor_number,
  int* g_neighbor_list,
  real* g_potential,
  real* g_hopping_real,
  real* g_hopping_imag,
  real* g_state_0_real,
  real* g_state_0_imag,
  real* g_state_1_real,
  real* g_state_1_imag,
  real* g_state_2_real,
  real* g_state_2_imag)
{
  int n = blockIdx.x * blockDim.x + threadIdx.x;
  if (n < number_of_atoms) {
    real temp_real = g_potential[n] * g_state_1_real[n]; // on-site
    real temp_imag = g_potential[n] * g_state_1_imag[n]; // on-site

    for (int m = 0; m < g_neighbor_number[n]; ++m) {
      int index_1 = m * number_of_atoms + n;
      int index_2 = g_neighbor_list[index_1];
      real a = g_hopping_real[index_1];
      real b = g_hopping_imag[index_1];
      real c = g_state_1_real[index_2];
      real d = g_state_1_imag[index_2];
      temp_real += a * c - b * d; // hopping
      temp_imag += a * d + b * c; // hopping
    }

    temp_real /= energy_max; // scale
    temp_imag /= energy_max; // scale

    temp_real = 2.0 * temp_real - g_state_0_real[n];
    temp_imag = 2.0 * temp_imag - g_state_0_imag[n];
    g_state_2_real[n] = temp_real;
    g_state_2_imag[n] = temp_imag;
  }
}

// Wrapper for the Chebyshev iteration
void Hamiltonian::kernel_polynomial(Vector& state_0, Vector& state_1, Vector& state_2)
{
  gpu_kernel_polynomial<<<grid_size, BLOCK_SIZE>>>(
    n,
    energy_max,
    neighbor_number,
    neighbor_list,
    potential,
    hopping_real,
    hopping_imag,
    state_0.real_part,
    state_0.imag_part,
    state_1.real_part,
    state_1.imag_part,
    state_2.real_part,
    state_2.imag_part);
  CHECK(hipGetLastError());
}
