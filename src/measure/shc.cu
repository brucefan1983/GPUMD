#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

/*----------------------------------------------------------------------------80
Spectral heat current (SHC) calculations. Referene:
[1] Z. Fan, H. Dong, A. Harju, T. Ala-Nissila, Homogeneous nonequilibrium
molecular dynamics method for heat transport and spectral decomposition
with many-body potentials, Phys. Rev. B 99, 064308 (2019).
------------------------------------------------------------------------------*/

#include "model/group.cuh"
#include "parse_utilities.cuh"
#include "shc.cuh"
#include "utilities/common.cuh"
#include "utilities/error.cuh"
#include "utilities/read_file.cuh"

const int BLOCK_SIZE_SHC = 128;

void SHC::preprocess(const int N, const std::vector<Group>& group)
{
  if (!compute) {
    return;
  }

  num_time_origins = 0;
  if (-1 == group_method) {
    group_size = N;
  } else {
    group_size = group[group_method].cpu_size[group_id];
  }

  vx.resize(group_size * Nc);
  vy.resize(group_size * Nc);
  vz.resize(group_size * Nc);
  sx.resize(group_size * Nc);
  sy.resize(group_size * Nc);
  sz.resize(group_size * Nc);
  ki_negative.resize(Nc, 0.0);
  ko_negative.resize(Nc, 0.0);
  ki_positive.resize(Nc, 0.0);
  ko_positive.resize(Nc, 0.0);
  ki.resize(Nc * 2 - 1);
  ko.resize(Nc * 2 - 1);
  ki.assign(Nc * 2 - 1, 0.0);
  ko.assign(Nc * 2 - 1, 0.0);
  shc_i.resize(num_omega);
  shc_o.resize(num_omega);
  shc_i.assign(num_omega, 0.0);
  shc_o.assign(num_omega, 0.0);
}

static __global__ void gpu_find_k(
  const int group_size,
  const int correlation_step,
  const double* g_sx,
  const double* g_sy,
  const double* g_sz,
  const double* g_vx,
  const double* g_vy,
  const double* g_vz,
  double* g_ki,
  double* g_ko)
{
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int size_sum = bid * group_size;
  int number_of_rounds = (group_size - 1) / BLOCK_SIZE_SHC + 1;
  __shared__ double s_ki[BLOCK_SIZE_SHC];
  __shared__ double s_ko[BLOCK_SIZE_SHC];
  double ki = 0.0;
  double ko = 0.0;

  for (int round = 0; round < number_of_rounds; ++round) {
    int n = tid + round * BLOCK_SIZE_SHC;
    if (n < group_size) {
      ki += g_sx[n] * g_vx[size_sum + n] + g_sy[n] * g_vy[size_sum + n];
      ko += g_sz[n] * g_vz[size_sum + n];
    }
  }
  s_ki[tid] = ki;
  s_ko[tid] = ko;
  __syncthreads();

  for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1) {
    if (tid < offset) {
      s_ki[tid] += s_ki[tid + offset];
      s_ko[tid] += s_ko[tid + offset];
    }
    __syncthreads();
  }

  if (tid == 0) {
    if (bid <= correlation_step) {
      g_ki[correlation_step - bid] += s_ki[0];
      g_ko[correlation_step - bid] += s_ko[0];
    } else {
      g_ki[correlation_step + gridDim.x - bid] += s_ki[0];
      g_ko[correlation_step + gridDim.x - bid] += s_ko[0];
    }
  }
}

static __global__ void gpu_copy_data(
  const int group_size,
  const int offset,
  const int* g_group_contents,
  double* g_sx_o,
  double* g_sy_o,
  double* g_sz_o,
  double* g_vx_o,
  double* g_vy_o,
  double* g_vz_o,
  const double* g_sx_i,
  const double* g_sy_i,
  const double* g_sz_i,
  const double* g_vx_i,
  const double* g_vy_i,
  const double* g_vz_i)
{
  int n = threadIdx.x + blockIdx.x * blockDim.x;

  if (n < group_size) {
    int m = g_group_contents[offset + n];
    g_sx_o[n] = g_sx_i[m];
    g_sy_o[n] = g_sy_i[m];
    g_sz_o[n] = g_sz_i[m];
    g_vx_o[n] = g_vx_i[m];
    g_vy_o[n] = g_vy_i[m];
    g_vz_o[n] = g_vz_i[m];
  }
}

void SHC::process(
  const int step,
  const std::vector<Group>& group,
  const GPU_Vector<double>& velocity_per_atom,
  const GPU_Vector<double>& virial_per_atom)
{
  if (!compute) {
    return;
  }
  if ((step + 1) % sample_interval != 0) {
    return;
  }
  int sample_step = step / sample_interval; // 0, 1, ..., Nc-1, Nc, Nc+1, ...
  int correlation_step = sample_step % Nc;  // 0, 1, ..., Nc-1, 0, 1, ...
  int offset = correlation_step * group_size;

  const int N = velocity_per_atom.size() / 3;

  const int tensor[3][3] = {0, 3, 4, 6, 1, 5, 7, 8, 2};
  const double* sx_tmp = virial_per_atom.data() + N * tensor[direction][0];
  const double* sy_tmp = virial_per_atom.data() + N * tensor[direction][1];
  const double* sz_tmp = virial_per_atom.data() + N * tensor[direction][2];
  const double* vx_tmp = velocity_per_atom.data();
  const double* vy_tmp = velocity_per_atom.data() + N;
  const double* vz_tmp = velocity_per_atom.data() + N * 2;

  if (-1 == group_method) {
    CHECK(hipMemcpy(sx.data() + offset, sx_tmp, sizeof(double) * N, hipMemcpyDeviceToDevice));
    CHECK(hipMemcpy(sy.data() + offset, sy_tmp, sizeof(double) * N, hipMemcpyDeviceToDevice));
    CHECK(hipMemcpy(sz.data() + offset, sz_tmp, sizeof(double) * N, hipMemcpyDeviceToDevice));
    CHECK(hipMemcpy(vx.data() + offset, vx_tmp, sizeof(double) * N, hipMemcpyDeviceToDevice));
    CHECK(hipMemcpy(vy.data() + offset, vy_tmp, sizeof(double) * N, hipMemcpyDeviceToDevice));
    CHECK(hipMemcpy(vz.data() + offset, vz_tmp, sizeof(double) * N, hipMemcpyDeviceToDevice));
  } else {
    gpu_copy_data<<<(group_size - 1) / BLOCK_SIZE_SHC + 1, BLOCK_SIZE_SHC>>>(
      group_size, group[group_method].cpu_size_sum[group_id], group[group_method].contents.data(),
      sx.data() + offset, sy.data() + offset, sz.data() + offset, vx.data() + offset,
      vy.data() + offset, vz.data() + offset, sx_tmp, sy_tmp, sz_tmp, vx_tmp, vy_tmp, vz_tmp);
    CUDA_CHECK_KERNEL
  }

  if (sample_step >= Nc - 1) {
    ++num_time_origins;

    gpu_find_k<<<Nc, BLOCK_SIZE_SHC>>>(
      group_size, correlation_step, sx.data() + offset, sy.data() + offset, sz.data() + offset,
      vx.data(), vy.data(), vz.data(), ki_negative.data(), ko_negative.data());
    CUDA_CHECK_KERNEL

    gpu_find_k<<<Nc, BLOCK_SIZE_SHC>>>(
      group_size, correlation_step, vx.data() + offset, vy.data() + offset, vz.data() + offset,
      sx.data(), sy.data(), sz.data(), ki_positive.data(), ko_positive.data());
    CUDA_CHECK_KERNEL
  }
}

void SHC::average_k()
{
  std::vector<double> ki_negative_cpu(Nc);
  std::vector<double> ko_negative_cpu(Nc);
  std::vector<double> ki_positive_cpu(Nc);
  std::vector<double> ko_positive_cpu(Nc);
  ki_negative.copy_to_host(ki_negative_cpu.data());
  ko_negative.copy_to_host(ko_negative_cpu.data());
  ki_positive.copy_to_host(ki_positive_cpu.data());
  ko_positive.copy_to_host(ko_positive_cpu.data());

  const double scalar = 1000.0 / TIME_UNIT_CONVERSION / num_time_origins;
  for (int nc = 0; nc < Nc - 1; ++nc) {
    ki[nc] = ki_negative_cpu[Nc - nc - 1] * scalar;
    ko[nc] = ko_negative_cpu[Nc - nc - 1] * scalar;
  }

  for (int nc = 0; nc < Nc; ++nc) {
    ki[nc + Nc - 1] = ki_positive_cpu[nc] * scalar;
    ko[nc + Nc - 1] = ko_positive_cpu[nc] * scalar;
  }
}

void SHC::find_shc(const double dt_in_ps, const double d_omega)
{
  for (int nc = 0; nc < Nc * 2 - 1; ++nc) {
    const double hann_window = (cos(PI * (nc + 1 - Nc) / Nc) + 1.0) * 0.5;
    ki[nc] *= hann_window;
    ko[nc] *= hann_window;
  }

  for (int nw = 0; nw < num_omega; ++nw) {
    const double omega = (nw + 1) * d_omega;
    for (int nc = 0; nc < Nc * 2 - 1; nc++) {
      const double t_in_ps = (nc + 1 - Nc) * dt_in_ps;
      const double cos_factor = cos(omega * t_in_ps);
      shc_i[nw] += ki[nc] * cos_factor;
      shc_o[nw] += ko[nc] * cos_factor;
    }
    shc_i[nw] *= 2.0 * dt_in_ps;
    shc_o[nw] *= 2.0 * dt_in_ps;
  }
}

void SHC::postprocess(const char* input_dir, const double time_step)
{
  if (!compute) {
    return;
  }

  const double dt_in_ps = time_step * sample_interval * TIME_UNIT_CONVERSION / 1000.0;
  const double d_omega = max_omega / num_omega;

  char file_shc[200];
  strcpy(file_shc, input_dir);
  strcat(file_shc, "/shc.out");
  FILE* fid = my_fopen(file_shc, "a");

  // ki and ko are in units of A*eV/ps
  average_k();
  for (int nc = 0; nc < Nc * 2 - 1; ++nc) {
    fprintf(fid, "%g %g %g\n", (nc + 1 - Nc) * dt_in_ps, ki[nc], ko[nc]);
  }

  // shc_i and shc_o are in units of A*eV = A*eV/ps/THz
  find_shc(dt_in_ps, d_omega);
  for (int nc = 0; nc < num_omega; ++nc) {
    fprintf(fid, "%g %g %g\n", (nc + 1) * d_omega, shc_i[nc], shc_o[nc]);
  }

  fflush(fid);
  fclose(fid);

  compute = 0;
  group_method = -1;
}

void SHC::parse(char** param, int num_param, const std::vector<Group>& groups)
{
  printf("Compute SHC.\n");
  compute = 1;

  if ((num_param != 6) && (num_param != 9)) {
    PRINT_INPUT_ERROR("compute_shc should have 5 or 8 parameters.");
  }

  if (!is_valid_int(param[1], &sample_interval)) {
    PRINT_INPUT_ERROR("Sampling interval for SHC should be an integer.");
  }
  if (sample_interval < 1) {
    PRINT_INPUT_ERROR("Sampling interval for SHC should >= 1.");
  }
  if (sample_interval > 50) {
    PRINT_INPUT_ERROR("Sampling interval for SHC should <= 50 (trust me).");
  }
  printf("    sampling interval for SHC is %d.\n", sample_interval);

  if (!is_valid_int(param[2], &Nc)) {
    PRINT_INPUT_ERROR("Nc for SHC should be an integer.");
  }
  if (Nc < 100) {
    PRINT_INPUT_ERROR("Nc for SHC should >= 100 (trust me).");
  }
  if (Nc > 1000) {
    PRINT_INPUT_ERROR("Nc for SHC should <= 1000 (trust me).");
  }
  printf("    number of correlation data is %d.\n", Nc);

  if (!is_valid_int(param[3], &direction)) {
    PRINT_INPUT_ERROR("direction for SHC should be an integer.");
  }
  if (direction == 0) {
    printf("    transport direction is x.\n");
  } else if (direction == 1) {
    printf("    transport direction is y.\n");
  } else if (direction == 2) {
    printf("    transport direction is z.\n");
  } else {
    PRINT_INPUT_ERROR("Transport direction should be x or y or z.");
  }

  if (!is_valid_int(param[4], &num_omega)) {
    PRINT_INPUT_ERROR("num_omega for SHC should be an integer.");
  }
  if (num_omega < 0) {
    PRINT_INPUT_ERROR("num_omega for SHC should >= 0.");
  }
  printf("    num_omega for SHC is %d.\n", num_omega);

  if (!is_valid_real(param[5], &max_omega)) {
    PRINT_INPUT_ERROR("max_omega for SHC should be a number.");
  }
  if (max_omega <= 0) {
    PRINT_INPUT_ERROR("max_omega for SHC should > 0.");
  }
  printf("    max_omega for SHC is %g.\n", max_omega);

  for (int k = 6; k < num_param; k++) {
    if (strcmp(param[k], "group") == 0) {
      parse_group(param, num_param, false, groups, k, group_method, group_id);
    } else {
      PRINT_INPUT_ERROR("Unrecognized argument in compute_shc.\n");
    }
  }
}
