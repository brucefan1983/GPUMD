#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan and GPUMD development team
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details. You should have received a copy of the GNU General
   Public License along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

/*-----------------------------------------------------------------------------------------------100
Run active learning on-the-fly during MD
--------------------------------------------------------------------------------------------------*/

#include "active.cuh"
#include "force/force.cuh"
#include "model/atom.cuh"
#include "model/box.cuh"
#include "parse_utilities.cuh"
#include "utilities/common.cuh"
#include "utilities/error.cuh"
#include "utilities/gpu_macro.cuh"
#include "utilities/read_file.cuh"
#include <iostream>
#include <vector>
#include <cstring>

static __global__ void gpu_sum(const int N, const double* g_data, double* g_data_sum)
{
  int number_of_rounds = (N - 1) / 1024 + 1;
  __shared__ double s_data[1024];
  s_data[threadIdx.x] = 0.0;
  for (int round = 0; round < number_of_rounds; ++round) {
    int n = threadIdx.x + round * 1024;
    if (n < N) {
      s_data[threadIdx.x] += g_data[n + blockIdx.x * N];
    }
  }
  __syncthreads();
  for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1) {
    if (threadIdx.x < offset) {
      s_data[threadIdx.x] += s_data[threadIdx.x + offset];
    }
    __syncthreads();
  }
  if (threadIdx.x == 0) {
    g_data_sum[blockIdx.x] = s_data[0];
  }
}

static __global__ void initialize_properties(
  int N, double* g_fx, double* g_fy, double* g_fz, double* g_pe, double* g_virial)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x;
  if (n1 < N) {
    g_fx[n1] = 0.0;
    g_fy[n1] = 0.0;
    g_fz[n1] = 0.0;
    g_pe[n1] = 0.0;
    g_virial[n1 + 0 * N] = 0.0;
    g_virial[n1 + 1 * N] = 0.0;
    g_virial[n1 + 2 * N] = 0.0;
    g_virial[n1 + 3 * N] = 0.0;
    g_virial[n1 + 4 * N] = 0.0;
    g_virial[n1 + 5 * N] = 0.0;
    g_virial[n1 + 6 * N] = 0.0;
    g_virial[n1 + 7 * N] = 0.0;
    g_virial[n1 + 8 * N] = 0.0;
  }
}

static __global__ void initialize_mean_vectors(int N, double* g_m, double* g_m_sq)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x;
  // 3*N since 3 cartesian directions
  if (n1 < 3 * N) {
    g_m[n1] = 0.0;
    g_m_sq[n1] = 0.0;
  }
}

static __global__ void
compute_mean(int N, int M, double* g_m, double* g_m_sq, double* g_fx, double* g_fy, double* g_fz)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x;
  if (n1 < N) {
    // Average over number of potentials, M
    g_m[n1 + 0 * N] += g_fx[n1] / M;
    g_m[n1 + 1 * N] += g_fy[n1] / M;
    g_m[n1 + 2 * N] += g_fz[n1] / M;
    g_m_sq[n1 + 0 * N] += g_fx[n1] * g_fx[n1] / M;
    g_m_sq[n1 + 1 * N] += g_fy[n1] * g_fy[n1] / M;
    g_m_sq[n1 + 2 * N] += g_fz[n1] * g_fz[n1] / M;
  }
}

static __global__ void compute_uncertainty(int N, double* g_m, double* g_m_sq, double* g_u)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x;
  if (n1 < N) {
    double var_x = g_m_sq[n1 + 0 * N] - g_m[n1 + 0 * N] * g_m[n1 + 0 * N];
    double var_y = g_m_sq[n1 + 1 * N] - g_m[n1 + 1 * N] * g_m[n1 + 1 * N];
    double var_z = g_m_sq[n1 + 2 * N] - g_m[n1 + 2 * N] * g_m[n1 + 2 * N];
    g_u[n1] = sqrt(var_x + var_y + var_z);
  }
}

Active::Active(const char** param, int num_param)
{
  parse(param, num_param);
  property_name = "active";
}

void Active::parse(const char** param, int num_param)
{
  check_ = true;
  printf("Active learning.\n");

  if (num_param != 5) {
    PRINT_INPUT_ERROR("active should have 4 parameters.");
  }
  if (!is_valid_int(param[1], &check_interval_)) {
    PRINT_INPUT_ERROR("check interval should be an integer.");
  }
  if (check_interval_ <= 0) {
    PRINT_INPUT_ERROR("check interval should > 0.");
  }
  printf("    check uncertainty every %d steps.\n", check_interval_);

  if (!is_valid_int(param[2], &has_velocity_)) {
    PRINT_INPUT_ERROR("has_velocity should be an integer.");
  }
  if (has_velocity_ == 0) {
    printf("    without velocity data.\n");
  } else {
    printf("    with velocity data.\n");
  }

  if (!is_valid_int(param[3], &has_force_)) {
    PRINT_INPUT_ERROR("has_force should be an integer.");
  }
  if (has_force_ == 0) {
    printf("    without force data.\n");
  } else {
    printf("    with force data.\n");
  }
  if (!is_valid_real(param[4], &threshold_)) {
    PRINT_INPUT_ERROR("threshold should be a real number.\n");
  }

  printf(
    "    will check if uncertainties exceed %f every %d iterations.\n",
    threshold_,
    check_interval_);
}

void Active::preprocess(
  const int number_of_steps,
  const double time_step,
  Integrate& integrate,
  std::vector<Group>& group,
  Atom& atom,
  Box& box,
  Force& force)
{
  // Always use mode "observe" with all other potentials for active learning.
  // Only propagate MD with the main potential.
  if (check_) {
    force.set_multiple_potentials_mode("observe");
    std::string exyz_filename = "active.xyz";
    std::string out_filename = "active.out";
    exyz_file_ = my_fopen(exyz_filename.c_str(), "a");
    out_file_ = my_fopen(out_filename.c_str(), "a");
    gpu_total_virial_.resize(6);
    cpu_total_virial_.resize(6);
    if (has_force_) {
      cpu_force_per_atom_.resize(atom.number_of_atoms * 3);
    }
    mean_force_.resize(atom.number_of_atoms * 3);
    mean_force_sq_.resize(atom.number_of_atoms * 3);
    gpu_uncertainty_.resize(atom.number_of_atoms);
    cpu_uncertainty_.resize(atom.number_of_atoms);
  }
}

void Active::process(
  const int number_of_steps,
  int step,
  const int fixed_group,
  const int move_group,
  const double global_time,
  const double temperature,
  Integrate& integrate,
  Box& box,
  std::vector<Group>& group,
  GPU_Vector<double>& thermo,
  Atom& atom,
  Force& force)
{
  // Only run if should check, since forces have to be recomputed with each potential.
  if (!check_)
    return;
  if ((step + 1) % check_interval_ != 0)
    return;

  const int number_of_potentials = force.potentials.size();
  const int number_of_atoms = atom.type.size();
  // Reset mean vectors to zero
  initialize_mean_vectors<<<(3 * number_of_atoms - 1) / 128 + 1, 128>>>(
    number_of_atoms, mean_force_.data(), mean_force_sq_.data());
  GPU_CHECK_KERNEL

  // Loop backwards over files to evaluate the main potential last, keeping it's properties intact
  for (int potential_index = number_of_potentials - 1; potential_index >= 0; potential_index--) {
    // Set potential/force/virials to zero
    initialize_properties<<<(number_of_atoms - 1) / 128 + 1, 128>>>(
      number_of_atoms,
      atom.force_per_atom.data(),
      atom.force_per_atom.data() + number_of_atoms,
      atom.force_per_atom.data() + number_of_atoms * 2,
      atom.potential_per_atom.data(),
      atom.virial_per_atom.data());
    GPU_CHECK_KERNEL
    // Compute new potential properties
    force.potentials[potential_index]->compute(
      box,
      atom.type,
      atom.position_per_atom,
      atom.potential_per_atom,
      atom.force_per_atom,
      atom.virial_per_atom);
    // Write properties to GPU vector
    compute_mean<<<(3 * number_of_atoms - 1) / 128 + 1, 128>>>(
      number_of_atoms,
      number_of_potentials,
      mean_force_.data(),
      mean_force_sq_.data(),
      atom.force_per_atom.data(),
      atom.force_per_atom.data() + number_of_atoms,
      atom.force_per_atom.data() + number_of_atoms * 2);
    GPU_CHECK_KERNEL
  }
  // Sum mean and mean_sq on GPU, move sum to CPU
  compute_uncertainty<<<(number_of_atoms - 1) / 128 + 1, 128>>>(
    number_of_atoms, mean_force_.data(), mean_force_sq_.data(), gpu_uncertainty_.data());
  GPU_CHECK_KERNEL
  gpu_uncertainty_.copy_to_host(cpu_uncertainty_.data());
  double uncertainty = -1.0;
  for (int i = 0; i < number_of_atoms; i++) {
    if (uncertainty < cpu_uncertainty_[i]) {
      uncertainty = cpu_uncertainty_[i];
    }
  }
  write_uncertainty(step, global_time, uncertainty);
  if (uncertainty > threshold_) {
    write_exyz(
      step,
      global_time,
      box,
      atom.cpu_atom_symbol,
      atom.cpu_type,
      atom.position_per_atom,
      atom.cpu_position_per_atom,
      atom.velocity_per_atom,
      atom.cpu_velocity_per_atom,
      atom.force_per_atom,
      atom.virial_per_atom,
      thermo,
      uncertainty);
  }
}

void Active::write_uncertainty(const int step, const double time, double uncertainty)
{
  if (!check_)
    return;
  if ((step + 1) % check_interval_ != 0)
    return;

  FILE* fid_ = out_file_;

  // Write time, uncertainty to file
  fprintf(fid_, "%20.10e%20.10e\n", time * TIME_UNIT_CONVERSION, uncertainty);
  fflush(fid_);
}

void Active::output_line2(
  const double time,
  const Box& box,
  const std::vector<std::string>& cpu_atom_symbol,
  GPU_Vector<double>& virial_per_atom,
  GPU_Vector<double>& gpu_thermo,
  double uncertainty,
  FILE* fid_)
{
  // time
  fprintf(fid_, "Time=%.8f", time * TIME_UNIT_CONVERSION); // output time is in units of fs

  // PBC
  fprintf(
    fid_, " pbc=\"%c %c %c\"", box.pbc_x ? 'T' : 'F', box.pbc_y ? 'T' : 'F', box.pbc_z ? 'T' : 'F');

  // Uncertainty
  fprintf(fid_, " uncertainty=%.8f", uncertainty);

  // box
  fprintf(
    fid_,
    " Lattice=\"%.8f %.8f %.8f %.8f %.8f %.8f %.8f %.8f %.8f\"",
    box.cpu_h[0],
    box.cpu_h[3],
    box.cpu_h[6],
    box.cpu_h[1],
    box.cpu_h[4],
    box.cpu_h[7],
    box.cpu_h[2],
    box.cpu_h[5],
    box.cpu_h[8]);

  // energy and virial (symmetric tensor) in eV, and stress (symmetric tensor) in eV/A^3
  double cpu_thermo[8];
  gpu_thermo.copy_to_host(cpu_thermo, 8);
  const int N = virial_per_atom.size() / 9;
  gpu_sum<<<6, 1024>>>(N, virial_per_atom.data(), gpu_total_virial_.data());
  gpu_total_virial_.copy_to_host(cpu_total_virial_.data());

  fprintf(fid_, " energy=%.8f", cpu_thermo[1]);
  fprintf(
    fid_,
    " virial=\"%.8f %.8f %.8f %.8f %.8f %.8f %.8f %.8f %.8f\"",
    cpu_total_virial_[0],
    cpu_total_virial_[3],
    cpu_total_virial_[4],
    cpu_total_virial_[3],
    cpu_total_virial_[1],
    cpu_total_virial_[5],
    cpu_total_virial_[4],
    cpu_total_virial_[5],
    cpu_total_virial_[2]);
  fprintf(
    fid_,
    " stress=\"%.8f %.8f %.8f %.8f %.8f %.8f %.8f %.8f %.8f\"",
    cpu_thermo[2],
    cpu_thermo[5],
    cpu_thermo[6],
    cpu_thermo[5],
    cpu_thermo[3],
    cpu_thermo[7],
    cpu_thermo[6],
    cpu_thermo[7],
    cpu_thermo[4]);

  // Properties
  fprintf(fid_, " Properties=species:S:1:pos:R:3");

  if (has_velocity_) {
    fprintf(fid_, ":vel:R:3");
  }
  if (has_force_) {
    fprintf(fid_, ":forces:R:3");
  }

  // Over
  fprintf(fid_, "\n");
}

void Active::write_exyz(
  const int step,
  const double global_time,
  const Box& box,
  const std::vector<std::string>& cpu_atom_symbol,
  const std::vector<int>& cpu_type,
  GPU_Vector<double>& position_per_atom,
  std::vector<double>& cpu_position_per_atom,
  GPU_Vector<double>& velocity_per_atom,
  std::vector<double>& cpu_velocity_per_atom,
  GPU_Vector<double>& force_per_atom,
  GPU_Vector<double>& virial_per_atom,
  GPU_Vector<double>& gpu_thermo,
  double uncertainty)
{
  if (!check_)
    return;
  if ((step + 1) % check_interval_ != 0)
    return;

  const int num_atoms_total = position_per_atom.size() / 3;
  FILE* fid_ = exyz_file_;
  position_per_atom.copy_to_host(cpu_position_per_atom.data());
  if (has_velocity_) {
    velocity_per_atom.copy_to_host(cpu_velocity_per_atom.data());
  }
  if (has_force_) {
    force_per_atom.copy_to_host(cpu_force_per_atom_.data());
  }

  // line 1
  fprintf(fid_, "%d\n", num_atoms_total);

  // line 2
  output_line2(global_time, box, cpu_atom_symbol, virial_per_atom, gpu_thermo, uncertainty, fid_);

  // other lines
  for (int n = 0; n < num_atoms_total; n++) {
    fprintf(fid_, "%s", cpu_atom_symbol[n].c_str());
    for (int d = 0; d < 3; ++d) {
      fprintf(fid_, " %.8f", cpu_position_per_atom[n + num_atoms_total * d]);
    }
    if (has_velocity_) {
      const double natural_to_A_per_fs = 1.0 / TIME_UNIT_CONVERSION;
      for (int d = 0; d < 3; ++d) {
        fprintf(
          fid_, " %.8f", cpu_velocity_per_atom[n + num_atoms_total * d] * natural_to_A_per_fs);
      }
    }
    if (has_force_) {
      for (int d = 0; d < 3; ++d) {
        fprintf(fid_, " %.8f", cpu_force_per_atom_[n + num_atoms_total * d]);
      }
    }
    fprintf(fid_, "\n");
  }

  fflush(fid_);
}

void Active::postprocess(
  Atom& atom,
  Box& box,
  Integrate& integrate,
  const int number_of_steps,
  const double time_step,
  const double temperature)
{
  if (check_) {
    fclose(exyz_file_);
    fclose(out_file_);
    check_ = false;
  }
}
