#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan and GPUMD development team
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

/*-----------------------------------------------------------------------------------------------100
Calculate:
    Radial distribution function (RDF)
--------------------------------------------------------------------------------------------------*/

#include "force/neighbor.cuh"
#include "integrate/integrate.cuh"
#include "model/atom.cuh"
#include "model/box.cuh"
#include "model/group.cuh"
#include "parse_utilities.cuh"
#include "rdf.cuh"
#include "utilities/common.cuh"
#include "utilities/error.cuh"
#include "utilities/gpu_macro.cuh"
#include "utilities/read_file.cuh"
#include <cstring>

namespace
{

static __global__ void gpu_find_rdf_ON1(
  const int N,
  const double density,
  const Box box,
  const int* __restrict__ cell_counts,
  const int* __restrict__ cell_count_sum,
  const int* __restrict__ cell_contents,
  const int nx,
  const int ny,
  const int nz,
  const double rc_inv,
  const double* __restrict__ x,
  const double* __restrict__ y,
  const double* __restrict__ z,
  const double* __restrict__ radial_,
  double* rdf_,
  const int rdf_bins_,
  const double r_step_)
{
  const int n1 = blockIdx.x * blockDim.x + threadIdx.x;
  double rdf_PI = 3.14159265358979323846;
  if (n1 < N) {
    const double x1 = x[n1];
    const double y1 = y[n1];
    const double z1 = z[n1];
    int cell_id;
    int cell_id_x;
    int cell_id_y;
    int cell_id_z;
    find_cell_id(box, x1, y1, z1, rc_inv, nx, ny, nz, cell_id_x, cell_id_y, cell_id_z, cell_id);

    const int z_lim = box.pbc_z ? 2 : 0;
    const int y_lim = box.pbc_y ? 2 : 0;
    const int x_lim = box.pbc_x ? 2 : 0;

    // get radial descriptors
    for (int k = -z_lim; k <= z_lim; ++k) {
      for (int j = -y_lim; j <= y_lim; ++j) {
        for (int i = -x_lim; i <= x_lim; ++i) {
          int neighbor_cell = cell_id + k * nx * ny + j * nx + i;
          if (cell_id_x + i < 0)
            neighbor_cell += nx;
          if (cell_id_x + i >= nx)
            neighbor_cell -= nx;
          if (cell_id_y + j < 0)
            neighbor_cell += ny * nx;
          if (cell_id_y + j >= ny)
            neighbor_cell -= ny * nx;
          if (cell_id_z + k < 0)
            neighbor_cell += nz * ny * nx;
          if (cell_id_z + k >= nz)
            neighbor_cell -= nz * ny * nx;

          const int num_atoms_neighbor_cell = cell_counts[neighbor_cell];
          const int num_atoms_previous_cells = cell_count_sum[neighbor_cell];

          for (int m = 0; m < num_atoms_neighbor_cell; ++m) {
            const int n2 = cell_contents[num_atoms_previous_cells + m];
            if (n2 >= 0 && n2 < N && n1 != n2) {

              double x12 = x[n2] - x1;
              double y12 = y[n2] - y1;
              double z12 = z[n2] - z1;
              apply_mic(box, x12, y12, z12);
              const double d2 = x12 * x12 + y12 * y12 + z12 * z12;

              for (int w = 0; w < rdf_bins_; w++) {
                double r_low = (radial_[w] - r_step_ / 2) * (radial_[w] - r_step_ / 2);
                double r_up = (radial_[w] + r_step_ / 2) * (radial_[w] + r_step_ / 2);
                double r_mid_sqaure = radial_[w] * radial_[w];
                if (d2 > r_low && d2 <= r_up) {
                  rdf_[n1 * rdf_bins_ + w] +=
                    1 / (N * density * r_mid_sqaure * 4 * rdf_PI * r_step_);
                }
              }
            }
          }
        }
      }
    }
  }
}

static __global__ void gpu_find_rdf_ON1(
  const int N,
  const double density1,
  const double density2,
  const double num_atom1_,
  const double num_atom2_,
  const double atom_id1_,
  const double atom_id2_,
  const Box box,
  const int* __restrict__ cell_counts,
  const int* __restrict__ cell_count_sum,
  const int* __restrict__ cell_contents,
  const int nx,
  const int ny,
  const int nz,
  const double rc_inv,
  const double* __restrict__ x,
  const double* __restrict__ y,
  const double* __restrict__ z,
  const int* __restrict__ type,
  const double* __restrict__ radial_,
  double* rdf_,
  const int rdf_bins_,
  const double r_step_)
{
  const int n1 = blockIdx.x * blockDim.x + threadIdx.x;
  double rdf_PI = 3.14159265358979323846;
  if (n1 < N && type[n1] == atom_id1_) {
    const double x1 = x[n1];
    const double y1 = y[n1];
    const double z1 = z[n1];
    int cell_id;
    int cell_id_x;
    int cell_id_y;
    int cell_id_z;
    find_cell_id(box, x1, y1, z1, rc_inv, nx, ny, nz, cell_id_x, cell_id_y, cell_id_z, cell_id);

    const int z_lim = box.pbc_z ? 2 : 0;
    const int y_lim = box.pbc_y ? 2 : 0;
    const int x_lim = box.pbc_x ? 2 : 0;

    // get radial descriptors
    for (int k = -z_lim; k <= z_lim; ++k) {
      for (int j = -y_lim; j <= y_lim; ++j) {
        for (int i = -x_lim; i <= x_lim; ++i) {
          int neighbor_cell = cell_id + k * nx * ny + j * nx + i;
          if (cell_id_x + i < 0)
            neighbor_cell += nx;
          if (cell_id_x + i >= nx)
            neighbor_cell -= nx;
          if (cell_id_y + j < 0)
            neighbor_cell += ny * nx;
          if (cell_id_y + j >= ny)
            neighbor_cell -= ny * nx;
          if (cell_id_z + k < 0)
            neighbor_cell += nz * ny * nx;
          if (cell_id_z + k >= nz)
            neighbor_cell -= nz * ny * nx;

          const int num_atoms_neighbor_cell = cell_counts[neighbor_cell];
          const int num_atoms_previous_cells = cell_count_sum[neighbor_cell];

          for (int m = 0; m < num_atoms_neighbor_cell; ++m) {
            const int n2 = cell_contents[num_atoms_previous_cells + m];
            if (n2 >= 0 && n2 < N && n1 != n2 && type[n2] == atom_id2_) {
              double x12 = x[n2] - x1;
              double y12 = y[n2] - y1;
              double z12 = z[n2] - z1;
              apply_mic(box, x12, y12, z12);
              const double d2 = x12 * x12 + y12 * y12 + z12 * z12;
              for (int w = 0; w < rdf_bins_; w++) {
                double r_low = (radial_[w] - r_step_ / 2) * (radial_[w] - r_step_ / 2);
                double r_up = (radial_[w] + r_step_ / 2) * (radial_[w] + r_step_ / 2);
                double r_mid_sqaure = radial_[w] * radial_[w];
                if (d2 > r_low && d2 <= r_up) {
                  rdf_[n1 * rdf_bins_ + w] +=
                    1 / (num_atom1_ * density2 * r_mid_sqaure * 4 * rdf_PI * r_step_);
                }
              }
            }
          }
        }
      }
    }
  } else if (n1 < N && type[n1] == atom_id2_) {
    const double x1 = x[n1];
    const double y1 = y[n1];
    const double z1 = z[n1];
    int cell_id;
    int cell_id_x;
    int cell_id_y;
    int cell_id_z;
    find_cell_id(box, x1, y1, z1, rc_inv, nx, ny, nz, cell_id_x, cell_id_y, cell_id_z, cell_id);

    const int z_lim = box.pbc_z ? 2 : 0;
    const int y_lim = box.pbc_y ? 2 : 0;
    const int x_lim = box.pbc_x ? 2 : 0;

    // get radial descriptors
    for (int k = -z_lim; k <= z_lim; ++k) {
      for (int j = -y_lim; j <= y_lim; ++j) {
        for (int i = -x_lim; i <= x_lim; ++i) {
          int neighbor_cell = cell_id + k * nx * ny + j * nx + i;
          if (cell_id_x + i < 0)
            neighbor_cell += nx;
          if (cell_id_x + i >= nx)
            neighbor_cell -= nx;
          if (cell_id_y + j < 0)
            neighbor_cell += ny * nx;
          if (cell_id_y + j >= ny)
            neighbor_cell -= ny * nx;
          if (cell_id_z + k < 0)
            neighbor_cell += nz * ny * nx;
          if (cell_id_z + k >= nz)
            neighbor_cell -= nz * ny * nx;

          const int num_atoms_neighbor_cell = cell_counts[neighbor_cell];
          const int num_atoms_previous_cells = cell_count_sum[neighbor_cell];

          for (int m = 0; m < num_atoms_neighbor_cell; ++m) {
            const int n2 = cell_contents[num_atoms_previous_cells + m];
            if (n2 >= 0 && n2 < N && n1 != n2 && type[n2] == atom_id1_) {
              double x12 = x[n2] - x1;
              double y12 = y[n2] - y1;
              double z12 = z[n2] - z1;
              apply_mic(box, x12, y12, z12);
              const double d2 = x12 * x12 + y12 * y12 + z12 * z12;
              for (int w = 0; w < rdf_bins_; w++) {
                double r_low = (radial_[w] - r_step_ / 2) * (radial_[w] - r_step_ / 2);
                double r_up = (radial_[w] + r_step_ / 2) * (radial_[w] + r_step_ / 2);
                double r_mid_sqaure = radial_[w] * radial_[w];
                if (d2 > r_low && d2 <= r_up) {
                  rdf_[n1 * rdf_bins_ + w] +=
                    1 / (num_atom2_ * density1 * r_mid_sqaure * 4 * rdf_PI * r_step_);
                }
              }
            }
          }
        }
      }
    }
  }
}
} // namespace

void RDF::find_rdf(
  const int bead,
  const int rdf_atom_count,
  const int rdf_atom_,
  int* atom_id1_,
  int* atom_id2_,
  std::vector<int>& atom_id1_typesize,
  std::vector<int>& atom_id2_typesize,
  std::vector<double>& density1,
  std::vector<double>& density2,
  double rc,
  Box& box,
  const GPU_Vector<int>& type,
  const GPU_Vector<double>& position_per_atom,
  GPU_Vector<int>& cell_count,
  GPU_Vector<int>& cell_count_sum,
  GPU_Vector<int>& cell_contents,
  int num_bins_0,
  int num_bins_1,
  int num_bins_2,
  const double rc_inv_cell_list,
  GPU_Vector<double>& radial_,
  GPU_Vector<double>& rdf_g_,
  const int rdf_bins_,
  const double r_step_)
{
  const int N = position_per_atom.size() / 3;
  const int block_size = 256;
  const int grid_size = (N - 1) / block_size + 1;
  const double* x = position_per_atom.data();
  const double* y = position_per_atom.data() + N;
  const double* z = position_per_atom.data() + N * 2;

  double* rdf_g_ind =
    rdf_g_.data() + bead * rdf_atom_count * N * rdf_bins_ + rdf_atom_ * N * rdf_bins_;

  if (rdf_atom_ == 0) {
    gpu_find_rdf_ON1<<<grid_size, block_size>>>(
      N,
      density1[rdf_atom_],
      box,
      cell_count.data(),
      cell_count_sum.data(),
      cell_contents.data(),
      num_bins_0,
      num_bins_1,
      num_bins_2,
      rc_inv_cell_list,
      x,
      y,
      z,
      radial_.data(),
      rdf_g_ind,
      rdf_bins_,
      r_step_);
    GPU_CHECK_KERNEL

  } else {
    gpu_find_rdf_ON1<<<grid_size, block_size>>>(
      N,
      density1[rdf_atom_],
      density2[rdf_atom_],
      atom_id1_typesize[rdf_atom_ - 1],
      atom_id2_typesize[rdf_atom_ - 1],
      atom_id1_[rdf_atom_ - 1],
      atom_id2_[rdf_atom_ - 1],
      box,
      cell_count.data(),
      cell_count_sum.data(),
      cell_contents.data(),
      num_bins_0,
      num_bins_1,
      num_bins_2,
      rc_inv_cell_list,
      x,
      y,
      z,
      type.data(),
      radial_.data(),
      rdf_g_ind,
      rdf_bins_,
      r_step_);
    GPU_CHECK_KERNEL
  }
}

void RDF::preprocess(
  const int number_of_steps,
  const double time_step,
  Integrate& integrate,
  std::vector<Group>& group,
  Atom& atom,
  Box& box,
  Force& force)
{
  if (!compute_)
    return;
  r_step_ = r_cut_ / rdf_bins_;
  std::vector<double> radial_cpu(rdf_bins_);
  for (int i = 0; i < rdf_bins_; i++) {
    radial_cpu[i] = i * r_step_ + r_step_ / 2;
  }
  radial_.resize(rdf_bins_);
  radial_.copy_from_host(radial_cpu.data());
  rdf_N_ = atom.number_of_atoms;
  num_atoms_ = atom.number_of_atoms * rdf_atom_count;
  density1.resize(rdf_atom_count);
  density2.resize(rdf_atom_count);
  atom_id1_typesize.resize(rdf_atom_count - 1);
  atom_id2_typesize.resize(rdf_atom_count - 1);
  for (int a = 0; a < rdf_atom_count - 1; a++) {
    atom_id1_typesize[a] = atom.cpu_type_size[atom_id1_[a]];
    atom_id2_typesize[a] = atom.cpu_type_size[atom_id2_[a]];
  }

  if (integrate.type >= 31) {
    rdf_g_.resize(atom.number_of_beads * num_atoms_ * rdf_bins_, 0);
    rdf_.resize(atom.number_of_beads * num_atoms_ * rdf_bins_, 0);
    cell_count.resize(atom.number_of_atoms);
    cell_count_sum.resize(atom.number_of_atoms);
    cell_contents.resize(atom.number_of_atoms);
  } else {
    rdf_g_.resize(num_atoms_ * rdf_bins_, 0);
    rdf_.resize(num_atoms_ * rdf_bins_, 0);
    cell_count.resize(atom.number_of_atoms);
    cell_count_sum.resize(atom.number_of_atoms);
    cell_contents.resize(atom.number_of_atoms);
  }
}

void RDF::process(
  const int number_of_steps,
  int step,
  const int fixed_group,
  const int move_group,
  const double global_time,
  const double temperature,
  Integrate& integrate,
  Box& box,
  std::vector<Group>& group,
  GPU_Vector<double>& thermo,
  Atom& atom,
  Force& force)
{
  if (!compute_)
    return;
  if ((step + 1) % num_interval_ != 0) {
    return;
  }
  num_repeat_++;
  density1[0] = rdf_N_ / box.get_volume();
  density2[0] = rdf_N_ / box.get_volume();
  for (int a = 0; a < rdf_atom_count - 1; a++) {
    density1[a + 1] = atom_id1_typesize[a] / box.get_volume();
    density2[a + 1] = atom_id2_typesize[a] / box.get_volume();
  }

  if (integrate.type >= 31) {

    for (int k = 0; k < atom.number_of_beads; k++) {
      const double rc_cell_list = 0.5 * r_cut_;
      const double rc_inv_cell_list = 2.0 / r_cut_;
      int num_bins[3];
      box.get_num_bins(rc_cell_list, num_bins);
      find_cell_list(
        rc_cell_list,
        num_bins,
        box,
        atom.position_beads[k],
        cell_count,
        cell_count_sum,
        cell_contents);

      for (int a = 0; a < rdf_atom_count; a++) {
        find_rdf(
          k,
          rdf_atom_count,
          a,
          atom_id1_,
          atom_id2_,
          atom_id1_typesize,
          atom_id2_typesize,
          density1,
          density2,
          r_cut_,
          box,
          atom.type,
          atom.position_beads[k],
          cell_count,
          cell_count_sum,
          cell_contents,
          num_bins[0],
          num_bins[1],
          num_bins[2],
          rc_inv_cell_list,
          radial_,
          rdf_g_,
          rdf_bins_,
          r_step_);
      }
    }
  } else {
    int classical = 0;
    const double rc_cell_list = 0.5 * r_cut_;
    const double rc_inv_cell_list = 2.0 / r_cut_;
    int num_bins[3];
    box.get_num_bins(rc_cell_list, num_bins);
    find_cell_list(
      rc_cell_list,
      num_bins,
      box,
      atom.position_per_atom,
      cell_count,
      cell_count_sum,
      cell_contents);

    for (int a = 0; a < rdf_atom_count; a++) {
      find_rdf(
        classical,
        rdf_atom_count,
        a,
        atom_id1_,
        atom_id2_,
        atom_id1_typesize,
        atom_id2_typesize,
        density1,
        density2,
        r_cut_,
        box,
        atom.type,
        atom.position_per_atom,
        cell_count,
        cell_count_sum,
        cell_contents,
        num_bins[0],
        num_bins[1],
        num_bins[2],
        rc_inv_cell_list,
        radial_,
        rdf_g_,
        rdf_bins_,
        r_step_);
    }
  }
}

void RDF::postprocess(
  Atom& atom,
  Box& box,
  Integrate& integrate,
  const int number_of_steps,
  const double time_step,
  const double temperature)
{
  if (!compute_)
    return;

  if (integrate.type >= 31) {

    CHECK(gpuMemcpy(
      rdf_.data(),
      rdf_g_.data(),
      sizeof(double) * atom.number_of_beads * num_atoms_ * rdf_bins_,
      gpuMemcpyDeviceToHost));
    CHECK(gpuDeviceSynchronize()); // needed for pre-Pascal GPU

    std::vector<double> rdf_average(atom.number_of_beads * rdf_atom_count * rdf_bins_, 0.0);
    for (int k = 0; k < atom.number_of_beads; k++) {
      for (int a = 0; a < rdf_atom_count; a++) {
        for (int m = 0; m < rdf_N_; m++) {
          for (int x = 0; x < rdf_bins_; x++) {
            rdf_average[k * rdf_atom_count * rdf_bins_ + a * rdf_bins_ + x] +=
              rdf_[k * num_atoms_ * rdf_bins_ + a * rdf_N_ * rdf_bins_ + m * rdf_bins_ + x] /
              num_repeat_;
          }
        }
      }
    }

    std::vector<double> rdf_centroid(rdf_atom_count * rdf_bins_, 0.0);
    for (int k = 0; k < atom.number_of_beads; k++) {
      for (int a = 0; a < rdf_atom_count; a++) {
        for (int x = 0; x < rdf_bins_; x++) {
          rdf_centroid[a * rdf_bins_ + x] +=
            rdf_average[k * rdf_atom_count * rdf_bins_ + a * rdf_bins_ + x] / atom.number_of_beads;
        }
      }
    }

    FILE* fid = fopen("rdf.out", "a");
    fprintf(fid, "#radius");
    for (int a = 0; a < rdf_atom_count; a++) {
      if (a == 0) {
        fprintf(fid, " total");
      } else {
        fprintf(fid, " type_%d_%d", atom_id1_[a - 1], atom_id2_[a - 1]);
      }
    }
    fprintf(fid, "\n");
    for (int nc = 0; nc < rdf_bins_; nc++) {
      fprintf(fid, "%.5f", nc * r_step_ + r_step_ / 2);
      for (int a = 0; a < rdf_atom_count; a++) {
        if (a == 0) {
          fprintf(fid, " %.5f", rdf_centroid[nc]);
        } else {
          fprintf(
            fid,
            " %.5f",
            (atom_id1_[a - 1] == atom_id2_[a - 1]) ? rdf_centroid[a * rdf_bins_ + nc]
                                                   : rdf_centroid[a * rdf_bins_ + nc] / 2);
        }
      }
      fprintf(fid, "\n");
    }
    fflush(fid);
    fclose(fid);

  } else {

    CHECK(gpuMemcpy(
      rdf_.data(), rdf_g_.data(), sizeof(double) * num_atoms_ * rdf_bins_, gpuMemcpyDeviceToHost));
    CHECK(gpuDeviceSynchronize()); // needed for pre-Pascal GPU

    std::vector<double> rdf_average(rdf_atom_count * rdf_bins_, 0.0);
    for (int a = 0; a < rdf_atom_count; a++) {
      for (int m = 0; m < rdf_N_; m++) {
        for (int x = 0; x < rdf_bins_; x++) {
          rdf_average[a * rdf_bins_ + x] +=
            rdf_[a * rdf_N_ * rdf_bins_ + m * rdf_bins_ + x] / num_repeat_;
        }
      }
    }

    FILE* fid = fopen("rdf.out", "a");
    fprintf(fid, "#radius");
    for (int a = 0; a < rdf_atom_count; a++) {
      if (a == 0) {
        fprintf(fid, " total");
      } else {
        fprintf(fid, " type_%d_%d", atom_id1_[a - 1], atom_id2_[a - 1]);
      }
    }
    fprintf(fid, "\n");
    for (int nc = 0; nc < rdf_bins_; nc++) {
      fprintf(fid, "%.5f", nc * r_step_ + r_step_ / 2);
      for (int a = 0; a < rdf_atom_count; a++) {
        if (a == 0) {
          fprintf(fid, " %.5f", rdf_average[nc]);
        } else {
          fprintf(
            fid,
            " %.5f",
            (atom_id1_[a - 1] == atom_id2_[a - 1]) ? rdf_average[a * rdf_bins_ + nc]
                                                   : rdf_average[a * rdf_bins_ + nc] / 2);
        }
      }
      fprintf(fid, "\n");
    }
    fflush(fid);
    fclose(fid);
  }

  compute_ = false;
  for (int s = 0; s < 6; s++) {
    atom_id1_[s] = -1;
    atom_id2_[s] = -1;
  }
  rdf_atom_count = 1;
  num_repeat_ = 0;
}

RDF::RDF(
  const char** param,
  const int num_param,
  Box& box,
  const int number_of_types,
  const int number_of_steps)
{
  parse(param, num_param, box, number_of_types, number_of_steps);
  property_name = "compute_rdf";
}

void RDF::parse(
  const char** param,
  const int num_param,
  Box& box,
  const int number_of_types,
  const int number_of_steps)
{
  printf("Compute radial distribution function (RDF).\n");
  compute_ = true;

  if (num_param < 4) {
    PRINT_INPUT_ERROR("compute_rdf should have at least 3 parameters.\n");
  }
  if (num_param > 22) {
    PRINT_INPUT_ERROR("compute_rdf has too many parameters.\n");
  }

  // radial cutoff
  if (!is_valid_real(param[1], &r_cut_)) {
    PRINT_INPUT_ERROR("radial cutoff should be a number.\n");
  }
  if (r_cut_ <= 0) {
    PRINT_INPUT_ERROR("radial cutoff should be positive.\n");
  }
  double thickness_half[3] = {
    box.get_volume() / box.get_area(0) / 2.5,
    box.get_volume() / box.get_area(1) / 2.5,
    box.get_volume() / box.get_area(2) / 2.5};
  if (r_cut_ > thickness_half[0] || r_cut_ > thickness_half[1] || r_cut_ > thickness_half[2]) {
    std::string message =
      "The box has a thickness < 2.5 RDF radial cutoffs in a periodic direction.\n"
      "                Please increase the periodic direction(s).\n";
    PRINT_INPUT_ERROR(message.c_str());
  }
  printf("    radial cutoff %g.\n", r_cut_);

  // number of bins
  if (!is_valid_int(param[2], &rdf_bins_)) {
    PRINT_INPUT_ERROR("number of bins should be an integer.\n");
  }
  if (rdf_bins_ <= 20) {
    PRINT_INPUT_ERROR("A larger nbins is recommended.\n");
  }

  if (rdf_bins_ > 500) {
    PRINT_INPUT_ERROR("A smaller nbins is recommended.\n");
  }

  printf("    radial cutoff will be divided into %d bins.\n", rdf_bins_);

  // sample interval
  if (!is_valid_int(param[3], &num_interval_)) {
    PRINT_INPUT_ERROR("interval step per sample should be an integer.\n");
  }
  if (num_interval_ <= 0) {
    PRINT_INPUT_ERROR("interval step per sample should be positive.\n");
  }
  printf("    RDF sample interval is %d step.\n", num_interval_);

  // Process optional arguments
  for (int k = 4; k < num_param; k += 3) {
    if (strcmp(param[k], "atom") == 0) {
      int k_a = ((k + 2) / 3) - 2;
      rdf_atom_count++;
      if (!is_valid_int(param[k + 1], &atom_id1_[k_a])) {
        PRINT_INPUT_ERROR("atom type index1 should be an integer.\n");
      }
      if (atom_id1_[k_a] < 0) {
        PRINT_INPUT_ERROR("atom type index1 should be non-negative.\n");
      }
      if (atom_id1_[k_a] > number_of_types) {
        PRINT_INPUT_ERROR("atom type index1 should be less than number of atomic types.\n");
      }
      if (!is_valid_int(param[k + 2], &atom_id2_[k_a])) {
        PRINT_INPUT_ERROR("atom type index2 should be an integer.\n");
      }
      if (atom_id2_[k_a] < 0) {
        PRINT_INPUT_ERROR("atom type index2 should be non-negative.\n");
      }
      if (atom_id2_[k_a] > number_of_types) {
        PRINT_INPUT_ERROR("atom type index1 should be less than number of atomic types.\n");
      }
    } else {
      PRINT_INPUT_ERROR("Unrecognized argument in compute_rdf.\n");
    }
  }
}
