#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan and GPUMD development team
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

#include "run.cuh"
#include "utilities/error.cuh"
#include "utilities/main_common.cuh"
#include <chrono>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

void print_welcome_information();

int main(int argc, char* argv[])
{
  print_welcome_information();
  print_compile_information();
  print_gpu_information();

  print_line_1();
  printf("Started running GPUMD.\n");
  print_line_2();

  // hipDeviceSynchronize();
  torch::cuda::synchronize();
  clock_t time_begin = clock();

  Run run;

  // hipDeviceSynchronize();
    torch::cuda::synchronize();
  clock_t time_finish = clock();
  double time_used = (time_finish - time_begin) / double(CLOCKS_PER_SEC);

  print_line_1();
  printf("Time used = %f s.\n", time_used);
  print_line_2();

  print_line_1();
  printf("Finished running GPUMD.\n");
  print_line_2();

  return EXIT_SUCCESS;
}

void print_welcome_information(void)
{
  printf("\n");
  printf("***************************************************************\n");
  printf("*                 Welcome to use GPUMD                        *\n");
  printf("*     (Graphics Processing Units Molecular Dynamics)          *\n");
  printf("*               Master version after 3.9.5                    *\n");
  printf("*              This is the gpumd executable                   *\n");
  printf("***************************************************************\n");
  printf("\n");
}
