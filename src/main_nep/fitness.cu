#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

/*----------------------------------------------------------------------------80
Get the fitness
------------------------------------------------------------------------------*/

#include "fitness.cuh"
#include "nep.cuh"
#include "parameters.cuh"
#include "structure.cuh"
#include "utilities/error.cuh"
#include "utilities/gpu_vector.cuh"
#include <algorithm>
#include <chrono>
#include <random>
#include <vector>

Fitness::Fitness(char* input_dir, Parameters& para)
{
  print_line_1();
  printf("Started reading train.in.\n");
  print_line_2();
  std::vector<Structure> structures_train;
  read_structures(true, input_dir, para, structures_train);
  num_batches = (structures_train.size() - 1) / para.batch_size + 1;
  printf("Number of batches = %d\n", num_batches);
  int batch_size_old = para.batch_size;
  para.batch_size = (structures_train.size() - 1) / num_batches + 1;
  if (batch_size_old != para.batch_size) {
    printf("Hello, I changed the batch_size from %d to %d.\n", batch_size_old, para.batch_size);
  }
  train_set.resize(num_batches);
  for (int batch_id = 0; batch_id < num_batches; ++batch_id) {
    printf("\nBatch %d:\n", batch_id);
    int n1 = batch_id * para.batch_size;
    int n2 = std::min(int(structures_train.size()), n1 + para.batch_size);
    printf("Number of configurations = %d.\n", n2 - n1);
    train_set[batch_id].construct(input_dir, para, structures_train, n1, n2);
  }

  print_line_1();
  printf("Started reading test.in.\n");
  print_line_2();
  std::vector<Structure> structures_test;
  read_structures(false, input_dir, para, structures_test);
  test_set.construct(input_dir, para, structures_test, 0, structures_test.size());

  int N = test_set.N;
  int N_times_max_NN_radial = test_set.N * test_set.max_NN_radial;
  int N_times_max_NN_angular = test_set.N * test_set.max_NN_angular;
  for (int n = 0; n < num_batches; ++n) {
    if (train_set[n].N > N) {
      N = train_set[n].N;
    };
    if (train_set[n].N * train_set[n].max_NN_radial > N_times_max_NN_radial) {
      N_times_max_NN_radial = train_set[n].N * train_set[n].max_NN_radial;
    };
    if (train_set[n].N * train_set[n].max_NN_angular > N_times_max_NN_angular) {
      N_times_max_NN_angular = train_set[n].N * train_set[n].max_NN_angular;
    };
  }
  potential.reset(new NEP2(input_dir, para, N, N_times_max_NN_radial, N_times_max_NN_angular));

  char file_loss_out[200];
  strcpy(file_loss_out, input_dir);
  strcat(file_loss_out, "/loss.out");
  fid_loss_out = my_fopen(file_loss_out, "a");
}

Fitness::~Fitness() { fclose(fid_loss_out); }

void Fitness::compute(
  const int generation, Parameters& para, const float* population, float* fitness)
{
  if (generation == 0) {
    std::vector<float> dummy_solution(para.number_of_variables, 1.0f);
    for (int n = 0; n < num_batches; ++n) {
      potential->find_force(para, dummy_solution.data(), train_set[n], true);
    }
  } else {
    int batch_id = generation % num_batches;
    for (int n = 0; n < para.population_size; ++n) {
      const float* individual = population + n * para.number_of_variables;
      potential->find_force(para, individual, train_set[batch_id], false);
      float energy_shift_per_structure_not_used;
      fitness[n + 0 * para.population_size] =
        para.lambda_e * train_set[batch_id].get_rmse_energy(energy_shift_per_structure_not_used);
      fitness[n + 1 * para.population_size] =
        para.lambda_f *
        train_set[batch_id].get_rmse_force(para, true /*is_weighted*/, true /*has_delta*/);
      fitness[n + 2 * para.population_size] = para.lambda_v * train_set[batch_id].get_rmse_virial();
    }
  }
}

void Fitness::predict_energy_or_stress(FILE* fid, float* data, float* ref, Dataset& dataset)
{
  for (int nc = 0; nc < dataset.Nc; ++nc) {
    int offset = dataset.Na_sum_cpu[nc];
    float data_nc = 0.0f;
    for (int m = 0; m < dataset.Na_cpu[nc]; ++m) {
      data_nc += data[offset + m];
    }
    fprintf(fid, "%g %g\n", data_nc / dataset.Na_cpu[nc], ref[nc]);
  }
}

void Fitness::report_error(
  char* input_dir,
  Parameters& para,
  const int generation,
  const float loss_total,
  const float loss_L1,
  const float loss_L2,
  float* elite)
{
  if (0 == (generation + 1) % 100) {
    int batch_id = generation % num_batches;

    potential->find_force(para, elite, train_set[batch_id], false);
    float energy_shift_per_structure;
    float rmse_energy_train = train_set[batch_id].get_rmse_energy(energy_shift_per_structure);
    float rmse_force_train =
      train_set[batch_id].get_rmse_force(para, false /*is_weighted*/, false /*has_delta*/);
    float rmse_virial_train = train_set[batch_id].get_rmse_virial();

    potential->find_force(para, elite, test_set, false);
    float rmse_energy_test = test_set.get_rmse_energy(energy_shift_per_structure);
    float rmse_force_test =
      test_set.get_rmse_force(para, false /*is_weighted*/, false /*has_delta*/);
    float rmse_virial_test = test_set.get_rmse_virial();

    // correct the last bias parameter in the NN
    elite[para.number_of_variables_ann - 1] += energy_shift_per_structure;

    // re-calculate the test set
    potential->find_force(para, elite, test_set, false);

    char file_nep[200];
    strcpy(file_nep, input_dir);
    strcat(file_nep, "/nep.txt");
    FILE* fid_nep = my_fopen(file_nep, "w");

    fprintf(fid_nep, "nep %d ", para.num_types);
    for (int n = 0; n < para.num_types; ++n) {
      fprintf(fid_nep, "%s ", para.elements[n].c_str());
    }
    fprintf(fid_nep, "\n");
    fprintf(fid_nep, "cutoff %g %g\n", para.rc_radial, para.rc_angular);
    fprintf(fid_nep, "n_max %d %d\n", para.n_max_radial, para.n_max_angular);
    fprintf(fid_nep, "l_max %d\n", para.L_max);
    fprintf(fid_nep, "ANN %d %d\n", para.num_neurons1, 0);
    for (int m = 0; m < para.number_of_variables; ++m) {
      fprintf(fid_nep, "%15.7e\n", elite[m]);
    }
    para.q_scaler_gpu.copy_to_host(para.q_scaler_cpu.data());
    for (int d = 0; d < para.q_scaler_cpu.size(); ++d) {
      fprintf(fid_nep, "%15.7e\n", para.q_scaler_cpu[d]);
    }
    fclose(fid_nep);

    printf(
      "%-8d%-11.5f%-11.5f%-11.5f%-13.5f%-13.5f%-13.5f%-13.5f%-13.5f%-13.5f\n", generation + 1,
      loss_total, loss_L1, loss_L2, rmse_energy_train, rmse_force_train, rmse_virial_train,
      rmse_energy_test, rmse_force_test, rmse_virial_test);
    fflush(stdout);
    fprintf(
      fid_loss_out, "%-8d%-11.5f%-11.5f%-11.5f%-13.5f%-13.5f%-13.5f%-13.5f%-13.5f%-13.5f\n",
      generation + 1, loss_total, loss_L1, loss_L2, rmse_energy_train, rmse_force_train,
      rmse_virial_train, rmse_energy_test, rmse_force_test, rmse_virial_test);
    fflush(fid_loss_out);

    char file_force[200];
    strcpy(file_force, input_dir);
    strcat(file_force, "/force_test.out");
    FILE* fid_force = my_fopen(file_force, "w");

    char file_energy[200];
    strcpy(file_energy, input_dir);
    strcat(file_energy, "/energy_test.out");
    FILE* fid_energy = my_fopen(file_energy, "w");

    char file_virial[200];
    strcpy(file_virial, input_dir);
    strcat(file_virial, "/virial_test.out");
    FILE* fid_virial = my_fopen(file_virial, "w");

    update_energy_force_virial(fid_energy, fid_force, fid_virial, test_set);

    fclose(fid_energy);
    fclose(fid_force);
    fclose(fid_virial);

    if (0 == (generation + 1) % 1000) {
      char file_force[200];
      strcpy(file_force, input_dir);
      strcat(file_force, "/force_train.out");
      FILE* fid_force = my_fopen(file_force, "w");

      char file_energy[200];
      strcpy(file_energy, input_dir);
      strcat(file_energy, "/energy_train.out");
      FILE* fid_energy = my_fopen(file_energy, "w");

      char file_virial[200];
      strcpy(file_virial, input_dir);
      strcat(file_virial, "/virial_train.out");
      FILE* fid_virial = my_fopen(file_virial, "w");

      for (int batch_id = 0; batch_id < num_batches; ++batch_id) {
        potential->find_force(para, elite, train_set[batch_id], false);
        update_energy_force_virial(fid_energy, fid_force, fid_virial, train_set[batch_id]);
      }

      fclose(fid_energy);
      fclose(fid_force);
      fclose(fid_virial);
    }
  }
}

void Fitness::update_energy_force_virial(
  FILE* fid_energy, FILE* fid_force, FILE* fid_virial, Dataset& dataset)
{
  dataset.energy.copy_to_host(dataset.energy_cpu.data());
  dataset.virial.copy_to_host(dataset.virial_cpu.data());
  dataset.force.copy_to_host(dataset.force_cpu.data());

  // update force.out
  for (int nc = 0; nc < dataset.Nc; ++nc) {
    int offset = dataset.Na_sum_cpu[nc];
    for (int m = 0; m < dataset.structures[nc].num_atom; ++m) {
      int n = offset + m;
      fprintf(
        fid_force, "%g %g %g %g %g %g\n", dataset.force_cpu[n], dataset.force_cpu[n + dataset.N],
        dataset.force_cpu[n + dataset.N * 2], dataset.force_ref_cpu[n],
        dataset.force_ref_cpu[n + dataset.N], dataset.force_ref_cpu[n + dataset.N * 2]);
    }
  }

  // update energy.out
  predict_energy_or_stress(
    fid_energy, dataset.energy_cpu.data(), dataset.energy_ref_cpu.data(), dataset);

  // update virial.out
  predict_energy_or_stress(
    fid_virial, dataset.virial_cpu.data(), dataset.virial_ref_cpu.data(), dataset);

  predict_energy_or_stress(
    fid_virial, dataset.virial_cpu.data() + dataset.N, dataset.virial_ref_cpu.data() + dataset.Nc,
    dataset);

  predict_energy_or_stress(
    fid_virial, dataset.virial_cpu.data() + dataset.N * 2,
    dataset.virial_ref_cpu.data() + dataset.Nc * 2, dataset);

  predict_energy_or_stress(
    fid_virial, dataset.virial_cpu.data() + dataset.N * 3,
    dataset.virial_ref_cpu.data() + dataset.Nc * 3, dataset);

  predict_energy_or_stress(
    fid_virial, dataset.virial_cpu.data() + dataset.N * 4,
    dataset.virial_ref_cpu.data() + dataset.Nc * 4, dataset);

  predict_energy_or_stress(
    fid_virial, dataset.virial_cpu.data() + dataset.N * 5,
    dataset.virial_ref_cpu.data() + dataset.Nc * 5, dataset);
}
