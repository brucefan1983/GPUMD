#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

/*----------------------------------------------------------------------------80
Get the fitness
------------------------------------------------------------------------------*/

#include "fitness.cuh"
#include "nep3.cuh"
#include "parameters.cuh"
#include "structure.cuh"
#include "utilities/error.cuh"
#include "utilities/gpu_vector.cuh"
#include <algorithm>
#include <chrono>
#include <ctime>
#include <iostream>
#include <random>
#include <sstream>
#include <vector>

Fitness::Fitness(Parameters& para)
{
  print_line_1();
  printf("Started reading train.xyz.\n");
  print_line_2();

  int deviceCount;
  CHECK(hipGetDeviceCount(&deviceCount));

  std::vector<Structure> structures_train;
  read_structures(true, para, structures_train);
  num_batches = (structures_train.size() - 1) / para.batch_size + 1;
  printf("Number of devices = %d\n", deviceCount);
  printf("Number of batches = %d\n", num_batches);
  int batch_size_old = para.batch_size;
  para.batch_size = (structures_train.size() - 1) / num_batches + 1;
  if (batch_size_old != para.batch_size) {
    printf("Hello, I changed the batch_size from %d to %d.\n", batch_size_old, para.batch_size);
  }

  train_set.resize(num_batches);
  for (int batch_id = 0; batch_id < num_batches; ++batch_id) {
    train_set[batch_id].resize(deviceCount);
  }
  for (int batch_id = 0; batch_id < num_batches; ++batch_id) {
    int n1 = batch_id * para.batch_size;
    int n2 = std::min(int(structures_train.size()), n1 + para.batch_size);
    printf("\nBatch %d:\n", batch_id);
    printf("Number of configurations = %d.\n", n2 - n1);
    for (int device_id = 0; device_id < deviceCount; ++device_id) {
      print_line_1();
      printf("Constructing train_set in device  %d.\n", device_id);
      CHECK(hipSetDevice(device_id));
      train_set[batch_id][device_id].construct(para, structures_train, n1, n2, device_id);
      print_line_2();
    }
  }
  print_line_1();
  printf("Started reading test.xyz.\n");
  print_line_2();
  std::vector<Structure> structures_test;
  read_structures(false, para, structures_test);
  test_set.resize(deviceCount);
  for (int device_id = 0; device_id < deviceCount; ++device_id) {
    print_line_1();
    printf("Constructing test_set in device  %d.\n", device_id);
    CHECK(hipSetDevice(device_id));
    test_set[device_id].construct(para, structures_test, 0, structures_test.size(), device_id);
    print_line_2();
  }

  int N = test_set[0].N;
  int N_times_max_NN_radial = test_set[0].N * test_set[0].max_NN_radial;
  int N_times_max_NN_angular = test_set[0].N * test_set[0].max_NN_angular;
  max_NN_radial = test_set[0].max_NN_radial;
  max_NN_angular = test_set[0].max_NN_angular;
  for (int n = 0; n < num_batches; ++n) {
    if (train_set[n][0].N > N) {
      N = train_set[n][0].N;
    };
    if (train_set[n][0].N * train_set[n][0].max_NN_radial > N_times_max_NN_radial) {
      N_times_max_NN_radial = train_set[n][0].N * train_set[n][0].max_NN_radial;
    };
    if (train_set[n][0].N * train_set[n][0].max_NN_angular > N_times_max_NN_angular) {
      N_times_max_NN_angular = train_set[n][0].N * train_set[n][0].max_NN_angular;
    };

    if (train_set[n][0].max_NN_radial > max_NN_radial) {
      max_NN_radial = train_set[n][0].max_NN_radial;
    }
    if (train_set[n][0].max_NN_angular > max_NN_angular) {
      max_NN_angular = train_set[n][0].max_NN_angular;
    }
  }

  potential.reset(
    new NEP3(para, N, N_times_max_NN_radial, N_times_max_NN_angular, para.version, deviceCount));

  fid_loss_out = my_fopen("loss.out", "a");
}

Fitness::~Fitness() { fclose(fid_loss_out); }

void Fitness::compute(
  const int generation, Parameters& para, const float* population, float* fitness)
{
  int deviceCount;
  CHECK(hipGetDeviceCount(&deviceCount));
  int population_iter = (para.population_size - 1) / deviceCount + 1;

  if (generation == 0) {
    std::vector<float> dummy_solution(para.number_of_variables * deviceCount, 1.0f);
    for (int n = 0; n < num_batches; ++n) {
      potential->find_force(para, dummy_solution.data(), train_set[n], true, deviceCount);
    }

  } else {
    int batch_id = generation % num_batches;
    for (int n = 0; n < population_iter; ++n) {
      const float* individual = population + deviceCount * n * para.number_of_variables;
      potential->find_force(para, individual, train_set[batch_id], false, deviceCount);
      for (int m = 0; m < deviceCount; ++m) {
        std::vector<float> energy_shift(para.num_types);
        fitness[deviceCount * n + m + 0 * para.population_size] =
          para.lambda_e * train_set[batch_id][m].get_rmse_energy(energy_shift, true, true, m);
        fitness[deviceCount * n + m + 1 * para.population_size] =
          para.lambda_f * train_set[batch_id][m].get_rmse_force(para, true, m);
        fitness[deviceCount * n + m + 2 * para.population_size] =
          para.lambda_v * train_set[batch_id][m].get_rmse_virial(para, true, m);
      }
    }
  }
}

void Fitness::predict_energy_or_stress(FILE* fid, float* data, float* ref, Dataset& dataset)
{
  for (int nc = 0; nc < dataset.Nc; ++nc) {
    int offset = dataset.Na_sum_cpu[nc];
    float data_nc = 0.0f;
    for (int m = 0; m < dataset.Na_cpu[nc]; ++m) {
      data_nc += data[offset + m];
    }
    fprintf(fid, "%g %g\n", data_nc / dataset.Na_cpu[nc], ref[nc]);
  }
}

void Fitness::write_nep_txt(FILE* fid_nep, Parameters& para, float* elite)
{
  if (para.version == 2) {
    if (para.enable_zbl) {
      fprintf(fid_nep, "nep_zbl %d ", para.num_types);
    } else {
      fprintf(fid_nep, "nep %d ", para.num_types);
    }
  } else if (para.version == 3) {
    if (para.enable_zbl) {
      fprintf(fid_nep, "nep3_zbl %d ", para.num_types);
    } else {
      fprintf(fid_nep, "nep3 %d ", para.num_types);
    }
  } else if (para.version == 4) {
    if (para.enable_zbl) {
      fprintf(fid_nep, "nep4_zbl %d ", para.num_types);
    } else {
      fprintf(fid_nep, "nep4 %d ", para.num_types);
    }
  }

  for (int n = 0; n < para.num_types; ++n) {
    fprintf(fid_nep, "%s ", para.elements[n].c_str());
  }
  fprintf(fid_nep, "\n");
  if (para.enable_zbl) {
    fprintf(fid_nep, "zbl %g %g\n", para.zbl_rc_inner, para.zbl_rc_outer);
  }
  fprintf(
    fid_nep, "cutoff %g %g %d %d\n", para.rc_radial, para.rc_angular, max_NN_radial,
    max_NN_angular);
  fprintf(fid_nep, "n_max %d %d\n", para.n_max_radial, para.n_max_angular);
  if (para.version >= 3) {
    fprintf(fid_nep, "basis_size %d %d\n", para.basis_size_radial, para.basis_size_angular);
    fprintf(fid_nep, "l_max %d %d %d\n", para.L_max, para.L_max_4body, para.L_max_5body);
  } else {
    fprintf(fid_nep, "l_max %d\n", para.L_max);
  }

  fprintf(fid_nep, "ANN %d %d\n", para.num_neurons1, 0);
  for (int m = 0; m < para.number_of_variables; ++m) {
    fprintf(fid_nep, "%15.7e\n", elite[m]);
  }
  CHECK(hipSetDevice(0));
  para.q_scaler_gpu[0].copy_to_host(para.q_scaler_cpu.data());
  for (int d = 0; d < para.q_scaler_cpu.size(); ++d) {
    fprintf(fid_nep, "%15.7e\n", para.q_scaler_cpu[d]);
  }
}

void Fitness::report_error(
  Parameters& para,
  const int generation,
  const float loss_total,
  const float loss_L1,
  const float loss_L2,
  float* elite)
{
  if (0 == (generation + 1) % 100) {
    int batch_id = generation % num_batches;
    potential->find_force(para, elite, train_set[batch_id], false, 1);
    std::vector<float> energy_shift(para.num_types);
    float rmse_energy_train = train_set[batch_id][0].get_rmse_energy(energy_shift, false, true, 0);
    float rmse_force_train = train_set[batch_id][0].get_rmse_force(para, false, 0);
    float rmse_virial_train = train_set[batch_id][0].get_rmse_virial(para, false, 0);

    // correct the last bias parameter in the NN
    if (para.train_mode == 0) {
      elite[para.number_of_variables_ann - 1] += energy_shift[0];
    }

    potential->find_force(para, elite, test_set, false, 1);
    float rmse_energy_test = test_set[0].get_rmse_energy(energy_shift, false, false, 0);
    float rmse_force_test = test_set[0].get_rmse_force(para, false, 0);
    float rmse_virial_test = test_set[0].get_rmse_virial(para, false, 0);

    FILE* fid_nep = my_fopen("nep.txt", "w");
    write_nep_txt(fid_nep, para, elite);
    fclose(fid_nep);

    if (0 == (generation + 1) % 100000) {
      time_t rawtime;
      time(&rawtime);
      struct tm* timeinfo = localtime(&rawtime);
      char buffer[200];
      strftime(buffer, sizeof(buffer), "nep_y%Y_m%m_d%d_h%H_m%M_s%S_generation", timeinfo);
      std::string filename(buffer + std::to_string(generation + 1) + ".txt");

      FILE* fid_nep = my_fopen(filename.c_str(), "w");
      write_nep_txt(fid_nep, para, elite);
      fclose(fid_nep);
    }

    printf(
      "%-8d%-11.5f%-11.5f%-11.5f%-13.5f%-13.5f%-13.5f%-13.5f%-13.5f%-13.5f\n", generation + 1,
      loss_total, loss_L1, loss_L2, rmse_energy_train, rmse_force_train, rmse_virial_train,
      rmse_energy_test, rmse_force_test, rmse_virial_test);
    fflush(stdout);
    fprintf(
      fid_loss_out, "%-8d%-11.5f%-11.5f%-11.5f%-13.5f%-13.5f%-13.5f%-13.5f%-13.5f%-13.5f\n",
      generation + 1, loss_total, loss_L1, loss_L2, rmse_energy_train, rmse_force_train,
      rmse_virial_train, rmse_energy_test, rmse_force_test, rmse_virial_test);
    fflush(fid_loss_out);

    FILE* fid_force = my_fopen("force_test.out", "w");
    FILE* fid_energy = my_fopen("energy_test.out", "w");
    FILE* fid_virial = my_fopen("virial_test.out", "w");

    update_energy_force_virial(fid_energy, fid_force, fid_virial, test_set[0]);

    fclose(fid_energy);
    fclose(fid_force);
    fclose(fid_virial);

    if (0 == (generation + 1) % 1000) {
      FILE* fid_force = my_fopen("force_train.out", "w");
      FILE* fid_energy = my_fopen("energy_train.out", "w");
      FILE* fid_virial = my_fopen("virial_train.out", "w");

      for (int batch_id = 0; batch_id < num_batches; ++batch_id) {
        potential->find_force(para, elite, train_set[batch_id], false, 1);
        update_energy_force_virial(fid_energy, fid_force, fid_virial, train_set[batch_id][0]);
      }

      fclose(fid_energy);
      fclose(fid_force);
      fclose(fid_virial);
    }
  }
}

void Fitness::update_energy_force_virial(
  FILE* fid_energy, FILE* fid_force, FILE* fid_virial, Dataset& dataset)
{
  dataset.energy.copy_to_host(dataset.energy_cpu.data());
  dataset.virial.copy_to_host(dataset.virial_cpu.data());
  dataset.force.copy_to_host(dataset.force_cpu.data());

  // update force.out
  for (int nc = 0; nc < dataset.Nc; ++nc) {
    int offset = dataset.Na_sum_cpu[nc];
    for (int m = 0; m < dataset.structures[nc].num_atom; ++m) {
      int n = offset + m;
      fprintf(
        fid_force, "%g %g %g %g %g %g\n", dataset.force_cpu[n], dataset.force_cpu[n + dataset.N],
        dataset.force_cpu[n + dataset.N * 2], dataset.force_ref_cpu[n],
        dataset.force_ref_cpu[n + dataset.N], dataset.force_ref_cpu[n + dataset.N * 2]);
    }
  }

  // update energy.out
  predict_energy_or_stress(
    fid_energy, dataset.energy_cpu.data(), dataset.energy_ref_cpu.data(), dataset);

  // update virial.out
  predict_energy_or_stress(
    fid_virial, dataset.virial_cpu.data(), dataset.virial_ref_cpu.data(), dataset);

  predict_energy_or_stress(
    fid_virial, dataset.virial_cpu.data() + dataset.N, dataset.virial_ref_cpu.data() + dataset.Nc,
    dataset);

  predict_energy_or_stress(
    fid_virial, dataset.virial_cpu.data() + dataset.N * 2,
    dataset.virial_ref_cpu.data() + dataset.Nc * 2, dataset);

  predict_energy_or_stress(
    fid_virial, dataset.virial_cpu.data() + dataset.N * 3,
    dataset.virial_ref_cpu.data() + dataset.Nc * 3, dataset);

  predict_energy_or_stress(
    fid_virial, dataset.virial_cpu.data() + dataset.N * 4,
    dataset.virial_ref_cpu.data() + dataset.Nc * 4, dataset);

  predict_energy_or_stress(
    fid_virial, dataset.virial_cpu.data() + dataset.N * 5,
    dataset.virial_ref_cpu.data() + dataset.Nc * 5, dataset);
}
