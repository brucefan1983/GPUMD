#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

/*----------------------------------------------------------------------------80
The neuroevolution potential (NEP)
Ref: Zheyong Fan et al., Neuroevolution machine learning potentials:
Combining high accuracy and low cost in atomistic simulations and application to
heat transport, Phys. Rev. B. 104, 104309 (2021).
------------------------------------------------------------------------------*/

#include "dataset.cuh"
#include "mic.cuh"
#include "nep.cuh"
#include "parameters.cuh"
#include "utilities/error.cuh"
#include "utilities/gpu_vector.cuh"
#include "utilities/nep_utilities.cuh"

static __global__ void gpu_find_neighbor_list(
  const int N,
  const int* Na,
  const int* Na_sum,
  const float rc2_radial,
  const float rc2_angular,
  const float* __restrict__ g_box,
  const float* __restrict__ g_box_original,
  const int* __restrict__ g_num_cell,
  const float* x,
  const float* y,
  const float* z,
  int* NN_radial,
  int* NL_radial,
  int* NN_angular,
  int* NL_angular,
  float* x12_radial,
  float* y12_radial,
  float* z12_radial,
  float* x12_angular,
  float* y12_angular,
  float* z12_angular)
{
  int N1 = Na_sum[blockIdx.x];
  int N2 = N1 + Na[blockIdx.x];
  for (int n1 = N1 + threadIdx.x; n1 < N2; n1 += blockDim.x) {
    const float* __restrict__ box = g_box + 18 * blockIdx.x;
    const float* __restrict__ box_original = g_box_original + 9 * blockIdx.x;
    const int* __restrict__ num_cell = g_num_cell + 3 * blockIdx.x;
    float x1 = x[n1];
    float y1 = y[n1];
    float z1 = z[n1];
    int count_radial = 0;
    int count_angular = 0;
    for (int n2 = N1; n2 < N2; ++n2) {
      for (int ia = 0; ia < num_cell[0]; ++ia) {
        for (int ib = 0; ib < num_cell[1]; ++ib) {
          for (int ic = 0; ic < num_cell[2]; ++ic) {
            if (ia == 0 && ib == 0 && ic == 0 && n1 == n2) {
              continue; // exclude self
            }
            float delta_x = box_original[0] * ia + box_original[1] * ib + box_original[2] * ic;
            float delta_y = box_original[3] * ia + box_original[4] * ib + box_original[5] * ic;
            float delta_z = box_original[6] * ia + box_original[7] * ib + box_original[8] * ic;
            float x12 = x[n2] + delta_x - x1;
            float y12 = y[n2] + delta_y - y1;
            float z12 = z[n2] + delta_z - z1;
            dev_apply_mic(box, x12, y12, z12);
            float distance_square = x12 * x12 + y12 * y12 + z12 * z12;
            if (distance_square < rc2_radial) {
              NL_radial[count_radial * N + n1] = n2;
              x12_radial[count_radial * N + n1] = x12;
              y12_radial[count_radial * N + n1] = y12;
              z12_radial[count_radial * N + n1] = z12;
              count_radial++;
            }
            if (distance_square < rc2_angular) {
              NL_angular[count_angular * N + n1] = n2;
              x12_angular[count_angular * N + n1] = x12;
              y12_angular[count_angular * N + n1] = y12;
              z12_angular[count_angular * N + n1] = z12;
              count_angular++;
            }
          }
        }
      }
    }
    NN_radial[n1] = count_radial;
    NN_angular[n1] = count_angular;
  }
}

static __global__ void find_descriptors_radial(
  const int N,
  const int* g_NN,
  const int* g_NL,
  const NEP2::ParaMB paramb,
  const NEP2::ANN annmb,
  const int* __restrict__ g_type,
  const float* __restrict__ g_x12,
  const float* __restrict__ g_y12,
  const float* __restrict__ g_z12,
  float* g_descriptors)
{
  int n1 = threadIdx.x + blockIdx.x * blockDim.x;
  if (n1 < N) {
    int t1 = g_type[n1];
    int neighbor_number = g_NN[n1];
    float q[MAX_DIM] = {0.0f};
    for (int i1 = 0; i1 < neighbor_number; ++i1) {
      int index = n1 + N * i1;
      int n2 = g_NL[index];
      float x12 = g_x12[index];
      float y12 = g_y12[index];
      float z12 = g_z12[index];
      float d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12);
      float fc12;
      find_fc(paramb.rc_radial, paramb.rcinv_radial, d12, fc12);
      int t2 = g_type[n2];
      float fn12[MAX_NUM_N];
      find_fn(paramb.n_max_radial, paramb.rcinv_radial, d12, fc12, fn12);
      for (int n = 0; n <= paramb.n_max_radial; ++n) {
        float c = (paramb.num_types == 1)
                    ? 1.0f
                    : annmb.c[(n * paramb.num_types + t1) * paramb.num_types + t2];
        q[n] += fn12[n] * c;
      }
    }
    for (int n = 0; n <= paramb.n_max_radial; ++n) {
      g_descriptors[n1 + n * N] = q[n];
    }
  }
}

static __global__ void find_descriptors_angular(
  const int N,
  const int* g_NN,
  const int* g_NL,
  NEP2::ParaMB paramb,
  const NEP2::ANN annmb,
  const int* __restrict__ g_type,
  const float* __restrict__ g_x12,
  const float* __restrict__ g_y12,
  const float* __restrict__ g_z12,
  float* g_descriptors,
  float* g_sum_fxyz)
{
  int n1 = threadIdx.x + blockIdx.x * blockDim.x;
  if (n1 < N) {
    int t1 = g_type[n1];
    int neighbor_number = g_NN[n1];
    float q[MAX_DIM] = {0.0f};

    for (int n = 0; n <= paramb.n_max_angular; ++n) {
      float s[NUM_OF_ABC] = {0.0f};
      for (int i1 = 0; i1 < neighbor_number; ++i1) {
        int index = n1 + N * i1;
        int n2 = g_NL[n1 + N * i1];
        float x12 = g_x12[index];
        float y12 = g_y12[index];
        float z12 = g_z12[index];
        float d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12);
        float fc12;
        find_fc(paramb.rc_angular, paramb.rcinv_angular, d12, fc12);
        int t2 = g_type[n2];
        float fn;
        find_fn(n, paramb.rcinv_angular, d12, fc12, fn);
        fn *=
          (paramb.num_types == 1)
            ? 1.0f
            : annmb
                .c[((paramb.n_max_radial + 1 + n) * paramb.num_types + t1) * paramb.num_types + t2];
        accumulate_s(d12, x12, y12, z12, fn, s);
      }
      find_q(paramb.n_max_angular + 1, n, s, q);
      for (int abc = 0; abc < NUM_OF_ABC; ++abc) {
        g_sum_fxyz[(n * NUM_OF_ABC + abc) * N + n1] = s[abc] * YLM[abc];
      }
    }

    for (int n = 0; n <= paramb.n_max_angular; ++n) {
      for (int l = 0; l < paramb.L_max; ++l) {
        int ln = l * (paramb.n_max_angular + 1) + n;
        g_descriptors[n1 + ((paramb.n_max_radial + 1) + ln) * N] = q[ln];
      }
    }
  }
}

NEP2::NEP2(
  char* input_dir, Parameters& para, int N, int N_times_max_NN_radial, int N_times_max_NN_angular)
{
  paramb.rc_radial = para.rc_radial;
  paramb.rcinv_radial = 1.0f / paramb.rc_radial;
  paramb.rc_angular = para.rc_angular;
  paramb.rcinv_angular = 1.0f / paramb.rc_angular;
  annmb.dim = (para.n_max_radial + 1) + (para.n_max_angular + 1) * para.L_max;
  annmb.num_neurons1 = para.num_neurons1;
  paramb.num_types = para.num_types;
  annmb.num_para = para.number_of_variables;
  paramb.n_max_radial = para.n_max_radial;
  paramb.n_max_angular = para.n_max_angular;
  paramb.L_max = para.L_max;

  nep_data.NN_radial.resize(N);
  nep_data.NN_angular.resize(N);
  nep_data.NL_radial.resize(N_times_max_NN_radial);
  nep_data.NL_angular.resize(N_times_max_NN_angular);
  nep_data.x12_radial.resize(N_times_max_NN_radial);
  nep_data.y12_radial.resize(N_times_max_NN_radial);
  nep_data.z12_radial.resize(N_times_max_NN_radial);
  nep_data.x12_angular.resize(N_times_max_NN_angular);
  nep_data.y12_angular.resize(N_times_max_NN_angular);
  nep_data.z12_angular.resize(N_times_max_NN_angular);
  nep_data.descriptors.resize(N * annmb.dim);
  nep_data.Fp.resize(N * annmb.dim);
  nep_data.sum_fxyz.resize(N * (paramb.n_max_angular + 1) * NUM_OF_ABC);
}

void NEP2::update_potential(const float* parameters, ANN& ann)
{
  ann.w0 = parameters;
  ann.b0 = ann.w0 + ann.num_neurons1 * ann.dim;
  ann.w1 = ann.b0 + ann.num_neurons1;
  ann.b1 = ann.w1 + ann.num_neurons1;
  if (paramb.num_types > 1) {
    ann.c = ann.b1 + 1;
  }
}

static void __global__ find_max_min(const int N, const float* g_q, float* g_q_scaler)
{
  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  __shared__ float s_max[1024];
  __shared__ float s_min[1024];
  s_max[tid] = -1000000.0f; // a small number
  s_min[tid] = +1000000.0f; // a large number
  const int stride = 1024;
  const int number_of_rounds = (N - 1) / stride + 1;
  for (int round = 0; round < number_of_rounds; ++round) {
    const int n = round * stride + tid;
    if (n < N) {
      const int m = n + N * bid;
      float q = g_q[m];
      if (q > s_max[tid]) {
        s_max[tid] = q;
      }
      if (q < s_min[tid]) {
        s_min[tid] = q;
      }
    }
  }
  __syncthreads();
  for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1) {
    if (tid < offset) {
      if (s_max[tid] < s_max[tid + offset]) {
        s_max[tid] = s_max[tid + offset];
      }
      if (s_min[tid] > s_min[tid + offset]) {
        s_min[tid] = s_min[tid + offset];
      }
    }
    __syncthreads();
  }
  if (tid == 0) {
    g_q_scaler[bid] = min(g_q_scaler[bid], 1.0f / (s_max[0] - s_min[0]));
  }
}

static __device__ void
apply_ann_one_layer(const NEP2::ANN& ann, float* q, float& energy, float* energy_derivative)
{
  for (int n = 0; n < ann.num_neurons1; ++n) {
    float w0_times_q = 0.0f;
    for (int d = 0; d < ann.dim; ++d) {
      w0_times_q += ann.w0[n * ann.dim + d] * q[d];
    }
    float x1 = tanh(w0_times_q - ann.b0[n]);
    energy += ann.w1[n] * x1;
    for (int d = 0; d < ann.dim; ++d) {
      float y1 = (1.0f - x1 * x1) * ann.w0[n * ann.dim + d];
      energy_derivative[d] += ann.w1[n] * y1;
    }
  }
  energy -= ann.b1[0];
}

static __global__ void apply_ann(
  const int N,
  const NEP2::ParaMB paramb,
  const NEP2::ANN annmb,
  const float* __restrict__ g_descriptors,
  const float* __restrict__ g_q_scaler,
  float* g_pe,
  float* g_Fp)
{
  int n1 = threadIdx.x + blockIdx.x * blockDim.x;
  if (n1 < N) {
    // get descriptors
    float q[MAX_DIM] = {0.0f};
    for (int d = 0; d < annmb.dim; ++d) {
      q[d] = g_descriptors[n1 + d * N] * g_q_scaler[d];
    }
    // get energy and energy gradient
    float F = 0.0f, Fp[MAX_DIM] = {0.0f};
    apply_ann_one_layer(annmb, q, F, Fp);
    g_pe[n1] = F;
    for (int d = 0; d < annmb.dim; ++d) {
      g_Fp[n1 + d * N] = Fp[d] * g_q_scaler[d];
    }
  }
}

static __global__ void zero_force(const int N, float* g_fx, float* g_fy, float* g_fz)
{
  int n1 = threadIdx.x + blockIdx.x * blockDim.x;
  if (n1 < N) {
    g_fx[n1] = 0.0f;
    g_fy[n1] = 0.0f;
    g_fz[n1] = 0.0f;
  }
}

static __global__ void find_force_radial(
  const int N,
  const int* g_NN,
  const int* g_NL,
  const NEP2::ParaMB paramb,
  const NEP2::ANN annmb,
  const int* __restrict__ g_type,
  const float* __restrict__ g_x12,
  const float* __restrict__ g_y12,
  const float* __restrict__ g_z12,
  const float* __restrict__ g_Fp,
  float* g_fx,
  float* g_fy,
  float* g_fz,
  float* g_virial)
{
  int n1 = threadIdx.x + blockIdx.x * blockDim.x;
  if (n1 < N) {
    int neighbor_number = g_NN[n1];
    float s_virial_xx = 0.0f;
    float s_virial_yy = 0.0f;
    float s_virial_zz = 0.0f;
    float s_virial_xy = 0.0f;
    float s_virial_yz = 0.0f;
    float s_virial_zx = 0.0f;
    int t1 = g_type[n1];
    for (int i1 = 0; i1 < neighbor_number; ++i1) {
      int index = i1 * N + n1;
      int n2 = g_NL[index];
      int t2 = g_type[n2];
      float r12[3] = {g_x12[index], g_y12[index], g_z12[index]};
      float d12 = sqrt(r12[0] * r12[0] + r12[1] * r12[1] + r12[2] * r12[2]);
      float d12inv = 1.0f / d12;
      float fc12, fcp12;
      find_fc_and_fcp(paramb.rc_radial, paramb.rcinv_radial, d12, fc12, fcp12);
      float fn12[MAX_NUM_N];
      float fnp12[MAX_NUM_N];
      find_fn_and_fnp(paramb.n_max_radial, paramb.rcinv_radial, d12, fc12, fcp12, fn12, fnp12);
      float f12[3] = {0.0f};
      for (int n = 0; n <= paramb.n_max_radial; ++n) {
        float tmp12 = g_Fp[n1 + n * N] * fnp12[n] * d12inv;
        tmp12 *= (paramb.num_types == 1)
                   ? 1.0f
                   : annmb.c[(n * paramb.num_types + t1) * paramb.num_types + t2];
        for (int d = 0; d < 3; ++d) {
          f12[d] += tmp12 * r12[d];
        }
      }
      atomicAdd(&g_fx[n1], f12[0]);
      atomicAdd(&g_fy[n1], f12[1]);
      atomicAdd(&g_fz[n1], f12[2]);
      atomicAdd(&g_fx[n2], -f12[0]);
      atomicAdd(&g_fy[n2], -f12[1]);
      atomicAdd(&g_fz[n2], -f12[2]);

      s_virial_xx -= r12[0] * f12[0];
      s_virial_yy -= r12[1] * f12[1];
      s_virial_zz -= r12[2] * f12[2];
      s_virial_xy -= r12[0] * f12[1];
      s_virial_yz -= r12[1] * f12[2];
      s_virial_zx -= r12[2] * f12[0];
    }
    g_virial[n1] = s_virial_xx;
    g_virial[n1 + N] = s_virial_yy;
    g_virial[n1 + N * 2] = s_virial_zz;
    g_virial[n1 + N * 3] = s_virial_xy;
    g_virial[n1 + N * 4] = s_virial_yz;
    g_virial[n1 + N * 5] = s_virial_zx;
  }
}

static __global__ void find_force_angular(
  const int N,
  const int* g_NN,
  const int* g_NL,
  const NEP2::ParaMB paramb,
  const NEP2::ANN annmb,
  const int* __restrict__ g_type,
  const float* __restrict__ g_x12,
  const float* __restrict__ g_y12,
  const float* __restrict__ g_z12,
  const float* __restrict__ g_Fp,
  const float* __restrict__ g_sum_fxyz,
  float* g_fx,
  float* g_fy,
  float* g_fz,
  float* g_virial)
{
  int n1 = threadIdx.x + blockIdx.x * blockDim.x;
  if (n1 < N) {

    float s_virial_xx = 0.0f;
    float s_virial_yy = 0.0f;
    float s_virial_zz = 0.0f;
    float s_virial_xy = 0.0f;
    float s_virial_yz = 0.0f;
    float s_virial_zx = 0.0f;

    float Fp[MAX_DIM_ANGULAR] = {0.0f};
    float sum_fxyz[NUM_OF_ABC * MAX_NUM_N];
    for (int d = 0; d < (paramb.n_max_angular + 1) * paramb.L_max; ++d) {
      Fp[d] = g_Fp[(paramb.n_max_radial + 1 + d) * N + n1];
    }
    for (int d = 0; d < (paramb.n_max_angular + 1) * NUM_OF_ABC; ++d) {
      sum_fxyz[d] = g_sum_fxyz[d * N + n1];
    }
    int neighbor_number = g_NN[n1];
    int t1 = g_type[n1];
    for (int i1 = 0; i1 < neighbor_number; ++i1) {
      int index = i1 * N + n1;
      int n2 = g_NL[index];
      float r12[3] = {g_x12[index], g_y12[index], g_z12[index]};
      float d12 = sqrt(r12[0] * r12[0] + r12[1] * r12[1] + r12[2] * r12[2]);
      float fc12, fcp12;
      find_fc_and_fcp(paramb.rc_angular, paramb.rcinv_angular, d12, fc12, fcp12);
      int t2 = g_type[n2];
      float f12[3] = {0.0f};
      for (int n = 0; n <= paramb.n_max_angular; ++n) {
        float fn;
        float fnp;
        find_fn_and_fnp(n, paramb.rcinv_angular, d12, fc12, fcp12, fn, fnp);
        const float c =
          (paramb.num_types == 1)
            ? 1.0f
            : annmb
                .c[((paramb.n_max_radial + 1 + n) * paramb.num_types + t1) * paramb.num_types + t2];
        fn *= c;
        fnp *= c;
        accumulate_f12(
          n, n1, paramb.n_max_radial + 1, paramb.n_max_angular + 1, d12, r12, fn, fnp, Fp, sum_fxyz,
          f12);
      }
      f12[0] *= 2.0f;
      f12[1] *= 2.0f;
      f12[2] *= 2.0f;

      atomicAdd(&g_fx[n1], f12[0]);
      atomicAdd(&g_fy[n1], f12[1]);
      atomicAdd(&g_fz[n1], f12[2]);
      atomicAdd(&g_fx[n2], -f12[0]);
      atomicAdd(&g_fy[n2], -f12[1]);
      atomicAdd(&g_fz[n2], -f12[2]);

      s_virial_xx -= r12[0] * f12[0];
      s_virial_yy -= r12[1] * f12[1];
      s_virial_zz -= r12[2] * f12[2];
      s_virial_xy -= r12[0] * f12[1];
      s_virial_yz -= r12[1] * f12[2];
      s_virial_zx -= r12[2] * f12[0];
    }
    g_virial[n1] += s_virial_xx;
    g_virial[n1 + N] += s_virial_yy;
    g_virial[n1 + N * 2] += s_virial_zz;
    g_virial[n1 + N * 3] += s_virial_xy;
    g_virial[n1 + N * 4] += s_virial_yz;
    g_virial[n1 + N * 5] += s_virial_zx;
  }
}

#ifdef USE_ZBL
static __device__ void find_phi_and_phip(float a, float b, float x, float& phi, float& phip)
{
  phi = a * exp(-b * x);
  phip = -b * phi;
}

static __device__ void find_f_and_fp(float d12, float& f, float& fp)
{
  float d12inv = 1 / d12;
  float d12inv_p = -1 / (d12 * d12);
  float Zbl_para[8] = {0.18175, 3.1998, 0.50986, 0.94229, 0.28022, 0.4029, 0.02817, 0.20162};
  float Z = 26;
  float e = 1.6023 * 1e-19;
  float ep = 8.8542 * 1e-12;
  double a = 0.46848 / (2 * powf(Z, 0.23f));
  float x = d12 / a;
  float A = (1 / (4 * 3.1415927f * ep) * Z * Z * e * e) * 1e10;
  float phi[4], phip[4];
  find_phi_and_phip(Zbl_para[0], Zbl_para[1], x, phi[0], phip[0]);
  find_phi_and_phip(Zbl_para[2], Zbl_para[3], x, phi[1], phip[1]);
  find_phi_and_phip(Zbl_para[4], Zbl_para[5], x, phi[2], phip[2]);
  find_phi_and_phip(Zbl_para[6], Zbl_para[7], x, phi[3], phip[3]);
  float PHI = phi[0] + phi[1] + phi[2] + phi[3];
  float PHIP = (phip[0] + phip[1] + phip[2] + phip[3]) / a;
  float fc, fcp;
  float r1 = 1.0;
  float r2 = 2.2;
  find_fc_and_fcp_zbl(r1, r2, d12, fc, fcp);
  f = fc * A * PHI * d12inv;
  fp = A * (fcp * PHI * d12inv + fc * PHIP * d12inv + fc * PHI * d12inv_p);
}

static __global__ void find_force_ZBL(
  const int N,
  const int* g_NN,
  const int* g_NL,
  const float* __restrict__ g_x12,
  const float* __restrict__ g_y12,
  const float* __restrict__ g_z12,
  float* g_fx,
  float* g_fy,
  float* g_fz,
  float* g_virial,
  float* g_pe)
{
  int n1 = threadIdx.x + blockIdx.x * blockDim.x;
  if (n1 < N) {
    float s_pe = 0.0f;
    float s_virial_xx = 0.0f;
    float s_virial_yy = 0.0f;
    float s_virial_zz = 0.0f;
    float s_virial_xy = 0.0f;
    float s_virial_yz = 0.0f;
    float s_virial_zx = 0.0f;
    int neighbor_number = g_NN[n1];
    for (int i1 = 0; i1 < neighbor_number; ++i1) {
      int index = i1 * N + n1;
      int n2 = g_NL[index];
      float r12[3] = {g_x12[index], g_y12[index], g_z12[index]};
      float d12 = sqrt(r12[0] * r12[0] + r12[1] * r12[1] + r12[2] * r12[2]);
      float d12inv = 1.0f / d12;
      float f, fp;
      find_f_and_fp(d12, f, fp);
      float f2 = fp * d12inv * 0.5f;
      float f12[3] = {r12[0] * f2, r12[1] * f2, r12[2] * f2};
      atomicAdd(&g_fx[n1], f12[0]);
      atomicAdd(&g_fy[n1], f12[1]);
      atomicAdd(&g_fz[n1], f12[2]);
      atomicAdd(&g_fx[n2], -f12[0]);
      atomicAdd(&g_fy[n2], -f12[1]);
      atomicAdd(&g_fz[n2], -f12[2]);
      s_virial_xx -= r12[0] * f12[0];
      s_virial_yy -= r12[1] * f12[1];
      s_virial_zz -= r12[2] * f12[2];
      s_virial_xy -= r12[0] * f12[1];
      s_virial_yz -= r12[1] * f12[2];
      s_virial_zx -= r12[2] * f12[0];
      s_pe += f * 0.5f;
    }
    g_virial[n1 + N * 0] += s_virial_xx;
    g_virial[n1 + N * 1] += s_virial_yy;
    g_virial[n1 + N * 2] += s_virial_zz;
    g_virial[n1 + N * 3] += s_virial_xy;
    g_virial[n1 + N * 4] += s_virial_yz;
    g_virial[n1 + N * 5] += s_virial_zx;
    g_pe[n1] += s_pe;
  }
}
#endif

void NEP2::find_force(
  Parameters& para, const float* parameters, Dataset& dataset, bool calculate_q_scaler)
{
  CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_parameters), parameters, sizeof(float) * annmb.num_para));
  float* address_c_parameters;
  CHECK(hipGetSymbolAddress((void**)&address_c_parameters, c_parameters));
  update_potential(address_c_parameters, annmb);

  float rc2_radial = para.rc_radial * para.rc_radial;
  float rc2_angular = para.rc_angular * para.rc_angular;

  gpu_find_neighbor_list<<<dataset.Nc, 256>>>(
    dataset.N, dataset.Na.data(), dataset.Na_sum.data(), rc2_radial, rc2_angular,
    dataset.box.data(), dataset.box_original.data(), dataset.num_cell.data(), dataset.r.data(),
    dataset.r.data() + dataset.N, dataset.r.data() + dataset.N * 2, nep_data.NN_radial.data(),
    nep_data.NL_radial.data(), nep_data.NN_angular.data(), nep_data.NL_angular.data(),
    nep_data.x12_radial.data(), nep_data.y12_radial.data(), nep_data.z12_radial.data(),
    nep_data.x12_angular.data(), nep_data.y12_angular.data(), nep_data.z12_angular.data());
  CUDA_CHECK_KERNEL

  const int block_size = 32;
  const int grid_size = (dataset.N - 1) / block_size + 1;

  find_descriptors_radial<<<grid_size, block_size>>>(
    dataset.N, nep_data.NN_radial.data(), nep_data.NL_radial.data(), paramb, annmb,
    dataset.type.data(), nep_data.x12_radial.data(), nep_data.y12_radial.data(),
    nep_data.z12_radial.data(), nep_data.descriptors.data());
  CUDA_CHECK_KERNEL

  find_descriptors_angular<<<grid_size, block_size>>>(
    dataset.N, nep_data.NN_angular.data(), nep_data.NL_angular.data(), paramb, annmb,
    dataset.type.data(), nep_data.x12_angular.data(), nep_data.y12_angular.data(),
    nep_data.z12_angular.data(), nep_data.descriptors.data(), nep_data.sum_fxyz.data());
  CUDA_CHECK_KERNEL

  if (calculate_q_scaler) {
    find_max_min<<<annmb.dim, 1024>>>(
      dataset.N, nep_data.descriptors.data(), para.q_scaler_gpu.data());
    CUDA_CHECK_KERNEL
  }

  apply_ann<<<grid_size, block_size>>>(
    dataset.N, paramb, annmb, nep_data.descriptors.data(), para.q_scaler_gpu.data(),
    dataset.energy.data(), nep_data.Fp.data());
  CUDA_CHECK_KERNEL

  zero_force<<<grid_size, block_size>>>(
    dataset.N, dataset.force.data(), dataset.force.data() + dataset.N,
    dataset.force.data() + dataset.N * 2);
  CUDA_CHECK_KERNEL

  find_force_radial<<<grid_size, block_size>>>(
    dataset.N, nep_data.NN_radial.data(), nep_data.NL_radial.data(), paramb, annmb,
    dataset.type.data(), nep_data.x12_radial.data(), nep_data.y12_radial.data(),
    nep_data.z12_radial.data(), nep_data.Fp.data(), dataset.force.data(),
    dataset.force.data() + dataset.N, dataset.force.data() + dataset.N * 2, dataset.virial.data());
  CUDA_CHECK_KERNEL

  find_force_angular<<<grid_size, block_size>>>(
    dataset.N, nep_data.NN_angular.data(), nep_data.NL_angular.data(), paramb, annmb,
    dataset.type.data(), nep_data.x12_angular.data(), nep_data.y12_angular.data(),
    nep_data.z12_angular.data(), nep_data.Fp.data(), nep_data.sum_fxyz.data(), dataset.force.data(),
    dataset.force.data() + dataset.N, dataset.force.data() + dataset.N * 2, dataset.virial.data());
  CUDA_CHECK_KERNEL

#ifdef USE_ZBL
  find_force_ZBL<<<grid_size, block_size>>>(
    dataset.N, nep_data.NN_angular.data(), nep_data.NL_angular.data(), nep_data.x12_angular.data(),
    nep_data.y12_angular.data(), nep_data.z12_angular.data(), dataset.force.data(),
    dataset.force.data() + dataset.N, dataset.force.data() + dataset.N * 2, dataset.virial.data(),
    dataset.energy.data());
  CUDA_CHECK_KERNEL
#endif
}
